#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <sys/time.h>

#include <filesystem>
#include <fstream>
#include <functional>

#include "HugeCTR/include/data_generator.hpp"
#include "HugeCTR/include/data_readers/data_reader.hpp"
#include "HugeCTR/include/embeddings/localized_slot_sparse_embedding_hash.hpp"
#include "HugeCTR/include/resource_managers/resource_manager_ext.hpp"
#include "gtest/gtest.h"
#include "nvToolsExt.h"
#include "utest/embedding/embedding_test_utils.hpp"
#include "utest/embedding/sparse_embedding_hash_cpu.hpp"
#include "utest/test_utils.h"

using namespace HugeCTR;
using namespace embedding_test;

namespace {
//---------------------------------------------------------------------------------------
// global params for all testing
const int train_batch_num = 10;  // can not more than 32
const int test_batch_num = 1;
const int train_batchsize = 1024;
const int test_batchsize = 2560;
const int slot_num = 26;
const int max_nnz_per_slot = 1;
const int max_feature_num = max_nnz_per_slot * slot_num;  // max_feature_num in a sample
const long long vocabulary_size = slot_num * 100;
const int embedding_vec_size = 128;
const int combiner = 0;  // 0-sum, 1-mean
const long long label_dim = 1;
const long long dense_dim = 0;
typedef long long T;

const float scaler = 1.0f;  // used in mixed precision training

// In order to not allocate the total size of hash table on each GPU, the users need to set the
// size of max_vocabulary_size_per_gpu, which should be more than vocabulary_size/gpu_count,
// eg: 1.25x of that.

const int num_threads = 1;  // must be 1 for CPU and GPU results comparation
const int num_files = 1;
const Check_t CHK = Check_t::Sum;  // Check_t::Sum
const char *train_file_list_name = "train_file_list.txt";
const char *test_file_list_name = "test_file_list.txt";
const char *prefix = "./data_reader_test_data/temp_dataset_";

const char *sparse_model_file = "localized_hash_table";
const char *opt_file_name = "localized_opt.bin";

std::vector<size_t> slot_sizes;  // null means use vocabulary_size/gpu_count/load_factor as
                                 // max_vocabulary_size_per_gpu

// CAUSION: must match vocabulary_size
// std::vector<size_t> slot_sizes = {39884406,39043,17289,7420,20263,3,7120,1543,63,38532951,
//   2953546,403346,10,2208,11938,155,4,976,14,39979771,25641295,39664984,585935,12972,108,36}; //
//   for cretio dataset
// std::vector<size_t> slot_sizes =
// {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100};
// // just for verify

//-----------------------------------------------------------------------------------------
auto load_sparse_model_to_map = [](std::vector<T> &key_vec, std::vector<size_t> &slot_vec,
                                   std::vector<float> &vec_vec, const std::string &sparse_model) {
  const std::string key_file(sparse_model + "/key");
  const std::string slot_file(sparse_model + "/slot_id");
  const std::string vec_file(sparse_model + "/emb_vector");

  std::ifstream fs_key(key_file, std::ifstream::binary);
  std::ifstream fs_slot(slot_file, std::ifstream::binary);
  std::ifstream fs_vec(vec_file, std::ifstream::binary);

  const size_t key_file_size_in_B = std::filesystem::file_size(key_file);
  const size_t slot_file_size_in_B = std::filesystem::file_size(slot_file);
  const size_t vec_file_size_in_B = std::filesystem::file_size(vec_file);
  const long long num_key = key_file_size_in_B / sizeof(long long);
  const long long num_slot = slot_file_size_in_B / sizeof(size_t);
  const long long num_vec = vec_file_size_in_B / (sizeof(float) * embedding_vec_size);

  if (num_key != num_vec || num_key != num_slot || num_key != vocabulary_size) {
    CK_THROW_(Error_t::BrokenFile, "num_key != num_vec (num_slot) || num_key != vocabulary_size");
  }

  key_vec.clear();
  key_vec.resize(num_key);
  slot_vec.clear();
  slot_vec.resize(num_key);
  vec_vec.clear();
  vec_vec.resize(num_vec * embedding_vec_size);

  using TypeKey = typename std::decay<decltype(*key_vec.begin())>::type;
  if (std::is_same<TypeKey, long long>::value) {
    fs_key.read(reinterpret_cast<char *>(key_vec.data()), key_file_size_in_B);
  } else {
    std::vector<long long> i64_key_vec(num_key, 0);
    fs_key.read(reinterpret_cast<char *>(i64_key_vec.data()), key_file_size_in_B);
    std::transform(i64_key_vec.begin(), i64_key_vec.end(), key_vec.begin(),
                   [](long long key) { return static_cast<unsigned>(key); });
  }
  fs_slot.read(reinterpret_cast<char *>(slot_vec.data()), slot_file_size_in_B);
  fs_vec.read(reinterpret_cast<char *>(vec_vec.data()), vec_file_size_in_B);
};

void init_sparse_model(const char *sparse_model) {
  std::cout << "Init hash table";
  // init hash table file: <key, solt_id, value>
  if (!std::filesystem::exists(sparse_model)) {
    std::filesystem::create_directories(sparse_model);
  }
  const std::string key_file = std::string(sparse_model) + "/key";
  const std::string slot_file = std::string(sparse_model) + "/slot_id";
  const std::string vec_file = std::string(sparse_model) + "/emb_vector";

  std::ofstream fs_key(key_file);
  std::ofstream fs_slot(slot_file);
  std::ofstream fs_vec(vec_file);
  if (!fs_key.is_open() || !fs_slot.is_open() || !fs_vec.is_open()) {
    ERROR_MESSAGE_("Error: file not open for writing");
  }

  // UnifiedDataSimulator<T> ldata_sim(0, slot_num-1); // for slot_id
  test::UniformDataSimulator fdata_sim;  // for value
  std::unique_ptr<float[]> buf(new float[embedding_vec_size]);
  for (long long i = 0; i < vocabulary_size; i++) {
    T key = (T)i;
    // T key = ldata_sim.get_num();
    // CAUSION: can not set random keys here, because we need to ensure that:
    // 1) we can find keys in the data file from this hash table
    // 2) there are no repeated keys
    fs_key.write((char *)&key, sizeof(T));
    T slot_id;
    if (slot_sizes.size() == 0) {
      slot_id = key % slot_num;  // CAUSION: need to dedicate the slot_id for each key for
                                 // correctness verification
    } else {
      size_t offset = 0;
      for (size_t j = 0; j < slot_sizes.size(); j++) {
        if ((key >= static_cast<T>(offset)) && (key < static_cast<T>(offset + slot_sizes[j]))) {
          slot_id = (T)j;
          break;
        }
        offset += slot_sizes[j];
      }
    }
    fs_slot.write((char *)&slot_id, sizeof(T));
    fdata_sim.fill(buf.get(), embedding_vec_size, -0.1f, 0.1f);
    fs_vec.write(reinterpret_cast<const char *>(buf.get()), embedding_vec_size * sizeof(float));
  }
  std::cout << " Done" << std::endl;
}

template <typename TypeEmbeddingComp>
void train_and_test(const std::vector<int> &device_list, const Optimizer_t &optimizer,
                    const Update_t &update_type) {
  OptHyperParams hyper_params;
  hyper_params.adam.beta1 = 0.9f;
  hyper_params.adam.beta2 = 0.999f;
  float tolerance;
  if (std::is_same<TypeEmbeddingComp, __half>::value) {
    hyper_params.adam.epsilon = 1e-4f;
    hyper_params.adagrad.epsilon = 1e-4f;
    tolerance = 5e-3f;
  } else {
    hyper_params.adam.epsilon = 1e-7f;
    hyper_params.adagrad.epsilon = 1e-7f;
    tolerance = 1e-4f;
  }
  hyper_params.momentum.factor = 0.9f;
  hyper_params.nesterov.mu = 0.9f;
  hyper_params.adagrad.initial_accu_value = 0.f;

  const float lr = optimizer == Optimizer_t::Adam ? 0.001f : 0.01f;

  const OptParams opt_params = {optimizer, lr, hyper_params, update_type, scaler};

  test::mpi_init();
  int numprocs = 1;
#ifdef ENABLE_MPI
  MPI_Comm_size(MPI_COMM_WORLD, &numprocs);
#endif

  // if there are multi-node, we assume each node has the same gpu device_list
  std::vector<std::vector<int>> vvgpu;
  for (int i = 0; i < numprocs; i++) {
    vvgpu.push_back(device_list);
  }
  const auto &resource_manager = ResourceManagerExt::create(vvgpu, 0);
  if (resource_manager->is_master_process()) {
    std::cout << "rank " << resource_manager->get_process_id() << " is generating data"
              << std::endl;
    // re-generate the dataset files
    {
      std::ifstream file(train_file_list_name);
      if (file.good()) {
        std::remove(train_file_list_name);
      }
    }
    {
      std::ifstream file(test_file_list_name);
      if (file.good()) {
        std::remove(test_file_list_name);
      }
    }
    // data generation: key's corresponding slot_id=(key%slot_num)
    if (slot_sizes.size() > 0) {
      HugeCTR::data_generation_for_localized_test<T, CHK>(
          train_file_list_name, prefix, num_files, train_batchsize * train_batch_num, slot_num,
          vocabulary_size, label_dim, dense_dim, max_nnz_per_slot, slot_sizes);
      HugeCTR::data_generation_for_localized_test<T, CHK>(
          test_file_list_name, prefix, num_files, test_batchsize * test_batch_num, slot_num,
          vocabulary_size, label_dim, dense_dim, max_nnz_per_slot, slot_sizes);
    } else {
      HugeCTR::data_generation_for_localized_test<T, CHK>(
          train_file_list_name, prefix, num_files, train_batchsize * train_batch_num, slot_num,
          vocabulary_size, label_dim, dense_dim, max_nnz_per_slot);
      HugeCTR::data_generation_for_localized_test<T, CHK>(
          test_file_list_name, prefix, num_files, test_batchsize * test_batch_num, slot_num,
          vocabulary_size, label_dim, dense_dim, max_nnz_per_slot);
    }
  }

#ifdef ENABLE_MPI
  MPI_Barrier(MPI_COMM_WORLD);
  std::cout << "This is rank: " << resource_manager->get_process_id() << std::endl;
#endif

  // setup a data reader
  const DataReaderSparseParam param = {"localized", max_nnz_per_slot, true, slot_num};
  std::vector<DataReaderSparseParam> params;
  params.push_back(param);

  std::unique_ptr<DataReader<T>> train_data_reader(new DataReader<T>(
      train_batchsize, label_dim, dense_dim, params, resource_manager, true, num_threads, false));

  train_data_reader->create_drwg_norm(train_file_list_name, CHK);

  std::unique_ptr<DataReader<T>> test_data_reader(new DataReader<T>(
      test_batchsize, label_dim, dense_dim, params, resource_manager, true, num_threads, false));

  test_data_reader->create_drwg_norm(test_file_list_name, CHK);

  slot_sizes.clear();  // don't init hashtable when doing training correctness checking.
                       // Because we will upload hashtable to GPUs.

  // generate hashtable
  if (resource_manager->is_master_process()) {
    init_sparse_model(sparse_model_file);
  }

#ifdef ENABLE_MPI
  MPI_Barrier(MPI_COMM_WORLD);
#endif

  const SparseEmbeddingHashParams embedding_params = {train_batchsize,
                                                      test_batchsize,
                                                      vocabulary_size,
                                                      {},
                                                      embedding_vec_size,
                                                      max_feature_num,
                                                      slot_num,
                                                      combiner,
                                                      opt_params,
                                                      true,
                                                      false};

  auto copy = [](const std::vector<SparseTensorBag> &tensorbags, SparseTensors<T> &sparse_tensors) {
    sparse_tensors.resize(tensorbags.size());
    for (size_t j = 0; j < tensorbags.size(); ++j) {
      sparse_tensors[j] = SparseTensor<T>::stretch_from(tensorbags[j]);
    }
  };
  SparseTensors<T> train_input;
  copy(train_data_reader->get_sparse_tensors("localized"), train_input);
  SparseTensors<T> test_input;
  copy(test_data_reader->get_sparse_tensors("localized"), test_input);

  std::unique_ptr<LocalizedSlotSparseEmbeddingHash<T, TypeEmbeddingComp>> embedding(
      new LocalizedSlotSparseEmbeddingHash<T, TypeEmbeddingComp>(
          train_input, test_input, embedding_params, resource_manager));

  // upload hash table to device
  embedding->load_parameters(sparse_model_file);

  // for SparseEmbeddingCpu
  std::unique_ptr<SparseEmbeddingHashCpu<T, TypeEmbeddingComp>> embedding_cpu(
      new SparseEmbeddingHashCpu<T, TypeEmbeddingComp>(
          train_batchsize, max_feature_num, vocabulary_size, embedding_vec_size, slot_num,
          label_dim, dense_dim, CHK, train_batch_num * train_batchsize, combiner, opt_params,
          train_file_list_name, sparse_model_file, SparseEmbedding_t::Localized));

  TypeEmbeddingComp *embedding_feature_from_cpu = embedding_cpu->get_forward_results();
  TypeEmbeddingComp *wgrad_from_cpu = embedding_cpu->get_backward_results();
  T *hash_table_key_from_cpu = embedding_cpu->get_hash_table_key_ptr();
  float *hash_table_value_from_cpu = embedding_cpu->get_hash_table_value_ptr();

  // for results check
  std::shared_ptr<GeneralBuffer2<HostAllocator>> buf = GeneralBuffer2<HostAllocator>::create();

  Tensor2<TypeEmbeddingComp> embedding_feature_from_gpu;
  buf->reserve({train_batchsize * slot_num * embedding_vec_size}, &embedding_feature_from_gpu);

  Tensor2<TypeEmbeddingComp> wgrad_from_gpu;
  buf->reserve({train_batchsize * slot_num * embedding_vec_size}, &wgrad_from_gpu);

  Tensor2<T> hash_table_key_from_gpu;
  buf->reserve({vocabulary_size}, &hash_table_key_from_gpu);

  Tensor2<float> hash_table_value_from_gpu;
  buf->reserve({vocabulary_size * embedding_vec_size}, &hash_table_value_from_gpu);

  Tensor2<TypeEmbeddingComp> embedding_feature_from_gpu_eval;
  buf->reserve({test_batchsize * slot_num * embedding_vec_size}, &embedding_feature_from_gpu_eval);

  buf->allocate();

  typedef struct TypeHashValue_ {
    float data[embedding_vec_size];
  } TypeHashValue;

  for (int i = 0; i < train_batch_num; i++) {
    printf("Rank%d: Round %d start training:\n", resource_manager->get_process_id(), i);

    // call read a batch
    printf("Rank%d: data_reader->read_a_batch_to_device()\n", resource_manager->get_process_id());
    train_data_reader->read_a_batch_to_device();

    // GPU forward
    printf("Rank%d: embedding->forward()\n", resource_manager->get_process_id());
    embedding->forward(true);

    // check the result of forward
    printf("Rank%d: embedding->get_forward_results()\n", resource_manager->get_process_id());
    embedding->get_forward_results(true, embedding_feature_from_gpu);  // memcpy from GPU to CPU

    if (resource_manager->is_master_process()) {
      // CPU forward
      printf("Rank0: embedding_cpu->forward()\n");
      embedding_cpu->forward();

      printf("Rank0: check forward results\n");
      ASSERT_TRUE(compare_embedding_feature(train_batchsize * slot_num * embedding_vec_size,
                                            embedding_feature_from_gpu.get_ptr(),
                                            embedding_feature_from_cpu, tolerance));
    }

#ifdef ENABLE_MPI
    MPI_Barrier(MPI_COMM_WORLD);
#endif

    // GPU backward
    printf("Rank%d: embedding->backward()\n", resource_manager->get_process_id());
    embedding->backward();

    // check the result of backward
    printf("Rank%d: embedding->get_backward_results()\n", resource_manager->get_process_id());
    embedding->get_backward_results(wgrad_from_gpu, 0);

    if (resource_manager->is_master_process()) {
      // CPU backward
      printf("Rank0: embedding_cpu->backward()\n");
      embedding_cpu->backward();

      printf("Rank0: check backward results: GPU and CPU\n");
      ASSERT_TRUE(compare_wgrad(train_batchsize * slot_num * embedding_vec_size,
                                wgrad_from_gpu.get_ptr(), wgrad_from_cpu, tolerance));
    }

#ifdef ENABLE_MPI
    MPI_Barrier(MPI_COMM_WORLD);
#endif

    // GPU update_params
    printf("Rank%d: embedding->update_params()\n", resource_manager->get_process_id());
    embedding->update_params();

    // check the results of update params
    printf("Rank%d: embedding->get_update_params_results()\n", resource_manager->get_process_id());
    embedding->get_update_params_results(hash_table_key_from_gpu,
                                         hash_table_value_from_gpu);  // memcpy from GPU to CPU

    if (resource_manager->is_master_process()) {
      // CPU update_params
      printf("Rank0: embedding_cpu->update_params()\n");
      embedding_cpu->update_params();

      printf("Rank0: check update_params results\n");
      ASSERT_TRUE(compare_hash_table(
          vocabulary_size, hash_table_key_from_gpu.get_ptr(),
          reinterpret_cast<TypeHashValue *>(hash_table_value_from_gpu.get_ptr()),
          hash_table_key_from_cpu, reinterpret_cast<TypeHashValue *>(hash_table_value_from_cpu),
          tolerance));
    }

#ifdef ENABLE_MPI
    MPI_Barrier(MPI_COMM_WORLD);
#endif
  }

  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  // create new obj for eval()
  embedding->dump_parameters(sparse_model_file, DataSourceParams());

  {
    printf("Rank%d: embedding->dump_opt_states()\n", resource_manager->get_process_id());
    std::ofstream fs(opt_file_name);
    embedding->dump_opt_states(fs, opt_file_name, DataSourceParams());
    fs.close();
  }

  {
    printf("Rank%d: embedding->load_opt_states()\n", resource_manager->get_process_id());
    std::ifstream fs(opt_file_name);
    embedding->load_opt_states(fs);
    fs.close();
  }

  // for SparseEmbeddingCpu eval
  std::unique_ptr<SparseEmbeddingHashCpu<T, TypeEmbeddingComp>> test_embedding_cpu(
      new SparseEmbeddingHashCpu<T, TypeEmbeddingComp>(
          test_batchsize, max_feature_num, vocabulary_size, embedding_vec_size, slot_num, label_dim,
          dense_dim, CHK, test_batch_num * test_batchsize, combiner, opt_params,
          test_file_list_name, sparse_model_file, SparseEmbedding_t::Localized));

  TypeEmbeddingComp *embedding_feature_from_cpu_eval = test_embedding_cpu->get_forward_results();

  /////////////////////////////////////////////////////////////////////////////////////////////
  // eval
  {
    printf("\nRank%d: Round start eval:\n", resource_manager->get_process_id());

    // call read a batch
    printf("Rank%d: data_reader_eval->read_a_batch_to_device()\n",
           resource_manager->get_process_id());
    test_data_reader->read_a_batch_to_device();

    // GPU forward
    printf("Rank%d: embedding_eval->forward()\n", resource_manager->get_process_id());
    embedding->forward(false);

    // check the result of forward
    printf("Rank%d: embedding_eval->get_forward_results()\n", resource_manager->get_process_id());
    embedding->get_forward_results(false,
                                   embedding_feature_from_gpu_eval);  // memcpy from GPU to CPU

    if (resource_manager->is_master_process()) {
      // CPU forward
      printf("Rank0: embedding_cpu_eval->forward()\n");
      test_embedding_cpu->forward();

      printf("Rank0: check forward results\n");
      ASSERT_TRUE(compare_embedding_feature(test_batchsize * slot_num * embedding_vec_size,
                                            embedding_feature_from_gpu_eval.get_ptr(),
                                            embedding_feature_from_cpu_eval, tolerance));
    }

#ifdef ENABLE_MPI
    MPI_Barrier(MPI_COMM_WORLD);
#endif

    printf("Rank%d: Round end:\n", resource_manager->get_process_id());
  }

  test::mpi_finalize();
}

template <typename TypeEmbeddingComp>
void load_and_dump(const std::vector<int> &device_list, const Optimizer_t &optimizer,
                   const Update_t &update_type) {
  using TypeKey = T;
  OptHyperParams hyper_params;
  hyper_params.adam.beta1 = 0.9f;
  hyper_params.adam.beta2 = 0.999f;
  float tolerance;
  if (std::is_same<TypeEmbeddingComp, __half>::value) {
    hyper_params.adam.epsilon = 1e-4f;
    tolerance = 5e-3f;
  } else {
    hyper_params.adam.epsilon = 1e-7f;
    tolerance = 1e-4f;
  }
  if (std::is_same<TypeEmbeddingComp, __half>::value) {
    hyper_params.adam.epsilon = 1e-4f;
    hyper_params.adagrad.epsilon = 1e-4f;
  } else {
    hyper_params.adam.epsilon = 1e-7f;
    hyper_params.adagrad.epsilon = 1e-7f;
  }
  hyper_params.momentum.factor = 0.9f;
  hyper_params.nesterov.mu = 0.9f;
  hyper_params.adagrad.initial_accu_value = 0.f;

  const float lr = optimizer == Optimizer_t::Adam ? 0.001f : 0.01f;

  const OptParams opt_params = {optimizer, lr, hyper_params, update_type, scaler};

  std::vector<std::vector<int>> vvgpu;
  vvgpu.push_back(device_list);
  const auto &resource_manager = ResourceManagerExt::create(vvgpu, 0);

  // re-generate the dataset files
  {
    std::ifstream fs(train_file_list_name);
    if (fs.good()) {
      std::remove(train_file_list_name);
    }
  }

  // data generation
  if (slot_sizes.size() > 0) {
    HugeCTR::data_generation_for_localized_test<T, CHK>(
        train_file_list_name, prefix, num_files, train_batchsize * train_batch_num, slot_num,
        vocabulary_size, label_dim, dense_dim, max_nnz_per_slot, slot_sizes);
  } else {
    HugeCTR::data_generation_for_localized_test<T, CHK>(
        train_file_list_name, prefix, num_files, train_batchsize * train_batch_num, slot_num,
        vocabulary_size, label_dim, dense_dim, max_nnz_per_slot);
  }

  // setup a data reader
  const DataReaderSparseParam param = {"localized", max_nnz_per_slot, true, slot_num};
  std::vector<DataReaderSparseParam> params;
  params.push_back(param);

  std::unique_ptr<DataReader<T>> train_data_reader(new DataReader<T>(
      train_batchsize, label_dim, dense_dim, params, resource_manager, true, num_threads, false));

  train_data_reader->create_drwg_norm(train_file_list_name, CHK);

  slot_sizes.clear();  // don't init hashtable when doing training correctness checking.
                       // Because we will upload hashtable to GPUs.

  // init hash table file
  init_sparse_model(sparse_model_file);

  const SparseEmbeddingHashParams embedding_params = {train_batchsize,
                                                      test_batchsize,
                                                      vocabulary_size,
                                                      {},
                                                      embedding_vec_size,
                                                      max_feature_num,
                                                      slot_num,
                                                      combiner,
                                                      opt_params,
                                                      true,
                                                      false};

  auto copy = [](const std::vector<SparseTensorBag> &tensorbags, SparseTensors<T> &sparse_tensors) {
    sparse_tensors.resize(tensorbags.size());
    for (size_t j = 0; j < tensorbags.size(); ++j) {
      sparse_tensors[j] = SparseTensor<T>::stretch_from(tensorbags[j]);
    }
  };
  SparseTensors<T> train_input;
  copy(train_data_reader->get_sparse_tensors("localized"), train_input);

  std::unique_ptr<LocalizedSlotSparseEmbeddingHash<T, TypeEmbeddingComp>> embedding(
      new LocalizedSlotSparseEmbeddingHash<T, TypeEmbeddingComp>(
          train_input, train_input, embedding_params, resource_manager));

  // upload hash table to device
  embedding->load_parameters(sparse_model_file);

  printf("max_vocabulary_size=%zu, vocabulary_size=%zu\n", embedding->get_max_vocabulary_size(),
         embedding->get_vocabulary_size());

  BufferBag buf_bag;
  {
    size_t buffer_size = embedding->get_max_vocabulary_size();
    size_t max_voc_size_per_gpu = embedding_params.max_vocabulary_size_per_gpu;

    auto host_blobs_buff = GeneralBuffer2<CudaHostAllocator>::create();

    Tensor2<TypeKey> tensor_keys;
    Tensor2<size_t> tensor_slot_id;
    host_blobs_buff->reserve({buffer_size}, &tensor_keys);
    host_blobs_buff->reserve({buffer_size}, &tensor_slot_id);
    host_blobs_buff->reserve({buffer_size, embedding_vec_size}, &(buf_bag.embedding));

    buf_bag.keys = tensor_keys.shrink();
    buf_bag.slot_id = tensor_slot_id.shrink();

    const size_t local_gpu_count = resource_manager->get_local_gpu_count();

    for (size_t id = 0; id < local_gpu_count; id++) {
      Tensor2<float> tensor;
      host_blobs_buff->reserve({max_voc_size_per_gpu, embedding_vec_size}, &tensor);
      buf_bag.h_value_tensors.push_back(tensor);

      Tensor2<size_t> tensor_slot_id;
      host_blobs_buff->reserve({max_voc_size_per_gpu}, &tensor_slot_id);
      buf_bag.h_slot_id_tensors.push_back(tensor_slot_id);
    }
    host_blobs_buff->allocate();

    CudaDeviceContext context;
    for (size_t id = 0; id < local_gpu_count; id++) {
      context.set_device(resource_manager->get_local_gpu(id)->get_device_id());
      {
        auto uvm_blobs_buff = GeneralBuffer2<CudaManagedAllocator>::create();
        Tensor2<TypeKey> tensor;
        uvm_blobs_buff->reserve({max_voc_size_per_gpu}, &tensor);
        buf_bag.uvm_key_tensor_bags.push_back(tensor.shrink());
        uvm_blobs_buff->allocate();
      }
      {
        auto hbm_blobs_buff = GeneralBuffer2<CudaAllocator>::create();
        Tensor2<size_t> tensor;
        hbm_blobs_buff->reserve({max_voc_size_per_gpu}, &tensor);
        buf_bag.d_value_index_tensors.push_back(tensor);
        hbm_blobs_buff->allocate();
      }
    }
  }

  size_t dump_size;
  embedding->dump_parameters(buf_bag, &dump_size);

  printf("dump_size=%zu, max_vocabulary_size=%zu, vocabulary_size=%zu\n", dump_size,
         embedding->get_max_vocabulary_size(), embedding->get_vocabulary_size());

  embedding->dump_parameters(buf_bag, &dump_size);

  printf("dump_size=%zu, max_vocabulary_size=%zu, vocabulary_size=%zu\n", dump_size,
         embedding->get_max_vocabulary_size(), embedding->get_vocabulary_size());

  embedding->reset();

  printf("max_vocabulary_size=%zu, vocabulary_size=%zu\n", embedding->get_max_vocabulary_size(),
         embedding->get_vocabulary_size());

  embedding->load_parameters(buf_bag, dump_size);

  printf("max_vocabulary_size=%zu, vocabulary_size=%zu\n", embedding->get_max_vocabulary_size(),
         embedding->get_vocabulary_size());

  embedding->dump_parameters(buf_bag, &dump_size);

  printf("dump_size=%zu, max_vocabulary_size=%zu, vocabulary_size=%zu\n", dump_size,
         embedding->get_max_vocabulary_size(), embedding->get_vocabulary_size());

  std::string tmp_sparse_model_file{"tmp_sparse_model"};
  embedding->dump_parameters(tmp_sparse_model_file, DataSourceParams());

  std::vector<T> hash_table_key_from_cpu;
  std::vector<size_t> slot_id_from_cpu;
  std::vector<float> hash_table_value_from_cpu;
  load_sparse_model_to_map(hash_table_key_from_cpu, slot_id_from_cpu, hash_table_value_from_cpu,
                           sparse_model_file);

  std::vector<T> hash_table_key_from_gpu;
  std::vector<size_t> slot_id_from_gpu;
  std::vector<float> hash_table_value_from_gpu;
  load_sparse_model_to_map(hash_table_key_from_gpu, slot_id_from_gpu, hash_table_value_from_gpu,
                           tmp_sparse_model_file);

  typedef struct TypeHashValue_ {
    float data[embedding_vec_size];
  } TypeHashValue;

  ASSERT_TRUE(compare_hash_table(
      vocabulary_size, hash_table_key_from_gpu.data(),
      reinterpret_cast<TypeHashValue *>(hash_table_value_from_gpu.data()),
      hash_table_key_from_cpu.data(),
      reinterpret_cast<TypeHashValue *>(hash_table_value_from_cpu.data()), tolerance));

  ASSERT_TRUE(compare_key_slot(vocabulary_size, hash_table_key_from_gpu.data(),
                               slot_id_from_gpu.data(), hash_table_key_from_cpu.data(),
                               slot_id_from_cpu.data()));
}

template <typename TypeEmbeddingComp>
void load_and_dump_file(const std::vector<int> &device_list, const Optimizer_t &optimizer,
                        const Update_t &update_type) {
  std::string sparse_model_src("sparse_model_src");
  std::string sparse_model_dst("sparse_model_dst");

  OptHyperParams hyper_params;
  hyper_params.adam.beta1 = 0.9f;
  hyper_params.adam.beta2 = 0.999f;
  float tolerance;
  if (std::is_same<TypeEmbeddingComp, __half>::value) {
    hyper_params.adam.epsilon = 1e-4f;
    tolerance = 5e-3f;
  } else {
    hyper_params.adam.epsilon = 1e-7f;
    tolerance = 1e-4f;
  }
  hyper_params.momentum.factor = 0.9f;
  hyper_params.nesterov.mu = 0.9f;

  const float lr = optimizer == Optimizer_t::Adam ? 0.001f : 0.01f;
  const OptParams opt_params = {optimizer, lr, hyper_params, update_type, scaler};

  int numprocs = 1, pid = 0;
  std::vector<std::vector<int>> vvgpu;
  test::mpi_init();
  for (int i = 0; i < numprocs; i++) {
    vvgpu.push_back(device_list);
  }
  const auto &resource_manager = ResourceManagerExt::create(vvgpu, 0);

  if (pid == 0) {
    // re-generate the dataset files
    if (std::filesystem::exists(train_file_list_name)) {
      std::filesystem::remove(train_file_list_name);
    }

    // data generation
    if (slot_sizes.size() > 0) {
      HugeCTR::data_generation_for_localized_test<T, CHK>(
          train_file_list_name, prefix, num_files, train_batchsize * train_batch_num, slot_num,
          vocabulary_size, label_dim, dense_dim, max_nnz_per_slot, slot_sizes);
    } else {
      HugeCTR::data_generation_for_localized_test<T, CHK>(
          train_file_list_name, prefix, num_files, train_batchsize * train_batch_num, slot_num,
          vocabulary_size, label_dim, dense_dim, max_nnz_per_slot);
    }
  }

#ifdef ENABLE_MPI
  MPI_Barrier(MPI_COMM_WORLD);
#endif

  // setup a data reader
  const DataReaderSparseParam param = {"localized", max_nnz_per_slot, true, slot_num};
  std::vector<DataReaderSparseParam> params;
  params.push_back(param);

  std::unique_ptr<DataReader<T>> train_data_reader(new DataReader<T>(
      train_batchsize, label_dim, dense_dim, params, resource_manager, true, num_threads, false));

  train_data_reader->create_drwg_norm(train_file_list_name, CHK);

  slot_sizes.clear();  // don't init hashtable when doing training correctness checking.
                       // Because we will upload hashtable to GPUs.

  const SparseEmbeddingHashParams embedding_params = {train_batchsize,
                                                      test_batchsize,
                                                      vocabulary_size,
                                                      {},
                                                      embedding_vec_size,
                                                      max_feature_num,
                                                      slot_num,
                                                      combiner,
                                                      opt_params,
                                                      true,
                                                      false};

  auto copy = [](const std::vector<SparseTensorBag> &tensorbags, SparseTensors<T> &sparse_tensors) {
    sparse_tensors.resize(tensorbags.size());
    for (size_t j = 0; j < tensorbags.size(); ++j) {
      sparse_tensors[j] = SparseTensor<T>::stretch_from(tensorbags[j]);
    }
  };
  SparseTensors<T> train_input;
  copy(train_data_reader->get_sparse_tensors("localized"), train_input);

  std::unique_ptr<LocalizedSlotSparseEmbeddingHash<T, TypeEmbeddingComp>> embedding(
      new LocalizedSlotSparseEmbeddingHash<T, TypeEmbeddingComp>(
          train_input, train_input, embedding_params, resource_manager));

  // init hash table file
  if (pid == 0) {
    init_sparse_model(sparse_model_src.c_str());
  }

#ifdef ENABLE_MPI
  MPI_Barrier(MPI_COMM_WORLD);
#endif

  // upload hash table to device
  embedding->load_parameters(sparse_model_src);

  if (pid == 0) {
    printf("max_vocabulary_size=%zu, vocabulary_size=%zu\n", embedding->get_max_vocabulary_size(),
           embedding->get_vocabulary_size());
  }

  // dump sparse model to file
  embedding->dump_parameters(sparse_model_dst, DataSourceParams());

#ifdef ENABLE_MPI
  MPI_Barrier(MPI_COMM_WORLD);
#endif

  std::vector<T> hash_table_key_from_cpu;
  std::vector<size_t> slot_id_from_cpu;
  std::vector<float> hash_table_value_from_cpu;
  load_sparse_model_to_map(hash_table_key_from_cpu, slot_id_from_cpu, hash_table_value_from_cpu,
                           sparse_model_src);

  std::vector<T> hash_table_key_from_gpu;
  std::vector<size_t> slot_id_from_gpu;
  std::vector<float> hash_table_value_from_gpu;
  load_sparse_model_to_map(hash_table_key_from_gpu, slot_id_from_gpu, hash_table_value_from_gpu,
                           sparse_model_dst);

  typedef struct TypeHashValue_ {
    float data[embedding_vec_size];
  } TypeHashValue;

  ASSERT_TRUE(compare_hash_table(
      vocabulary_size, hash_table_key_from_gpu.data(),
      reinterpret_cast<TypeHashValue *>(hash_table_value_from_gpu.data()),
      hash_table_key_from_cpu.data(),
      reinterpret_cast<TypeHashValue *>(hash_table_value_from_cpu.data()), tolerance));

  ASSERT_TRUE(compare_key_slot(vocabulary_size, hash_table_key_from_gpu.data(),
                               slot_id_from_gpu.data(), hash_table_key_from_cpu.data(),
                               slot_id_from_cpu.data()));

  test::mpi_finalize();
}

}  // namespace

TEST(localized_sparse_embedding_hash_test, fp32_sgd_1gpu) {
  train_and_test<float>({0}, Optimizer_t::SGD, Update_t::Local);
}

TEST(localized_sparse_embedding_hash_test, fp32_sgd_8gpu) {
  train_and_test<float>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::SGD, Update_t::Local);
}

TEST(localized_sparse_embedding_hash_test, fp32_sgd_global_update_1gpu) {
  train_and_test<float>({0}, Optimizer_t::SGD, Update_t::Global);
}

TEST(localized_sparse_embedding_hash_test, fp32_sgd_global_update_8gpu) {
  train_and_test<float>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::SGD, Update_t::Global);
}

TEST(localized_sparse_embedding_hash_test, fp16_sgd_1gpu) {
  train_and_test<__half>({0}, Optimizer_t::SGD, Update_t::Local);
}

TEST(localized_sparse_embedding_hash_test, fp16_sgd_8gpu) {
  train_and_test<__half>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::SGD, Update_t::Local);
}

TEST(localized_sparse_embedding_hash_test, fp16_sgd_global_update_1gpu) {
  train_and_test<__half>({0}, Optimizer_t::SGD, Update_t::Global);
}

TEST(localized_sparse_embedding_hash_test, fp16_sgd_global_update_8gpu) {
  train_and_test<__half>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::SGD, Update_t::Global);
}

TEST(localized_sparse_embedding_hash_test, fp32_adam_1gpu) {
  train_and_test<float>({0}, Optimizer_t::Adam, Update_t::Local);
}

TEST(localized_sparse_embedding_hash_test, fp32_adam_8gpu) {
  train_and_test<float>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::Adam, Update_t::Local);
}

TEST(localized_sparse_embedding_hash_test, fp32_adam_global_update_1gpu) {
  train_and_test<float>({0}, Optimizer_t::Adam, Update_t::Global);
}

TEST(localized_sparse_embedding_hash_test, fp32_adam_global_update_8gpu) {
  train_and_test<float>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::Adam, Update_t::Global);
}

TEST(localized_sparse_embedding_hash_test, fp32_adam_lazyglobal_update_1gpu) {
  train_and_test<float>({0}, Optimizer_t::Adam, Update_t::LazyGlobal);
}

TEST(localized_sparse_embedding_hash_test, fp32_adam_lazyglobal_update_8gpu) {
  train_and_test<float>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::Adam, Update_t::LazyGlobal);
}

TEST(localized_sparse_embedding_hash_test, fp16_adam_1gpu) {
  train_and_test<__half>({0}, Optimizer_t::Adam, Update_t::Local);
}

TEST(localized_sparse_embedding_hash_test, fp16_adam_8gpu) {
  train_and_test<__half>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::Adam, Update_t::Local);
}

TEST(localized_sparse_embedding_hash_test, fp16_adam_global_update_1gpu) {
  train_and_test<__half>({0}, Optimizer_t::Adam, Update_t::Global);
}

TEST(localized_sparse_embedding_hash_test, fp16_adam_global_update_8gpu) {
  train_and_test<__half>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::Adam, Update_t::Global);
}

TEST(localized_sparse_embedding_hash_test, fp16_adam_lazyglobal_update_1gpu) {
  train_and_test<__half>({0}, Optimizer_t::Adam, Update_t::LazyGlobal);
}

TEST(localized_sparse_embedding_hash_test, fp16_adam_lazyglobal_update_8gpu) {
  train_and_test<__half>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::Adam, Update_t::LazyGlobal);
}

TEST(localized_sparse_embedding_hash_test, fp32_adagrad_1gpu) {
  train_and_test<float>({0}, Optimizer_t::AdaGrad, Update_t::Local);
}

TEST(localized_sparse_embedding_hash_test, fp32_adagrad_8gpu) {
  train_and_test<float>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::AdaGrad, Update_t::Local);
}

TEST(localized_sparse_embedding_hash_test, fp16_adagrad_1gpu) {
  train_and_test<__half>({0}, Optimizer_t::AdaGrad, Update_t::Local);
}

TEST(localized_sparse_embedding_hash_test, fp16_adagrad_8gpu) {
  train_and_test<__half>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::AdaGrad, Update_t::Local);
}

TEST(localized_sparse_embedding_hash_test, load_and_dump_1gpu) {
  load_and_dump<float>({0}, Optimizer_t::SGD, Update_t::Global);
}

TEST(localized_sparse_embedding_hash_test, load_and_dump_8gpu) {
  load_and_dump<float>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::SGD, Update_t::Global);
}

TEST(localized_sparse_embedding_hash_test, load_and_dump_file_1gpu) {
  load_and_dump_file<float>({0}, Optimizer_t::SGD, Update_t::Global);
}

TEST(localized_sparse_embedding_hash_test, load_and_dump_file_8gpu) {
  load_and_dump_file<float>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::SGD, Update_t::Global);
}
