#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>

#include "embedding_training_cache/hmem_cache/hmem_cache.hpp"
#include "embedding_training_cache/parameter_server.hpp"
#include "etc_test_utils.hpp"

using namespace HugeCTR;
using namespace etc_test;

namespace {

const char* prefix = "./embedding_training_cache_test_data/tmp_";
const char* file_list_name_train = "file_list_train.txt";
const char* file_list_name_eval = "file_list_eval.txt";
const char* snapshot_src_file = "distributed_snapshot_src";
const char* snapshot_dst_file = "distributed_snapshot_dst";
const char* snapshot_bkp_file_unsigned = "distributed_snapshot_unsigned";
const char* snapshot_bkp_file_longlong = "distributed_snapshot_longlong";
const char* keyset_file_name = "keyset_file.bin";

const int batchsize = 4096;
const long long label_dim = 1;
const long long dense_dim = 0;
const int slot_num = 128;
const int max_nnz_per_slot = 1;
const int max_feature_num = max_nnz_per_slot * slot_num;
const long long vocabulary_size = 100000;
const int emb_vec_size = 64;
const int combiner = 0;
const float scaler = 1.0f;
const int num_workers = 1;
const int num_files = 1;

const Check_t check = Check_t::Sum;
const Update_t update_type = Update_t::Local;

// const int batch_num_train = 10;
const int batch_num_eval = 1;

template <typename TypeKey>
void do_upload_and_download_snapshot(int batch_num_train, TrainPSType_t ps_type,
                                     bool is_distributed, Optimizer_t opt_type = Optimizer_t::Adam,
                                     std::string local_path = "./",
                                     HMemCacheConfig hc_config = HMemCacheConfig()) {
  Embedding_t embedding_type = is_distributed ? Embedding_t::DistributedSlotSparseEmbeddingHash
                                              : Embedding_t::LocalizedSlotSparseEmbeddingHash;
  // create a resource manager for a single GPU
  std::vector<std::vector<int>> vvgpu;
  vvgpu.push_back({0});
  const auto resource_manager{ResourceManagerExt::create(vvgpu, 0)};

  generate_sparse_model<TypeKey, check>(
      snapshot_src_file, snapshot_dst_file, snapshot_bkp_file_unsigned, snapshot_bkp_file_longlong,
      file_list_name_train, file_list_name_eval, prefix, num_files, label_dim, dense_dim, slot_num,
      max_nnz_per_slot, max_feature_num, vocabulary_size, emb_vec_size, combiner, scaler,
      num_workers, batchsize, batch_num_train, batch_num_eval, update_type, resource_manager);
  generate_opt_state(snapshot_src_file, opt_type);
  if (std::filesystem::exists(snapshot_dst_file)) {
    std::filesystem::remove_all(snapshot_dst_file);
  }
  std::filesystem::copy(snapshot_src_file, snapshot_dst_file,
                        std::filesystem::copy_options::recursive);

  auto get_ext_file = [](const std::string& sparse_model_file, std::string ext) {
    return std::string(sparse_model_file) + "/" + ext;
  };

  // Create a ParameterServer
  hc_config.block_capacity = vocabulary_size;
  ParameterServer<TypeKey> parameter_server(ps_type, snapshot_dst_file, embedding_type, opt_type,
                                            emb_vec_size, resource_manager, local_path, hc_config);

  // Make a synthetic keyset files
  std::vector<long long> keys_in_file;
  {
    size_t key_file_size_in_byte =
        std::filesystem::file_size(get_ext_file(snapshot_dst_file, "key"));
    size_t num_keys = key_file_size_in_byte / sizeof(long long);
    keys_in_file.resize(num_keys);
    std::ifstream key_ifs(get_ext_file(snapshot_dst_file, "key"));
    key_ifs.read(reinterpret_cast<char*>(keys_in_file.data()), key_file_size_in_byte);
    TypeKey* key_ptr = nullptr;
    std::vector<TypeKey> key_vec;
    if (std::is_same<TypeKey, long long>::value) {
      key_ptr = reinterpret_cast<TypeKey*>(keys_in_file.data());
    } else {
      key_vec.resize(num_keys);
      std::transform(keys_in_file.begin(), keys_in_file.end(), key_vec.begin(),
                     [](long long key) { return static_cast<unsigned>(key); });
      key_ptr = key_vec.data();
    }
    std::ofstream key_ofs(keyset_file_name, std::ofstream::binary | std::ofstream::trunc);
    key_ofs.write(reinterpret_cast<char*>(key_ptr), num_keys * sizeof(TypeKey));
  }

  BufferBag buf_bag;
  {
    auto blobs_buff{GeneralBuffer2<CudaHostAllocator>::create()};
    buf_bag.opt_states.resize(vec_per_line[opt_type] - 1);

    Tensor2<TypeKey> tensor_keys;
    Tensor2<size_t> tensor_slot_id;
    blobs_buff->reserve({vocabulary_size}, &tensor_keys);
    blobs_buff->reserve({vocabulary_size}, &tensor_slot_id);
    blobs_buff->reserve({vocabulary_size, emb_vec_size}, &(buf_bag.embedding));
    for (auto& opt_state : buf_bag.opt_states) {
      blobs_buff->reserve({vocabulary_size, emb_vec_size}, &opt_state);
    }
    blobs_buff->allocate();

    buf_bag.keys = tensor_keys.shrink();
    buf_bag.slot_id = tensor_slot_id.shrink();
  }

  Timer timer_ps;
  timer_ps.start();

  parameter_server.load_keyset_from_file(keyset_file_name);

  size_t size_tmp = 0;
  parameter_server.pull(buf_bag, size_tmp);
  parameter_server.push(buf_bag, size_tmp);
  parameter_server.flush_emb_tbl_to_ssd();

  MESSAGE_("Batch_num=" + std::to_string(batch_num_train) +
           ", embedding_vec_size=" + std::to_string(emb_vec_size) +
           ", elapsed time=" + std::to_string(timer_ps.elapsedSeconds()) + "s");

  // Check if the result is correct
  std::vector<std::string> data_files{"key"};
  if (!is_distributed) data_files.push_back("slot_id");
  auto vec_files{get_data_file(opt_type)};
  if (ps_type == TrainPSType_t::Cached) {
    for (auto const& vec_file : vec_files) data_files.push_back(vec_file);
  } else {
    data_files.push_back(vec_files[0]);
  }
  for (const auto& data_file : data_files) {
    std::string dst_name(snapshot_dst_file);
    MESSAGE_(std::string("check ") + dst_name + "/" + data_file, true, false);
    ASSERT_TRUE(check_vector_equality(snapshot_src_file, dst_name.c_str(), data_file.c_str()));
    MESSAGE_(" [DONE]", true, true, false);
  }

  auto key_vec_pair{parameter_server.pull(keys_in_file)};
  std::string vec_file_name("./emb_vector");
  std::ofstream vec_ofs(vec_file_name, std::ofstream::binary | std::ofstream::trunc);
  vec_ofs.write(reinterpret_cast<char*>(key_vec_pair.second.data()),
                key_vec_pair.second.size() * sizeof(float));

  ASSERT_EQ(key_vec_pair.first.size(), keys_in_file.size());
  ASSERT_TRUE(check_vector_equality(snapshot_src_file, "./", "emb_vector"));
}

TEST(parameter_server_test, unsigned_host_distributed) {
  do_upload_and_download_snapshot<unsigned>(20, TrainPSType_t::Staged, true);
}
TEST(parameter_server_test, long_long_cache_distributed_Adam) {
  HMemCacheConfig hc_config(1, 0.5, 0);
  do_upload_and_download_snapshot<long long>(20, TrainPSType_t::Cached, true, Optimizer_t::Adam,
                                             "./", hc_config);
}

TEST(parameter_server_test, unsigned_host_localized) {
  do_upload_and_download_snapshot<unsigned>(20, TrainPSType_t::Staged, false);
}
TEST(parameter_server_test, unsigned_cache_localized_SGD) {
  HMemCacheConfig hc_config(1, 0.5, 0);
  do_upload_and_download_snapshot<unsigned>(20, TrainPSType_t::Cached, false, Optimizer_t::SGD,
                                            "./", hc_config);
}

}  // namespace
