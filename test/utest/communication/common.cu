
#include <hip/hip_runtime.h>
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#ifdef ENABLE_MPI
#include <gtest/gtest.h>

#include <utest/test_utils.hpp>

using namespace HugeCTR;

class MPIEnvironment : public ::testing::Environment {
 protected:
  virtual void SetUp() {
    test::mpi_init();
    HCTR_LIB_THROW(nvmlInit_v2());
  };
  virtual void TearDown() { test::mpi_finalize(); }
  virtual ~MPIEnvironment(){};
};

::testing::Environment* const mpi_env = ::testing::AddGlobalTestEnvironment(new MPIEnvironment);
#endif
