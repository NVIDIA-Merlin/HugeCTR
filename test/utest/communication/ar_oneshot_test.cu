#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#ifndef ENABLE_MPI
#include <gtest/gtest.h>

#include <collectives/collective.hpp>
#include <common.hpp>
#include <general_buffer2.hpp>
#include <random>
#include <resource_managers/resource_manager_core.hpp>
#include <tensor2.hpp>
#include <type_traits>
#include <utest/test_utils.hpp>
#include <utils.hpp>

using namespace HugeCTR;

#define TIMEIT(function, bench_time)                                                       \
  {                                                                                        \
    int warmup_iters = 10;                                                                 \
    for (int i = 0; i < warmup_iters; i++) {                                               \
      function;                                                                            \
    }                                                                                      \
    stream_sync_all();                                                                     \
                                                                                           \
    int iters = 1000;                                                                      \
    auto t0 = std::chrono::high_resolution_clock::now();                                   \
    for (int i = 0; i < iters; i++) {                                                      \
      function;                                                                            \
    }                                                                                      \
    stream_sync_all();                                                                     \
    auto t1 = std::chrono::high_resolution_clock::now();                                   \
    bench_time =                                                                           \
        1.e6 * std::chrono::duration_cast<std::chrono::duration<double>>(t1 - t0).count(); \
    bench_time = bench_time / iters;                                                       \
  }

namespace {

template <bool is_integral, typename T>
struct uniform_distribution_selector;
template <typename T>
struct uniform_distribution_selector<true, T> {
  using type = typename std::uniform_int_distribution<T>;
};
template <typename T>
struct uniform_distribution_selector<false, T> {
  using type = typename std::uniform_real_distribution<T>;
};
template <typename T>
using uniform_distribution_t =
    typename uniform_distribution_selector<std::is_integral<T>::value, T>::type;

template <typename T>
ncclDataType_t get_nccl_type();
template <>
ncclDataType_t get_nccl_type<float>() {
  return ncclFloat32;
}
template <>
ncclDataType_t get_nccl_type<__half>() {
  return ncclFloat16;
}
template <>
ncclDataType_t get_nccl_type<uint32_t>() {
  return ncclUint32;
}

template <typename T>
bool compare_values(T a, T b, std::false_type const&) {
  T epsilon = 0.1;
  if (a > b) {
    return (a - b) < epsilon;
  } else {
    return (b - a) < epsilon;
  }
  return false;
}

template <typename T>
bool compare_values(T a, T b, std::true_type const&) {
  return (a == b);
}

template <typename T>
bool compare_values(T a, T b) {
  return compare_values<T>(a, b, std::is_integral<T>{});
}

template <typename TypeEmbeddingComp>
struct arTest {
 public:
  arTest(const std::vector<int>& device_list, size_t max_size)
      : num_gpus_(device_list.size()), max_size_(max_size) {
    max_elems_ = max_size_ / sizeof(TypeEmbeddingComp);

    std::vector<std::vector<int>> vvgpu;
    vvgpu.push_back(device_list);

    if (std::is_same<TypeEmbeddingComp, float>::value) {
      use_mixed_precision_ = false;
    } else {
      use_mixed_precision_ = true;
    }

    resource_manager_ = ResourceManagerCore::create(vvgpu, 0, DeviceMap::LOCAL_FIRST);
    collective_manager_ = std::make_shared<CollectiveManager>(resource_manager_);
    collective_manager_->set_ar_comm(AllReduceAlgo::ONESHOT, use_mixed_precision_);
    ar_comm_ = collective_manager_->get_ar_comm();
    init_buffers();
  }

 private:
  size_t num_gpus_;
  size_t max_size_;
  size_t max_elems_;
  bool use_mixed_precision_;
  AllReduceInPlaceComm* ar_comm_;
  std::shared_ptr<ResourceManager> resource_manager_;
  std::shared_ptr<CollectiveManager> collective_manager_;

  std::vector<Tensor2<TypeEmbeddingComp>> h_ar_buff_;
  std::vector<Tensor2<TypeEmbeddingComp>> d_ar_buff_;
  std::vector<Tensor2<TypeEmbeddingComp>> d_ar_buff_ref_;
  std::vector<Tensor2<TypeEmbeddingComp>> h_ar_buff_out_;
  std::vector<Tensor2<TypeEmbeddingComp>> h_ar_buff_out_ref_;

  std::vector<std::shared_ptr<GeneralBuffer2<CudaAllocator>>> dev_bufs_;
  std::vector<std::shared_ptr<GeneralBuffer2<CudaHostAllocator>>> host_bufs_;

  std::vector<size_t> ar_sizes_;
  std::vector<AllReduceInPlaceComm::Handle> ar_handles_;

  void init_buffers() {
    dev_bufs_.resize(num_gpus_);
    host_bufs_.resize(num_gpus_);

    h_ar_buff_.resize(num_gpus_);
    d_ar_buff_.resize(num_gpus_);
    d_ar_buff_ref_.resize(num_gpus_);
    h_ar_buff_out_.resize(num_gpus_);
    h_ar_buff_out_ref_.resize(num_gpus_);

    CudaDeviceContext context;
    for (size_t g = 0; g < num_gpus_; g++) {
      auto& device_list = resource_manager_->get_local_gpu_device_id_list();
      context.set_device(device_list[g]);
      dev_bufs_[g] = GeneralBuffer2<CudaAllocator>::create();
      host_bufs_[g] = GeneralBuffer2<CudaHostAllocator>::create();

      dev_bufs_[g]->reserve({max_elems_}, &d_ar_buff_[g]);
      dev_bufs_[g]->reserve({max_elems_}, &d_ar_buff_ref_[g]);
      dev_bufs_[g]->allocate();

      host_bufs_[g]->reserve({max_elems_}, &h_ar_buff_[g]);
      host_bufs_[g]->reserve({max_elems_}, &h_ar_buff_out_[g]);
      host_bufs_[g]->reserve({max_elems_}, &h_ar_buff_out_ref_[g]);
      host_bufs_[g]->allocate();
    }
  }

  void fill_buffers() {
    // reset output buffers
    for (size_t g = 0; g < num_gpus_; g++) {
      memset(h_ar_buff_out_[g].get_ptr(), 0, max_size_);
      memset(h_ar_buff_out_ref_[g].get_ptr(), 0, max_size_);
    }

    for (size_t g = 0; g < num_gpus_; g++) {
      for (size_t s = 0; s < max_elems_; s++) {
        float tmp_num = s % 31 + 41 / (g + 1);
        *(h_ar_buff_[g].get_ptr() + s) = tmp_num;
        // *(h_ar_buff_[g].get_ptr() + s) = g;
      }
    }

    auto& device_list = resource_manager_->get_local_gpu_device_id_list();
    for (size_t g = 0; g < num_gpus_; g++) {
      HCTR_LIB_THROW(hipSetDevice(device_list[g]));
      HCTR_LIB_THROW(hipMemcpy(d_ar_buff_[g].get_ptr(), h_ar_buff_[g].get_ptr(), max_size_,
                                hipMemcpyHostToDevice));
      HCTR_LIB_THROW(hipMemcpy(d_ar_buff_ref_[g].get_ptr(), h_ar_buff_[g].get_ptr(), max_size_,
                                hipMemcpyHostToDevice));
    }
  }

  void gen_uniform_sizes() {
    for (size_t mysize = 1024; mysize <= max_size_; mysize = mysize * 2) {
      // make sure size is aligned to 16B* num_gpus_
      auto size_aligned = (mysize / (16 * num_gpus_)) * (16 * num_gpus_);
      ar_sizes_.push_back(size_aligned);
    }
  }

  void gen_rand_sizes() {
    int max_rand_sizes = 20;
    int rand_sizes = 0;
    while (rand_sizes < max_rand_sizes) {
      std::default_random_engine generator;
      uniform_distribution_t<size_t> distribution(1, max_size_);
      size_t size = distribution(generator);
      auto size_aligned = (size / (16 * num_gpus_)) * (16 * num_gpus_);
      if (size_aligned > 0) {
        ar_sizes_.push_back(size_aligned);
        rand_sizes++;
      }
    }
  }

  void register_buffers() {
    for (auto& size : ar_sizes_) {
      auto handle = ar_comm_->register_coll();
      ar_handles_.push_back(handle);
      for (size_t g = 0; g < num_gpus_; g++) {
        ar_comm_->set_coll_buf(handle, d_ar_buff_[g].get_ptr(), size, g);
      }
      ar_comm_->register_coll_buf(handle);
    }
  }

  void stream_sync_all() {
    auto& device_list = resource_manager_->get_local_gpu_device_id_list();
    for (size_t g = 0; g < num_gpus_; g++) {
      const auto& local_gpu = resource_manager_->get_local_gpu(g);
      HCTR_LIB_THROW(hipSetDevice(device_list[g]));
      HCTR_LIB_THROW(hipStreamSynchronize(local_gpu->get_stream()));
    }
  }

  void do_nccl_ar(int i) {
    size_t size = ar_sizes_[i];
    auto& device_list = resource_manager_->get_local_gpu_device_id_list();
#pragma omp parallel for num_threads(num_gpus_)
    for (size_t g = 0; g < num_gpus_; g++) {
      const auto& local_gpu = resource_manager_->get_local_gpu(g);
      HCTR_LIB_THROW(hipSetDevice(device_list[g]));
      HCTR_LIB_THROW(ncclAllReduce(
          (const void*)d_ar_buff_ref_[g].get_ptr(), (void*)d_ar_buff_ref_[g].get_ptr(),
          size / sizeof(TypeEmbeddingComp), get_nccl_type<TypeEmbeddingComp>(), ncclSum,
          local_gpu->get_nccl(), local_gpu->get_stream()));
    }
  }

  void do_custom_ar(int i) {
    auto handle = ar_handles_[i];
    auto& device_list = resource_manager_->get_local_gpu_device_id_list();
#pragma omp parallel for num_threads(num_gpus_)
    for (size_t g = 0; g < num_gpus_; g++) {
      const auto& local_gpu = resource_manager_->get_local_gpu(g);
      HCTR_LIB_THROW(hipSetDevice(device_list[g]));
      ar_comm_->all_reduce(handle, local_gpu->get_stream(), g);
    }
  }

  void compare_outputs() {
    auto& device_list = resource_manager_->get_local_gpu_device_id_list();
    for (size_t g = 0; g < num_gpus_; g++) {
      const auto& local_gpu = resource_manager_->get_local_gpu(g);
      HCTR_LIB_THROW(hipSetDevice(device_list[g]));
      HCTR_LIB_THROW(hipMemcpyAsync(h_ar_buff_out_[g].get_ptr(), d_ar_buff_[g].get_ptr(),
                                     max_size_, hipMemcpyDeviceToHost, local_gpu->get_stream()));

      HCTR_LIB_THROW(hipMemcpyAsync(h_ar_buff_out_ref_[g].get_ptr(), d_ar_buff_ref_[g].get_ptr(),
                                     max_size_, hipMemcpyDeviceToHost, local_gpu->get_stream()));
    }
    stream_sync_all();
    for (size_t g = 0; g < num_gpus_; g++) {
      for (size_t e = 0; e < max_elems_; e++) {
        bool match = compare_values(*(h_ar_buff_out_[g].get_ptr() + e),
                                    *(h_ar_buff_out_ref_[g].get_ptr() + e));
        if (!match) {
          size_t my_proc = resource_manager_->get_process_id();
          float expected_num = *(h_ar_buff_out_ref_[g].get_ptr() + e);
          float wrong_num = *(h_ar_buff_out_[g].get_ptr() + e);

          HCTR_LOG_S(DEBUG, WORLD)
              << my_proc << ": Data mismatch at gpu " << g << " element: " << e
              << " expected: " << expected_num << " got: " << wrong_num << std::endl;
        }
      }
    }
  }

 public:
  void test() {
    gen_uniform_sizes();
    gen_rand_sizes();
    register_buffers();
    int repeat = 2;
    for (int r = 0; r < repeat; r++) {
      for (size_t s = 0; s < ar_sizes_.size(); s++) {
        fill_buffers();
        do_nccl_ar(s);
        stream_sync_all();
        do_custom_ar(s);
        stream_sync_all();
        if (std::is_same<TypeEmbeddingComp, float>::value) {
          compare_outputs();
        }
      }
    }
  }

  void perf_test() {
    size_t my_proc = resource_manager_->get_process_id();
    gen_uniform_sizes();
    register_buffers();
    for (size_t s = 0; s < ar_sizes_.size(); s++) {
      double bench_time;
      auto size = ar_sizes_[s];
      TIMEIT(do_custom_ar(s), bench_time);
      if (my_proc == 0) {
        HCTR_LOG_S(DEBUG, WORLD) << size << " " << bench_time << std::endl;
      }
    }
  }
};

template <typename TypeEmbeddingComp>
void test_ar_comm(const std::vector<int>& device_list) {
  const size_t MAX_SIZE = 64 * 1024 * 1024;
  arTest<TypeEmbeddingComp> test(device_list, MAX_SIZE);
  test.test();
}

template <typename TypeEmbeddingComp>
void test_ar_comm_perf(const std::vector<int>& device_list) {
  const size_t MAX_SIZE = 64 * 1024 * 1024;
  arTest<TypeEmbeddingComp> test(device_list, MAX_SIZE);
  test.perf_test();
}

TEST(ar_oneshot_test, half_2gpu) { test_ar_comm<__half>({0, 1}); }
TEST(ar_oneshot_test, half_4gpu) { test_ar_comm<__half>({0, 1, 2, 3}); }
TEST(ar_oneshot_test, half_8gpu) { test_ar_comm<__half>({0, 1, 2, 3, 4, 5, 6, 7}); }
TEST(ar_oneshot_test, float_2gpu) { test_ar_comm<float>({0, 1}); }
TEST(ar_oneshot_test, float_4gpu) { test_ar_comm<float>({0, 1, 2, 3}); }
TEST(ar_oneshot_test, float_8gpu) { test_ar_comm<float>({0, 1, 2, 3, 4, 5, 6, 7}); }
TEST(ar_oneshot_perf, float_2gpu) { test_ar_comm_perf<float>({0, 1}); }
TEST(ar_oneshot_perf, float_4gpu) { test_ar_comm_perf<float>({0, 1, 2, 3}); }
TEST(ar_oneshot_perf, float_8gpu) { test_ar_comm_perf<float>({0, 1, 2, 3, 4, 5, 6, 7}); }

}  // namespace

#endif
