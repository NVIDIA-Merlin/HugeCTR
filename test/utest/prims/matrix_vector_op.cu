#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <utest/prims/matrix_vector_op.h>

#include <linalg/matrix_vector_op.cuh>
#include <linalg/unary_op.cuh>
#include <random/rng.cuh>

namespace MLCommon {
namespace LinAlg {
//! we intentionally avoid using third_party/cuml/cpp/test/prims/test_utils.h
//! because its misuse of shared_ptr of array. We define used util here
// CompareApprox
template <typename T>
struct CompareApprox {
  CompareApprox(T eps_) : eps(eps_) {}
  bool operator()(const T &a, const T &b) const {
    T diff = abs(a - b);
    T m = std::max(abs(a), abs(b));
    T ratio = diff >= eps ? diff / m : diff;

    return (ratio <= eps);
  }

 private:
  T eps;
};
// match
template <typename T, typename L>
::testing::AssertionResult devArrMatch(const T *expected, const T *actual, size_t size,
                                       L eq_compare, hipStream_t stream = 0) {
  std::shared_ptr<T[]> exp_h(new T[size]);
  std::shared_ptr<T[]> act_h(new T[size]);
  updateHost<T>(exp_h.get(), expected, size, stream);
  updateHost<T>(act_h.get(), actual, size, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  for (size_t i(0); i < size; ++i) {
    auto exp = exp_h.get()[i];
    auto act = act_h.get()[i];
    if (!eq_compare(exp, act)) {
      return ::testing::AssertionFailure()
             << "actual=" << act << " != expected=" << exp << " @" << i;
    }
  }
  return ::testing::AssertionSuccess();
}

template <typename T, typename IdxType = int>
struct MatVecOpInputs {
  T tolerance;
  IdxType rows, cols;
  bool rowMajor, bcastAlongRows, useTwoVectors;
  unsigned long long int seed;
};

template <typename T, typename IdxType>
::std::ostream &operator<<(::std::ostream &os, const MatVecOpInputs<T, IdxType> &dims) {
  return os;
}

// Or else, we get the following compilation error
// for an extended __device__ lambda cannot have private or protected access
// within its class
template <typename T, typename IdxType>
void matrixVectorOpLaunch1(T *out, const T *in, const T *vec1, IdxType D, IdxType N, bool rowMajor,
                           bool bcastAlongRows, hipStream_t stream) {
  matrixVectorOp(
      out, in, vec1, D, N, rowMajor, bcastAlongRows,
      [] __device__(T a, T b) {
        T in = a + b;
        return (in < 0) ? 0 : in;
      },
      stream);
}

template <typename T, typename IdxType>
void matrixVectorOpLaunch2(T *out, const T *in, const T *vec1, IdxType D, IdxType N, bool rowMajor,
                           bool bcastAlongRows, hipStream_t stream) {
  matrixVectorOp(
      out, in, vec1, D, N, rowMajor, bcastAlongRows, [] __device__(T a, T b) { return a + b; },
      stream);

  unaryOp(
      out, out, D * N, [] __device__(T in) { return (in < 0) ? 0 : in; }, stream);
}

template <typename T, typename IdxType>
class MatVecOpTest : public ::testing::TestWithParam<MatVecOpInputs<T, IdxType>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<MatVecOpInputs<T, IdxType>>::GetParam();
    Random::Rng r(params.seed);
    IdxType N = params.rows, D = params.cols;
    IdxType len = N * D;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate(in, len);
    allocate(out_ref, len);
    allocate(out, len);
    IdxType vecLen = params.bcastAlongRows ? D : N;
    allocate(vec1, vecLen);
    allocate(vec2, vecLen);
    r.uniform(in, len, (T)-1.0, (T)1.0, stream);
    r.uniform(vec1, vecLen, (T)-1.0, (T)1.0, stream);
    r.uniform(vec2, vecLen, (T)-1.0, (T)1.0, stream);

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start));
    matrixVectorOpLaunch1(out_ref, in, vec1, D, N, params.rowMajor, params.bcastAlongRows, stream);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    // HCTR_LOG(INFO, WORLD, "Fused: %f\n", milliseconds);

    CUDA_CHECK(hipEventRecord(start));
    matrixVectorOpLaunch2(out, in, vec1, D, N, params.rowMajor, params.bcastAlongRows, stream);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    // HCTR_LOG(INFO, WORLD, "Normal: %f\n", milliseconds);

    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(vec1));
    CUDA_CHECK(hipFree(vec2));
    CUDA_CHECK(hipFree(out));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(in));
  }

 protected:
  MatVecOpInputs<T, IdxType> params;
  T *in, *out, *out_ref, *vec1, *vec2;
};

const std::vector<MatVecOpInputs<float, int>> inputsf_i32 = {
    {0.00001f, 10024, 32, true, true, false, 1234ULL},
    {0.00001f, 10024, 64, true, true, false, 1234ULL},
    {0.00001f, 10024, 32, true, false, false, 1234ULL},
    {0.00001f, 10024, 64, true, false, false, 1234ULL},
    {0.00001f, 10024, 32, false, true, false, 1234ULL},
    {0.00001f, 10024, 64, false, true, false, 1234ULL},
    {0.00001f, 10024, 32, false, false, false, 1234ULL},
    {0.00001f, 10024, 64, false, false, false, 1234ULL},
    {0.00001f, 10024, 32, true, true, true, 1234ULL},
    {0.00001f, 10024, 64, true, true, true, 1234ULL},
    {0.00001f, 10024, 32, true, false, true, 1234ULL},
    {0.00001f, 10024, 64, true, false, true, 1234ULL},
    {0.00001f, 10024, 32, false, true, true, 1234ULL},
    {0.00001f, 10024, 64, false, true, true, 1234ULL},
    {0.00001f, 10024, 32, false, false, true, 1234ULL},
    {0.00001f, 10024, 64, false, false, true, 1234ULL}};

typedef MatVecOpTest<float, int> MatVecOpTestF_i32;
TEST_P(MatVecOpTestF_i32, Result) {
  ASSERT_TRUE(
      devArrMatch(out_ref, out, params.rows * params.cols, CompareApprox<float>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(MatVecOpTests, MatVecOpTestF_i32, ::testing::ValuesIn(inputsf_i32));

const std::vector<MatVecOpInputs<float, size_t>> inputsf_i64 = {
    {0.00001f, 2500, 250, false, false, false, 1234ULL},
    {0.00001f, 2500, 250, false, false, true, 1234ULL}};

typedef MatVecOpTest<float, size_t> MatVecOpTestF_i64;
TEST_P(MatVecOpTestF_i64, Result) {
  ASSERT_TRUE(
      devArrMatch(out_ref, out, params.rows * params.cols, CompareApprox<float>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(MatVecOpTests, MatVecOpTestF_i64, ::testing::ValuesIn(inputsf_i64));

const std::vector<MatVecOpInputs<double, int>> inputsd_i32 = {
    {0.0000001, 10024, 32, true, true, false, 1234ULL},
    {0.0000001, 10024, 64, true, true, false, 1234ULL},
    {0.0000001, 10024, 32, true, false, false, 1234ULL},
    {0.0000001, 10024, 64, true, false, false, 1234ULL},
    {0.0000001, 10024, 32, false, true, false, 1234ULL},
    {0.0000001, 10024, 64, false, true, false, 1234ULL},
    {0.0000001, 10024, 32, false, false, false, 1234ULL},
    {0.0000001, 10024, 64, false, false, false, 1234ULL},
    {0.0000001, 10024, 32, true, true, true, 1234ULL},
    {0.0000001, 10024, 64, true, true, true, 1234ULL},
    {0.0000001, 10024, 32, true, false, true, 1234ULL},
    {0.0000001, 10024, 64, true, false, true, 1234ULL},
    {0.0000001, 10024, 32, false, true, true, 1234ULL},
    {0.0000001, 10024, 64, false, true, true, 1234ULL},
    {0.0000001, 10024, 32, false, false, true, 1234ULL},
    {0.0000001, 10024, 64, false, false, true, 1234ULL}};

typedef MatVecOpTest<double, int> MatVecOpTestD_i32;
TEST_P(MatVecOpTestD_i32, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.rows * params.cols,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(MatVecOpTests, MatVecOpTestD_i32, ::testing::ValuesIn(inputsd_i32));

const std::vector<MatVecOpInputs<double, size_t>> inputsd_i64 = {
    {0.0000001, 2500, 250, false, false, false, 1234ULL},
    {0.0000001, 2500, 250, false, false, true, 1234ULL}};

typedef MatVecOpTest<double, size_t> MatVecOpTestD_i64;
TEST_P(MatVecOpTestD_i64, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.rows * params.cols,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(MatVecOpTests, MatVecOpTestD_i64, ::testing::ValuesIn(inputsd_i64));

}  // end namespace LinAlg
}  // end namespace MLCommon
