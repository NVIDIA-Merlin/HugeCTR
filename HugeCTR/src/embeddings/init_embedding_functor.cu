#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/data_simulator.hpp"
#include "HugeCTR/include/embeddings/sparse_embedding_functors.hpp"
#include "HugeCTR/include/general_buffer2.hpp"
#include "HugeCTR/include/utils.hpp"

namespace HugeCTR {

void SparseEmbeddingFunctors::init_embedding_per_gpu(size_t gid, size_t total_gpu_count,
                                                     const std::vector<size_t> &slot_sizes,
                                                     size_t embedding_vec_size,
                                                     Tensors2<float> &embedding_tables,
                                                     Tensor2<size_t> &slot_ids,
                                                     const GPUResource &gpu_resource) {
  CudaDeviceContext context(gpu_resource.get_device_id());

  size_t *slot_ids_ptr = slot_ids.get_ptr();

  size_t key_offset = 0;
  size_t value_index_offset = 0;
  for (size_t i = 0, j = 0; i < slot_sizes.size(); i++) {
    size_t slot_size = slot_sizes[i];
    if ((i % total_gpu_count) == gid) {
      MESSAGE_("gpu" + std::to_string(gid) + " start to init embedding of slot" +
               std::to_string(i) + " , slot_size=" + std::to_string(slot_size) +
               ", key_offset=" + std::to_string(key_offset) +
               ", value_index_offset=" + std::to_string(value_index_offset));

      float up_bound = sqrt(1.f / slot_size);
      HugeCTR::UniformGenerator::fill(embedding_tables[j++], -up_bound, up_bound, gpu_resource);

      memset_const(slot_ids_ptr, i, slot_size, gpu_resource.get_stream());

      value_index_offset += slot_size;
      slot_ids_ptr += slot_size;
    }
    key_offset += slot_size;
  }
}

}  // namespace HugeCTR