#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/embeddings/sparse_embedding_functors.hpp"
#include "HugeCTR/include/utils.hpp"
namespace HugeCTR {
template <typename TypeEmbeddingComp>
std::vector<Tensors2<TypeEmbeddingComp>> SparseEmbeddingFunctors::get_opt_states(
    const std::vector<OptimizerTensor<TypeEmbeddingComp>>& opt_tensors_, Optimizer_t optimizer_type,
    size_t local_gpu_count) {
  std::vector<Tensors2<TypeEmbeddingComp>> opt_states;
  opt_states.resize(local_gpu_count);

  for (size_t i = 0; i < local_gpu_count; ++i) {
    switch (optimizer_type) {
      case Optimizer_t::Adam:  // adam
      {
        opt_states[i].push_back(opt_tensors_[i].opt_m_tensors_);
        opt_states[i].push_back(opt_tensors_[i].opt_v_tensors_);
        break;
      }

      case Optimizer_t::AdaGrad:  // nesterov
      {
        opt_states[i].push_back(opt_tensors_[i].opt_accm_tensors_);
        break;
      }
      case Optimizer_t::MomentumSGD:  // momentum_sgd
      {
        opt_states[i].push_back(opt_tensors_[i].opt_momentum_tensors_);
        break;
      }

      case Optimizer_t::Nesterov:  // nesterov
      {
        opt_states[i].push_back(opt_tensors_[i].opt_accm_tensors_);
        break;
      }

      case Optimizer_t::SGD:
        break;

      default:
        throw std::runtime_error(
            std::string("[HCDEBUG][ERROR] Runtime error: Invalid optimizer type\n"));
    }
  }

  std::vector<Tensors2<TypeEmbeddingComp>> transpose_opt_states;
  if (opt_states[0].size() > 0) {
    transpose_opt_states.resize(opt_states[0].size());
    for (size_t i = 0; i < opt_states[0].size(); ++i) {
      transpose_opt_states[i].resize(opt_states.size());
      for (size_t j = 0; j < opt_states.size(); ++j) {
        transpose_opt_states[i][j] = opt_states[j][i];
      }
    }
  }
  return transpose_opt_states;
}

template <typename TypeEmbeddingComp>
void SparseEmbeddingFunctors::dump_opt_states(
    std::ofstream& stream, std::string& write_path, DataSourceParams data_source_params,
    const ResourceManager& resource_manager, std::vector<Tensors2<TypeEmbeddingComp>>& opt_states) {
  size_t local_gpu_count = resource_manager.get_local_gpu_count();

  bool hdfs_append_flag = false;
  CudaDeviceContext context;
  for (auto& opt_state : opt_states) {
    size_t total_size = 0;
    for (size_t id = 0; id < local_gpu_count; id++) {
      total_size += opt_state[id].get_size_in_bytes();
    }
    size_t max_size = total_size;

#ifdef ENABLE_MPI
    bool is_master_process = resource_manager.is_master_process();
    CK_MPI_THROW_(MPI_Reduce(is_master_process ? MPI_IN_PLACE : &max_size, &max_size,
                             sizeof(size_t), MPI_CHAR, MPI_MAX,
                             resource_manager.get_master_process_id(), MPI_COMM_WORLD));
#endif

    std::unique_ptr<char[]> h_opt_state(new char[max_size]);
    size_t offset = 0;
    for (size_t id = 0; id < local_gpu_count; id++) {
      size_t local_size = opt_state[id].get_size_in_bytes();
      auto& local_gpu = resource_manager.get_local_gpu(id);
      context.set_device(local_gpu->get_device_id());
      CK_CUDA_THROW_(hipMemcpyAsync(h_opt_state.get() + offset, opt_state[id].get_ptr(),
                                     local_size, hipMemcpyDeviceToHost, local_gpu->get_stream()));
      offset += local_size;
    }
    sync_all_gpus(resource_manager);
    int pid = resource_manager.get_process_id();
    if (resource_manager.is_master_process()) {
      MESSAGE_("Rank" + std::to_string(pid) + ": Write optimzer state to file", true);
      if (data_source_params.use_hdfs) {
        HdfsService hs = HdfsService(data_source_params.namenode, data_source_params.port);
        if (!hdfs_append_flag) {
          hs.write(write_path, h_opt_state.get(), total_size, true);
          hdfs_append_flag = true;
        } else {
          hs.write(write_path, h_opt_state.get(), total_size, false);
        }
      } else {
        stream.write(h_opt_state.get(), total_size);
      }
    }
#ifdef ENABLE_MPI
    else {
      MESSAGE_("Rank" + std::to_string(pid) + ": Send optimzer state to master node", true);
      int tag = (pid << 8) | 0xBA;
      CK_MPI_THROW_(MPI_Send(h_opt_state.get(), total_size, MPI_CHAR,
                             resource_manager.get_master_process_id(), tag, MPI_COMM_WORLD));
    }

    if (resource_manager.is_master_process()) {
      for (int r = 1; r < resource_manager.get_num_process(); r++) {
        MESSAGE_("Rank" + std::to_string(pid) + ": Recv optimzer state from rank" +
                     std::to_string(r) + ", and write to file",
                 true);
        int tag = (r << 8) | 0xBA;
        int recv_size = 0;
        MPI_Status status;
        CK_MPI_THROW_(MPI_Probe(r, tag, MPI_COMM_WORLD, &status));
        CK_MPI_THROW_(MPI_Get_count(&status, MPI_CHAR, &recv_size));
        CK_MPI_THROW_(MPI_Recv(h_opt_state.get(), recv_size, MPI_CHAR, r, tag, MPI_COMM_WORLD,
                               MPI_STATUS_IGNORE));
        if (data_source_params.use_hdfs) {
          HdfsService hs = HdfsService(data_source_params.namenode, data_source_params.port);
          if (!hdfs_append_flag) {
            hs.write(write_path, h_opt_state.get(), recv_size, true);
            hdfs_append_flag = true;
          } else {
            hs.write(write_path, h_opt_state.get(), recv_size, false);
          }
        } else {
          stream.write(h_opt_state.get(), recv_size);
        }
      }
    }
#endif
    MESSAGE_("Done");
  }
}

template <typename TypeEmbeddingComp>
void SparseEmbeddingFunctors::load_opt_states(
    std::ifstream& stream, const ResourceManager& resource_manager,
    std::vector<Tensors2<TypeEmbeddingComp>>& opt_states) {
  size_t local_gpu_count = resource_manager.get_local_gpu_count();

  CudaDeviceContext context;
  for (auto& opt_state : opt_states) {
    size_t total_size = 0;
    for (size_t id = 0; id < local_gpu_count; id++) {
      total_size += opt_state[id].get_size_in_bytes();
    }
    int pid = resource_manager.get_process_id();

    auto h2d_op = [&opt_state, &resource_manager, &context](char* h_opt_state) {
      size_t offset = 0;
      for (size_t id = 0; id < resource_manager.get_local_gpu_count(); id++) {
        size_t local_size = opt_state[id].get_size_in_bytes();
        auto& local_gpu = resource_manager.get_local_gpu(id);
        context.set_device(local_gpu->get_device_id());
        CK_CUDA_THROW_(hipMemcpyAsync(opt_state[id].get_ptr(), h_opt_state + offset, local_size,
                                       hipMemcpyHostToDevice, local_gpu->get_stream()));
        offset += local_size;
      }
    };

    std::unique_ptr<size_t[]> proc_sizes(new size_t[resource_manager.get_num_process()]);
    proc_sizes[0] = total_size;
#ifdef ENABLE_MPI
    CK_MPI_THROW_(MPI_Gather(&total_size, sizeof(size_t), MPI_CHAR, proc_sizes.get(),
                             sizeof(size_t), MPI_CHAR, 0, MPI_COMM_WORLD));
#endif

    if (resource_manager.is_master_process()) {
      size_t sum_sizes = 0;
      size_t max_size = 0;
      for (int i = 0; i < resource_manager.get_num_process(); ++i) {
        sum_sizes += proc_sizes[i];
        if (proc_sizes[i] > max_size) {
          max_size = proc_sizes[i];
        }
      }
      size_t cur_pos = stream.tellg();
      stream.seekg(0, stream.end);
      size_t remaining_file_size = stream.tellg() - cur_pos;
      if (remaining_file_size < sum_sizes) {
        CK_THROW_(Error_t::WrongInput,
                  "optimizer state file size is incompatible with the embedding!");
      }
      stream.seekg(cur_pos);

      std::unique_ptr<char[]> h_opt_state(new char[max_size]);
      MESSAGE_("Rank" + std::to_string(pid) + ": Read optimzer state from file", true);
      stream.read(h_opt_state.get(), total_size);

      h2d_op(h_opt_state.get());
      sync_all_gpus(resource_manager);

#ifdef ENABLE_MPI
      for (int r = 1; r < resource_manager.get_num_process(); r++) {
        MESSAGE_("Rank" + std::to_string(pid) + ": Read from file" +
                     ", and send optimzer state to rank" + std::to_string(r),
                 true);
        stream.read(h_opt_state.get(), proc_sizes[r]);
        int tag = (r << 8) | 0xAB;
        CK_MPI_THROW_(MPI_Send(h_opt_state.get(), proc_sizes[r], MPI_CHAR, r, tag, MPI_COMM_WORLD));
      }
#endif
    }
#ifdef ENABLE_MPI
    else {
      MESSAGE_("Rank" + std::to_string(pid) + ": Recv optimzer state from master node" +
                   ", and write to GPUs",
               true);
      int mid = resource_manager.get_master_process_id();
      int tag = (pid << 8) | 0xAB;
      int recv_size = 0;
      MPI_Status status;
      CK_MPI_THROW_(MPI_Probe(mid, tag, MPI_COMM_WORLD, &status));
      CK_MPI_THROW_(MPI_Get_count(&status, MPI_CHAR, &recv_size));
      std::unique_ptr<char[]> h_opt_state(new char[recv_size]);
      stream.read(h_opt_state.get(), recv_size);
      CK_MPI_THROW_(MPI_Recv(h_opt_state.get(), recv_size, MPI_CHAR, mid, tag, MPI_COMM_WORLD,
                             MPI_STATUS_IGNORE));

      h2d_op(h_opt_state.get());
      sync_all_gpus(resource_manager);
    }
#endif
    MESSAGE_("Done");
  }
}
template std::vector<Tensors2<float>> SparseEmbeddingFunctors::get_opt_states(
    const std::vector<OptimizerTensor<float>>& opt_tensors_, Optimizer_t optimizer_type,
    size_t local_gpu_count);

template std::vector<Tensors2<__half>> SparseEmbeddingFunctors::get_opt_states(
    const std::vector<OptimizerTensor<__half>>& opt_tensors_, Optimizer_t optimizer_type,
    size_t local_gpu_count);

template void SparseEmbeddingFunctors::dump_opt_states<float>(
    std::ofstream& stream, std::string& write_path, DataSourceParams data_source_params,
    const ResourceManager& resource_manager, std::vector<Tensors2<float>>& opt_states);

template void SparseEmbeddingFunctors::dump_opt_states<__half>(
    std::ofstream& stream, std::string& write_path, DataSourceParams data_source_params,
    const ResourceManager& resource_manager, std::vector<Tensors2<__half>>& opt_states);

template void SparseEmbeddingFunctors::load_opt_states<float>(
    std::ifstream& stream, const ResourceManager& resource_manager,
    std::vector<Tensors2<float>>& opt_states);

template void SparseEmbeddingFunctors::load_opt_states<__half>(
    std::ifstream& stream, const ResourceManager& resource_manager,
    std::vector<Tensors2<__half>>& opt_states);

}  // namespace HugeCTR
