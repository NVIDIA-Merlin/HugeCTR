/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/embeddings/sparse_embedding_functors.hpp"
#include "HugeCTR/include/utils.hpp"

namespace HugeCTR {
/**
 * collection communication: all_reduce.
 * @param send_count the count of elements will be sent.
 * @param send_tensors the send tensors of multi GPUs.
 * @param recv_tensors the recv tensors of multi GPUs.
 * @param device_resources all gpus device resources.
 * @param context gpu device context, for switching device.
 */
template <typename TypeHashKey>
void SparseEmbeddingFunctors::all_reduce(size_t send_count,
                                         const Tensors2<TypeHashKey> &send_tensors,
                                         Tensors2<TypeHashKey> &recv_tensors,
                                         const ResourceManager &resource_manager) {
  size_t local_gpu_count = resource_manager.get_local_gpu_count();
  size_t total_gpu_count = resource_manager.get_global_gpu_count();

  // need to know the type of Type here
  ncclDataType_t type;
  switch (sizeof(TypeHashKey)) {
    case 4:
      type = ncclUint32;
      break;
    case 8:
      type = ncclUint64;
      break;
    default:
      CK_THROW_(Error_t::WrongInput, "Error: Type not support by now");
  }

  // for multi GPUs, use NCCL to do all_reduce (supporting multi-node GPU servers)
  if (total_gpu_count > 1) {
    CK_NCCL_THROW_(ncclGroupStart());
    for (size_t id = 0; id < local_gpu_count; id++) {
      const auto &local_gpu = resource_manager.get_local_gpu(id);
      CK_NCCL_THROW_(ncclAllReduce(send_tensors[id].get_ptr(), recv_tensors[id].get_ptr(),
                                   send_count, type, ncclSum, local_gpu->get_nccl(),
                                   local_gpu->get_stream()));
    }
    CK_NCCL_THROW_(ncclGroupEnd());
  }
  // for single GPU, just do memcpyD2D
  else {  // total_gpu_count == 1
    const auto &local_gpu = resource_manager.get_local_gpu(0);
    CudaDeviceContext context(local_gpu->get_device_id());
    CK_CUDA_THROW_(hipMemcpyAsync(recv_tensors[0].get_ptr(), send_tensors[0].get_ptr(),
                                   send_count * sizeof(TypeHashKey), hipMemcpyDeviceToDevice,
                                   local_gpu->get_stream()));
  }

  return;
}

template void SparseEmbeddingFunctors::all_reduce<unsigned int>(
    size_t send_count, const Tensors2<unsigned int> &send_tensors,
    Tensors2<unsigned int> &recv_tensors, const ResourceManager &resource_manager);

template void SparseEmbeddingFunctors::all_reduce<long long>(
    size_t send_count, const Tensors2<long long> &send_tensors, Tensors2<long long> &recv_tensors,
    const ResourceManager &resource_manager);

}  // namespace HugeCTR