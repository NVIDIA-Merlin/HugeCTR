#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <thrust/execution_policy.h>
#include <thrust/sort.h>

#include <algorithm>
#include <hipcub/hipcub.hpp>
#include <filesystem>
#include <numeric>

#include "HugeCTR/include/data_simulator.hpp"
#include "HugeCTR/include/embeddings/distributed_slot_sparse_embedding_hash.hpp"
#include "HugeCTR/include/utils.cuh"

namespace HugeCTR {

namespace distributed_embedding_kernels {

template <typename Key>
__global__ void select_rowoffset(const Key *rowoffset_ptr, const size_t num, const Key *value_ptr,
                                 Key *select_rowoffset_ptr, size_t global_id, size_t global_num) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid == 0) {
    select_rowoffset_ptr[tid] = 0;
  }
  if (tid < num) {
    Key count = 0;
    Key start = rowoffset_ptr[tid];
    for (Key i = start; i < rowoffset_ptr[tid + 1]; ++i) {
      if ((value_ptr[i] % global_num) == global_id) {
        ++count;
      }
    }
    select_rowoffset_ptr[tid + 1] = count;
  }
}

template <typename Key>
struct HashOp {
  size_t global_id;
  size_t global_num;

  __host__ __device__ __forceinline__ HashOp(size_t global_id_, size_t global_num_)
      : global_id(global_id_), global_num(global_num_) {}

  __device__ __forceinline__ bool operator()(const Key &key) const {
    return ((size_t)key % global_num) == global_id;
  }
};

}  // namespace distributed_embedding_kernels

template <typename TypeHashKey>
DistributedFilterKeyStorage<TypeHashKey>::DistributedFilterKeyStorage(
    const std::shared_ptr<GeneralBuffer2<CudaAllocator>> &buf, size_t max_nnz,
    size_t rowoffset_count, size_t global_id, size_t global_num) {
  buf->reserve({1}, &value_select_num);
  // select value
  {
    distributed_embedding_kernels::HashOp<TypeHashKey> select_op{global_id, global_num};
    size_t size_in_bytes = 0;
    hipcub::DeviceSelect::If(nullptr, size_in_bytes, (TypeHashKey *)nullptr, (TypeHashKey *)nullptr,
                          (size_t *)nullptr, max_nnz, select_op);
    buf->reserve({size_in_bytes}, &temp_value_select_storage);
  }

  // count rowoffset
  {
    size_t size_in_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(nullptr, size_in_bytes, (TypeHashKey *)nullptr,
                                  (TypeHashKey *)nullptr, rowoffset_count);
    buf->reserve({size_in_bytes}, &temp_rowoffset_select_scan_storage);
  }
  buf->reserve({rowoffset_count}, &rowoffset_select);
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void DistributedSlotSparseEmbeddingHash<TypeHashKey, TypeEmbeddingComp>::filter_keys_per_gpu(
    bool is_train, size_t id, size_t global_id, size_t global_num) {
  const SparseTensor<TypeHashKey> &all_gather_key = embedding_data_.get_input_keys(is_train)[id];
  Tensor2<TypeHashKey> rowoffset_tensor = embedding_data_.get_row_offsets_tensors(is_train)[id];
  Tensor2<TypeHashKey> value_tensor = embedding_data_.get_value_tensors(is_train)[id];
  std::shared_ptr<size_t> nnz_ptr = embedding_data_.get_nnz_array(is_train)[id];
  auto &filter_keys_storage = filter_keys_storage_[id];

  auto &stream = embedding_data_.get_local_gpu(id).get_stream();

  if (all_gather_key.get_dimensions().size() != 2) {
    CK_THROW_(Error_t::WrongInput, "distributed embedding all gather key dimension != 2");
  }
  size_t batch_size = embedding_data_.embedding_params_.get_batch_size(is_train);
  size_t slot_num = (all_gather_key.rowoffset_count() - 1) / batch_size;
  size_t rowoffset_num = batch_size * slot_num + 1;
  size_t rowoffset_num_without_zero = rowoffset_num - 1;
  if (rowoffset_tensor.get_num_elements() != rowoffset_num) {
    std::cout << rowoffset_tensor.get_num_elements() << " " << rowoffset_num << std::endl;
    CK_THROW_(Error_t::WrongInput, "filter rowoffset size not match.");
  }

  // select value
  {
    distributed_embedding_kernels::HashOp<TypeHashKey> select_op{global_id, global_num};

    size_t size_in_bytes = filter_keys_storage.temp_value_select_storage.get_size_in_bytes();
    hipcub::DeviceSelect::If(filter_keys_storage.temp_value_select_storage.get_ptr(), size_in_bytes,
                          all_gather_key.get_value_ptr(), value_tensor.get_ptr(),
                          filter_keys_storage.value_select_num.get_ptr(), all_gather_key.nnz(),
                          select_op, stream);
  }

  // select rowoffset
  {
    hipMemsetAsync(filter_keys_storage.rowoffset_select.get_ptr(), 0,
                    filter_keys_storage.rowoffset_select.get_size_in_bytes(), stream);
    {
      constexpr int block_size = 512;
      int grid_size = (rowoffset_num_without_zero - 1) / block_size + 1;
      distributed_embedding_kernels::select_rowoffset<<<grid_size, block_size, 0, stream>>>(
          all_gather_key.get_rowoffset_ptr(), rowoffset_num_without_zero,
          all_gather_key.get_value_ptr(), filter_keys_storage.rowoffset_select.get_ptr(), global_id,
          global_num);
    }
    {
      size_t size_in_bytes =
          filter_keys_storage.temp_rowoffset_select_scan_storage.get_size_in_bytes();
      hipcub::DeviceScan::InclusiveSum(
          filter_keys_storage.temp_rowoffset_select_scan_storage.get_ptr(), size_in_bytes,
          filter_keys_storage.rowoffset_select.get_ptr(), rowoffset_tensor.get_ptr(), rowoffset_num,
          stream);
    }

    // select nnz
    hipMemcpyAsync(nnz_ptr.get(), filter_keys_storage.value_select_num.get_ptr(), sizeof(size_t),
                    hipMemcpyDeviceToHost, stream);
  }
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
DistributedSlotSparseEmbeddingHash<TypeHashKey, TypeEmbeddingComp>::
    DistributedSlotSparseEmbeddingHash(
        const Tensors2<TypeHashKey> &train_row_offsets_tensors,
        const Tensors2<TypeHashKey> &train_value_tensors,
        const std::vector<std::shared_ptr<size_t>> &train_nnz_array,
        const Tensors2<TypeHashKey> &evaluate_row_offsets_tensors,
        const Tensors2<TypeHashKey> &evaluate_value_tensors,
        const std::vector<std::shared_ptr<size_t>> &evaluate_nnz_array,
        const SparseEmbeddingHashParams &embedding_params,
        const std::shared_ptr<ResourceManager> &resource_manager)
    : embedding_data_(train_row_offsets_tensors, train_value_tensors, train_nnz_array,
                      evaluate_row_offsets_tensors, evaluate_value_tensors, evaluate_nnz_array,
                      Embedding_t::DistributedSlotSparseEmbeddingHash, embedding_params,
                      resource_manager) {
  embedding_data_.embedding_params_.is_data_parallel =
      false;  // this ctor is only used for embedding plugin
  try {
    // CAUSION: can not decide how many <key,value> pairs in each GPU, because the GPU
    // distribution is computed by (key%gpu_count). In order to not allocate the total size of
    // hash table on each GPU, meanwhile get a better performance by a unfull hash table, the
    // users need to set the param "load_factor"(load_factor<1).
    max_vocabulary_size_per_gpu_ = embedding_data_.embedding_params_.max_vocabulary_size_per_gpu;
    max_vocabulary_size_ = max_vocabulary_size_per_gpu_ *
                           embedding_data_.get_resource_manager().get_global_gpu_count();

    MESSAGE_("max_vocabulary_size_per_gpu_=" + std::to_string(max_vocabulary_size_per_gpu_));
    CudaDeviceContext context;
    for (size_t id = 0; id < embedding_data_.get_resource_manager().get_local_gpu_count(); id++) {
      context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

      // new GeneralBuffer objects
      const std::shared_ptr<GeneralBuffer2<CudaAllocator>> &buf = embedding_data_.get_buffer(id);
      embedding_optimizers_.emplace_back(max_vocabulary_size_per_gpu_,
                                         embedding_data_.embedding_params_, buf);

      // new hash table value vectors
      {
        Tensor2<float> tensor;
        buf->reserve(
            {max_vocabulary_size_per_gpu_, embedding_data_.embedding_params_.embedding_vec_size},
            &tensor);
        hash_table_value_tensors_.push_back(tensor);
      }

      // new hash table value_index that get() from HashTable
      {
        Tensor2<size_t> tensor;
        buf->reserve({1, embedding_data_.embedding_params_.get_universal_batch_size() *
                             embedding_data_.embedding_params_.max_feature_num},
                     &tensor);
        hash_value_index_tensors_.push_back(tensor);
      }

      // new embedding features reduced by hash table values(results of forward)
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_universal_batch_size() *
                          embedding_data_.embedding_params_.slot_num,
                      embedding_data_.embedding_params_.embedding_vec_size},
                     &tensor);
        embedding_feature_tensors_.push_back(tensor);
      }

      // new wgrad used by backward
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_batch_size(true) *
                          embedding_data_.embedding_params_.slot_num,
                      embedding_data_.embedding_params_.embedding_vec_size},
                     &tensor);
        wgrad_tensors_.push_back(tensor);
      }

      // new temp tensors used by update_params
      {
        Tensor2<TypeHashKey> tensor;
        buf->reserve({1, embedding_data_.embedding_params_.get_universal_batch_size() *
                                 embedding_data_.embedding_params_.slot_num +
                             1},
                     &tensor);
        row_offset_allreduce_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_universal_batch_size() *
                          embedding_data_.embedding_params_.slot_num,
                      embedding_data_.embedding_params_.embedding_vec_size},
                     &tensor);
        utest_forward_temp_tensors_.push_back(tensor);
      }
// init GenenralBuffers to do real allocation
#ifndef NDEBUG
      std::cout << " max_feature_num_:" << embedding_data_.embedding_params_.max_feature_num
                << std::endl;
#endif
    }

    hash_tables_.resize(embedding_data_.get_resource_manager().get_local_gpu_count());
#pragma omp parallel num_threads(embedding_data_.get_resource_manager().get_local_gpu_count())
    {
      size_t id = omp_get_thread_num();
      CudaDeviceContext context(embedding_data_.get_local_gpu(id).get_device_id());
      // construct HashTable object: used to store hash table <key, value_index>
      hash_tables_[id].reset(new NvHashTable(max_vocabulary_size_per_gpu_));
      embedding_data_.get_buffer(id)->allocate();
    }

    for (size_t id = 0; id < embedding_data_.get_resource_manager().get_local_gpu_count(); id++) {
      context.set_device(embedding_data_.get_local_gpu(id).get_device_id());
      embedding_optimizers_[id].initialize(embedding_data_.get_local_gpu(id));

    }  // end of for(int id = 0; id < embedding_data_.get_local_gpu_count(); id++)

    functors_.sync_all_gpus(embedding_data_.get_resource_manager());

  } catch (const std::runtime_error &rt_err) {
    std::cerr << rt_err.what() << std::endl;
    throw;
  }

  return;
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
DistributedSlotSparseEmbeddingHash<TypeHashKey, TypeEmbeddingComp>::
    DistributedSlotSparseEmbeddingHash(const SparseTensors<TypeHashKey> &train_keys,
                                       const SparseTensors<TypeHashKey> &evaluate_keys,
                                       const SparseEmbeddingHashParams &embedding_params,
                                       const std::shared_ptr<ResourceManager> &resource_manager)
    : embedding_data_(Embedding_t::DistributedSlotSparseEmbeddingHash, train_keys, evaluate_keys,
                      embedding_params, resource_manager) {
  try {
    // CAUSION: can not decide how many <key,value> pairs in each GPU, because the GPU
    // distribution is computed by (key%gpu_count). In order to not allocate the total size of
    // hash table on each GPU, meanwhile get a better performance by a unfull hash table, the
    // users need to set the param "load_factor"(load_factor<1).
    max_vocabulary_size_per_gpu_ = embedding_data_.embedding_params_.max_vocabulary_size_per_gpu;
    max_vocabulary_size_ = max_vocabulary_size_per_gpu_ *
                           embedding_data_.get_resource_manager().get_global_gpu_count();

    MESSAGE_("max_vocabulary_size_per_gpu_=" + std::to_string(max_vocabulary_size_per_gpu_));
    CudaDeviceContext context;
    for (size_t id = 0; id < embedding_data_.get_resource_manager().get_local_gpu_count(); id++) {
      context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

      // new GeneralBuffer objects
      const std::shared_ptr<GeneralBuffer2<CudaAllocator>> &buf = embedding_data_.get_buffer(id);
      embedding_optimizers_.emplace_back(max_vocabulary_size_per_gpu_,
                                         embedding_data_.embedding_params_, buf);

      {
        Tensor2<TypeHashKey> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_batch_size(true),
                      embedding_data_.embedding_params_.max_feature_num},
                     &tensor);
        embedding_data_.train_value_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeHashKey> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_batch_size(false),
                      embedding_data_.embedding_params_.max_feature_num},
                     &tensor);
        embedding_data_.evaluate_value_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeHashKey> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_batch_size(true) *
                          embedding_data_.embedding_params_.slot_num +
                      1},
                     &tensor);
        embedding_data_.train_row_offsets_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeHashKey> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_batch_size(false) *
                          embedding_data_.embedding_params_.slot_num +
                      1},
                     &tensor);
        embedding_data_.evaluate_row_offsets_tensors_.push_back(tensor);
      }
      { embedding_data_.train_nnz_array_.push_back(std::make_shared<size_t>(0)); }
      { embedding_data_.evaluate_nnz_array_.push_back(std::make_shared<size_t>(0)); }
      // new hash table value vectors
      {
        Tensor2<float> tensor;
        buf->reserve(
            {max_vocabulary_size_per_gpu_, embedding_data_.embedding_params_.embedding_vec_size},
            &tensor);
        hash_table_value_tensors_.push_back(tensor);
      }

      // new hash table value_index that get() from HashTable
      {
        Tensor2<size_t> tensor;
        buf->reserve({1, embedding_data_.embedding_params_.get_universal_batch_size() *
                             embedding_data_.embedding_params_.max_feature_num},
                     &tensor);
        hash_value_index_tensors_.push_back(tensor);
      }

      // new embedding features reduced by hash table values(results of forward)
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_universal_batch_size() *
                          embedding_data_.embedding_params_.slot_num,
                      embedding_data_.embedding_params_.embedding_vec_size},
                     &tensor);
        embedding_feature_tensors_.push_back(tensor);
      }

      // new wgrad used by backward
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_batch_size(true) *
                          embedding_data_.embedding_params_.slot_num,
                      embedding_data_.embedding_params_.embedding_vec_size},
                     &tensor);
        wgrad_tensors_.push_back(tensor);
      }

      // new temp tensors used by update_params
      {
        Tensor2<TypeHashKey> tensor;
        buf->reserve({1, embedding_data_.embedding_params_.get_universal_batch_size() *
                                 embedding_data_.embedding_params_.slot_num +
                             1},
                     &tensor);
        row_offset_allreduce_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_universal_batch_size() *
                          embedding_data_.embedding_params_.slot_num,
                      embedding_data_.embedding_params_.embedding_vec_size},
                     &tensor);
        utest_forward_temp_tensors_.push_back(tensor);
      }
      // temp storage for filter keys
      {
        size_t max_nnz = embedding_data_.embedding_params_.get_universal_batch_size() *
                         embedding_data_.embedding_params_.max_feature_num;
        size_t rowoffset_count = embedding_data_.embedding_params_.slot_num *
                                     embedding_data_.embedding_params_.get_universal_batch_size() +
                                 1;

        filter_keys_storage_.emplace_back(
            buf, max_nnz, rowoffset_count, embedding_data_.get_local_gpu(id).get_global_id(),
            embedding_data_.get_resource_manager().get_global_gpu_count());
      }
// init GenenralBuffers to do real allocation
#ifndef NDEBUG
      std::cout << " max_feature_num_:" << embedding_data_.embedding_params_.max_feature_num
                << std::endl;
#endif
    }

    hash_tables_.resize(embedding_data_.get_resource_manager().get_local_gpu_count());
#pragma omp parallel num_threads(embedding_data_.get_resource_manager().get_local_gpu_count())
    {
      size_t id = omp_get_thread_num();
      CudaDeviceContext context(embedding_data_.get_local_gpu(id).get_device_id());
      // construct HashTable object: used to store hash table <key, value_index>
      hash_tables_[id].reset(new NvHashTable(max_vocabulary_size_per_gpu_));
      embedding_data_.get_buffer(id)->allocate();
    }

    for (size_t id = 0; id < embedding_data_.get_resource_manager().get_local_gpu_count(); id++) {
      context.set_device(embedding_data_.get_local_gpu(id).get_device_id());
      embedding_optimizers_[id].initialize(embedding_data_.get_local_gpu(id));

    }  // end of for(int id = 0; id < embedding_data_.get_local_gpu_count(); id++)

    if (!embedding_data_.embedding_params_.slot_size_array.empty()) {
      std::vector<TypeHashKey> embedding_offsets;
      TypeHashKey slot_sizes_prefix_sum = 0;
      for (size_t i = 0; i < embedding_data_.embedding_params_.slot_size_array.size(); i++) {
        embedding_offsets.push_back(slot_sizes_prefix_sum);
        slot_sizes_prefix_sum += embedding_data_.embedding_params_.slot_size_array[i];
      }
      for (size_t id = 0; id < embedding_data_.get_resource_manager().get_local_gpu_count(); ++id) {
        CudaDeviceContext context(embedding_data_.get_local_gpu(id).get_device_id());

        CK_CUDA_THROW_(
            hipMemcpy(embedding_data_.embedding_offsets_[id].get_ptr(), embedding_offsets.data(),
                       embedding_offsets.size() * sizeof(TypeHashKey), hipMemcpyHostToDevice));
      }
    }
    functors_.sync_all_gpus(embedding_data_.get_resource_manager());

  } catch (const std::runtime_error &rt_err) {
    std::cerr << rt_err.what() << std::endl;
    throw;
  }

  return;
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void DistributedSlotSparseEmbeddingHash<TypeHashKey, TypeEmbeddingComp>::load_parameters(
    std::string sparse_model) {
  if (!std::filesystem::exists(sparse_model)) {
    CK_THROW_(Error_t::WrongInput, std::string("Folder ") + sparse_model + " doesn't exist");
  }
  const std::string key_file(sparse_model + "/key");
  const std::string vec_file(sparse_model + "/emb_vector");

  std::ifstream key_stream(key_file, std::ifstream::binary);
  std::ifstream vec_stream(vec_file, std::ifstream::binary);
  // check if file is opened successfully
  if (!vec_stream.is_open() || !key_stream.is_open()) {
    CK_THROW_(Error_t::WrongInput, "Error: file not open for reading");
  }

  size_t key_file_size_in_byte = std::filesystem::file_size(key_file);
  size_t vec_file_size_in_byte = std::filesystem::file_size(vec_file);

  size_t key_size = sizeof(long long);
  size_t vec_size = sizeof(float) * embedding_data_.embedding_params_.embedding_vec_size;
  size_t key_num = key_file_size_in_byte / key_size;
  size_t vec_num = vec_file_size_in_byte / vec_size;

  if (key_num != vec_num || key_file_size_in_byte % key_size != 0 ||
      vec_file_size_in_byte % vec_size != 0) {
    CK_THROW_(Error_t::WrongInput, "Error: file size is not correct");
  }

  auto blobs_buff = GeneralBuffer2<CudaHostAllocator>::create();

  Tensor2<TypeHashKey> keys;
  blobs_buff->reserve({key_num}, &keys);

  Tensor2<float> embeddings;
  blobs_buff->reserve({vec_num, embedding_data_.embedding_params_.embedding_vec_size}, &embeddings);

  blobs_buff->allocate();

  TypeHashKey *key_ptr = keys.get_ptr();
  float *embedding_ptr = embeddings.get_ptr();

  if (std::is_same<TypeHashKey, long long>::value) {
    key_stream.read(reinterpret_cast<char *>(key_ptr), key_file_size_in_byte);
  } else {
    std::vector<long long> i64_key_vec(key_num, 0);
    key_stream.read(reinterpret_cast<char *>(i64_key_vec.data()), key_file_size_in_byte);
    std::transform(i64_key_vec.begin(), i64_key_vec.end(), key_ptr,
                   [](long long key) { return static_cast<unsigned>(key); });
  }
  vec_stream.read(reinterpret_cast<char *>(embedding_ptr), vec_file_size_in_byte);

  load_parameters(keys, embeddings, key_num, max_vocabulary_size_,
                  embedding_data_.embedding_params_.embedding_vec_size,
                  max_vocabulary_size_per_gpu_, hash_table_value_tensors_, hash_tables_);

  return;
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void DistributedSlotSparseEmbeddingHash<TypeHashKey, TypeEmbeddingComp>::load_parameters(
    BufferBag &buf_bag, size_t num) {
  load_parameters(buf_bag, num, max_vocabulary_size_,
                  embedding_data_.embedding_params_.embedding_vec_size,
                  max_vocabulary_size_per_gpu_, hash_table_value_tensors_, hash_tables_);
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void DistributedSlotSparseEmbeddingHash<TypeHashKey, TypeEmbeddingComp>::load_parameters(
    BufferBag &buf_bag, size_t num, size_t vocabulary_size, size_t embedding_vec_size,
    size_t max_vocabulary_size_per_gpu, Tensors2<float> &embedding_tensors,
    std::vector<std::shared_ptr<NvHashTable>> &hash_tables) {
  if (num == 0) return;

  const TensorBag2 &keys_bag = buf_bag.keys;
  const Tensor2<float> &embeddings = buf_bag.embedding;
  const Tensor2<TypeHashKey> keys = Tensor2<TypeHashKey>::stretch_from(keys_bag);

  if (keys.get_dimensions()[0] < num || embeddings.get_dimensions()[0] < num) {
    CK_THROW_(Error_t::WrongInput, "The rows of keys and embeddings are not consistent.");
  }

  if (num > vocabulary_size) {
    CK_THROW_(Error_t::WrongInput,
              "num_key to be loaded is larger than hash table vocabulary_size");
  }

  const TypeHashKey *key_ptr = keys.get_ptr();
  const float *embedding_ptr = embeddings.get_ptr();

  const int my_rank = embedding_data_.get_resource_manager().get_process_id();

  CudaDeviceContext context;
  const size_t local_gpu_count = embedding_data_.get_resource_manager().get_local_gpu_count();

  std::unique_ptr<size_t[]> counter_per_gpu(new size_t[local_gpu_count]);
  memset(counter_per_gpu.get(), 0, sizeof(size_t) * local_gpu_count);

  const size_t num_thread = std::thread::hardware_concurrency();
  std::vector<std::vector<std::vector<TypeHashKey>>> chunk_keys(num_thread);
  std::vector<std::vector<std::vector<size_t>>> chunk_src_indexs(num_thread);

  for (size_t tid = 0; tid < num_thread; tid++) {
    chunk_keys[tid].resize(local_gpu_count);
    chunk_src_indexs[tid].resize(local_gpu_count);
  }

#pragma omp parallel num_threads(num_thread)
  {
    const size_t tid = omp_get_thread_num();
    const size_t thread_num = omp_get_num_threads();
    size_t sub_chunk_size = num / thread_num;
    size_t res_chunk_size = num % thread_num;
    const size_t idx = tid * sub_chunk_size;

    if (tid == thread_num - 1) sub_chunk_size += res_chunk_size;

    for (size_t i = 0; i < sub_chunk_size; i++) {
      auto key = key_ptr[idx + i];
      auto gid = key % embedding_data_.get_resource_manager().get_global_gpu_count();
      auto id = embedding_data_.get_resource_manager().get_gpu_local_id_from_global_id(gid);
      auto dst_rank = embedding_data_.get_resource_manager().get_process_id_from_gpu_global_id(gid);

      if (dst_rank != my_rank) {
        CK_THROW_(Error_t::UnspecificError, "ETC selected keys error");
      }

      chunk_keys[tid][id].push_back(key);
      chunk_src_indexs[tid][id].push_back(idx + i);
    }
  }

  std::vector<std::vector<size_t>> offset_per_thread(local_gpu_count);
#pragma omp parallel for num_threads(local_gpu_count)
  for (size_t id = 0; id < local_gpu_count; id++) {
    offset_per_thread[id].resize(num_thread, 0);

    std::vector<size_t> num_per_thread(num_thread, 0);
    for (size_t tid = 0; tid < num_thread; tid++) {
      counter_per_gpu[id] += chunk_keys[tid][id].size();
      num_per_thread[tid] = chunk_keys[tid][id].size();
    }
    std::exclusive_scan(num_per_thread.begin(), num_per_thread.end(), offset_per_thread[id].begin(),
                        0);
  }

  size_t total_count =
      std::accumulate(counter_per_gpu.get(), counter_per_gpu.get() + local_gpu_count, 0);
  if (total_count != num) {
    CK_THROW_(Error_t::UnspecificError, "total_count != num_of_keys");
  }

  std::unique_ptr<TypeHashKey *[]> uvm_key_per_gpu(new TypeHashKey *[local_gpu_count]);
  std::unique_ptr<float *[]> h_value_per_gpu(new float *[local_gpu_count]);
  std::unique_ptr<size_t *[]> d_value_index_per_gpu(new size_t *[local_gpu_count]);

  for (size_t id = 0; id < local_gpu_count; id++) {
    uvm_key_per_gpu[id] =
        Tensor2<TypeHashKey>::stretch_from(buf_bag.uvm_key_tensor_bags[id]).get_ptr();
    d_value_index_per_gpu[id] = buf_bag.d_value_index_tensors[id].get_ptr();
    h_value_per_gpu[id] = buf_bag.h_value_tensors[id].get_ptr();

    size_t value_index_size_in_B = counter_per_gpu[id] * sizeof(size_t);
    CK_CUDA_THROW_(hipMemsetAsync(d_value_index_per_gpu[id], 0, value_index_size_in_B,
                                   embedding_data_.get_local_gpu(id).get_stream()));

    size_t key_size_in_B = counter_per_gpu[id] * sizeof(TypeHashKey);
    CK_CUDA_THROW_(hipMemPrefetchAsync(uvm_key_per_gpu[id], key_size_in_B, hipCpuDeviceId,
                                        embedding_data_.get_local_gpu(id).get_stream()));
  }
  functors_.sync_all_gpus(embedding_data_.get_resource_manager());

  std::vector<std::vector<size_t>> src_indexs(local_gpu_count);
  for (size_t id = 0; id < local_gpu_count; id++) {
    src_indexs[id].resize(counter_per_gpu[id]);

#pragma omp parallel for num_threads(num_thread)
    for (size_t tid = 0; tid < num_thread; tid++) {
      TypeHashKey *key_dst_ptr = uvm_key_per_gpu[id] + offset_per_thread[id][tid];
      TypeHashKey *key_src_ptr = chunk_keys[tid][id].data();
      size_t key_size_in_B = chunk_keys[tid][id].size() * sizeof(TypeHashKey);
      memcpy(key_dst_ptr, key_src_ptr, key_size_in_B);

      size_t *idx_dst_ptr = src_indexs[id].data() + offset_per_thread[id][tid];
      size_t *idx_src_ptr = chunk_src_indexs[tid][id].data();
      size_t idx_size_in_B = chunk_src_indexs[tid][id].size() * sizeof(size_t);
      memcpy(idx_dst_ptr, idx_src_ptr, idx_size_in_B);
    }

#pragma omp parallel for num_threads(num_thread)
    for (size_t i = 0; i < src_indexs[id].size(); i++) {
      float *vec_dst_ptr = h_value_per_gpu[id] + i * embedding_vec_size;
      const float *vec_src_ptr = embedding_ptr + src_indexs[id][i] * embedding_vec_size;
      size_t vec_size_in_B = embedding_vec_size * sizeof(float);
      memcpy(vec_dst_ptr, vec_src_ptr, vec_size_in_B);
    }
  }

// do HashTable insert <key,value_index>
#pragma omp parallel for num_threads(local_gpu_count)
  for (size_t id = 0; id < local_gpu_count; id++) {
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

    const size_t counter = counter_per_gpu[id];

    // memcpy hash_table_key from CPU to GPU
    size_t key_size_in_B = counter * sizeof(TypeHashKey);
    CK_CUDA_THROW_(hipMemPrefetchAsync(uvm_key_per_gpu[id], key_size_in_B, id,
                                        embedding_data_.get_local_gpu(id).get_stream()));

    // set hash_table_value_index on GPU
    functors_.memset_liner(d_value_index_per_gpu[id], 0ul, 1ul, counter,
                           embedding_data_.get_local_gpu(id).get_stream());

    // do hash table insert <key, value_index> on GPU
    hash_tables[id]->insert(uvm_key_per_gpu[id], d_value_index_per_gpu[id], counter,
                            embedding_data_.get_local_gpu(id).get_stream());
    hash_tables[id]->set_value_head(counter, embedding_data_.get_local_gpu(id).get_stream());

    // memcpy hash_table_value from CPU to GPU
    size_t vec_block_in_B = counter * embedding_vec_size * sizeof(float);
    CK_CUDA_THROW_(hipMemcpyAsync(embedding_tensors[id].get_ptr(), h_value_per_gpu[id],
                                   vec_block_in_B, hipMemcpyHostToDevice,
                                   embedding_data_.get_local_gpu(id).get_stream()));
  }
  functors_.sync_all_gpus(embedding_data_.get_resource_manager());
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void DistributedSlotSparseEmbeddingHash<TypeHashKey, TypeEmbeddingComp>::load_parameters(
    const Tensor2<TypeHashKey> &keys, const Tensor2<float> &embeddings, size_t num,
    size_t vocabulary_size, size_t embedding_vec_size, size_t max_vocabulary_size_per_gpu,
    Tensors2<float> &embedding_tensors, std::vector<std::shared_ptr<NvHashTable>> &hash_tables) {
  if (num == 0) return;

  if (keys.get_dimensions()[0] < num || embeddings.get_dimensions()[0] < num) {
    CK_THROW_(Error_t::WrongInput, "The rows of keys and embeddings are not consistent.");
  }

  if (num > vocabulary_size) {
    CK_THROW_(Error_t::WrongInput,
              "Error: hash table file size is larger than hash table vocabulary_size");
  }

  const TypeHashKey *key_ptr = keys.get_ptr();
  const float *embedding_ptr = embeddings.get_ptr();

  int my_rank = embedding_data_.get_resource_manager().get_process_id();
  int n_ranks = embedding_data_.get_resource_manager().get_num_process();

  // define size
  size_t local_gpu_count = embedding_data_.get_resource_manager().get_local_gpu_count();
  const size_t chunk_size = 1000;
  size_t hash_table_key_tile_size = 1;
  size_t hash_table_key_tile_size_in_B = hash_table_key_tile_size * sizeof(TypeHashKey);
  size_t hash_table_key_chunk_size = hash_table_key_tile_size * chunk_size;
  size_t hash_table_key_chunk_size_in_B = hash_table_key_chunk_size * sizeof(TypeHashKey);
  size_t hash_table_value_index_chunk_size_in_B = hash_table_key_chunk_size * sizeof(size_t);
  size_t hash_table_value_tile_size = embedding_vec_size;
  size_t hash_table_value_tile_size_in_B = hash_table_value_tile_size * sizeof(float);
  size_t hash_table_value_chunk_size = hash_table_value_tile_size * chunk_size;
  size_t hash_table_value_chunk_size_in_B = hash_table_value_chunk_size * sizeof(float);

  // CAUSION: can not decide how many values for each GPU, so need to allocate enough memory
  // for each GPU allocate GPU memory for hash_table_value_index
  std::unique_ptr<size_t[]> tile_counter_per_gpu(
      new size_t[local_gpu_count]);  // <= hash_table_value_index_per_gpu_size
  memset(tile_counter_per_gpu.get(), 0, sizeof(size_t) * local_gpu_count);
  std::unique_ptr<size_t[]> tile_counter_in_chunk_per_gpu(new size_t[local_gpu_count]);
  memset(tile_counter_in_chunk_per_gpu.get(), 0, sizeof(size_t) * local_gpu_count);
  std::unique_ptr<size_t *[]> d_hash_table_value_index_chunk_per_gpu(new size_t *[local_gpu_count]);

  CudaDeviceContext context;
  for (size_t id = 0; id < local_gpu_count; id++) {
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());
    CK_CUDA_THROW_(hipMalloc(&d_hash_table_value_index_chunk_per_gpu[id],
                              hash_table_value_index_chunk_size_in_B));
    // initalize to zeros
    CK_CUDA_THROW_(hipMemsetAsync(d_hash_table_value_index_chunk_per_gpu[id], 0,
                                   hash_table_value_index_chunk_size_in_B,
                                   embedding_data_.get_local_gpu(id).get_stream()));
  }

  // sync wait
  functors_.sync_all_gpus(embedding_data_.get_resource_manager());

  // CAUSION: can not decide how many values for each GPU, so need to allocate enough memory
  // for each GPU allocate CPU/GPU memory for hash_table/key/value chunk
  std::unique_ptr<TypeHashKey *[]> h_hash_table_key_chunk_per_gpu(
      new TypeHashKey *[local_gpu_count]);
  for (size_t id = 0; id < local_gpu_count; id++) {
    CK_CUDA_THROW_(
        hipHostMalloc(&h_hash_table_key_chunk_per_gpu[id], hash_table_key_chunk_size_in_B));
  }
  std::unique_ptr<TypeHashKey *[]> d_hash_table_key_chunk_per_gpu(
      new TypeHashKey *[local_gpu_count]);
  for (size_t id = 0; id < local_gpu_count; id++) {
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());
    CK_CUDA_THROW_(hipMalloc(&d_hash_table_key_chunk_per_gpu[id], hash_table_key_chunk_size_in_B));
  }
  std::unique_ptr<float *[]> h_hash_table_value_chunk_per_gpu(new float *[local_gpu_count]);
  for (size_t id = 0; id < local_gpu_count; id++) {
    CK_CUDA_THROW_(
        hipHostMalloc(&h_hash_table_value_chunk_per_gpu[id], hash_table_value_chunk_size_in_B));
  }

  // do upload
  size_t loop_num = num / chunk_size;
  for (size_t i = 0; i < loop_num; i++) {
    TypeHashKey *key_dst_buf;
    float *value_dst_buf;
    for (size_t k = 0; k < chunk_size; k++) {  // process a tile in each loop
      TypeHashKey key = key_ptr[i * chunk_size + k];
      size_t gid =
          key % embedding_data_.get_resource_manager().get_global_gpu_count();  // global GPU ID
      size_t id = embedding_data_.get_resource_manager().get_gpu_local_id_from_global_id(
          gid);  // local GPU ID (not gpudevice id)
      int dst_rank =
          embedding_data_.get_resource_manager().get_process_id_from_gpu_global_id(gid);  // node id

      if (my_rank == dst_rank) {
        // memcpy hash_table_key to corresponding GPU
        key_dst_buf = h_hash_table_key_chunk_per_gpu[id] +
                      tile_counter_in_chunk_per_gpu[id] * hash_table_key_tile_size;

        *key_dst_buf = key;

        // memcpy hash_table_value to corresponding GPU
        value_dst_buf = h_hash_table_value_chunk_per_gpu[id] +
                        tile_counter_in_chunk_per_gpu[id] * hash_table_value_tile_size;

        memcpy(value_dst_buf, embedding_ptr + (i * chunk_size + k) * embedding_vec_size,
               hash_table_value_tile_size_in_B);

        tile_counter_in_chunk_per_gpu[id] += 1;
      } else {
        continue;
      }
    }  // end of for(int k = 0; k < (chunk_loop * local_gpu_count); k++)

    // do HashTable insert <key,value_index>
    for (size_t id = 0; id < local_gpu_count; id++) {
      context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

      size_t tile_count = tile_counter_in_chunk_per_gpu[id];

      // memcpy hash_table_key from CPU to GPU
      CK_CUDA_THROW_(hipMemcpyAsync(d_hash_table_key_chunk_per_gpu[id],
                                     h_hash_table_key_chunk_per_gpu[id],
                                     tile_count * sizeof(TypeHashKey), hipMemcpyHostToDevice,
                                     embedding_data_.get_local_gpu(id).get_stream()));

      size_t value_index_offset = tile_counter_per_gpu[id];
      size_t *value_index_buf = d_hash_table_value_index_chunk_per_gpu[id];

      if (tile_count > 0) {
        // set hash_table_value_index on GPU
        functors_.memset_liner(value_index_buf, value_index_offset, 1ul, tile_count,
                               embedding_data_.get_local_gpu(id).get_stream());
      }

      // do hash table insert <key, value_index> on GPU
      hash_tables[id]->insert(d_hash_table_key_chunk_per_gpu[id], value_index_buf, tile_count,
                              embedding_data_.get_local_gpu(id).get_stream());
      size_t value_head = hash_tables[id]->get_and_add_value_head(
          tile_count, embedding_data_.get_local_gpu(id).get_stream());
    }

    // memcpy hash_table_value from CPU to GPU
    for (size_t id = 0; id < local_gpu_count; id++) {
      context.set_device(embedding_data_.get_local_gpu(id).get_device_id());
      size_t value_chunk_size = tile_counter_in_chunk_per_gpu[id] * embedding_vec_size;
      size_t value_chunk_offset = tile_counter_per_gpu[id] * embedding_vec_size;
      float *src_buf = h_hash_table_value_chunk_per_gpu[id];
      float *dst_buf = embedding_tensors[id].get_ptr() + value_chunk_offset;
      CK_CUDA_THROW_(hipMemcpyAsync(dst_buf, src_buf, value_chunk_size * sizeof(float),
                                     hipMemcpyHostToDevice,
                                     embedding_data_.get_local_gpu(id).get_stream()));
    }

    functors_.sync_all_gpus(embedding_data_.get_resource_manager());

    // set counter value
    for (size_t id = 0; id < local_gpu_count; id++) {
      tile_counter_per_gpu[id] += tile_counter_in_chunk_per_gpu[id];
      tile_counter_in_chunk_per_gpu[id] = 0;  // reset chunk counter to zero

      if (tile_counter_per_gpu[id] > max_vocabulary_size_per_gpu) {
        char msg[100]{0};
        sprintf(msg, "The size of hash table on GPU %zu is out of range %zu\n", id,
                max_vocabulary_size_per_gpu);
        CK_THROW_(Error_t::OutOfBound, msg);
      }
    }
  }  // end of for(int i = 0; i < loop_num; i++)

  // process the remaining data(less than a chunk)
  size_t remain_loop_num = num - loop_num * chunk_size;
  TypeHashKey *key_dst_buf;
  size_t *value_index_buf;
  float *value_dst_buf;
  for (size_t i = 0; i < remain_loop_num; i++) {
    TypeHashKey key = key_ptr[loop_num * chunk_size + i];
    size_t gid =
        key % embedding_data_.get_resource_manager().get_global_gpu_count();  // global GPU ID
    size_t id = embedding_data_.get_resource_manager().get_gpu_local_id_from_global_id(
        gid);  // local GPU ID (not gpudevice id)
    int dst_rank = embedding_data_.get_resource_manager().get_process_id_from_gpu_global_id(gid);

    if (my_rank == dst_rank) {
      context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

      // memcpy hash_table_key from CPU to GPU
      key_dst_buf = d_hash_table_key_chunk_per_gpu[id];
      CK_CUDA_THROW_(hipMemcpyAsync(key_dst_buf, &key, hash_table_key_tile_size_in_B,
                                     hipMemcpyHostToDevice,
                                     embedding_data_.get_local_gpu(id).get_stream()));

      // set value_index
      size_t value_index_offset = tile_counter_per_gpu[id];
      value_index_buf = d_hash_table_value_index_chunk_per_gpu[id];
      functors_.memset_liner(value_index_buf, value_index_offset, 1ul, 1ul,
                             embedding_data_.get_local_gpu(id).get_stream());

      // do hash table insert <key, value_index> on GPU
      hash_tables[id]->insert(d_hash_table_key_chunk_per_gpu[id], value_index_buf,
                              hash_table_key_tile_size,
                              embedding_data_.get_local_gpu(id).get_stream());
      size_t value_head = hash_tables[id]->get_and_add_value_head(
          hash_table_key_tile_size, embedding_data_.get_local_gpu(id).get_stream());

      // memcpy hash_table_value from CPU to GPU
      size_t value_offset = tile_counter_per_gpu[id] * embedding_vec_size;
      value_dst_buf = embedding_tensors[id].get_ptr() + value_offset;
      CK_CUDA_THROW_(hipMemcpyAsync(
          value_dst_buf, embedding_ptr + (loop_num * chunk_size + i) * embedding_vec_size,
          hash_table_value_tile_size_in_B, hipMemcpyHostToDevice,
          embedding_data_.get_local_gpu(id).get_stream()));

      // set counter
      tile_counter_per_gpu[id] += hash_table_key_tile_size;
    } else {
      continue;
    }

    // sync wait
    functors_.sync_all_gpus(embedding_data_.get_resource_manager());

  }  // end of if(remain_loop_num)

  // release resources
  for (size_t id = 0; id < local_gpu_count; id++) {
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());
    CK_CUDA_THROW_(hipFree(d_hash_table_value_index_chunk_per_gpu[id]));
    CK_CUDA_THROW_(hipFree(d_hash_table_key_chunk_per_gpu[id]));
  }
  for (size_t id = 0; id < local_gpu_count; id++) {
    CK_CUDA_THROW_(hipHostFree(h_hash_table_key_chunk_per_gpu[id]));
    CK_CUDA_THROW_(hipHostFree(h_hash_table_value_chunk_per_gpu[id]));
  }
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void DistributedSlotSparseEmbeddingHash<TypeHashKey, TypeEmbeddingComp>::dump_parameters(
    std::string sparse_model, DataSourceParams data_source_params) const {
  dump_parameters(sparse_model, data_source_params, max_vocabulary_size_,
                  embedding_data_.embedding_params_.embedding_vec_size, hash_table_value_tensors_,
                  hash_tables_);
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void DistributedSlotSparseEmbeddingHash<TypeHashKey, TypeEmbeddingComp>::dump_parameters(
    BufferBag &buf_bag, size_t *num) const {
  TensorBag2 keys_bag = buf_bag.keys;
  Tensor2<float> &embeddings = buf_bag.embedding;
  Tensor2<TypeHashKey> keys = Tensor2<TypeHashKey>::stretch_from(keys_bag);
  dump_parameters(keys, embeddings, num, max_vocabulary_size_,
                  embedding_data_.embedding_params_.embedding_vec_size, hash_table_value_tensors_,
                  hash_tables_);
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void DistributedSlotSparseEmbeddingHash<TypeHashKey, TypeEmbeddingComp>::dump_parameters(
    const std::string &sparse_model, DataSourceParams data_source_params, size_t vocabulary_size,
    size_t embedding_vec_size, const Tensors2<float> &hash_table_value_tensors,
    const std::vector<std::shared_ptr<HashTable<TypeHashKey, size_t>>> &hash_tables) const {
  CudaDeviceContext context;
  size_t local_gpu_count = embedding_data_.get_resource_manager().get_local_gpu_count();

  if (!data_source_params.use_hdfs && !std::filesystem::exists(sparse_model)) {
    std::filesystem::create_directories(sparse_model);
  }
  const std::string key_file(sparse_model + "/key");
  const std::string vec_file(sparse_model + "/emb_vector");

#ifdef ENABLE_MPI
  MPI_File key_fh, vec_fh;
  CK_MPI_THROW_(MPI_File_open(MPI_COMM_WORLD, key_file.c_str(), MPI_MODE_CREATE | MPI_MODE_WRONLY,
                              MPI_INFO_NULL, &key_fh));
  CK_MPI_THROW_(MPI_File_open(MPI_COMM_WORLD, vec_file.c_str(), MPI_MODE_CREATE | MPI_MODE_WRONLY,
                              MPI_INFO_NULL, &vec_fh));
#endif

  // memory allocation
  std::unique_ptr<size_t[]> count(new size_t[local_gpu_count]);
  size_t total_count = 0;

  for (size_t id = 0; id < local_gpu_count; id++) {
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());
    auto count_tmp_1 = hash_tables[id]->get_size(embedding_data_.get_local_gpu(id).get_stream());
    auto count_tmp_2 =
        hash_tables[id]->get_value_head(embedding_data_.get_local_gpu(id).get_stream());
    if (count_tmp_1 != count_tmp_2) {
      CK_THROW_(Error_t::WrongInput,
                "Error: hash_table get_value_head() size not equal to get_size()");
    }
    count[id] = count_tmp_1;
    total_count += count[id];
  }

  if (total_count > (size_t)vocabulary_size) {
    CK_THROW_(Error_t::WrongInput,
              "Error: required download size is larger than hash table vocabulary_size");
  }

  std::vector<size_t> offset_host(local_gpu_count, 0);
  std::exclusive_scan(count.get(), count.get() + local_gpu_count, offset_host.begin(), 0);

  TypeHashKey *h_hash_table_key;
  float *h_hash_table_value;
  CK_CUDA_THROW_(hipHostMalloc(&h_hash_table_key, total_count * sizeof(TypeHashKey)));
  CK_CUDA_THROW_(
      hipHostMalloc(&h_hash_table_value, total_count * embedding_vec_size * sizeof(float)));

  std::unique_ptr<TypeHashKey *[]> d_hash_table_key(new TypeHashKey *[local_gpu_count]);
  std::unique_ptr<size_t *[]> d_hash_table_value_index(new size_t *[local_gpu_count]);
  std::unique_ptr<size_t *[]> d_dump_counter(new size_t *[local_gpu_count]);

  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) continue;
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

    CK_CUDA_THROW_(hipMallocManaged(&d_hash_table_key[id], count[id] * sizeof(TypeHashKey)));
    CK_CUDA_THROW_(hipMallocManaged(&d_hash_table_value_index[id], count[id] * sizeof(size_t)));
    CK_CUDA_THROW_(hipMalloc(&d_dump_counter[id], sizeof(size_t)));
  }

  // dump hash table from GPUs
  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) continue;
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

    hash_tables[id]->dump(d_hash_table_key[id], d_hash_table_value_index[id], d_dump_counter[id],
                          embedding_data_.get_local_gpu(id).get_stream());

    CK_CUDA_THROW_(hipMemcpyAsync(
        h_hash_table_value + offset_host[id] * embedding_vec_size,
        hash_table_value_tensors[id].get_ptr(), count[id] * embedding_vec_size * sizeof(float),
        hipMemcpyDeviceToHost, embedding_data_.get_local_gpu(id).get_stream()));
  }
  functors_.sync_all_gpus(embedding_data_.get_resource_manager());

  // sort key according to memory index
  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) continue;
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

    thrust::sort_by_key(thrust::device, d_hash_table_value_index[id],
                        d_hash_table_value_index[id] + count[id], d_hash_table_key[id]);

    CK_CUDA_THROW_(hipMemcpyAsync(h_hash_table_key + offset_host[id], d_hash_table_key[id],
                                   count[id] * sizeof(TypeHashKey), hipMemcpyDeviceToHost,
                                   embedding_data_.get_local_gpu(id).get_stream()));
  }
  functors_.sync_all_gpus(embedding_data_.get_resource_manager());

  long long *h_key_ptr;
  std::vector<long long> i64_key_vec;
  if (std::is_same<TypeHashKey, long long>::value) {
    h_key_ptr = reinterpret_cast<long long *>(h_hash_table_key);
  } else {
    i64_key_vec.resize(total_count);
    std::transform(h_hash_table_key, h_hash_table_key + total_count, i64_key_vec.begin(),
                   [](unsigned key) { return static_cast<long long>(key); });
    h_key_ptr = i64_key_vec.data();
  }

  const size_t key_size = sizeof(long long);
  const size_t vec_size = sizeof(float) * embedding_vec_size;

  // write sparse model to file
  MESSAGE_("Rank" + std::to_string(embedding_data_.get_resource_manager().get_process_id()) +
               ": Write hash table to file",
           true);
#ifdef ENABLE_MPI
  MPI_Datatype TYPE_EMB_VECTOR;
  CK_MPI_THROW_(MPI_Type_contiguous(embedding_vec_size, MPI_FLOAT, &TYPE_EMB_VECTOR));
  CK_MPI_THROW_(MPI_Type_commit(&TYPE_EMB_VECTOR));

  int my_rank = embedding_data_.get_resource_manager().get_process_id();
  int n_ranks = embedding_data_.get_resource_manager().get_num_process();

  std::vector<size_t> offset_per_rank(n_ranks, 0);
  CK_MPI_THROW_(MPI_Allgather(&total_count, sizeof(size_t), MPI_CHAR, offset_per_rank.data(),
                              sizeof(size_t), MPI_CHAR, MPI_COMM_WORLD));
  std::exclusive_scan(offset_per_rank.begin(), offset_per_rank.end(), offset_per_rank.begin(), 0);

  size_t key_offset = offset_per_rank[my_rank] * key_size;
  size_t vec_offset = offset_per_rank[my_rank] * vec_size;

  CK_MPI_THROW_(MPI_Barrier(MPI_COMM_WORLD));
  MPI_Status status;
  CK_MPI_THROW_(
      MPI_File_write_at(key_fh, key_offset, h_key_ptr, total_count, MPI_LONG_LONG_INT, &status));
  CK_MPI_THROW_(MPI_File_write_at(vec_fh, vec_offset, h_hash_table_value, total_count,
                                  TYPE_EMB_VECTOR, &status));

  CK_MPI_THROW_(MPI_File_close(&key_fh));
  CK_MPI_THROW_(MPI_File_close(&vec_fh));
  CK_MPI_THROW_(MPI_Type_free(&TYPE_EMB_VECTOR));
#else
  if (data_source_params.use_hdfs) {
    HdfsService hs(data_source_params.namenode, data_source_params.port);
    hs.write(key_file, reinterpret_cast<char *>(h_key_ptr), total_count * key_size, true);
    hs.write(vec_file, reinterpret_cast<char *>(h_hash_table_value), total_count * vec_size, true);
  } else {
    std::ofstream key_stream(key_file, std::ofstream::binary | std::ofstream::trunc);
    std::ofstream vec_stream(vec_file, std::ofstream::binary | std::ofstream::trunc);
    // check if the file is opened successfully
    if (!vec_stream.is_open() || !key_stream.is_open()) {
      CK_THROW_(Error_t::WrongInput, "Error: file not open for writing");
      return;
    }
    key_stream.write(reinterpret_cast<char *>(h_key_ptr), total_count * key_size);
    vec_stream.write(reinterpret_cast<char *>(h_hash_table_value), total_count * vec_size);
  }
#endif

  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) continue;
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

    CK_CUDA_THROW_(hipFree(d_hash_table_key[id]));
    CK_CUDA_THROW_(hipFree(d_hash_table_value_index[id]));
    CK_CUDA_THROW_(hipFree(d_dump_counter[id]));
  }
  CK_CUDA_THROW_(hipHostFree(h_hash_table_key));
  CK_CUDA_THROW_(hipHostFree(h_hash_table_value));
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void DistributedSlotSparseEmbeddingHash<TypeHashKey, TypeEmbeddingComp>::dump_parameters(
    Tensor2<TypeHashKey> &keys, Tensor2<float> &embeddings, size_t *num, size_t vocabulary_size,
    size_t embedding_vec_size, const Tensors2<float> &embedding_tensors,
    const std::vector<std::shared_ptr<HashTable<TypeHashKey, size_t>>> &hash_tables) const {
  TypeHashKey *key_ptr = keys.get_ptr();
  float *embedding_ptr = embeddings.get_ptr();

  size_t local_gpu_count = embedding_data_.get_resource_manager().get_local_gpu_count();

  // memory allocation
  std::unique_ptr<size_t[]> count(new size_t[local_gpu_count]);
  size_t total_count = 0;

  CudaDeviceContext context;
  for (size_t id = 0; id < local_gpu_count; id++) {
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());
    auto count_tmp_1 = hash_tables[id]->get_size(embedding_data_.get_local_gpu(id).get_stream());
    auto count_tmp_2 =
        hash_tables[id]->get_value_head(embedding_data_.get_local_gpu(id).get_stream());
    if (count_tmp_1 != count_tmp_2) {
      CK_THROW_(Error_t::WrongInput,
                "Error: hash_table get_value_head() size not equal to get_size()");
    }
    count[id] = count_tmp_1;
    total_count += count[id];
  }

  if (total_count > (size_t)vocabulary_size) {
    CK_THROW_(Error_t::WrongInput, "Required download size > hash table vocabulary_size");
  }

  std::vector<size_t> offset_host(local_gpu_count, 0);
  std::exclusive_scan(count.get(), count.get() + local_gpu_count, offset_host.begin(), 0);
  *num = total_count;

  std::unique_ptr<TypeHashKey *[]> d_hash_table_key(new TypeHashKey *[local_gpu_count]);
  std::unique_ptr<size_t *[]> d_hash_table_value_index(new size_t *[local_gpu_count]);
  std::unique_ptr<size_t *[]> d_dump_counter(new size_t *[local_gpu_count]);
  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) continue;
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

    CK_CUDA_THROW_(hipMalloc(&d_hash_table_key[id], count[id] * sizeof(TypeHashKey)));
    CK_CUDA_THROW_(hipMalloc(&d_hash_table_value_index[id], count[id] * sizeof(size_t)));
    CK_CUDA_THROW_(hipMalloc(&d_dump_counter[id], sizeof(size_t)));
  }

  // dump hash table from GPUs
  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) continue;
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

    hash_tables[id]->dump(d_hash_table_key[id], d_hash_table_value_index[id], d_dump_counter[id],
                          embedding_data_.get_local_gpu(id).get_stream());

    CK_CUDA_THROW_(hipMemcpyAsync(
        embedding_ptr + offset_host[id] * embedding_vec_size,
        hash_table_value_tensors_[id].get_ptr(), count[id] * embedding_vec_size * sizeof(float),
        hipMemcpyDeviceToHost, embedding_data_.get_local_gpu(id).get_stream()));
  }
  functors_.sync_all_gpus(embedding_data_.get_resource_manager());

  // sort key according to memory index
  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) continue;
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

    thrust::sort_by_key(thrust::device, d_hash_table_value_index[id],
                        d_hash_table_value_index[id] + count[id], d_hash_table_key[id]);

    CK_CUDA_THROW_(hipMemcpyAsync(key_ptr + offset_host[id], d_hash_table_key[id],
                                   count[id] * sizeof(TypeHashKey), hipMemcpyDeviceToHost,
                                   embedding_data_.get_local_gpu(id).get_stream()));
  }
  functors_.sync_all_gpus(embedding_data_.get_resource_manager());

  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) continue;
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

    CK_CUDA_THROW_(hipFree(d_hash_table_key[id]));
    CK_CUDA_THROW_(hipFree(d_hash_table_value_index[id]));
    CK_CUDA_THROW_(hipFree(d_dump_counter[id]));
  }
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void DistributedSlotSparseEmbeddingHash<TypeHashKey, TypeEmbeddingComp>::dump_opt_states(
    std::ofstream &stream, std::string write_path, DataSourceParams data_source_params) {
  std::vector<OptimizerTensor<TypeEmbeddingComp>> opt_tensors_;
  for (auto &opt : embedding_optimizers_) {
    opt_tensors_.push_back(opt.opt_tensors_);
  }
  auto opt_states =
      functors_.get_opt_states(opt_tensors_, embedding_data_.embedding_params_.opt_params.optimizer,
                               embedding_data_.get_resource_manager().get_local_gpu_count());

  functors_.dump_opt_states(stream, write_path, data_source_params,
                            embedding_data_.get_resource_manager(), opt_states);
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void DistributedSlotSparseEmbeddingHash<TypeHashKey, TypeEmbeddingComp>::load_opt_states(
    std::ifstream &stream) {
  std::vector<OptimizerTensor<TypeEmbeddingComp>> opt_tensors_;
  for (auto &opt : embedding_optimizers_) {
    opt_tensors_.push_back(opt.opt_tensors_);
  }
  auto opt_states =
      functors_.get_opt_states(opt_tensors_, embedding_data_.embedding_params_.opt_params.optimizer,
                               embedding_data_.get_resource_manager().get_local_gpu_count());

  functors_.load_opt_states(stream, embedding_data_.get_resource_manager(), opt_states);
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void DistributedSlotSparseEmbeddingHash<TypeHashKey, TypeEmbeddingComp>::init_embedding(
    size_t max_vocabulary_size_per_gpu, size_t embedding_vec_size,
    Tensors2<float> &hash_table_value_tensors) {
#pragma omp parallel num_threads(embedding_data_.get_resource_manager().get_local_gpu_count())
  {
    size_t id = omp_get_thread_num();
    CudaDeviceContext context(embedding_data_.get_local_gpu(id).get_device_id());

    MESSAGE_("gpu" + std::to_string(id) + " start to init embedding");

    HugeCTR::UniformGenerator::fill(
        hash_table_value_tensors[id], -0.05f, 0.05f,
        embedding_data_.get_local_gpu(id).get_sm_count(),
        embedding_data_.get_local_gpu(id).get_replica_variant_curand_generator(),
        embedding_data_.get_local_gpu(id).get_stream());

    CK_CUDA_THROW_(hipStreamSynchronize(embedding_data_.get_local_gpu(id).get_stream()));
    MESSAGE_("gpu" + std::to_string(id) + " init embedding done");
  }
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void DistributedSlotSparseEmbeddingHash<TypeHashKey, TypeEmbeddingComp>::reset() {
  CudaDeviceContext context;
  for (size_t i = 0; i < embedding_data_.get_resource_manager().get_local_gpu_count(); i++) {
    context.set_device(embedding_data_.get_local_gpu(i).get_device_id());
    hash_tables_[i]->clear(embedding_data_.get_local_gpu(i).get_stream());
    HugeCTR::UniformGenerator::fill(
        hash_table_value_tensors_[i], -0.05f, 0.05f,
        embedding_data_.get_local_gpu(i).get_sm_count(),
        embedding_data_.get_local_gpu(i).get_replica_variant_curand_generator(),
        embedding_data_.get_local_gpu(i).get_stream());
  }

  for (size_t id = 0; id < embedding_data_.get_resource_manager().get_local_gpu_count(); id++) {
    CK_CUDA_THROW_(hipStreamSynchronize(embedding_data_.get_local_gpu(id).get_stream()));
  }
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void DistributedSlotSparseEmbeddingHash<TypeHashKey, TypeEmbeddingComp>::reset_optimizer() {
  CudaDeviceContext context;
  auto local_gpu_count{embedding_data_.get_resource_manager().get_local_gpu_count()};
  for (size_t id{0}; id < local_gpu_count; id++) {
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());
    embedding_optimizers_[id].reset(embedding_data_.get_local_gpu(id));
  }
}

template class DistributedSlotSparseEmbeddingHash<unsigned int, float>;
template class DistributedSlotSparseEmbeddingHash<long long, float>;
template class DistributedSlotSparseEmbeddingHash<unsigned int, __half>;
template class DistributedSlotSparseEmbeddingHash<long long, __half>;

}  // namespace HugeCTR
