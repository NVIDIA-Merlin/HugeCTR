#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>

#include <collectives/all_reduce_comm.hpp>
#include <vector>

#include "HugeCTR/include/common.hpp"
#include "HugeCTR/include/embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/calibration_data.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/data.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/frequent_embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/indices_container.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/infrequent_embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/model.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/statistics.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/utils.hpp"
#include "HugeCTR/include/embeddings/hybrid_sparse_embedding.hpp"
#include "HugeCTR/include/tensor2.hpp"

namespace HugeCTR {
template <typename dtype, typename emtype>
HybridSparseEmbedding<dtype, emtype>::HybridSparseEmbedding(
    const SparseTensors<dtype> &train_input_tensors,
    const SparseTensors<dtype> &evaluate_input_tensors,
    const HybridSparseEmbeddingParams &embedding_params,
    const std::vector<BuffPtr<emtype>> &grouped_wgrad_buff,
    const GpuLearningRateSchedulers lr_scheds, bool graph_mode,
    const std::shared_ptr<ResourceManager> &resource_manager, bool overlap_ar_a2a,
    bool eval_overlap)
    : train_input_tensors_(train_input_tensors),
      evaluate_input_tensors_(evaluate_input_tensors),
      embedding_params_(embedding_params),
      resource_manager_(resource_manager),
      stream_manager_(resource_manager->get_local_gpu_count()),
      grouped_wgrad_buff_(grouped_wgrad_buff),
      grouped_all_reduce_(grouped_wgrad_buff[0] != NULL),
      lr_scheds_(lr_scheds),
      graph_mode_(graph_mode),
      overlap_ar_a2a_(overlap_ar_a2a),
      eval_overlap_(eval_overlap) {
  try {
    // 0. Error check
    if (embedding_params_.train_batch_size < 1 || embedding_params_.evaluate_batch_size < 1 ||
        embedding_params_.slot_num < 1 || embedding_params_.embedding_vec_size < 1) {
      HCTR_OWN_THROW(Error_t::WrongInput,
                     "batchsize < 1 || slot_num < 1 || embedding_vec_size < 1");
    }

    if (embedding_params_.embedding_vec_size > 1024) {
      HCTR_OWN_THROW(Error_t::WrongInput,
                     "the embedding_vec_size can not be more than 1024 in embedding layer");
    }

    size_t total_gpu_count = resource_manager_->get_global_gpu_count();
    size_t local_gpu_count = resource_manager_->get_local_gpu_count();

    if (train_input_tensors.size() != local_gpu_count ||
        evaluate_input_tensors.size() != local_gpu_count) {
      HCTR_OWN_THROW(Error_t::WrongInput,
                     "either train_input_tensors.size() or evaluate_input_tensors.size() isn't "
                     "local_gpu_count_");
    }

    HCTR_LOG_S(INFO, ROOT) << "Using Hybrid Embedding with train batch " << get_batch_size(true)
                           << " and eval batch " << get_batch_size(false) << std::endl;

    // 1. initialize optimizer
    for (size_t id = 0; id < local_gpu_count; id++) {
      OptParams opt_params;
      opt_params.optimizer = embedding_params_.opt_params.optimizer;
      opt_params.lr = embedding_params_.opt_params.lr;
      opt_params.update_type = embedding_params_.opt_params.update_type;
      opt_params.scaler = embedding_params_.opt_params.scaler;
      opt_params_.emplace_back(opt_params);
    }
    // 2. reserve buffers for different tensors
    data_statistics_.reserve(local_gpu_count);
    data_train_.reserve(local_gpu_count);
    data_evaluate_.reserve(local_gpu_count);
    model_.reserve(local_gpu_count);
    calibration_.reserve(local_gpu_count);
    statistics_.reserve(local_gpu_count);
    train_output_tensors_.reserve(local_gpu_count);
    evaluate_output_tensors_.reserve(local_gpu_count);
    frequent_embeddings_.reserve(local_gpu_count);
    infrequent_embeddings_.reserve(local_gpu_count);
    infrequent_forward_comm_buffers_.reserve(local_gpu_count);
    infrequent_backward_comm_buffers_.reserve(local_gpu_count);

    assert(bufs_.empty());
    CudaDeviceContext context;
    // 2.1. construct data
    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);

      data_statistics_.emplace_back(embedding_params_.slot_size_array, get_batch_size(true),
                                    embedding_params_.num_iterations_statistics);
      if (!embedding_params_.use_train_precompute_indices) {
        data_train_.emplace_back(embedding_params_.slot_size_array, get_batch_size(true), 1);
      }
      if (!embedding_params_.use_eval_precompute_indices) {
        data_evaluate_.emplace_back(embedding_params_.slot_size_array, get_batch_size(false), 1);
      }
    }

    // 2.2 construct model
    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);

      std::vector<uint32_t> num_instances_per_node(resource_manager_->get_num_process(), 0);
      get_num_instances_per_node(num_instances_per_node);
      model_.emplace_back(embedding_params_.communication_type,
                          resource_manager_->get_local_gpu(i)->get_global_id(),
                          num_instances_per_node, get_categories_num());
    }

    // 2.3 construct calibration
    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);
      calibration_.emplace_back(resource_manager_->get_num_process(), embedding_params_.p_dup_max,
                                embedding_params_.max_all_reduce_bandwidth,
                                embedding_params_.max_all_to_all_bandwidth,
                                embedding_params_.efficiency_bandwidth_ratio);
    }

    // 2.4 construct Statistics
    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);
      const size_t num_samples_statistics = embedding_params_.num_iterations_statistics *
                                            get_batch_size(true) * embedding_params_.slot_num;
      statistics_.emplace_back((dtype)num_samples_statistics, embedding_params_.slot_num,
                               model_[i].num_instances, get_categories_num());
    }

    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);
      std::shared_ptr<GeneralBuffer2<CudaAllocator>> buf = GeneralBuffer2<CudaAllocator>::create();
      bufs_.emplace_back(buf);
      // 2.5. reserve for train output/ evaluate output tensors
      Tensor2<emtype> tensor;
      buf->reserve({get_batch_size_per_gpu(true), get_slot_num(), get_embedding_vec_size()},
                   &tensor);
      train_output_tensors_.emplace_back(tensor);
      buf->reserve({get_batch_size_per_gpu(false), get_slot_num(), get_embedding_vec_size()},
                   &tensor);
      evaluate_output_tensors_.emplace_back(tensor);

      // 2.6 construct frequent embedding
      frequent_embeddings_.emplace_back(model_[i], get_local_gpu(i), grouped_wgrad_buff_[i],
                                        get_embedding_vec_size(),
                                        embedding_params_.max_num_frequent_categories);
      if (!embedding_params_.use_train_precompute_indices) {
        frequent_embedding_train_indices_.emplace_back(
            embedding_params_.max_num_frequent_categories, data_train_[i], model_[i]);
      }
      if (!embedding_params_.use_eval_precompute_indices) {
        frequent_embedding_evaluate_indices_.emplace_back(
            embedding_params_.max_num_frequent_categories, data_evaluate_[i], model_[i]);
      }

      // 2.7 construct infrequent embedding
      infrequent_embeddings_.emplace_back(model_[i], get_local_gpu(i), get_embedding_vec_size());
      if (!embedding_params_.use_train_precompute_indices) {
        infrequent_embedding_train_indices_.emplace_back(data_train_[i], model_[i]);
      }
      if (!embedding_params_.use_eval_precompute_indices) {
        infrequent_embedding_evaluate_indices_.emplace_back(data_evaluate_[i], model_[i]);
      }

      // 2.8 construct communication
      if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
        size_t max_buf_size = embedding_params_.embedding_vec_size *
                              std::max(get_batch_size(true), get_batch_size(false)) *
                              embedding_params_.slot_num;
        infrequent_forward_comm_buffers_.emplace_back(buf.get(), max_buf_size);
        infrequent_backward_comm_buffers_.emplace_back(buf.get(), max_buf_size);
        infrequent_forward_comms_.emplace_back(std::make_unique<AllToAll_Multi_NCCL<emtype>>(
            infrequent_forward_comm_buffers_.back().send_buffer,
            infrequent_forward_comm_buffers_.back().recv_buffer,
            infrequent_embeddings_.back().get_model_indices_offsets_ptr(),
            infrequent_embeddings_.back().get_network_indices_offsets_ptr(), &get_local_gpu(i),
            embedding_params_.embedding_vec_size));
        infrequent_backward_comms_.emplace_back(std::make_unique<AllToAll_Multi_NCCL<emtype>>(
            infrequent_backward_comm_buffers_.back().send_buffer,
            infrequent_backward_comm_buffers_.back().recv_buffer,
            infrequent_embeddings_.back().get_network_indices_offsets_ptr(),
            infrequent_embeddings_.back().get_model_indices_offsets_ptr(), &get_local_gpu(i),
            embedding_params_.embedding_vec_size));
      }

      // Construct comm buffers
      if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
        double p_infrequent_samples = 1.0;
        if (embedding_params_.max_num_infrequent_samples >= 0) {
          p_infrequent_samples = (double)embedding_params_.max_num_infrequent_samples /
                                 ((double)get_batch_size(true) * embedding_params_.slot_num);
        }
        auto align = [this, i](size_t val) {
          auto alignment = model_[i].num_instances;
          return ((val + alignment - 1) / alignment) * alignment;
        };

        infrequent_embeddings_[i].max_num_infrequent_per_batch_ =
            align(std::max(get_batch_size(true), get_batch_size(false)) *
                  embedding_params_.slot_num * p_infrequent_samples);

        infrequent_embeddings_[i].max_num_infrequent_per_train_batch_ =
            align(get_batch_size(true) * embedding_params_.slot_num * p_infrequent_samples);

        size_t max_buf_size = embedding_params_.embedding_vec_size *
                              infrequent_embeddings_[i].max_num_infrequent_per_batch_;
        size_t max_back_buf_size = embedding_params_.embedding_vec_size *
                                   infrequent_embeddings_[i].max_num_infrequent_per_train_batch_;

        HCTR_LOG_S(INFO, ROOT) << "Allocating A2A buffers for infrequent categories. For training: "
                               << infrequent_embeddings_[i].max_num_infrequent_per_train_batch_
                               << ", for evaluation:  "
                               << infrequent_embeddings_[i].max_num_infrequent_per_batch_
                               << std::endl;

        infrequent_backward_comm_buffers_.emplace_back(buf.get(), max_back_buf_size);
        infrequent_forward_comm_buffers_.emplace_back(buf.get(), max_buf_size);
        buf->reserve({local_gpu_count}, &infrequent_forward_comm_buffers_.back().send_buffer_ptrs);
        buf->reserve({local_gpu_count}, &infrequent_backward_comm_buffers_.back().send_buffer_ptrs);
      }

      // For global barrier in eval
      {
        Tensor2<uint32_t> tensor;
        buf->reserve({1}, &tensor);
        d_barrier_store_.push_back(tensor);
      }
      buf->allocate();
    }

    // Frequent AR comm init
    if ((embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) ||
        (embedding_params_.communication_type == CommunicationType::IB_NVLink)) {
      if (!grouped_all_reduce_) {
        // Do your own all-reduce
        auto ar_comm = resource_manager_->get_ar_comm();
        frequent_embedding_handle_ = ar_comm->register_coll();

        // Frequent all reduce comm
        for (uint32_t i = 0; i < local_gpu_count; i++) {
          int cur_device = get_local_gpu(i).get_device_id();
          CudaDeviceContext context(cur_device);
          ar_comm->set_coll_buf(frequent_embedding_handle_,
                                frequent_embeddings_[i].get_gradients().get_ptr(),
                                frequent_embeddings_[i].get_gradients().get_size_in_bytes(), i);
          frequent_comms_.emplace_back(std::make_unique<AllReduceComm<emtype>>(
              ar_comm, frequent_embedding_handle_, &get_local_gpu(i)));
        }
        ar_comm->register_coll_buf(frequent_embedding_handle_);
      }
    }

    // Init after buffer allocation
    if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
#ifdef ENABLE_MPI
      ib_comm_ = resource_manager_->get_ib_comm();
      if (!ib_comm_) {
        resource_manager_->init_ib_comm();
        ib_comm_ = resource_manager_->get_ib_comm();
      }
      comm_stream_.resize(local_gpu_count);

      std::vector<size_t *> h_model_indices_sizes_ptrs(local_gpu_count);
      std::vector<size_t *> h_network_indices_sizes_ptrs(local_gpu_count);
      std::vector<emtype *> h_fwd_send_buffer_ptrs(local_gpu_count);
      std::vector<emtype *> h_bwd_send_buffer_ptrs(local_gpu_count);
      for (uint32_t i = 0; i < local_gpu_count; i++) {
        h_model_indices_sizes_ptrs[i] = infrequent_embeddings_[i].model_indices_sizes_.get_ptr();
        h_network_indices_sizes_ptrs[i] =
            infrequent_embeddings_[i].network_indices_sizes_.get_ptr();
        h_fwd_send_buffer_ptrs[i] = infrequent_forward_comm_buffers_[i].send_buffer.get_ptr();
        h_bwd_send_buffer_ptrs[i] = infrequent_backward_comm_buffers_[i].send_buffer.get_ptr();
      }

      // Forward coll init
      auto infrequent_forward_coll_handle = ib_comm_->register_hier_a2a_v_coll(true);
      auto ar_comm = resource_manager_->get_ar_comm();
      for (uint32_t i = 0; i < local_gpu_count; i++) {
        int cur_device = get_local_gpu(i).get_device_id();
        context.set_device(cur_device);

        // download pointers
        HCTR_LIB_THROW(
            hipMemcpyAsync(infrequent_embeddings_[i].model_indices_sizes_ptrs_.get_ptr(),
                            h_model_indices_sizes_ptrs.data(), sizeof(size_t *) * local_gpu_count,
                            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        HCTR_LIB_THROW(
            hipMemcpyAsync(infrequent_embeddings_[i].network_indices_sizes_ptrs_.get_ptr(),
                            h_network_indices_sizes_ptrs.data(), sizeof(size_t *) * local_gpu_count,
                            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        HCTR_LIB_THROW(
            hipMemcpyAsync(infrequent_forward_comm_buffers_[i].send_buffer_ptrs.get_ptr(),
                            h_fwd_send_buffer_ptrs.data(), sizeof(emtype *) * local_gpu_count,
                            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        HCTR_LIB_THROW(
            hipMemcpyAsync(infrequent_backward_comm_buffers_[i].send_buffer_ptrs.get_ptr(),
                            h_bwd_send_buffer_ptrs.data(), sizeof(emtype *) * local_gpu_count,
                            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        HCTR_LIB_THROW(hipStreamSynchronize(get_local_gpu(i).get_stream()));

        // Initialize IB comm
        HCTR_LIB_THROW(hipStreamCreateWithPriority(&comm_stream_[i], hipStreamNonBlocking, -100));
        ib_comm_->set_a2a_coll_stream(infrequent_forward_coll_handle, comm_stream_[i], i);

        ib_comm_->set_a2a_coll_buf(
            infrequent_forward_coll_handle,
            infrequent_forward_comm_buffers_[i].send_buffer.get_ptr(),
            infrequent_forward_comm_buffers_[i].send_buffer.get_size_in_bytes(),
            infrequent_forward_comm_buffers_[i].recv_buffer.get_ptr(),
            infrequent_forward_comm_buffers_[i].recv_buffer.get_size_in_bytes(), i);

        infrequent_forward_comms_.emplace_back(std::make_unique<HierAll2Allv_Multi_IB<emtype>>(
            i, infrequent_forward_coll_handle,
            infrequent_embeddings_[i].model_indices_sizes_ptrs_.get_ptr(), &get_local_gpu(i),
            ib_comm_, comm_stream_[i]));
      }
      ib_comm_->register_a2a_coll_buf(infrequent_forward_coll_handle);

      // Backward coll init
      auto infrequent_backward_coll_handle = ib_comm_->register_hier_a2a_v_coll(true);
      for (uint32_t i = 0; i < local_gpu_count; i++) {
        int cur_device = get_local_gpu(i).get_device_id();
        context.set_device(cur_device);

        ib_comm_->set_a2a_coll_stream(infrequent_backward_coll_handle, comm_stream_[i], i);
        ib_comm_->set_a2a_coll_buf(
            infrequent_backward_coll_handle,
            infrequent_backward_comm_buffers_[i].send_buffer.get_ptr(),
            infrequent_backward_comm_buffers_[i].send_buffer.get_size_in_bytes(),
            infrequent_backward_comm_buffers_[i].recv_buffer.get_ptr(),
            infrequent_backward_comm_buffers_[i].recv_buffer.get_size_in_bytes(), i);

        infrequent_backward_comms_.emplace_back(std::make_unique<HierAll2Allv_Multi_IB<emtype>>(
            i, infrequent_backward_coll_handle,
            infrequent_embeddings_[i].network_indices_sizes_ptrs_.get_ptr(), &get_local_gpu(i),
            ib_comm_, comm_stream_[i]));
      }
      ib_comm_->register_a2a_coll_buf(infrequent_backward_coll_handle);
#else
      HCTR_OWN_THROW(Error_t::WrongInput, "MPI is not enabled but trying to use IB_NVLink_Hier");
#endif
    }

    // 2.9 Single-node: copy some pointers arrays to device
    if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
      // Initialize GPU barrier
      gpu_barrier_ = std::make_unique<GPUBarrier>(resource_manager_->get_local_gpu_count(),
                                                  resource_manager_->get_local_gpu_device_id_list(),
                                                  graph_mode_);

      std::vector<const emtype *> frequent_vectors_cache_pointers(local_gpu_count);
      std::vector<emtype *> interaction_layer_input_pointers_train(local_gpu_count);
      std::vector<emtype *> interaction_layer_input_pointers_eval(local_gpu_count);
      std::vector<const emtype *> gradients_pointers(local_gpu_count);
      std::vector<const emtype *> frequent_partial_gradients_pointers(local_gpu_count);

      for (uint32_t i = 0; i < local_gpu_count; i++) {
        frequent_vectors_cache_pointers[i] =
            frequent_embeddings_[i].get_embedding_vectors_cache().get_ptr();
        interaction_layer_input_pointers_train[i] = train_output_tensors_[i].get_ptr();
        gradients_pointers[i] = train_output_tensors_[i].get_ptr();
        interaction_layer_input_pointers_eval[i] = evaluate_output_tensors_[i].get_ptr();
        frequent_partial_gradients_pointers[i] = frequent_embeddings_[i].get_gradients().get_ptr();
      }

      for (uint32_t i = 0; i < local_gpu_count; i++) {
        int cur_device = get_local_gpu(i).get_device_id();
        context.set_device(cur_device);

        HCTR_LIB_THROW(hipMemcpyAsync(
            frequent_embeddings_[i].embedding_vectors_cache_pointers_.get_ptr(),
            frequent_vectors_cache_pointers.data(), local_gpu_count * sizeof(float *),
            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));
        HCTR_LIB_THROW(hipMemcpyAsync(
            infrequent_embeddings_[i].interaction_layer_input_pointers_train_.get_ptr(),
            interaction_layer_input_pointers_train.data(), local_gpu_count * sizeof(emtype *),
            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));
        HCTR_LIB_THROW(hipMemcpyAsync(
            infrequent_embeddings_[i].interaction_layer_input_pointers_eval_.get_ptr(),
            interaction_layer_input_pointers_eval.data(), local_gpu_count * sizeof(emtype *),
            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));
        HCTR_LIB_THROW(hipMemcpyAsync(infrequent_embeddings_[i].gradients_pointers_.get_ptr(),
                                       gradients_pointers.data(),
                                       local_gpu_count * sizeof(emtype *), hipMemcpyHostToDevice,
                                       get_local_gpu(i).get_stream()));
        HCTR_LIB_THROW(hipMemcpyAsync(
            frequent_embeddings_[i].partial_gradients_pointers_.get_ptr(),
            frequent_partial_gradients_pointers.data(), local_gpu_count * sizeof(emtype *),
            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));
      }
    }
  } catch (const std::runtime_error &rt_err) {
    HCTR_LOG_S(ERROR, WORLD) << rt_err.what() << std::endl;
    throw;
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::init_model(const SparseTensors<dtype> &data,
                                                      size_t &wgrad_offset_in_bytes) {
  size_t local_gpu_count = resource_manager_->get_local_gpu_count();
  HCTR_LOG(INFO, ROOT, "Initializing Hybrid Embedding\n");
#pragma omp parallel for num_threads(local_gpu_count)
  for (size_t id = 0; id < local_gpu_count; ++id) {
    int cur_device = get_local_gpu(id).get_device_id();
    CudaDeviceContext context(cur_device);
    auto stream = get_local_gpu(id).get_stream();
    data_statistics_[id].data_to_unique_categories(data[id].get_value_tensor(), stream);
    model_[id].init_hybrid_model(calibration_[id], statistics_[id], data_statistics_[id], stream);
    frequent_embeddings_[id].initialize_embedding_vectors(data_statistics_[id].table_sizes,
                                                          wgrad_offset_in_bytes);
    infrequent_embeddings_[id].initialize_embedding_vectors(data_statistics_[id].table_sizes);

    if (embedding_params_.max_num_frequent_categories < (size_t)model_[id].num_frequent) {
      HCTR_OWN_THROW(
          Error_t::WrongInput,
          "Found too many frequent categories, please increase 'max_num_frequent_categories'");
    }
  }

  HCTR_LOG_S(INFO, ROOT) << "Initialized hybrid model with " << model_[0].num_frequent
                         << " frequent categories, probability of being frequent is "
                         << model_[0].frequent_probability << std::endl;

  size_t avg_train_infrequent = (1 - model_[0].frequent_probability) *
                                embedding_params_.slot_size_array.size() * get_batch_size(true);
  size_t avg_evaluate_infrequent = (1 - model_[0].frequent_probability) *
                                   embedding_params_.slot_size_array.size() * get_batch_size(false);

  HCTR_LOG_S(INFO, ROOT) << "Estimated number of infrequent categories per train batch: "
                         << avg_train_infrequent << ", eval batch: " << avg_evaluate_infrequent
                         << std::endl;

  size_t wgrad_size =
      model_[0].num_frequent * embedding_params_.embedding_vec_size * sizeof(emtype);
  if ((embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) ||
      (embedding_params_.communication_type == CommunicationType::IB_NVLink)) {
    if (!grouped_all_reduce_) {
      // Manage your own all-reduce
      auto ar_comm = resource_manager_->get_ar_comm();
      ar_comm->update_size(frequent_embedding_handle_, wgrad_size);
    } else {
      wgrad_offset_in_bytes += wgrad_size;
    }
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::setup_async_mode(AsyncReader<dtype> *train_data_reader,
                                                            AsyncReader<dtype> *eval_data_reader,
                                                            bool eval_overlap,
                                                            bool use_cuda_graph) {
  auto create_async_indices = [this](AsyncReader<dtype> *data_reader, bool is_train) {
    size_t batch_size = get_batch_size(is_train);
    size_t label_dim, dense_dim, sparse_dim, sample_size_items;
    data_reader->get_dimensions(label_dim, dense_dim, sparse_dim, sample_size_items);

    std::vector<FrequentEmbeddingBase<dtype> *> frequent_base_ptrs;
    for (auto &freq : frequent_embeddings_) {
      frequent_base_ptrs.push_back(dynamic_cast<FrequentEmbeddingBase<dtype> *>(&freq));
    }

    std::vector<InfrequentEmbeddingBase<dtype> *> infrequent_base_ptrs;
    for (auto &infreq : infrequent_embeddings_) {
      infrequent_base_ptrs.push_back(dynamic_cast<InfrequentEmbeddingBase<dtype> *>(&infreq));
    }

    return std::make_shared<IndexProcessor<dtype>>(
        model_, frequent_base_ptrs, infrequent_base_ptrs, resource_manager_,
        // double buffer for train, cache each batch for eval
        is_train ? 2 : data_reader->get_total_queue_size(), batch_size,
        embedding_params_.slot_size_array, embedding_params_.max_num_frequent_categories,
        data_reader->is_mixed_precision(), embedding_params_.communication_type, label_dim,
        dense_dim, sparse_dim, sample_size_items);
  };

  if (embedding_params_.use_train_precompute_indices) {
    train_async_indices_ = create_async_indices(train_data_reader, true);
    train_data_reader->register_extra_processing(train_async_indices_, false, use_cuda_graph);
  }
  if (embedding_params_.use_eval_precompute_indices) {
    eval_async_indices_ = create_async_indices(eval_data_reader, false);
    eval_data_reader->register_extra_processing(eval_async_indices_, eval_overlap, use_cuda_graph);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::index_calculation(bool is_train, bool is_first_batch,
                                                             int i, hipStream_t stream) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);

  if (is_train && embedding_params_.use_train_precompute_indices) {
    // Async indices, need to do nothing at all here
  } else if (!is_train && embedding_params_.use_eval_precompute_indices) {
    // Async indices, need to do nothing at all here
  } else {
    auto frequent_indices = (is_train) ? &frequent_embedding_train_indices_[i]
                                       : &frequent_embedding_evaluate_indices_[i];
    auto infrequent_indices = (is_train) ? &infrequent_embedding_train_indices_[i]
                                         : &infrequent_embedding_evaluate_indices_[i];

    auto data = (is_train) ? &data_train_[i] : &data_evaluate_[i];
    auto input = (is_train) ? train_input_tensors_[i].get_value_tensor()
                            : evaluate_input_tensors_[i].get_value_tensor();

    if (is_first_batch) {
      auto &before_idx_event = stream_manager_.get_event(i, "before_idx");
      auto &set_idx_stream = stream_manager_.get_stream(i, "set_idx_stream");
      HCTR_LIB_THROW(hipEventRecord(before_idx_event, stream));
      HCTR_LIB_THROW(hipStreamWaitEvent(set_idx_stream, before_idx_event));
    }

    PROFILE_RECORD("data_to_unique_categories.start", stream);
    data->data_to_unique_categories(input, stream);
    PROFILE_RECORD("data_to_unique_categories.stop", stream);

    compute_indices(*frequent_indices, *infrequent_indices, embedding_params_.communication_type,
                    is_train || is_first_batch, stream, stream_manager_, i,
                    resource_manager_->get_local_gpu(i)->get_sm_count());

    // Setting the indices involves hipMemcpy, so we'll only do that
    // for the first batch after we switch from train to eval (and from eval to train)
    if (is_first_batch) {
      auto &set_idx_stream = stream_manager_.get_stream(i, "set_idx_stream");
      auto &set_idx_event = stream_manager_.get_event(i, "set_idx");

      frequent_embeddings_[i].set_current_indices(frequent_indices, stream);
      infrequent_embeddings_[i].set_current_indices(infrequent_indices, stream);

      HCTR_LIB_THROW(hipEventRecord(set_idx_event, set_idx_stream));
      HCTR_LIB_THROW(hipStreamWaitEvent(stream, set_idx_event));
    }
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::forward(bool is_train, bool is_first_batch, int i,
                                                   hipStream_t stream, hipEvent_t *evt_ptr) {
  int cur_device = get_local_gpu(i).get_device_id();
  auto &gpu = get_local_gpu(i);
  CudaDeviceContext context(cur_device);

  auto &output = (is_train) ? train_output_tensors_[i] : evaluate_output_tensors_[i];

  PROFILE_RECORD("hybrid_embedding.forward.start", stream, false);
  if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
    // TODO: These copies need to be moved to the index computation
    HCTR_LIB_THROW(
        hipMemcpyAsync(infrequent_embeddings_[i].model_indices_offsets_.get_ptr(),
                        infrequent_embeddings_[i].indices_->model_indices_offsets_.get_ptr(),
                        infrequent_embeddings_[i].model_indices_offsets_.get_size_in_bytes(),
                        hipMemcpyDeviceToDevice, stream));

    HCTR_LIB_THROW(
        hipMemcpyAsync(infrequent_embeddings_[i].network_indices_offsets_.get_ptr(),
                        infrequent_embeddings_[i].indices_->network_indices_offsets_.get_ptr(),
                        infrequent_embeddings_[i].network_indices_offsets_.get_size_in_bytes(),
                        hipMemcpyDeviceToDevice, stream));

    HCTR_LIB_THROW(hipStreamSynchronize(stream));
    PROFILE_RECORD("multi_node_fre_forward_network.start", stream, false);
    frequent_embeddings_[i].forward_network(output.get_ptr(), false, stream);
    PROFILE_RECORD("multi_node_fre_forward_network.stop", stream, false);

    PROFILE_RECORD("multi_node_inf_forward_model.start", stream, false);
    infrequent_embeddings_[i].forward_model(
        infrequent_forward_comm_buffers_[i].send_buffer.get_ptr(), stream);
    PROFILE_RECORD("multi_node_inf_forward_model.stop", stream, false);

    PROFILE_RECORD("multi_node_inf_forward_a2a.start", stream, false);
    infrequent_forward_comms_[i]->communicate(stream);
    PROFILE_RECORD("multi_node_inf_forward_a2a.stop", stream, false);

    PROFILE_RECORD("multi_node_inf_forward_network.start", stream, false);
    infrequent_embeddings_[i].forward_network(
        infrequent_forward_comm_buffers_[i].recv_buffer.get_ptr(), output.get_ptr(), stream);
    PROFILE_RECORD("multi_node_inf_forward_network.stop", stream);
    evt_ptr = nullptr;

  } else if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
    PROFILE_RECORD("multi_node_inf_calculate_model_indices_sizes_from_offsets.start", stream);
    infrequent_embeddings_[i].calculate_model_indices_sizes_from_offsets(stream);
    PROFILE_RECORD("multi_node_inf_calculate_model_indices_sizes_from_offsets.stop", stream);
    PROFILE_RECORD("multi_node_inf_calculate_network_indices_sizes_from_offsets.start", stream);
    infrequent_embeddings_[i].calculate_network_indices_sizes_from_offsets(stream);
    PROFILE_RECORD("multi_node_inf_calculate_network_indices_sizes_from_offsets.stop", stream);
    infrequent_forward_comms_[i]->update_sizes(stream);

    PROFILE_RECORD("multi_node_inf_fused_intra_forward_model.start", stream);
    infrequent_embeddings_[i].fused_intra_forward_model(
        infrequent_forward_comm_buffers_[i].send_buffer_ptrs.get_ptr(), stream);
    PROFILE_RECORD("multi_node_inf_fused_intra_forward_model.stop", stream);

    PROFILE_RECORD("multi_node_inf_forward_a2a_init.start", stream);
    infrequent_forward_comms_[i]->initiate_communication(stream);
    PROFILE_RECORD("multi_node_inf_forward_a2a_init.stop", stream);
    // Let's initiate the communication as soon as we can and start every other non-urgent work here
    // This is for network
    if (is_train) {
      HCTR_LIB_THROW(hipEventRecord(*evt_ptr, stream));
    }

    // This is for frequent forward network running in a side stream
    auto &stream_side = stream_manager_.get_stream(i, "stream_side");
    auto &ready_freq_fwd_net = stream_manager_.get_event(i, "ready_freq_fwd_net");
    auto &freq_fwd_net_completion = stream_manager_.get_event(i, "freq_fwd_net_completion");

    if (is_train) {
      HCTR_LIB_THROW(hipEventRecord(ready_freq_fwd_net, stream));
      HCTR_LIB_THROW(hipStreamWaitEvent(stream_side, ready_freq_fwd_net));
    }

    PROFILE_RECORD("multi_node_inf_forward_a2a_wait_completion.stop", stream);
    infrequent_forward_comms_[i]->wait_completion(stream);
    PROFILE_RECORD("multi_node_inf_forward_a2a_wait_completion.stop", stream);

    if (!is_train) {
      if (eval_overlap_) {
        HCTR_LIB_THROW(hipStreamWaitEvent(stream, gpu.get_event("eval_comm_wait")));
      }
      HCTR_LIB_THROW(hipEventRecord(ready_freq_fwd_net, stream));
      HCTR_LIB_THROW(hipStreamWaitEvent(stream_side, ready_freq_fwd_net));
    }

    PROFILE_RECORD("multi_node_fre_forward_network.start", stream_side);
    frequent_embeddings_[i].forward_network(output.get_ptr(), false, stream_side);
    PROFILE_RECORD("multi_node_fre_forward_network.stop", stream_side);

    PROFILE_RECORD("multi_node_inf_hier_forward_network.start", stream);
    infrequent_embeddings_[i].hier_forward_network(
        infrequent_forward_comm_buffers_[i].recv_buffer.get_ptr(), output.get_ptr(), stream);
    PROFILE_RECORD("multi_node_inf_hier_forward_network.stop", stream, false);

    // join back frequent forward network
    HCTR_LIB_THROW(hipEventRecord(freq_fwd_net_completion, stream_side));
    HCTR_LIB_THROW(hipStreamWaitEvent(stream, freq_fwd_net_completion));

    if (!is_train) {
      if (eval_overlap_) {
        HCTR_LIB_THROW(hipEventRecord(gpu.get_event("eval_comp_wait"), stream));
      }

      // Global barrier
      HCTR_LIB_THROW(ncclAllReduce((const void *)d_barrier_store_[i].get_ptr(),
                                   d_barrier_store_[i].get_ptr(), sizeof(uint32_t),
                                   NcclDataType<uint32_t>::getType(), ncclSum,
                                   get_local_gpu(i).get_nccl(), stream));
    }
  } else {  // Assuming single node

    PROFILE_RECORD("single_node_inf_forward_network_direct.start", stream, false);
    infrequent_embeddings_[i].forward_network_direct(is_train, stream);
    PROFILE_RECORD("single_node_inf_forward_network_direct.stop", stream, false);

    PROFILE_RECORD("single_node_fre_forward_model.start", stream, false);
    // we just need to update frequent cache once in eval
    if (is_train) {
      frequent_embeddings_[i].forward_model(stream);
    } else {
      if (is_first_batch) {
        frequent_embeddings_[i].forward_model_eval(stream);
      }
    }
    PROFILE_RECORD("single_node_fre_forward_model.stop", stream, false);

    // This barrier is needed for two reasons:
    // - Ensure all infrequent vectors have been pushed before mlp
    // - Ensure all frequent vectors have been pushed before forward_network
    gpu_barrier_->sync_all_gpus(stream, i);

    PROFILE_RECORD("single_node_fre_forward_network.start", stream, false);
    frequent_embeddings_[i].forward_network(output.get_ptr(), true, stream);
    PROFILE_RECORD("single_node_fre_forward_network.stop", stream);
    evt_ptr = nullptr;
  }
  PROFILE_RECORD("hybrid_embedding.forward.stop", stream, false);
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::forward(bool is_train, bool is_first_batch) {
  size_t local_gpu_count = resource_manager_->get_local_gpu_count();

// Index calculations
#pragma omp parallel for num_threads(local_gpu_count)
  for (size_t i = 0; i < local_gpu_count; i++) {
    auto &gpu = get_local_gpu(i);
    CudaDeviceContext context(gpu.get_device_id());
    auto stream = is_train || !eval_overlap_ ? gpu.get_stream() : gpu.get_stream("eval_comms", -1);
    index_calculation(is_train, is_first_batch, i, stream);
    forward(is_train, is_first_batch, i, stream, nullptr);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::frequent_local_reduce(int i, hipStream_t stream) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);

  PROFILE_RECORD("fre_local_reduce.start", stream);
  bool reset_all = ((embedding_params_.communication_type == CommunicationType::IB_NVLink) ||
                    (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier));
  frequent_embeddings_[i].local_reduce(train_output_tensors_[i].get_ptr(), stream, reset_all);
  PROFILE_RECORD("fre_local_reduce.stop", stream);
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::backward_pre_communication(int i, hipStream_t stream) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);

  if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
    PROFILE_RECORD("multi_node_inf_update_network.start", stream);
    infrequent_embeddings_[i].update_network(
        train_output_tensors_[i].get_ptr(),
        infrequent_backward_comm_buffers_[i].send_buffer.get_ptr(), stream);
    PROFILE_RECORD("multi_node_inf_update_network.stop", stream);
  } else if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
    infrequent_backward_comms_[i]->update_sizes(stream);

    PROFILE_RECORD("multi_node_inf_fused_intra_update_network.start", stream);
    infrequent_embeddings_[i].fused_intra_update_network(
        train_output_tensors_[i].get_ptr(),
        infrequent_backward_comm_buffers_[i].send_buffer_ptrs.get_ptr(), stream);
    PROFILE_RECORD("multi_node_inf_fused_intra_update_network.stop", stream, false);
  }
}

// Everything that involves network and can be better overlapped with compute
template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::backward_communications(int i, hipStream_t stream) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);
  if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
    // Synchronize all GPUs before pulling the reduced gradients
    gpu_barrier_->sync_all_gpus(stream, i);

    float *dev_lr = lr_scheds_[i]->get_learning_rate();
    float scale = opt_params_[i].scaler;
    PROFILE_RECORD("single_node_fre_update_model_direct.start", stream, false);
    frequent_embeddings_[i].update_model_direct(dev_lr, scale, stream);
    PROFILE_RECORD("single_node_fre_update_model_direct.stop", stream, false);

    PROFILE_RECORD("single_node_inf_update_model_direct.start", stream, false);
    infrequent_embeddings_[i].update_model_direct(dev_lr, scale, stream);
    PROFILE_RECORD("single_node_inf_update_model_direct.stop", stream, false);
  } else {
    PROFILE_RECORD("multi_node_fre_backward_allreduce.start", stream, false);
    if (!grouped_all_reduce_) {
      frequent_comms_[i]->communicate(stream);
    }
    PROFILE_RECORD("multi_node_fre_backward_allreduce.stop", stream, false);

    PROFILE_RECORD("multi_node_inf_backward_a2a.start", stream, false);
    infrequent_backward_comms_[i]->communicate(stream);
    PROFILE_RECORD("multi_node_inf_backward_a2a.stop", stream, false);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::frequent_update(int i, hipStream_t stream) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);
  float *dev_lr = lr_scheds_[i]->get_learning_rate();
  float scale = opt_params_[i].scaler;

  if (embedding_params_.communication_type != CommunicationType::NVLink_SingleNode) {
    PROFILE_RECORD("multi_node_fre_update_model.start", stream, false);
    frequent_embeddings_[i].update_model(dev_lr, scale, stream);
    PROFILE_RECORD("multi_node_fre_update_model.stop", stream, false);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::backward_post_communication(int i, hipStream_t stream) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);
  float *dev_lr = lr_scheds_[i]->get_learning_rate();
  float scale = opt_params_[i].scaler;

  if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
    PROFILE_RECORD("multi_node_inf_update_model.start", stream, false);
    infrequent_embeddings_[i].update_model(
        infrequent_backward_comm_buffers_[i].recv_buffer.get_ptr(), dev_lr, scale, stream);
    PROFILE_RECORD("multi_node_inf_update_model.stop", stream, false);
  }
  if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
#ifdef ENABLE_MPI
    PROFILE_RECORD("multi_node_inf_hier_update_model.start", stream, false);
    infrequent_embeddings_[i].hier_update_model(
        infrequent_backward_comm_buffers_[i].recv_buffer.get_ptr(), dev_lr, scale, stream);

    if (graph_mode_) {
      hipEvent_t update_comm_event = stream_manager_.get_event(i, "update_comm_event");
      HCTR_LIB_THROW(hipEventRecord(update_comm_event, comm_stream_[i]));
      HCTR_LIB_THROW(hipStreamWaitEvent(stream, update_comm_event));
    }

    PROFILE_RECORD("multi_node_inf_hier_update_model.stop", stream, false);
#else
    HCTR_OWN_THROW(Error_t::WrongInput, "MPI is not enabled but trying to use IB_NVLink_Hier");
#endif
  }

#ifdef ENABLE_PROFILING
  bool should_run = PROFILE_RECORD_DATA("hybrid_run_time_params.start", stream);
  if (should_run) {
    std::string general_info =
        std::string("{\"global_batch_size\":") +
        std::to_string(embedding_params_.train_batch_size) + std::string(",") +
        std::string("\"slots_num\":") + std::to_string(embedding_params_.slot_num) +
        std::string(",") + std::string("\"total_gpu_count\":") +
        std::to_string(resource_manager_->get_global_gpu_count()) + std::string(",") +
        std::string("\"local_gpu_count\":") +
        std::to_string(resource_manager_->get_local_gpu_count()) + std::string(",") +
        std::string("\"total_categories\":") + std::to_string(model_[0].num_categories) +
        std::string(",") + std::string("\"bytes_of_dtype\":") + std::to_string(sizeof(dtype)) +
        std::string(",") + std::string("\"bytes_of_emtype\":") + std::to_string(sizeof(emtype)) +
        std::string(",") + std::string("\"embedding_vec_size\":") +
        std::to_string(embedding_params_.embedding_vec_size) + std::string(",");
    std::vector<uint32_t> num_frequent_categories;
    download_tensor(num_frequent_categories, frequent_embeddings_[i].d_num_frequent_sample_indices_,
                    stream);
    std::vector<uint32_t> infrequent_model_indices_offset;
    download_tensor(infrequent_model_indices_offset,
                    infrequent_embeddings_[i].model_indices_offsets_, stream);
    std::vector<uint32_t> infrequent_network_indices_offset;
    download_tensor(infrequent_network_indices_offset,
                    infrequent_embeddings_[i].network_indices_offsets_, stream);
    std::vector<uint32_t> network_cache_indices_offsets_;
    download_tensor(network_cache_indices_offsets_,
                    frequent_embeddings_[i].network_cache_indices_offsets_, stream);
    std::string device_info = std::string("\"num_frequent\":") +
                              std::to_string(model_[i].num_frequent) + std::string(",");
    device_info =
        device_info + std::string("\"num_infrequent\":") +
        std::to_string(model_[i].h_infrequent_model_table_offsets[embedding_params_.slot_num]) +
        std::string(",");
    device_info = device_info + std::string("\"num_frequent_samples\":") +
                  std::to_string(num_frequent_categories[0]) + std::string(",");
    device_info = device_info + std::string("\"infrequent_model_indices_offset\": [");
    for (auto size : infrequent_model_indices_offset) {
      device_info = device_info + std::to_string(size) + std::string(",");
    }
    device_info.pop_back();
    device_info = device_info + std::string("],");

    device_info = device_info + std::string("\"infrequent_network_indices_offset\": [");
    for (auto size : infrequent_network_indices_offset) {
      device_info = device_info + std::to_string(size) + std::string(",");
    }
    device_info.pop_back();
    device_info = device_info + std::string("],");

    device_info = device_info + std::string("\"network_cache_indices_offsets_\": [");
    for (auto size : network_cache_indices_offsets_) {
      device_info = device_info + std::to_string(size) + std::string(",");
    }
    device_info.pop_back();
    device_info = device_info + std::string("]");
    std::string run_time_info = general_info + device_info + std::string("}");

    PROFILE_RECORD_DATA("hybrid_run_time_params.stop", stream, run_time_info);
  }
#endif
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::backward() {
  size_t local_gpu_count = resource_manager_->get_local_gpu_count();

#pragma omp parallel for num_threads(local_gpu_count)
  for (size_t i = 0; i < local_gpu_count; i++) {
    auto stream = get_local_gpu(i).get_stream();
    auto cur_device = get_local_gpu(i).get_device_id();
    CudaDeviceContext context(cur_device);
    PROFILE_RECORD("hybrid_embedding.backward.start", stream, false);
    frequent_local_reduce(i, stream);
    backward_pre_communication(i, stream);
    backward_communications(i, stream);
    PROFILE_RECORD("hybrid_embedding.backward.stop", stream, false);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::update_params() {
  size_t local_gpu_count = resource_manager_->get_local_gpu_count();

#pragma omp parallel for num_threads(local_gpu_count)
  for (size_t i = 0; i < local_gpu_count; i++) {
    auto stream = get_local_gpu(i).get_stream();
    auto cur_device = get_local_gpu(i).get_device_id();
    CudaDeviceContext context(cur_device);
    PROFILE_RECORD("hybrid_embedding.update_params.start", stream, false);
    frequent_update(i, stream);
    backward_post_communication(i, stream);
    PROFILE_RECORD("hybrid_embedding.update_params.stop", stream, false);
  }
}

template <typename dtype, typename emtype>
TrainState HybridSparseEmbedding<dtype, emtype>::train(bool is_train, int i, TrainState state) {
  auto &stream = stream_manager_.get_stream(i, "main_stream");
  auto &ready_bot_mlp_fprop = stream_manager_.get_event(i, "ready_bot_mlp_fprop");
  auto &ready_top_mlp_fprop = stream_manager_.get_event(i, "ready_top_mlp_fprop");
  auto &finish_backward_pre = stream_manager_.get_event(i, "finish_backward_pre");
  auto &finish_iteration = stream_manager_.get_event(i, "finish_iteration");

  auto sync = [&state, &stream]() {
    if (state.event) {
      HCTR_LIB_THROW(hipStreamWaitEvent(stream, *state.event));
    }
  };

  hipEvent_t *event_ptr = nullptr;
  switch (state.state) {
    case TrainState_t::Init:
      sync();
      index_calculation(is_train, -1, i, stream);
      forward(is_train, -1, i, stream, &ready_bot_mlp_fprop);
      event_ptr = &ready_bot_mlp_fprop;
      break;
    case TrainState_t::BottomMLPFprop:
      sync();
      break;
    case TrainState_t::TopMLPFprop:
      HCTR_LIB_THROW(hipEventRecord(ready_top_mlp_fprop, stream));
      event_ptr = &ready_top_mlp_fprop;
      break;
    case TrainState_t::TopMLPBprop:
      break;
    case TrainState_t::BottomMLPBprop:
      if (overlap_ar_a2a_) {
        sync();
        frequent_local_reduce(i, stream);
      }
      break;
    case TrainState_t::MLPExchangeWgrad:
      if (!overlap_ar_a2a_) {
        sync();
        frequent_local_reduce(i, stream);
        backward_pre_communication(i, stream);
      }
      if (grouped_all_reduce_) {
        HCTR_LIB_THROW(hipEventRecord(finish_backward_pre, stream));
        event_ptr = &finish_backward_pre;
      }
      if (overlap_ar_a2a_) {
        backward_pre_communication(i, stream);
        backward_communications(i, stream);
        backward_post_communication(i, stream);
      }
      break;
    case TrainState_t::MLPUpdate:
      if (!overlap_ar_a2a_) {
        sync();
        backward_communications(i, stream);
        frequent_update(i, stream);
        backward_post_communication(i, stream);
      } else {
        sync();
        frequent_update(i, stream);
      }
      break;
    case TrainState_t::Finalize:
      HCTR_LIB_THROW(hipEventRecord(finish_iteration, stream));
      event_ptr = &finish_iteration;
      break;
    default:
      HCTR_OWN_THROW(Error_t::InvalidEnv, "hybrid embedding train reach invalid status");
  }
  state.event = event_ptr;
  return state;
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::init_params() {
  // TODO: create init_params()
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::load_parameters(
    std::string sparse_model, const DataSourceParams &data_source_params) {
  // TODO: create load_parameters()
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::dump_parameters(
    std::string sparse_model, const DataSourceParams &data_source_params) const {
  // TODO: create dump_parameters()
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::set_learning_rate(float lr) {
  HCTR_OWN_THROW(Error_t::WrongInput, "HybridSparseEmbedding only supports GPU LR scheduler");
}

template <typename dtype, typename emtype>
GpuLearningRateSchedulers HybridSparseEmbedding<dtype, emtype>::get_learning_rate_schedulers()
    const {
  return lr_scheds_;
}

template <typename dtype, typename emtype>
size_t HybridSparseEmbedding<dtype, emtype>::get_params_num() const {
  return 0;
}

template <typename dtype, typename emtype>
size_t HybridSparseEmbedding<dtype, emtype>::get_vocabulary_size() const {
  // TODO: create get_vocabulary_size()
  return 0;
}

template <typename dtype, typename emtype>
size_t HybridSparseEmbedding<dtype, emtype>::get_max_vocabulary_size() const {
  // TODO: create get_max_vocabulary_size()
  return 0;
}

template <typename dtype, typename emtype>
std::vector<TensorBag2> HybridSparseEmbedding<dtype, emtype>::get_train_output_tensors() const {
  return tensors_to_bags(train_output_tensors_);
}

template <typename dtype, typename emtype>
std::vector<TensorBag2> HybridSparseEmbedding<dtype, emtype>::get_evaluate_output_tensors() const {
  return tensors_to_bags(evaluate_output_tensors_);
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::compute_indices(
    FrequentEmbeddingCompression<dtype> &compression,
    InfrequentEmbeddingSelection<dtype> &selection, CommunicationType communication_type,
    bool compute_network_cache_indices, hipStream_t main_stream, StreamManager &manager,
    int raw_device_id, int sm_count) {
  hipStream_t stream_frequent_sample_indices =
      manager.get_stream(raw_device_id, "stream_frequent_sample_indices");
  hipStream_t stream_model_indices = manager.get_stream(raw_device_id, "stream_model_indices");
  hipStream_t stream_network_indices = manager.get_stream(raw_device_id, "stream_network_indices");

  hipEvent_t event_main = manager.get_event(raw_device_id, "event_main");
  hipEvent_t event_frequent_sample_indices =
      manager.get_event(raw_device_id, "event_frequent_sample_indices");
  hipEvent_t event_model_indices = manager.get_event(raw_device_id, "event_model_indices");
  hipEvent_t event_network_indices = manager.get_event(raw_device_id, "event_network_indices");

  // The new streams can only start after previous work in the main stream has completed
  HCTR_LIB_THROW(hipEventRecord(event_main, main_stream));
  HCTR_LIB_THROW(hipStreamWaitEvent(stream_frequent_sample_indices, event_main));
  HCTR_LIB_THROW(hipStreamWaitEvent(stream_model_indices, event_main));
  HCTR_LIB_THROW(hipStreamWaitEvent(stream_network_indices, event_main));

  // PROFILE_RECORD("index_calculation.start", main_stream);
  // PROFILE_RECORD("calculate_frequent_sample_indices.start", stream_frequent_sample_indices);
  compression.calculate_frequent_sample_indices(stream_frequent_sample_indices);
  // PROFILE_RECORD("calculate_frequent_sample_indices.stop", stream_frequent_sample_indices, true,
  //               -1, std::string("num_frequent: ") + std::to_string(model_.num_frequent));
  HCTR_LIB_THROW(hipEventRecord(event_frequent_sample_indices, stream_frequent_sample_indices));

  // PROFILE_RECORD("inf_calculate_model_indices.start", stream_model_indices);
  selection.calculate_model_indices(stream_model_indices);
  // PROFILE_RECORD("inf_calculate_model_indices.stop", stream_model_indices);
  HCTR_LIB_THROW(hipEventRecord(event_model_indices, stream_model_indices));

  if (communication_type != CommunicationType::NVLink_SingleNode) {
    // PROFILE_RECORD("inf_calculate_network_indices.start", stream_network_indices);
    selection.calculate_network_indices(sm_count, stream_network_indices);
    // PROFILE_RECORD("inf_calculate_network_indices.stop", stream_network_indices);
    HCTR_LIB_THROW(hipEventRecord(event_network_indices, stream_network_indices));
    HCTR_LIB_THROW(hipStreamWaitEvent(main_stream, event_network_indices));

  } else {
    hipStream_t stream_cache_masks = manager.get_stream(raw_device_id, "stream_cache_masks");
    hipStream_t stream_network_cache_indices =
        manager.get_stream(raw_device_id, "stream_network_cache_indices");
    hipStream_t stream_model_cache_indices =
        manager.get_stream(raw_device_id, "stream_model_cache_indices");
    hipEvent_t event_cache_masks = manager.get_event(raw_device_id, "event_cache_masks");
    hipEvent_t event_network_cache_indices =
        manager.get_event(raw_device_id, "event_network_cache_indices");
    hipEvent_t event_model_cache_indices =
        manager.get_event(raw_device_id, "event_model_cache_indices");

    HCTR_LIB_THROW(hipStreamWaitEvent(stream_cache_masks, event_main));

    // PROFILE_RECORD("single_node_fre_calculate_cache_masks.start", stream_cache_masks);
    compression.calculate_cache_masks(stream_cache_masks);
    // PROFILE_RECORD("single_node_fre_calculate_cache_masks.stop", stream_cache_masks);
    HCTR_LIB_THROW(hipEventRecord(event_cache_masks, stream_cache_masks));

    HCTR_LIB_THROW(hipStreamWaitEvent(stream_network_cache_indices, event_cache_masks));
    HCTR_LIB_THROW(hipStreamWaitEvent(stream_model_cache_indices, event_cache_masks));

    // PROFILE_RECORD("single_node_fre_calculate_network_cache_indices.start",
    //                stream_network_cache_indices);
    // we don't need to calculate cache indices during eval
    if (compute_network_cache_indices) {
      compression.calculate_network_cache_indices(stream_network_cache_indices);
    }
    // PROFILE_RECORD("single_node_fre_calculate_network_cache_indices.stop",
    //               stream_network_cache_indices);
    HCTR_LIB_THROW(hipEventRecord(event_network_cache_indices, stream_network_cache_indices));
    HCTR_LIB_THROW(hipStreamWaitEvent(main_stream, event_network_cache_indices));

    // PROFILE_RECORD("single_node_fre_calculate_model_cache_indices.start",
    //                stream_model_cache_indices);
    compression.calculate_model_cache_indices(sm_count, stream_model_cache_indices);
    // PROFILE_RECORD("single_node_fre_calculate_model_cache_indices.stop",
    // stream_model_cache_indices);
    HCTR_LIB_THROW(hipEventRecord(event_model_cache_indices, stream_model_cache_indices));
    HCTR_LIB_THROW(hipStreamWaitEvent(main_stream, event_model_cache_indices));
  }

  // Join streams to the main stream
  HCTR_LIB_THROW(hipStreamWaitEvent(main_stream, event_frequent_sample_indices));
  HCTR_LIB_THROW(hipStreamWaitEvent(main_stream, event_model_indices));

  // PROFILE_RECORD("index_calculation.stop", main_stream);
}

template class HybridSparseEmbedding<uint32_t, __half>;
template class HybridSparseEmbedding<uint32_t, float>;
template class HybridSparseEmbedding<long long, __half>;
template class HybridSparseEmbedding<long long, float>;
}  // namespace HugeCTR
