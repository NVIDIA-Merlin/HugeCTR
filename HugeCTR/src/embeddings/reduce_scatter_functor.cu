#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/embeddings/sparse_embedding_functors.hpp"
#include "HugeCTR/include/utils.hpp"

namespace HugeCTR {

template <typename TypeEmbeddingComp>
void SparseEmbeddingFunctors::reduce_scatter(size_t recv_count,
                                             const Tensors2<TypeEmbeddingComp> &send_tensors,
                                             Tensors2<TypeEmbeddingComp> &recv_tensors,
                                             const ResourceManager &resource_manager) {
  size_t local_gpu_count = resource_manager.get_local_gpu_count();
  size_t total_gpu_count = resource_manager.get_global_gpu_count();

  // need to know the type of TypeHashKey here
  ncclDataType_t type;
  switch (sizeof(TypeEmbeddingComp)) {
    case 2:
      type = ncclHalf;
      break;
    case 4:
      type = ncclFloat;
      break;
    default:
      CK_THROW_(Error_t::WrongInput, "Error: TypeHashKey not support by now");
  }

  // for multi GPUs, use NCCL to do Reduce-Scatter(supporting multi-node GPU servers)
  if (total_gpu_count > 1) {
    CK_NCCL_THROW_(ncclGroupStart());
    for (size_t id = 0; id < local_gpu_count; id++) {
      const auto &local_gpu = resource_manager.get_local_gpu(id);
      CK_NCCL_THROW_(ncclReduceScatter(send_tensors[id].get_ptr(),  // send buf
                                       recv_tensors[id].get_ptr(),  // recv buff
                                       recv_count, type, ncclSum, local_gpu->get_nccl(),
                                       local_gpu->get_stream()));
    }
    CK_NCCL_THROW_(ncclGroupEnd());
  }
  // for single GPU, just do memcpyD2D
  else {  // total_gpu_count == 1
    const auto &local_gpu = resource_manager.get_local_gpu(0);
    CudaDeviceContext context(local_gpu->get_device_id());
    CK_CUDA_THROW_(hipMemcpyAsync(recv_tensors[0].get_ptr(), send_tensors[0].get_ptr(),
                                   recv_count * sizeof(TypeEmbeddingComp), hipMemcpyDeviceToDevice,
                                   local_gpu->get_stream()));
  }

  return;
}

template void SparseEmbeddingFunctors::reduce_scatter<float>(
    size_t recv_count, const Tensors2<float> &send_tensors, Tensors2<float> &recv_tensors,
    const ResourceManager &resource_manager);

template void SparseEmbeddingFunctors::reduce_scatter<__half>(
    size_t recv_count, const Tensors2<__half> &send_tensors, Tensors2<__half> &recv_tensors,
    const ResourceManager &resource_manager);

}  // namespace HugeCTR