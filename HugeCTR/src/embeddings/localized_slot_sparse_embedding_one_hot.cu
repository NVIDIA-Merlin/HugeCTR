#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/embeddings/localized_slot_sparse_embedding_one_hot.hpp"

#ifdef ENABLE_MPI
#include <mpi.h>
#endif

#include <filesystem>
#include <numeric>

namespace HugeCTR {
namespace localized_onehot_filter_keys_kernel {

template <typename TypeKey>
__global__ void select_value_by_slot_id_kernel(const TypeKey *value, size_t num,
                                               TypeKey *filter_value, size_t slot_num_per_gpu,
                                               size_t slot_num, size_t global_id,
                                               size_t global_num) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num) {
    int batch_size = tid / slot_num;
    int slot_id = tid % slot_num;
    if (slot_id % global_num == global_id) {
      int res_slot_id = slot_id / global_num;
      filter_value[batch_size * slot_num_per_gpu + res_slot_id] = __ldg(value + tid);
    }
  }
}
}  // namespace localized_onehot_filter_keys_kernel

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::filter_keys_per_gpu(
    bool is_train, size_t id, size_t global_id, size_t global_num) {
  const SparseTensor<TypeHashKey> &all_gather_key = embedding_data_.get_input_keys(is_train)[id];
  auto &local_gpu = embedding_data_.get_local_gpu(id);
  Tensor2<TypeHashKey> value_tensor = embedding_data_.get_value_tensors(is_train)[id];
  std::shared_ptr<size_t> nnz_ptr = embedding_data_.get_nnz_array(is_train)[id];

  if (all_gather_key.get_dimensions().size() != 2) {
    CK_THROW_(Error_t::WrongInput, "localized embedding all gather key dimension != 2");
  }

  size_t batch_size = embedding_data_.embedding_params_.get_batch_size(is_train);
  size_t slot_num_per_gpu = slot_num_per_gpu_[id];
  size_t slot_num = (all_gather_key.rowoffset_count() - 1) / batch_size;

  constexpr size_t block_size = 256;
  size_t grid_size = (all_gather_key.nnz() - 1) / block_size + 1;
  localized_onehot_filter_keys_kernel::
      select_value_by_slot_id_kernel<<<grid_size, block_size, 0, local_gpu.get_stream()>>>(
          all_gather_key.get_value_ptr(), all_gather_key.nnz(), value_tensor.get_ptr(),
          slot_num_per_gpu, slot_num, global_id, global_num);

  *nnz_ptr = (all_gather_key.nnz() / slot_num) * slot_num_per_gpu;
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<
    TypeHashKey, TypeEmbeddingComp>::data_to_unique_categories_per_gpu(bool is_train, size_t id) {
  SparseTensor<TypeHashKey> &all_gather_key = embedding_data_.get_input_keys(is_train)[id];
  auto &local_gpu = embedding_data_.get_local_gpu(id);

  if (all_gather_key.get_dimensions().size() != 2) {
    CK_THROW_(Error_t::WrongInput, "localized embedding all gather key dimension != 2");
  }

  size_t batch_size = embedding_data_.embedding_params_.get_batch_size(is_train);
  size_t nnz = all_gather_key.nnz();
  size_t slot_num = (all_gather_key.rowoffset_count() - 1) / batch_size;

  data_to_unique_categories(all_gather_key.get_value_ptr(),
                            embedding_data_.embedding_offsets_[id].get_ptr(), slot_num, nnz,
                            local_gpu.get_stream());
}

namespace {

template <typename value_type>
__global__ void upload_value_tensor_kernel(value_type *value_buf, size_t *index_buf,
                                           value_type *dst_tensor, int emb_vec_size, size_t len) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < len) {
    size_t src_offset = gid * emb_vec_size;
    size_t dst_offset = index_buf[gid] * emb_vec_size;
    for (int i = 0; i < emb_vec_size; i++) {
      dst_tensor[dst_offset + i] = value_buf[src_offset + i];
    }
  }
}

}  // namespace

template <typename TypeHashKey, typename TypeEmbeddingComp>
LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::
    LocalizedSlotSparseEmbeddingOneHot(
        const Tensors2<TypeHashKey> &train_row_offsets_tensors,
        const Tensors2<TypeHashKey> &train_value_tensors,
        const std::vector<std::shared_ptr<size_t>> &train_nnz_array,
        const Tensors2<TypeHashKey> &evaluate_row_offsets_tensors,
        const Tensors2<TypeHashKey> &evaluate_value_tensors,
        const std::vector<std::shared_ptr<size_t>> &evaluate_nnz_array,
        const SparseEmbeddingHashParams &embedding_params,
        const std::shared_ptr<ResourceManager> &resource_manager)
    : embedding_data_(train_row_offsets_tensors, train_value_tensors, train_nnz_array,
                      evaluate_row_offsets_tensors, evaluate_value_tensors, evaluate_nnz_array,
                      Embedding_t::LocalizedSlotSparseEmbeddingOneHot, embedding_params,
                      resource_manager),
      slot_size_array_(embedding_params.slot_size_array) {
  embedding_data_.embedding_params_.is_data_parallel =
      false;  // this ctor is only used for embedding plugin
  try {
    max_vocabulary_size_ = 0;
    for (size_t slot_size : slot_size_array_) {
      max_vocabulary_size_ += slot_size;
    }

    max_vocabulary_size_per_gpu_ =
        cal_max_voc_size_per_gpu(slot_size_array_, embedding_data_.get_resource_manager());

    MESSAGE_("max_vocabulary_size_per_gpu_=" + std::to_string(max_vocabulary_size_per_gpu_));

    CudaDeviceContext context;
    for (size_t id = 0; id < embedding_data_.get_resource_manager().get_local_gpu_count(); id++) {
      context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

      size_t gid = embedding_data_.get_local_gpu(id).get_global_id();
      size_t slot_num_per_gpu =
          embedding_data_.embedding_params_.slot_num /
              embedding_data_.get_resource_manager().get_global_gpu_count() +
          ((gid < embedding_data_.embedding_params_.slot_num %
                      embedding_data_.get_resource_manager().get_global_gpu_count())
               ? 1
               : 0);
      slot_num_per_gpu_.push_back(slot_num_per_gpu);

      // new GeneralBuffer objects
      const std::shared_ptr<GeneralBuffer2<CudaAllocator>> &buf = embedding_data_.get_buffer(id);

      // new hash table value vectors
      {
        const std::shared_ptr<BufferBlock2<float>> &block = buf->create_block<float>();
        Tensors2<float> tensors;
        for (size_t i = 0; i < slot_size_array_.size(); i++) {
          if ((i % embedding_data_.get_resource_manager().get_global_gpu_count()) == gid) {
            Tensor2<float> tensor;
            block->reserve(
                {slot_size_array_[i], embedding_data_.embedding_params_.embedding_vec_size},
                &tensor);
            tensors.push_back(tensor);
          }
        }
        value_table_tensors_.push_back(tensors);
        hash_table_value_tensors_.push_back(block->as_tensor());
      }

      // list of top categories, from single iteration worth of data, so max size is same as
      // hash_table_value_index_ array
      {
        std::cout << "Initializing size_top_categories_ and top_categories.." << std::endl;
        Tensor2<size_t> tensor;
        buf->reserve({1, embedding_data_.embedding_params_.get_universal_batch_size() *
                             embedding_data_.embedding_params_.max_feature_num},
                     &tensor);
        size_top_categories_.push_back(0);
        top_categories_.push_back(tensor);
        // std::cout << "top_categories size : " << Base::get_universal_batch_size() *
        // Base::get_max_feature_num()
        // << std::endl;
      }

      // new hash table value_index that get() from HashTable
      {
        Tensor2<size_t> tensor;
        buf->reserve({1, embedding_data_.embedding_params_.get_universal_batch_size() *
                             embedding_data_.embedding_params_.max_feature_num},
                     &tensor);
        hash_value_index_tensors_.push_back(tensor);
      }

      // new embedding features reduced by hash table values(results of forward)
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve(
            {embedding_data_.embedding_params_.get_universal_batch_size() * slot_num_per_gpu,
             embedding_data_.embedding_params_.embedding_vec_size},
            &tensor);
        embedding_feature_tensors_.push_back(tensor);
      }

      // new wgrad used by backward
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_batch_size(true) * slot_num_per_gpu,
                      embedding_data_.embedding_params_.embedding_vec_size},
                     &tensor);
        wgrad_tensors_.push_back(tensor);
      }

      // new optimizer params used by update_params
      switch (embedding_data_.embedding_params_.opt_params.optimizer) {
        case Optimizer_t::SGD:
          break;

        default:
          throw std::runtime_error(
              std::string("[HCDEBUG][ERROR] Runtime error: Invalid optimizer type\n"));
      }

      // the tenosrs for storing slot ids
      // TODO: init to -1 ?
      {
        Tensor2<size_t> tensor;
        buf->reserve({max_vocabulary_size_per_gpu_, 1}, &tensor);
        hash_table_slot_id_tensors_.push_back(tensor);
      }

      // temp tensors for all2all
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({embedding_data_.get_universal_batch_size_per_gpu() *
                          embedding_data_.embedding_params_.slot_num,
                      embedding_data_.embedding_params_.embedding_vec_size},
                     &tensor);
        all2all_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_universal_batch_size() *
                          embedding_data_.embedding_params_.slot_num,
                      embedding_data_.embedding_params_.embedding_vec_size},
                     &tensor);
        utest_forward_temp_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({embedding_data_.get_batch_size_per_gpu(true) *
                          embedding_data_.embedding_params_.slot_num,
                      embedding_data_.embedding_params_.embedding_vec_size},
                     &tensor);
        utest_all2all_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({embedding_data_.get_batch_size_per_gpu(true) *
                          embedding_data_.embedding_params_.slot_num,
                      embedding_data_.embedding_params_.embedding_vec_size},
                     &tensor);
        utest_reorder_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_batch_size(true) *
                          embedding_data_.embedding_params_.slot_num,
                      embedding_data_.embedding_params_.embedding_vec_size},
                     &tensor);
        utest_backward_temp_tensors_.push_back(tensor);
      }
      {
        Tensor2<uint32_t> tensor;
        buf->reserve({1, slot_num_per_gpu}, &tensor);
        mapping_offsets_per_gpu_tensors_.push_back(tensor);
      }

// init GenenralBuffers to do real allocation
#ifndef NDEBUG
      std::cout << " max_feature_num_:" << embedding_data_.embedding_params_.max_feature_num
                << std::endl;
#endif

    }  // end of for(int id = 0; id < embedding_data_.get_local_gpu_count(); id++)

#pragma omp parallel num_threads(embedding_data_.get_resource_manager().get_local_gpu_count())
    {
      size_t id = omp_get_thread_num();
      CudaDeviceContext context(embedding_data_.get_local_gpu(id).get_device_id());
      embedding_data_.get_buffer(id)->allocate();
      CK_CUDA_THROW_(hipStreamSynchronize(embedding_data_.get_local_gpu(id).get_stream()));
    }

    // get the mapping table between local value_index and input value_index
    for (size_t id = 0; id < embedding_data_.get_resource_manager().get_local_gpu_count(); id++) {
      context.set_device(embedding_data_.get_local_gpu(id).get_device_id());
      uint32_t slot_sizes_prefix_sum = 0;
      uint32_t slot_sizes_prefix_sum_local = 0;
      int slot_num = 0;
      for (size_t i = 0; i < slot_size_array_.size(); i++) {
        size_t global_id = embedding_data_.get_local_gpu(id).get_global_id();
        size_t slot_size = slot_size_array_[i];
        if (i % embedding_data_.get_resource_manager().get_global_gpu_count() == global_id) {
          uint32_t mapping_offset = slot_sizes_prefix_sum - slot_sizes_prefix_sum_local;
          CK_CUDA_THROW_(hipMemcpy(&((mapping_offsets_per_gpu_tensors_[id].get_ptr())[slot_num]),
                                    &mapping_offset, sizeof(uint32_t), hipMemcpyHostToDevice));
          slot_sizes_prefix_sum_local += slot_size;
          slot_num++;
        }
        slot_sizes_prefix_sum += slot_size;
      }
    }

    // Check whether the P2P access can be enabled
    if (embedding_data_.get_resource_manager().get_local_gpu_count() > 1 &&
        !embedding_data_.get_resource_manager().all_p2p_enabled()) {
      throw std::runtime_error(
          std::string("[HCDEBUG][ERROR] Runtime error: Localized_slot_sparse_embedding_one_hot "
                      "cannot be used on machine without GPU peer2peer access support. \n"));
    }
#ifdef ENABLE_MPI
    {
      int num_processor;
      MPI_Comm_size(MPI_COMM_WORLD, &num_processor);
      if (num_processor > 1) {
        throw std::runtime_error(
            std::string("[HCDEBUG][ERROR] Runtime error: Localized_slot_sparse_embedding_one_hot "
                        "cannot support multi-node currently. \n"));
      }
    }
#endif

    std::shared_ptr<GeneralBuffer2<CudaManagedAllocator>> unified_buf =
        GeneralBuffer2<CudaManagedAllocator>::create();
    unified_buf->reserve({embedding_data_.get_resource_manager().get_local_gpu_count()},
                         &train_embedding_features_);
    unified_buf->reserve({embedding_data_.get_resource_manager().get_local_gpu_count()},
                         &evaluate_embedding_features_);
    unified_buf->allocate();

    for (size_t id = 0; id < embedding_data_.get_resource_manager().get_local_gpu_count(); id++) {
      train_embedding_features_.get_ptr()[id] =
          embedding_data_.get_output_tensors(true)[id].get_ptr();
      evaluate_embedding_features_.get_ptr()[id] =
          embedding_data_.get_output_tensors(false)[id].get_ptr();
    }

  } catch (const std::runtime_error &rt_err) {
    std::cerr << rt_err.what() << std::endl;
    throw;
  }

  return;
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::
    LocalizedSlotSparseEmbeddingOneHot(const SparseTensors<TypeHashKey> &train_keys,
                                       const SparseTensors<TypeHashKey> &evaluate_keys,
                                       const SparseEmbeddingHashParams &embedding_params,
                                       const std::shared_ptr<ResourceManager> &resource_manager)
    : embedding_data_(Embedding_t::LocalizedSlotSparseEmbeddingOneHot, train_keys, evaluate_keys,
                      embedding_params, resource_manager),
      slot_size_array_(embedding_params.slot_size_array) {
  try {
    max_vocabulary_size_ = 0;
    for (size_t slot_size : slot_size_array_) {
      max_vocabulary_size_ += slot_size;
    }

    max_vocabulary_size_per_gpu_ =
        cal_max_voc_size_per_gpu(slot_size_array_, embedding_data_.get_resource_manager());

    MESSAGE_("max_vocabulary_size_per_gpu_=" + std::to_string(max_vocabulary_size_per_gpu_));

    CudaDeviceContext context;
    for (size_t id = 0; id < embedding_data_.get_resource_manager().get_local_gpu_count(); id++) {
      context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

      size_t gid = embedding_data_.get_local_gpu(id).get_global_id();
      size_t slot_num_per_gpu =
          embedding_data_.embedding_params_.slot_num /
              embedding_data_.get_resource_manager().get_global_gpu_count() +
          ((gid < embedding_data_.embedding_params_.slot_num %
                      embedding_data_.get_resource_manager().get_global_gpu_count())
               ? 1
               : 0);
      slot_num_per_gpu_.push_back(slot_num_per_gpu);

      // new GeneralBuffer objects
      const std::shared_ptr<GeneralBuffer2<CudaAllocator>> &buf = embedding_data_.get_buffer(id);

      // new hash table value vectors
      {
        const std::shared_ptr<BufferBlock2<float>> &block = buf->create_block<float>();
        Tensors2<float> tensors;
        for (size_t i = 0; i < slot_size_array_.size(); i++) {
          if ((i % embedding_data_.get_resource_manager().get_global_gpu_count()) == gid) {
            Tensor2<float> tensor;
            block->reserve(
                {slot_size_array_[i], embedding_data_.embedding_params_.embedding_vec_size},
                &tensor);
            tensors.push_back(tensor);
          }
        }
        value_table_tensors_.push_back(tensors);
        hash_table_value_tensors_.push_back(block->as_tensor());
      }
      {
        Tensor2<TypeHashKey> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_batch_size(true),
                      embedding_data_.embedding_params_.max_feature_num},
                     &tensor);
        embedding_data_.train_value_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeHashKey> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_batch_size(false),
                      embedding_data_.embedding_params_.max_feature_num},
                     &tensor);
        embedding_data_.evaluate_value_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeHashKey> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_batch_size(true) *
                          embedding_data_.embedding_params_.slot_num +
                      1},
                     &tensor);
        embedding_data_.train_row_offsets_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeHashKey> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_batch_size(false) *
                          embedding_data_.embedding_params_.slot_num +
                      1},
                     &tensor);
        embedding_data_.evaluate_row_offsets_tensors_.push_back(tensor);
      }
      { embedding_data_.train_nnz_array_.push_back(std::make_shared<size_t>(0)); }
      { embedding_data_.evaluate_nnz_array_.push_back(std::make_shared<size_t>(0)); }

      // list of top categories, from single iteration worth of data, so max size is same as
      {
        std::cout << "Initializing size_top_categories_ and top_categories.." << std::endl;
        Tensor2<size_t> tensor;
        buf->reserve({1, embedding_data_.embedding_params_.get_universal_batch_size() *
                             embedding_data_.embedding_params_.max_feature_num},
                     &tensor);
        size_top_categories_.push_back(0);
        top_categories_.push_back(tensor);
      }

      // new hash table value_index that get() from HashTable
      {
        Tensor2<size_t> tensor;
        buf->reserve({1, embedding_data_.embedding_params_.get_universal_batch_size() *
                             embedding_data_.embedding_params_.max_feature_num},
                     &tensor);
        hash_value_index_tensors_.push_back(tensor);
      }

      // new embedding features reduced by hash table values(results of forward)
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve(
            {embedding_data_.embedding_params_.get_universal_batch_size() * slot_num_per_gpu,
             embedding_data_.embedding_params_.embedding_vec_size},
            &tensor);
        embedding_feature_tensors_.push_back(tensor);
      }

      // new wgrad used by backward
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_batch_size(true) * slot_num_per_gpu,
                      embedding_data_.embedding_params_.embedding_vec_size},
                     &tensor);
        wgrad_tensors_.push_back(tensor);
      }

      // new optimizer params used by update_params
      switch (embedding_data_.embedding_params_.opt_params.optimizer) {
        case Optimizer_t::SGD:
          break;

        default:
          throw std::runtime_error(
              std::string("[HCDEBUG][ERROR] Runtime error: Invalid optimizer type\n"));
      }

      // the tenosrs for storing slot ids
      // TODO: init to -1 ?
      {
        Tensor2<size_t> tensor;
        buf->reserve({max_vocabulary_size_per_gpu_, 1}, &tensor);
        hash_table_slot_id_tensors_.push_back(tensor);
      }

      // temp tensors for all2all
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({embedding_data_.get_universal_batch_size_per_gpu() *
                          embedding_data_.embedding_params_.slot_num,
                      embedding_data_.embedding_params_.embedding_vec_size},
                     &tensor);
        all2all_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_universal_batch_size() *
                          embedding_data_.embedding_params_.slot_num,
                      embedding_data_.embedding_params_.embedding_vec_size},
                     &tensor);
        utest_forward_temp_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({embedding_data_.get_batch_size_per_gpu(true) *
                          embedding_data_.embedding_params_.slot_num,
                      embedding_data_.embedding_params_.embedding_vec_size},
                     &tensor);
        utest_all2all_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({embedding_data_.get_batch_size_per_gpu(true) *
                          embedding_data_.embedding_params_.slot_num,
                      embedding_data_.embedding_params_.embedding_vec_size},
                     &tensor);
        utest_reorder_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({embedding_data_.embedding_params_.get_batch_size(true) *
                          embedding_data_.embedding_params_.slot_num,
                      embedding_data_.embedding_params_.embedding_vec_size},
                     &tensor);
        utest_backward_temp_tensors_.push_back(tensor);
      }
      {
        Tensor2<uint32_t> tensor;
        buf->reserve({1, slot_num_per_gpu}, &tensor);
        mapping_offsets_per_gpu_tensors_.push_back(tensor);
      }

// init GenenralBuffers to do real allocation
#ifndef NDEBUG
      std::cout << " max_feature_num_:" << embedding_data_.embedding_params_.max_feature_num
                << std::endl;
#endif

    }  // end of for(int id = 0; id < embedding_data_.get_local_gpu_count(); id++)

#pragma omp parallel for num_threads(embedding_data_.get_resource_manager().get_local_gpu_count())
    for (size_t id = 0; id < embedding_data_.get_resource_manager().get_local_gpu_count(); ++id) {
      CudaDeviceContext context(embedding_data_.get_local_gpu(id).get_device_id());
      embedding_data_.get_buffer(id)->allocate();

      // filling rowoffset and slot_size_array
      hipStream_t stream = embedding_data_.get_local_gpu(id).get_stream();
      CK_CUDA_THROW_(hipStreamSynchronize(stream));
    }

    {
      std::vector<TypeHashKey> embedding_offsets;
      TypeHashKey slot_sizes_prefix_sum = 0;
      for (size_t i = 0; i < slot_size_array_.size(); i++) {
        embedding_offsets.push_back(slot_sizes_prefix_sum);
        slot_sizes_prefix_sum += slot_size_array_[i];
      }
      for (size_t id = 0; id < embedding_data_.get_resource_manager().get_local_gpu_count(); ++id) {
        CudaDeviceContext context(embedding_data_.get_local_gpu(id).get_device_id());

        CK_CUDA_THROW_(
            hipMemcpy(embedding_data_.embedding_offsets_[id].get_ptr(), embedding_offsets.data(),
                       embedding_offsets.size() * sizeof(TypeHashKey), hipMemcpyHostToDevice));

        size_t slot_num_per_gpu = slot_num_per_gpu_[id];
        {
          std::vector<TypeHashKey> rowoffset_host(
              embedding_data_.embedding_params_.get_batch_size(true) *
                  embedding_data_.embedding_params_.slot_num +
              1);
          std::iota(rowoffset_host.begin(), rowoffset_host.end(), 0);
          CK_CUDA_THROW_(hipMemcpy(
              embedding_data_.train_row_offsets_tensors_[id].get_ptr(), rowoffset_host.data(),
              rowoffset_host.size() * sizeof(TypeHashKey), hipMemcpyHostToDevice));
        }
        {
          std::vector<TypeHashKey> rowoffset_host(
              embedding_data_.embedding_params_.get_batch_size(false) *
                  embedding_data_.embedding_params_.slot_num +
              1);
          std::iota(rowoffset_host.begin(), rowoffset_host.end(), 0);
          CK_CUDA_THROW_(hipMemcpy(
              embedding_data_.evaluate_row_offsets_tensors_[id].get_ptr(), rowoffset_host.data(),
              rowoffset_host.size() * sizeof(TypeHashKey), hipMemcpyHostToDevice));
        }
      }
    }

    // get the mapping table between local value_index and input value_index
    for (size_t id = 0; id < embedding_data_.get_resource_manager().get_local_gpu_count(); id++) {
      context.set_device(embedding_data_.get_local_gpu(id).get_device_id());
      uint32_t slot_sizes_prefix_sum = 0;
      uint32_t slot_sizes_prefix_sum_local = 0;
      int slot_num = 0;
      for (size_t i = 0; i < slot_size_array_.size(); i++) {
        size_t global_id = embedding_data_.get_local_gpu(id).get_global_id();
        size_t slot_size = slot_size_array_[i];
        if (i % embedding_data_.get_resource_manager().get_global_gpu_count() == global_id) {
          uint32_t mapping_offset = slot_sizes_prefix_sum - slot_sizes_prefix_sum_local;
          CK_CUDA_THROW_(hipMemcpy(&((mapping_offsets_per_gpu_tensors_[id].get_ptr())[slot_num]),
                                    &mapping_offset, sizeof(uint32_t), hipMemcpyHostToDevice));
          slot_sizes_prefix_sum_local += slot_size;
          slot_num++;
        }
        slot_sizes_prefix_sum += slot_size;
      }
    }

    // Check whether the P2P access can be enabled
    if (embedding_data_.get_resource_manager().get_local_gpu_count() > 1 &&
        !embedding_data_.get_resource_manager().all_p2p_enabled()) {
      throw std::runtime_error(
          std::string("[HCDEBUG][ERROR] Runtime error: Localized_slot_sparse_embedding_one_hot "
                      "cannot be used on machine without GPU peer2peer access support. \n"));
    }
#ifdef ENABLE_MPI
    {
      int num_processor;
      MPI_Comm_size(MPI_COMM_WORLD, &num_processor);
      if (num_processor > 1) {
        throw std::runtime_error(
            std::string("[HCDEBUG][ERROR] Runtime error: Localized_slot_sparse_embedding_one_hot "
                        "cannot support multi-node currently. \n"));
      }
    }
#endif

    std::shared_ptr<GeneralBuffer2<CudaManagedAllocator>> unified_buf =
        GeneralBuffer2<CudaManagedAllocator>::create();
    unified_buf->reserve({embedding_data_.get_resource_manager().get_local_gpu_count()},
                         &train_embedding_features_);
    unified_buf->reserve({embedding_data_.get_resource_manager().get_local_gpu_count()},
                         &evaluate_embedding_features_);
    unified_buf->allocate();

    for (size_t id = 0; id < embedding_data_.get_resource_manager().get_local_gpu_count(); id++) {
      train_embedding_features_.get_ptr()[id] =
          embedding_data_.get_output_tensors(true)[id].get_ptr();
      evaluate_embedding_features_.get_ptr()[id] =
          embedding_data_.get_output_tensors(false)[id].get_ptr();
    }

  } catch (const std::runtime_error &rt_err) {
    std::cerr << rt_err.what() << std::endl;
    throw;
  }

  return;
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::load_parameters(
    std::string sparse_model) {
  if (!std::filesystem::exists(sparse_model)) {
    CK_THROW_(Error_t::WrongInput, std::string("Error: folder ") + sparse_model + " doesn't exist");
  }
  const std::string key_file(sparse_model + "/key");
  const std::string slot_file(sparse_model + "/slot_id");
  const std::string vec_file(sparse_model + "/emb_vector");

  std::ifstream key_stream(key_file, std::ifstream::binary);
  std::ifstream slot_stream(slot_file, std::ifstream::binary);
  std::ifstream vec_stream(vec_file, std::ifstream::binary);
  // check if file is opened successfully
  if (!vec_stream.is_open() || !key_stream.is_open() || !slot_stream.is_open()) {
    CK_THROW_(Error_t::WrongInput, "Error: file not open for reading");
  }

  size_t key_file_size_in_byte = std::filesystem::file_size(key_file);
  size_t slot_file_size_in_byte = std::filesystem::file_size(slot_file);
  size_t vec_file_size_in_byte = std::filesystem::file_size(vec_file);

  size_t key_size = sizeof(long long);
  size_t slot_size = sizeof(size_t);
  size_t vec_size = sizeof(float) * embedding_data_.embedding_params_.embedding_vec_size;
  size_t key_num = key_file_size_in_byte / key_size;
  size_t slot_num = slot_file_size_in_byte / slot_size;
  size_t vec_num = vec_file_size_in_byte / vec_size;

  if (key_num != vec_num || key_file_size_in_byte % key_size != 0 ||
      vec_file_size_in_byte % vec_size != 0 || key_num != slot_num ||
      slot_file_size_in_byte % slot_size != 0) {
    CK_THROW_(Error_t::WrongInput, "Error: file size is not correct");
  }

  auto blobs_buff = GeneralBuffer2<CudaHostAllocator>::create();

  Tensor2<TypeHashKey> keys;
  blobs_buff->reserve({key_num}, &keys);

  Tensor2<size_t> slot_id;
  blobs_buff->reserve({slot_num}, &slot_id);

  Tensor2<float> embeddings;
  blobs_buff->reserve({vec_num, embedding_data_.embedding_params_.embedding_vec_size}, &embeddings);

  blobs_buff->allocate();

  TypeHashKey *key_ptr = keys.get_ptr();
  size_t *slot_id_ptr = slot_id.get_ptr();
  float *embedding_ptr = embeddings.get_ptr();

  if (std::is_same<TypeHashKey, long long>::value) {
    key_stream.read(reinterpret_cast<char *>(key_ptr), key_file_size_in_byte);
  } else {
    std::vector<long long> i64_key_vec(key_num, 0);
    key_stream.read(reinterpret_cast<char *>(i64_key_vec.data()), key_file_size_in_byte);
    std::transform(i64_key_vec.begin(), i64_key_vec.end(), key_ptr,
                   [](long long key) { return static_cast<unsigned>(key); });
  }
  slot_stream.read(reinterpret_cast<char *>(slot_id_ptr), slot_file_size_in_byte);
  vec_stream.read(reinterpret_cast<char *>(embedding_ptr), vec_file_size_in_byte);

  load_parameters(keys, slot_id, embeddings, key_num,
                  embedding_data_.embedding_params_.embedding_vec_size, hash_table_value_tensors_,
                  slot_size_array_, mapping_offsets_per_gpu_tensors_);
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::load_parameters(
    BufferBag &buf_bag, size_t num) {
  const TensorBag2 &keys_bag = buf_bag.keys;
  const TensorBag2 &slot_id_bag = buf_bag.slot_id;
  const Tensor2<float> &embeddings = buf_bag.embedding;
  Tensor2<TypeHashKey> keys = Tensor2<TypeHashKey>::stretch_from(keys_bag);
  Tensor2<size_t> slot_id = Tensor2<size_t>::stretch_from(slot_id_bag);

  load_parameters(keys, slot_id, embeddings, num,
                  embedding_data_.embedding_params_.embedding_vec_size, hash_table_value_tensors_,
                  slot_size_array_, mapping_offsets_per_gpu_tensors_);
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::load_parameters(
    const Tensor2<TypeHashKey> &keys, const Tensor2<size_t> &slot_id,
    const Tensor2<float> &embeddings, size_t num, size_t embedding_vec_size,
    Tensors2<float> &hash_table_value_tensors, const std::vector<size_t> &slot_sizes,
    const Tensors2<uint32_t> &mapping_offsets_per_gpu_tensors) {
  if (num == 0) return;

  CudaDeviceContext context;
  if (keys.get_dimensions()[0] < num || embeddings.get_dimensions()[0] < num) {
    CK_THROW_(Error_t::WrongInput, "The rows of keys and embeddings are not consistent.");
  }

  const TypeHashKey *key_ptr = keys.get_ptr();
  const size_t *slot_id_ptr = slot_id.get_ptr();
  const float *embedding_ptr = embeddings.get_ptr();

  // define size
  size_t local_gpu_count = embedding_data_.get_resource_manager().get_local_gpu_count();
  size_t chunk_size = 1000;
  size_t tile_size = 1;  // must be 1, because we need to cal (key&local_gpu_count) to decide
                         // gpu_id for each <key,value>
  size_t hash_table_value_tile_size = tile_size * embedding_vec_size;
  size_t hash_table_value_tile_size_in_B = hash_table_value_tile_size * sizeof(float);
  size_t hash_table_value_chunk_size = hash_table_value_tile_size * chunk_size;
  size_t hash_table_value_chunk_size_in_B = hash_table_value_chunk_size * sizeof(float);
  size_t total_gpu_count = embedding_data_.get_resource_manager().get_global_gpu_count();

  // CAUSION: can not decide how many values for each GPU, so need to allocate enough memory for
  // each GPU allocate CPU/GPU memory for value/index chunk
  std::unique_ptr<float *[]> h_hash_table_value_chunk_per_gpu(new float *[local_gpu_count]);
  for (size_t id = 0; id < local_gpu_count; id++) {
    CK_CUDA_THROW_(
        hipHostMalloc(&h_hash_table_value_chunk_per_gpu[id], hash_table_value_chunk_size_in_B));
  }
  std::unique_ptr<float *[]> d_hash_table_value_chunk_per_gpu(new float *[local_gpu_count]);
  for (size_t id = 0; id < local_gpu_count; id++) {
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());
    CK_CUDA_THROW_(
        hipMalloc(&d_hash_table_value_chunk_per_gpu[id], hash_table_value_chunk_size_in_B));
  }
  std::unique_ptr<size_t *[]> h_hash_table_index_chunk_per_gpu(new size_t *[local_gpu_count]);
  for (size_t id = 0; id < local_gpu_count; id++) {
    CK_CUDA_THROW_(
        hipHostMalloc(&h_hash_table_index_chunk_per_gpu[id], chunk_size * sizeof(size_t)));
  }
  std::unique_ptr<size_t *[]> d_hash_table_index_chunk_per_gpu(new size_t *[local_gpu_count]);
  for (size_t id = 0; id < local_gpu_count; id++) {
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());
    CK_CUDA_THROW_(hipMalloc(&d_hash_table_index_chunk_per_gpu[id], chunk_size * sizeof(size_t)));
  }

  std::unique_ptr<size_t[]> tile_counter_in_chunk_per_gpu(new size_t[local_gpu_count]);
  memset(tile_counter_in_chunk_per_gpu.get(), 0, sizeof(size_t) * local_gpu_count);

  // The vector that store the relationship between slot_id and slot order on the specific GPU
  std::vector<size_t> local_slot_id(slot_sizes.size());
  std::vector<size_t> local_slot_num(local_gpu_count, 0);
  for (size_t i = 0; i < slot_sizes.size(); i++) {
    size_t gid = i % total_gpu_count;  // global GPU ID
    size_t id = embedding_data_.get_resource_manager().get_gpu_local_id_from_global_id(
        gid);  // local GPU ID (not gpudevice id)
    int dst_rank =
        embedding_data_.get_resource_manager().get_process_id_from_gpu_global_id(gid);  // node id
    if (embedding_data_.get_resource_manager().get_process_id() == dst_rank) {
      local_slot_id[i] = local_slot_num[id];
      local_slot_num[id]++;
    }
  }

  // Host buffer to keep mapping_offset
  std::vector<uint32_t *> h_mapping_offsets_per_gpu_tensors(local_gpu_count);
  for (size_t id = 0; id < local_gpu_count; id++) {
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());
    CK_CUDA_THROW_(hipHostMalloc(&h_mapping_offsets_per_gpu_tensors[id],
                                  local_slot_num[id] * sizeof(uint32_t)));
    // Copy the mapping offset from GPU to Host
    CK_CUDA_THROW_(hipMemcpyAsync(h_mapping_offsets_per_gpu_tensors[id],
                                   mapping_offsets_per_gpu_tensors[id].get_ptr(),
                                   local_slot_num[id] * sizeof(uint32_t), hipMemcpyDeviceToHost,
                                   embedding_data_.get_local_gpu(id).get_stream()));
  }

  // sync wait
  functors_.sync_all_gpus(embedding_data_.get_resource_manager());

  // do upload
  size_t loop_num = num / chunk_size;
  MESSAGE_("Start to upload embedding table file to GPUs, total loop_num: " +
           std::to_string(loop_num));
  for (size_t i = 0; i < loop_num; i++) {
    float *value_dst_buf;
    size_t *tensor_index_dst_buf;
    for (size_t k = 0; k < chunk_size; k++) {  // process a tile in each loop
      size_t slot_id = slot_id_ptr[i * chunk_size + k];
      size_t gid = slot_id % total_gpu_count;  // global GPU ID
      size_t id = embedding_data_.get_resource_manager().get_gpu_local_id_from_global_id(
          gid);  // local GPU ID (not gpudevice id)
      int dst_rank =
          embedding_data_.get_resource_manager().get_process_id_from_gpu_global_id(gid);  // node id

      if (embedding_data_.get_resource_manager().get_process_id() == dst_rank) {
        TypeHashKey tile_key = key_ptr[i * chunk_size + k];
        size_t tensor_index =
            tile_key - (h_mapping_offsets_per_gpu_tensors[id][local_slot_id[slot_id]]);

        // memcpy hash_table_value to corresponding GPU
        value_dst_buf = h_hash_table_value_chunk_per_gpu[id] +
                        tile_counter_in_chunk_per_gpu[id] * hash_table_value_tile_size;
        memcpy(value_dst_buf, embedding_ptr + (i * chunk_size + k) * embedding_vec_size,
               hash_table_value_tile_size_in_B);

        tensor_index_dst_buf =
            h_hash_table_index_chunk_per_gpu[id] + tile_counter_in_chunk_per_gpu[id];
        *tensor_index_dst_buf = tensor_index;
        tile_counter_in_chunk_per_gpu[id] += 1;
      } else {
        continue;
      }
    }  // end of for(int k = 0; k < (chunk_size * local_gpu_count); k++)

    // memcpy hash_table_slot_id and hash_table_value from CPU to GPU
    for (size_t id = 0; id < local_gpu_count; id++) {
      if (tile_counter_in_chunk_per_gpu[id] == 0) {
        continue;
      }

      context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

      // Copy value buffer and tensor_index buffer to GPU
      size_t value_chunk_size = tile_counter_in_chunk_per_gpu[id] * hash_table_value_tile_size;
      float *src_buf_value = h_hash_table_value_chunk_per_gpu[id];
      float *dst_buf_value = d_hash_table_value_chunk_per_gpu[id];
      CK_CUDA_THROW_(hipMemcpyAsync(dst_buf_value, src_buf_value, value_chunk_size * sizeof(float),
                                     hipMemcpyHostToDevice,
                                     embedding_data_.get_local_gpu(id).get_stream()));
      size_t *src_buf_index = h_hash_table_index_chunk_per_gpu[id];
      size_t *dst_buf_index = d_hash_table_index_chunk_per_gpu[id];
      value_chunk_size = tile_counter_in_chunk_per_gpu[id];
      CK_CUDA_THROW_(hipMemcpyAsync(dst_buf_index, src_buf_index,
                                     value_chunk_size * sizeof(size_t), hipMemcpyHostToDevice,
                                     embedding_data_.get_local_gpu(id).get_stream()));

      // Call kernel to insert the value into embedding value tensor
      const size_t grid_size = (tile_counter_in_chunk_per_gpu[id] - 1) / 256 + 1;
      upload_value_tensor_kernel<<<grid_size, 256, 0,
                                   embedding_data_.get_local_gpu(id).get_stream()>>>(
          d_hash_table_value_chunk_per_gpu[id], d_hash_table_index_chunk_per_gpu[id],
          hash_table_value_tensors[id].get_ptr(), hash_table_value_tile_size,
          tile_counter_in_chunk_per_gpu[id]);
    }

    functors_.sync_all_gpus(embedding_data_.get_resource_manager());

    // set counter value
    for (size_t id = 0; id < local_gpu_count; id++) {
      tile_counter_in_chunk_per_gpu[id] = 0;  // reset chunk counter to zero
    }
  }  // end of for(int i = 0; i < loop_num; i++)

  // std::cout << std::endl;

  // process the remaining data(less than a chunk)
  const size_t remain_loop_num = num - loop_num * chunk_size;
  float *value_dst_buf;
  size_t *tensor_index_dst_buf;
  for (size_t i = 0; i < remain_loop_num; i++) {  // process one tile in each loop

    size_t slot_id = slot_id_ptr[loop_num * chunk_size + i];
    size_t gid = slot_id % total_gpu_count;  // global GPU ID
    size_t id = embedding_data_.get_resource_manager().get_gpu_local_id_from_global_id(
        gid);  // local GPU ID (not gpudevice id)
    int dst_rank =
        embedding_data_.get_resource_manager().get_process_id_from_gpu_global_id(gid);  // node id

    if (embedding_data_.get_resource_manager().get_process_id() == dst_rank) {
      TypeHashKey tile_key = key_ptr[loop_num * chunk_size + i];
      size_t tensor_index =
          tile_key - (h_mapping_offsets_per_gpu_tensors[id][local_slot_id[slot_id]]);

      // memcpy hash_table_value to corresponding GPU
      value_dst_buf = h_hash_table_value_chunk_per_gpu[id] +
                      tile_counter_in_chunk_per_gpu[id] * hash_table_value_tile_size;
      memcpy(value_dst_buf, embedding_ptr + (loop_num * chunk_size + i) * embedding_vec_size,
             hash_table_value_tile_size_in_B);

      tensor_index_dst_buf =
          h_hash_table_index_chunk_per_gpu[id] + tile_counter_in_chunk_per_gpu[id];
      *tensor_index_dst_buf = tensor_index;
      tile_counter_in_chunk_per_gpu[id] += 1;

    } else {
      continue;
    }
  }

  // memcpy hash_table_slot_id and hash_table_value from CPU to GPU and insert into embedding
  // table
  for (size_t id = 0; id < local_gpu_count; id++) {
    if (tile_counter_in_chunk_per_gpu[id] == 0) {
      continue;
    }

    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

    // Copy value buffer and tensor_index buffer to GPU
    size_t value_chunk_size = tile_counter_in_chunk_per_gpu[id] * hash_table_value_tile_size;
    float *src_buf_value = h_hash_table_value_chunk_per_gpu[id];
    float *dst_buf_value = d_hash_table_value_chunk_per_gpu[id];
    CK_CUDA_THROW_(hipMemcpyAsync(dst_buf_value, src_buf_value, value_chunk_size * sizeof(float),
                                   hipMemcpyHostToDevice,
                                   embedding_data_.get_local_gpu(id).get_stream()));
    size_t *src_buf_index = h_hash_table_index_chunk_per_gpu[id];
    size_t *dst_buf_index = d_hash_table_index_chunk_per_gpu[id];
    value_chunk_size = tile_counter_in_chunk_per_gpu[id];
    CK_CUDA_THROW_(hipMemcpyAsync(dst_buf_index, src_buf_index, value_chunk_size * sizeof(size_t),
                                   hipMemcpyHostToDevice,
                                   embedding_data_.get_local_gpu(id).get_stream()));

    // Call kernel to insert the value into embedding value tensor
    const size_t grid_size = (tile_counter_in_chunk_per_gpu[id] - 1) / 256 + 1;
    upload_value_tensor_kernel<<<grid_size, 256, 0,
                                 embedding_data_.get_local_gpu(id).get_stream()>>>(
        d_hash_table_value_chunk_per_gpu[id], d_hash_table_index_chunk_per_gpu[id],
        hash_table_value_tensors[id].get_ptr(), hash_table_value_tile_size,
        tile_counter_in_chunk_per_gpu[id]);
  }

  // sync wait
  functors_.sync_all_gpus(embedding_data_.get_resource_manager());

  MESSAGE_("Done");

  // release resources
  for (size_t id = 0; id < local_gpu_count; id++) {
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());
    CK_CUDA_THROW_(hipFree(d_hash_table_value_chunk_per_gpu[id]));
    CK_CUDA_THROW_(hipFree(d_hash_table_index_chunk_per_gpu[id]));
  }
  for (size_t id = 0; id < local_gpu_count; id++) {
    CK_CUDA_THROW_(hipHostFree(h_hash_table_value_chunk_per_gpu[id]));
    CK_CUDA_THROW_(hipHostFree(h_hash_table_index_chunk_per_gpu[id]));
    CK_CUDA_THROW_(hipHostFree(h_mapping_offsets_per_gpu_tensors[id]));
  }
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::dump_parameters(
    std::string sparse_model, DataSourceParams data_source_params) const {
  dump_parameters(sparse_model, data_source_params,
                  embedding_data_.embedding_params_.embedding_vec_size, hash_table_value_tensors_,
                  slot_size_array_);
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::dump_parameters(
    BufferBag &buf_bag, size_t *num) const {
  TensorBag2 keys_bag = buf_bag.keys;
  TensorBag2 slot_id_bag = buf_bag.slot_id;
  Tensor2<float> &embeddings = buf_bag.embedding;
  Tensor2<TypeHashKey> keys = Tensor2<TypeHashKey>::stretch_from(keys_bag);
  Tensor2<size_t> slot_id = Tensor2<size_t>::stretch_from(slot_id_bag);

  dump_parameters(keys, slot_id, embeddings, num,
                  embedding_data_.embedding_params_.embedding_vec_size, hash_table_value_tensors_,
                  slot_size_array_);
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::dump_parameters(
    const std::string &sparse_model, DataSourceParams data_source_params, size_t embedding_vec_size,
    const Tensors2<float> &hash_table_value_tensors, const std::vector<size_t> &slot_sizes) const {
  CudaDeviceContext context;
  size_t local_gpu_count = embedding_data_.get_resource_manager().get_local_gpu_count();

  if (!data_source_params.use_hdfs && !std::filesystem::exists(sparse_model)) {
    std::filesystem::create_directories(sparse_model);
  }
  const std::string key_file(sparse_model + "/key");
  const std::string slot_file(sparse_model + "/slot_id");
  const std::string vec_file(sparse_model + "/emb_vector");

#ifdef ENABLE_MPI
  MPI_File key_fh, slot_fh, vec_fh;
  CK_MPI_THROW_(MPI_File_open(MPI_COMM_WORLD, key_file.c_str(), MPI_MODE_CREATE | MPI_MODE_WRONLY,
                              MPI_INFO_NULL, &key_fh));
  CK_MPI_THROW_(MPI_File_open(MPI_COMM_WORLD, slot_file.c_str(), MPI_MODE_CREATE | MPI_MODE_WRONLY,
                              MPI_INFO_NULL, &slot_fh));
  CK_MPI_THROW_(MPI_File_open(MPI_COMM_WORLD, vec_file.c_str(), MPI_MODE_CREATE | MPI_MODE_WRONLY,
                              MPI_INFO_NULL, &vec_fh));
#endif

  // memory allocation
  std::unique_ptr<size_t[]> count(new size_t[local_gpu_count]);
  size_t total_count = 0;

  for (size_t id = 0; id < local_gpu_count; id++) {
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());
    count[id] = 0;
    for (size_t i = 0; i < slot_sizes.size(); i++) {
      size_t global_id = embedding_data_.get_local_gpu(id).get_global_id();
      if ((i % embedding_data_.get_resource_manager().get_global_gpu_count()) == global_id) {
        count[id] += slot_sizes[i];
      }
    }
    total_count += count[id];
  }

  std::vector<size_t> offset_host(local_gpu_count, 0);
  std::exclusive_scan(count.get(), count.get() + local_gpu_count, offset_host.begin(), 0);

  TypeHashKey *h_hash_table_key;
  size_t *h_hash_table_slot_id;
  float *h_hash_table_value;
  CK_CUDA_THROW_(hipHostMalloc(&h_hash_table_key, total_count * sizeof(TypeHashKey)));
  CK_CUDA_THROW_(hipHostMalloc(&h_hash_table_slot_id, total_count * sizeof(size_t)));
  CK_CUDA_THROW_(
      hipHostMalloc(&h_hash_table_value, total_count * embedding_vec_size * sizeof(float)));

  std::unique_ptr<TypeHashKey *[]> d_hash_table_key(new TypeHashKey *[local_gpu_count]);
  std::unique_ptr<size_t *[]> d_hash_table_slot_id(new size_t *[local_gpu_count]);

  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) continue;
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

    CK_CUDA_THROW_(hipMalloc(&d_hash_table_key[id], count[id] * sizeof(TypeHashKey)));
    CK_CUDA_THROW_(hipMalloc(&d_hash_table_slot_id[id], count[id] * sizeof(size_t)));
  }

  // Generate key and slot_id tensor, dump value tensor on GPU
  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) continue;
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

    MESSAGE_("Rank" + std::to_string(embedding_data_.get_resource_manager().get_process_id()) +
                 ": Dump embedding table from GPU" + std::to_string(id),
             true);

    // Loop for each slot
    size_t buffer_offset = 0;
    for (size_t i = 0; i < slot_sizes.size(); i++) {
      size_t global_id = embedding_data_.get_local_gpu(id).get_global_id();
      if ((i % embedding_data_.get_resource_manager().get_global_gpu_count()) == global_id) {
        // Generate key buffer
        size_t key_offset = 0;
        for (size_t j = 0; j < i; j++) {
          key_offset += slot_sizes[j];
        }
        functors_.memset_liner(d_hash_table_key[id] + buffer_offset, (TypeHashKey)key_offset,
                               (TypeHashKey)1, slot_sizes[i],
                               embedding_data_.get_local_gpu(id).get_stream());

        // Generate slot_id
        functors_.memset_const(d_hash_table_slot_id[id] + buffer_offset, i, slot_sizes[i],
                               embedding_data_.get_local_gpu(id).get_stream());

        buffer_offset += slot_sizes[i];
      }
    }
    // Copy key buffer to host
    CK_CUDA_THROW_(hipMemcpyAsync(h_hash_table_key + offset_host[id], d_hash_table_key[id],
                                   count[id] * sizeof(TypeHashKey), hipMemcpyDeviceToHost,
                                   embedding_data_.get_local_gpu(id).get_stream()));
    // Copy value buffer to host
    CK_CUDA_THROW_(hipMemcpyAsync(
        h_hash_table_value + offset_host[id] * embedding_vec_size,
        hash_table_value_tensors[id].get_ptr(), count[id] * embedding_vec_size * sizeof(float),
        hipMemcpyDeviceToHost, embedding_data_.get_local_gpu(id).get_stream()));
    // Copy slot_id to host
    CK_CUDA_THROW_(hipMemcpyAsync(h_hash_table_slot_id + offset_host[id], d_hash_table_slot_id[id],
                                   count[id] * sizeof(size_t), hipMemcpyDeviceToHost,
                                   embedding_data_.get_local_gpu(id).get_stream()));
  }
  functors_.sync_all_gpus(embedding_data_.get_resource_manager());

  long long *h_key_ptr;
  std::vector<long long> i64_key_vec;
  if (std::is_same<TypeHashKey, long long>::value) {
    h_key_ptr = reinterpret_cast<long long *>(h_hash_table_key);
  } else {
    i64_key_vec.resize(total_count);
    std::transform(h_hash_table_key, h_hash_table_key + total_count, i64_key_vec.begin(),
                   [](unsigned key) { return static_cast<long long>(key); });
    h_key_ptr = i64_key_vec.data();
  }

  const size_t key_size = sizeof(long long);
  const size_t slot_size = sizeof(size_t);
  const size_t vec_size = sizeof(float) * embedding_vec_size;

  // write sparse model to file
  MESSAGE_("Rank" + std::to_string(embedding_data_.get_resource_manager().get_process_id()) +
               ": Write hash table <key,value> pairs to file",
           true);
#ifdef ENABLE_MPI
  MPI_Datatype TYPE_EMB_VECTOR;
  CK_MPI_THROW_(MPI_Type_contiguous(embedding_vec_size, MPI_FLOAT, &TYPE_EMB_VECTOR));
  CK_MPI_THROW_(MPI_Type_commit(&TYPE_EMB_VECTOR));

  int my_rank = embedding_data_.get_resource_manager().get_process_id();
  int n_ranks = embedding_data_.get_resource_manager().get_num_process();

  std::vector<size_t> offset_per_rank(n_ranks, 0);
  CK_MPI_THROW_(MPI_Allgather(&total_count, sizeof(size_t), MPI_CHAR, offset_per_rank.data(),
                              sizeof(size_t), MPI_CHAR, MPI_COMM_WORLD));
  std::exclusive_scan(offset_per_rank.begin(), offset_per_rank.end(), offset_per_rank.begin(), 0);

  size_t key_offset = offset_per_rank[my_rank] * key_size;
  size_t slot_offset = offset_per_rank[my_rank] * slot_size;
  size_t vec_offset = offset_per_rank[my_rank] * vec_size;

  CK_MPI_THROW_(MPI_Barrier(MPI_COMM_WORLD));
  MPI_Status status;
  CK_MPI_THROW_(
      MPI_File_write_at(key_fh, key_offset, h_key_ptr, total_count, MPI_LONG_LONG_INT, &status));
  CK_MPI_THROW_(MPI_File_write_at(slot_fh, slot_offset, h_hash_table_slot_id, total_count,
                                  MPI_SIZE_T, &status));
  CK_MPI_THROW_(MPI_File_write_at(vec_fh, vec_offset, h_hash_table_value, total_count,
                                  TYPE_EMB_VECTOR, &status));

  CK_MPI_THROW_(MPI_File_close(&key_fh));
  CK_MPI_THROW_(MPI_File_close(&slot_fh));
  CK_MPI_THROW_(MPI_File_close(&vec_fh));
  CK_MPI_THROW_(MPI_Type_free(&TYPE_EMB_VECTOR));
#else
  if (data_source_params.use_hdfs) {
    HdfsService hs(data_source_params.namenode, data_source_params.port);
    hs.write(key_file, reinterpret_cast<char *>(h_key_ptr), total_count * key_size, true);
    hs.write(slot_file, reinterpret_cast<char *>(h_hash_table_slot_id), total_count * slot_size,
             true);
    hs.write(vec_file, reinterpret_cast<char *>(h_hash_table_value), total_count * vec_size, true);
  } else {
    std::ofstream key_stream(key_file, std::ofstream::binary | std::ofstream::trunc);
    std::ofstream slot_stream(slot_file, std::ofstream::binary | std::ofstream::trunc);
    std::ofstream vec_stream(vec_file, std::ofstream::binary | std::ofstream::trunc);
    // check if the file is opened successfully
    if (!vec_stream.is_open() || !key_stream.is_open() || !slot_stream.is_open()) {
      CK_THROW_(Error_t::WrongInput, "Error: file not open for writing");
      return;
    }
    key_stream.write(reinterpret_cast<char *>(h_key_ptr), total_count * key_size);
    slot_stream.write(reinterpret_cast<char *>(h_hash_table_slot_id), total_count * slot_size);
    vec_stream.write(reinterpret_cast<char *>(h_hash_table_value), total_count * vec_size);
  }
#endif
  MESSAGE_("Done");

  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) continue;
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

    CK_CUDA_THROW_(hipFree(d_hash_table_key[id]));
    CK_CUDA_THROW_(hipFree(d_hash_table_slot_id[id]));
  }
  CK_CUDA_THROW_(hipHostFree(h_hash_table_key));
  CK_CUDA_THROW_(hipHostFree(h_hash_table_slot_id));
  CK_CUDA_THROW_(hipHostFree(h_hash_table_value));
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::dump_parameters(
    Tensor2<TypeHashKey> &keys, Tensor2<size_t> &slot_id, Tensor2<float> &embeddings, size_t *num,
    size_t embedding_vec_size, const Tensors2<float> &hash_table_value_tensors,
    const std::vector<size_t> &slot_sizes) const {
  TypeHashKey *key_ptr = keys.get_ptr();
  size_t *slot_id_ptr = slot_id.get_ptr();
  float *embedding_ptr = embeddings.get_ptr();

  size_t local_gpu_count = embedding_data_.get_resource_manager().get_local_gpu_count();

  // memory allocation
  std::unique_ptr<size_t[]> count(new size_t[local_gpu_count]);
  size_t total_count = 0;

  CudaDeviceContext context;
  for (size_t id = 0; id < local_gpu_count; id++) {
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());
    count[id] = 0;
    for (size_t i = 0; i < slot_sizes.size(); i++) {
      size_t global_id = embedding_data_.get_local_gpu(id).get_global_id();
      if ((i % embedding_data_.get_resource_manager().get_global_gpu_count()) == global_id) {
        count[id] += slot_sizes[i];
      }
    }
    total_count += count[id];
  }

  std::vector<size_t> offset_host(local_gpu_count, 0);
  std::exclusive_scan(count.get(), count.get() + local_gpu_count, offset_host.begin(), 0);
  *num = total_count;

  std::unique_ptr<TypeHashKey *[]> d_hash_table_key(new TypeHashKey *[local_gpu_count]);
  std::unique_ptr<size_t *[]> d_hash_table_slot_id(new size_t *[local_gpu_count]);

  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) continue;
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

    CK_CUDA_THROW_(hipMalloc(&d_hash_table_key[id], count[id] * sizeof(TypeHashKey)));
    CK_CUDA_THROW_(hipMalloc(&d_hash_table_slot_id[id], count[id] * sizeof(size_t)));
  }

  // Generate key and slot_id tensor, dump value tensor on GPU
  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) continue;
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

    // Loop for each slot
    size_t buffer_offset = 0;
    for (size_t i = 0; i < slot_sizes.size(); i++) {
      size_t global_id = embedding_data_.get_local_gpu(id).get_global_id();
      if ((i % embedding_data_.get_resource_manager().get_global_gpu_count()) == global_id) {
        // Generate key buffer
        size_t key_offset = 0;
        for (size_t j = 0; j < i; j++) {
          key_offset += slot_sizes[j];
        }
        functors_.memset_liner(d_hash_table_key[id] + buffer_offset,
                               static_cast<TypeHashKey>(key_offset), static_cast<TypeHashKey>(1),
                               slot_sizes[i], embedding_data_.get_local_gpu(id).get_stream());

        // Generate slot_id
        functors_.memset_const(d_hash_table_slot_id[id] + buffer_offset, i, slot_sizes[i],
                               embedding_data_.get_local_gpu(id).get_stream());

        buffer_offset += slot_sizes[i];
      }
    }
    // Copy key buffer to host
    CK_CUDA_THROW_(hipMemcpyAsync(key_ptr + offset_host[id], d_hash_table_key[id],
                                   count[id] * sizeof(TypeHashKey), hipMemcpyDeviceToHost,
                                   embedding_data_.get_local_gpu(id).get_stream()));
    // Copy value buffer to host
    CK_CUDA_THROW_(hipMemcpyAsync(
        embedding_ptr + offset_host[id] * embedding_vec_size,
        hash_table_value_tensors[id].get_ptr(), count[id] * embedding_vec_size * sizeof(float),
        hipMemcpyDeviceToHost, embedding_data_.get_local_gpu(id).get_stream()));
    // Copy slot_id to host
    CK_CUDA_THROW_(hipMemcpyAsync(slot_id_ptr + offset_host[id], d_hash_table_slot_id[id],
                                   count[id] * sizeof(size_t), hipMemcpyDeviceToHost,
                                   embedding_data_.get_local_gpu(id).get_stream()));
  }
  functors_.sync_all_gpus(embedding_data_.get_resource_manager());

  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) continue;
    context.set_device(embedding_data_.get_local_gpu(id).get_device_id());

    CK_CUDA_THROW_(hipFree(d_hash_table_key[id]));
    CK_CUDA_THROW_(hipFree(d_hash_table_slot_id[id]));
  }
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::init_embedding(
    const std::vector<size_t> slot_sizes, size_t embedding_vec_size,
    std::vector<Tensors2<float>> &hash_table_value_tensors,
    Tensors2<size_t> &hash_table_slot_id_tensors) {
  size_t local_gpu_count = embedding_data_.get_resource_manager().get_local_gpu_count();
  size_t total_gpu_count = embedding_data_.get_resource_manager().get_global_gpu_count();

#ifndef NDEBUG
  MESSAGE_("local_gpu_count=" + std::to_string(local_gpu_count) +
           ", total_gpu_count=" + std::to_string(total_gpu_count));
#endif

#pragma omp parallel num_threads(embedding_data_.get_resource_manager().get_local_gpu_count())
  {
    size_t id = omp_get_thread_num();
    size_t device_id = embedding_data_.get_local_gpu(id).get_device_id();
    size_t global_id = embedding_data_.get_local_gpu(id).get_global_id();

#ifndef NDEBUG
    MESSAGE_("id=" + std::to_string(id) + ", device_id=" + std::to_string(device_id) +
             ", global_id=" + std::to_string(global_id));
#endif

    functors_.init_embedding_per_gpu(global_id, total_gpu_count, slot_sizes, embedding_vec_size,
                                     hash_table_value_tensors[id], hash_table_slot_id_tensors[id],
                                     embedding_data_.get_local_gpu(id));

    CK_CUDA_THROW_(hipStreamSynchronize(embedding_data_.get_local_gpu(id).get_stream()));
    MESSAGE_("gpu" + std::to_string(id) + " init embedding done");
  }

  return;
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::reset() {
  CudaDeviceContext context;
  for (size_t i = 0; i < embedding_data_.get_resource_manager().get_local_gpu_count(); i++) {
    functors_.init_embedding_per_gpu(
        embedding_data_.get_local_gpu(i).get_global_id(),
        embedding_data_.get_resource_manager().get_global_gpu_count(), slot_size_array_,
        embedding_data_.embedding_params_.embedding_vec_size, value_table_tensors_[i],
        hash_table_slot_id_tensors_[i], embedding_data_.get_local_gpu(i));
  }

  for (size_t i = 0; i < embedding_data_.get_resource_manager().get_local_gpu_count(); i++) {
    CK_CUDA_THROW_(hipStreamSynchronize(embedding_data_.get_local_gpu(i).get_stream()));
  }
}

template class LocalizedSlotSparseEmbeddingOneHot<unsigned int, float>;
template class LocalizedSlotSparseEmbeddingOneHot<long long, float>;
template class LocalizedSlotSparseEmbeddingOneHot<unsigned int, __half>;
template class LocalizedSlotSparseEmbeddingOneHot<long long, __half>;

}  // namespace HugeCTR
