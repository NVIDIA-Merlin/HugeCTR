#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/embeddings/sparse_embedding_functors.hpp"
#include "HugeCTR/include/utils.hpp"

namespace HugeCTR {

template <typename TypeEmbeddingComp>
void SparseEmbeddingFunctors::get_forward_results(
    size_t memcpy_size, const Tensors2<TypeEmbeddingComp> &embedding_feature_tensors,
    Tensor2<TypeEmbeddingComp> &embedding_feature, Tensors2<TypeEmbeddingComp> &temp_tensors,
    const ResourceManager &resource_manager) {
  size_t total_gpu_count = resource_manager.get_global_gpu_count();
  const auto &local_gpu = resource_manager.get_local_gpu(0);

  CudaDeviceContext context;
  if (total_gpu_count > 1) {
    // nccl allGather
    all_gather(memcpy_size,
               embedding_feature_tensors,  // send
               temp_tensors,               // recv
               resource_manager);

    // memcpy D2H
    context.set_device(local_gpu->get_device_id());
    CK_CUDA_THROW_(hipMemcpyAsync(embedding_feature.get_ptr(), temp_tensors[0].get_ptr(),
                                   total_gpu_count * memcpy_size * sizeof(TypeEmbeddingComp),
                                   hipMemcpyDeviceToHost, local_gpu->get_stream()));
    CK_CUDA_THROW_(hipStreamSynchronize(local_gpu->get_stream()));
  } else {
    context.set_device(local_gpu->get_device_id());
    CK_CUDA_THROW_(hipMemcpyAsync(
        embedding_feature.get_ptr(), embedding_feature_tensors[0].get_ptr(),
        memcpy_size * sizeof(TypeEmbeddingComp), hipMemcpyDeviceToHost, local_gpu->get_stream()));
    CK_CUDA_THROW_(hipStreamSynchronize(local_gpu->get_stream()));
  }

  return;
}

template <typename TypeEmbeddingComp>
void SparseEmbeddingFunctors::get_forward_results(size_t memcpy_size,
                         const Tensors2<TypeEmbeddingComp> &embedding_feature_tensors,
                         void* const embedding_feature,
                         Tensors2<TypeEmbeddingComp> &temp_tensors,
                         const ResourceManager &resource_manager,
                         const bool on_gpu) {
  size_t total_gpu_count = resource_manager.get_global_gpu_count();
  const auto &local_gpu = resource_manager.get_local_gpu(0);

  hipMemcpyKind direction = (on_gpu ? hipMemcpyDeviceToDevice : hipMemcpyDeviceToHost);

  CudaDeviceContext context;
  if (total_gpu_count > 1) {
    // nccl allGather
    all_gather(memcpy_size,
               embedding_feature_tensors,  // send
               temp_tensors,               // recv
               resource_manager);

    // memcpy D2H
    context.set_device(local_gpu->get_device_id());
    CK_CUDA_THROW_(hipMemcpyAsync(embedding_feature, temp_tensors[0].get_ptr(),
                                  total_gpu_count * memcpy_size * sizeof(TypeEmbeddingComp),
                                  direction, local_gpu->get_stream())); 
    CK_CUDA_THROW_(hipStreamSynchronize(local_gpu->get_stream()));
  } else {
    context.set_device(local_gpu->get_device_id());
    CK_CUDA_THROW_(hipMemcpyAsync(
        embedding_feature, embedding_feature_tensors[0].get_ptr(),
        memcpy_size * sizeof(TypeEmbeddingComp), direction, local_gpu->get_stream())); 
    CK_CUDA_THROW_(hipStreamSynchronize(local_gpu->get_stream()));
  }

  return;
}


template void SparseEmbeddingFunctors::get_forward_results<float>(
    size_t memcpy_size, const Tensors2<float> &embedding_feature_tensors,
    Tensor2<float> &embedding_feature, Tensors2<float> &temp_tensors,
    const ResourceManager &resource_manager);

template void SparseEmbeddingFunctors::get_forward_results<__half>(
    size_t memcpy_size, const Tensors2<__half> &embedding_feature_tensors,
    Tensor2<__half> &embedding_feature, Tensors2<__half> &temp_tensors,
    const ResourceManager &resource_manager);

template void SparseEmbeddingFunctors::get_forward_results<float>(
    size_t memcpy_size,
    const Tensors2<float> &embedding_feature_tensors,
    void* const embedding_feature,
    Tensors2<float> &temp_tensors,
    const ResourceManager &resource_manager,
    const bool on_gpu);

template void SparseEmbeddingFunctors::get_forward_results<__half>(
    size_t memcpy_size,
    const Tensors2<__half> &embedding_feature_tensors,
    void* const embedding_feature,
    Tensors2<__half> &temp_tensors,
    const ResourceManager &resource_manager,
    const bool on_gpu);

}  // namespace HugeCTR