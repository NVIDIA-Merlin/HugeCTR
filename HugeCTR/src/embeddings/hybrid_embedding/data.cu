#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>

#include <algorithm>
#include <iostream>
#include <vector>

#include "HugeCTR/include/common.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/data.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/utils.hpp"
#include "HugeCTR/include/tensor2.hpp"

namespace HugeCTR {

namespace hybrid_embedding {

template <typename dtype>
size_t EmbeddingTableFunctors<dtype>::get_embedding_table_index(
    const std::vector<size_t>& table_sizes, dtype category) {
  size_t embedding = 0;
  dtype next_offset = (dtype)table_sizes[embedding];
  for (embedding = 0; embedding < table_sizes.size() - 1 && category >= next_offset; ++embedding)
    next_offset += table_sizes[embedding + 1];
  return embedding;
}

template <typename dtype>
void EmbeddingTableFunctors<dtype>::get_embedding_offsets(std::vector<dtype>& embedding_offsets,
                                                          const std::vector<size_t>& table_sizes) {
  const size_t num_tables = table_sizes.size();
  embedding_offsets.resize(num_tables);
  dtype embedding_offset = (dtype)0;
  for (size_t embedding = 0; embedding < num_tables; ++embedding) {
    embedding_offsets[embedding] = embedding_offset;
    embedding_offset += table_sizes[embedding];
  }
}

template <typename dtype>
dtype EmbeddingTableFunctors<dtype>::get_num_categories(const std::vector<size_t>& table_sizes) {
  dtype num_categories = (dtype)0;
  for (size_t i = 0; i < table_sizes.size(); ++i) num_categories += table_sizes[i];
  return num_categories;
}

template <typename dtype>
__global__ void data_to_unique_categories_kernel(dtype* data, dtype* embedding_offsets,
                                                 int num_tables, int num_data, dtype* samples,
                                                 int num_valid_data, dtype pad_val) {
  for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < num_data;
       idx += blockDim.x * gridDim.x) {
    samples[idx] =
        idx >= num_valid_data ? pad_val : data[idx] + embedding_offsets[idx % num_tables];
  }
}

template <typename dtype>
__global__ void data_to_unique_categories_align4_kernel(dtype* data, dtype* embedding_offsets,
                                                        int num_tables, int num_data,
                                                        dtype* samples, int num_valid_data,
                                                        dtype pad_val) {
  auto data4 = reinterpret_cast<uint4*>(data);
  auto samples4 = reinterpret_cast<uint4*>(samples);
  for (int idx4 = threadIdx.x + blockIdx.x * blockDim.x; idx4 < num_data / 4;
       idx4 += blockDim.x * gridDim.x) {
    uint4 load_data = data4[idx4];
    uint4 load_embedding_offsets;

    int idx = idx4 * 4;
    load_data.x += embedding_offsets[(idx) % num_tables];
    load_data.y += embedding_offsets[(idx + 1) % num_tables];
    load_data.z += embedding_offsets[(idx + 2) % num_tables];
    load_data.w += embedding_offsets[(idx + 3) % num_tables];

    load_data.x = idx >= num_valid_data ? pad_val : load_data.x;
    load_data.y = idx + 1 >= num_valid_data ? pad_val : load_data.y;
    load_data.z = idx + 2 >= num_valid_data ? pad_val : load_data.z;
    load_data.w = idx + 3 >= num_valid_data ? pad_val : load_data.w;

    samples4[idx4] = load_data;
  }
}

/// data_to_unique_categories converts the argument 'data' and stores
///        the result in member variable 'samples'.
///        Per network, the columns corresponding to embedding tables
///        are concatenated and categories get an unique index / label.
template <typename dtype>
void Data<dtype>::data_to_unique_categories(Tensor2<dtype> data, hipStream_t stream) {
  /// === TODO: PERFORM ON GPU ===
  /// ============================
  // HCTR_LOG_S(WARNING, WORLD) << "data_to_unique_categories() needs to be placed on the GPU!" <<
  // std::endl;
  // TODO : perform conversion by kernel (before start of iteration ? => see below)
  //        for batch_size = 55*1024
  //        batch_size * 26 * 4 / 1600e9 = 3.67 microseconds,
  //
  // Remark:
  //        Doesn't need to be before start of kernel.
  //        Would be nice to have just before calculating indices, since
  //        those would be in L2 cache already.
  size_t current_batch_size = data.get_dimensions()[0];
  size_t block_size = 256;
  size_t grid_size =
      std::min(static_cast<size_t>(4096),
               (table_sizes.size() * batch_size * num_iterations - 1) / block_size + 1);
  size_t num_samples = table_sizes.size() * batch_size * num_iterations;
  // Not all samples in a batch may be valid. I.e last iteration of evaluation may be incomplete.
  size_t num_valid_samples = table_sizes.size() * current_batch_size * num_iterations;
  assert(num_valid_samples > 0 && "Batch contained 0 valid samples");
  auto null_category = static_cast<dtype>(num_categories);

  if (num_samples % 4 == 0 && sizeof(dtype) == 4) {
    data_to_unique_categories_align4_kernel<<<grid_size, block_size, 0, stream>>>(
        data.get_ptr(), embedding_offsets.get_ptr(), table_sizes.size(), num_samples,
        samples.get_ptr(), num_valid_samples, null_category);
  } else {
    data_to_unique_categories_kernel<<<grid_size, block_size, 0, stream>>>(
        data.get_ptr(), embedding_offsets.get_ptr(), table_sizes.size(), num_samples,
        samples.get_ptr(), num_valid_samples, null_category);
  }
  HCTR_LIB_THROW(hipPeekAtLastError());
}

template class Data<uint32_t>;
template class Data<long long>;

template struct EmbeddingTableFunctors<uint32_t>;
template struct EmbeddingTableFunctors<long long>;
}  // namespace hybrid_embedding

}  // namespace HugeCTR
