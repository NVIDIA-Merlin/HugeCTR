#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>

#include <algorithm>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <utility>
#include <vector>

#include "HugeCTR/include/common.hpp"
#include "HugeCTR/include/data_simulator.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/data.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/infrequent_embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/model.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/update.cuh"
#include "HugeCTR/include/embeddings/hybrid_embedding/utils.cuh"
#include "HugeCTR/include/embeddings/hybrid_embedding/utils.hpp"
#include "HugeCTR/include/shuffle/shuffle.cuh"
#include "HugeCTR/include/tensor2.hpp"
#include "HugeCTR/include/utils.hpp"

namespace HugeCTR {

namespace hybrid_embedding {

namespace infrequent_embedding_kernels {

template <typename dtype, typename emtype>
__global__ void hier_update_model(InfrequentEmbeddingSelectionView<dtype>* indices,
                                  const dtype* __restrict__ category_location,
                                  const emtype* __restrict__ gradients,
                                  float* __restrict__ embedding_vectors,
                                  uint32_t embedding_vec_size, uint32_t num_instances,
                                  uint32_t local_samples_size, uint32_t local_comm_buff_size,
                                  const float* __restrict__ lr_ptr, const float scale) {
  float lr = __ldg(lr_ptr) / scale;
  const uint32_t num_indices = indices->model_indices_offsets[num_instances];

  // Load offset only when the network_id changes
  uint32_t previous_network_id = 0;
  uint32_t offset = 0;

  for (uint32_t i = blockIdx.x; i < num_indices; i += gridDim.x) {
    uint32_t index = indices->model_indices[i];
    dtype category = indices->samples[index];
    dtype location = category_location[2 * category + 1];
    uint32_t network_id = index / local_samples_size;
    if (network_id != previous_network_id) {
      offset = indices->model_indices_offsets[network_id];
      previous_network_id = network_id;
    }
    atomicAdd(
        embedding_vectors + location * embedding_vec_size + threadIdx.x,
        -lr * TypeConvertFunc<float, emtype>::convert(
                  gradients[embedding_vec_size * (network_id * local_comm_buff_size + i - offset) +
                            threadIdx.x]));
  }
}

template <typename dtype, typename emtype>
__global__ void infrequent_update_model_direct(
    const emtype* const* __restrict__ gradients_pointers, float* embedding_vectors,
    InfrequentEmbeddingSelectionView<dtype>* indices, const dtype* __restrict__ category_location,
    uint32_t num_instances, uint32_t model_id, uint32_t embedding_vec_size,
    uint32_t local_samples_size, const float* __restrict__ lr_ptr, const float scale) {
  float lr = __ldg(lr_ptr) / scale;
  // Shift pattern
  const uint32_t offset = indices->model_indices_offsets[model_id + 1];
  const uint32_t num_model_indices = indices->model_indices_offsets[num_instances];

  for (uint32_t i = blockIdx.x; i < num_model_indices; i += gridDim.x) {
    uint32_t vid = (i + offset) % num_model_indices;

    uint32_t index = indices->model_indices[vid];
    uint32_t network_id = index / local_samples_size;
    uint32_t local_index = index % local_samples_size;
    dtype category = indices->samples[index];
    uint32_t location = category_location[2 * category + 1];

    const emtype* gradients = gradients_pointers[network_id];

    atomicAdd(embedding_vectors + location * embedding_vec_size + threadIdx.x,
              -lr * TypeConvertFunc<float, emtype>::convert(
                        gradients[local_index * embedding_vec_size + threadIdx.x]));
  }
}

// template <typename dtype>
// __global__ void calculate_network_indices_mask(const dtype* __restrict__ local_samples,
//                                                const dtype* __restrict__ category_location,
//                                                bool* mask, uint32_t local_samples_size,
//                                                uint32_t num_instances) {
//   for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < local_samples_size;
//        i += gridDim.x * blockDim.x) {
//     dtype category = local_samples[i];
//     uint32_t model_id = static_cast<uint32_t>(category_location[2 * category]);
//     for (uint32_t section_id = 0; section_id < num_instances; section_id++) {
//       mask[local_samples_size * section_id + i] = (model_id == section_id);
//     }
//   }
// }

template <typename LambdaPtr>
static __global__ void offsets_to_sizes(size_t* sizes, LambdaPtr get_offsets_ptr,
                                        size_t element_size, uint32_t num_instances) {
  uint32_t* offsets = get_offsets_ptr();
  for (int t = blockIdx.x * blockDim.x + threadIdx.x; t < num_instances;
       t += gridDim.x * blockDim.x) {
    sizes[t] = (offsets[t + 1] - offsets[t]) * element_size;
  }
}

}  // namespace infrequent_embedding_kernels

template <typename dtype>
InfrequentEmbeddingBase<dtype>::InfrequentEmbeddingBase() {
  HCTR_LIB_THROW(hipMalloc(&indices_view_, sizeof(*indices_view_)));
}

template <typename dtype>
InfrequentEmbeddingBase<dtype>::~InfrequentEmbeddingBase() {
  hipFree(indices_view_);
}

template <typename dtype>
void InfrequentEmbeddingBase<dtype>::set_current_indices(
    InfrequentEmbeddingSelection<dtype>* indices, hipStream_t stream) {
  indices_ = indices;
  data_ = indices->get_data();
  HCTR_LIB_THROW(hipMemcpyAsync(indices_view_, indices->get_device_view(), sizeof(*indices_view_),
                                 hipMemcpyDeviceToDevice, stream));
}

template <typename dtype, typename emtype>
InfrequentEmbedding<dtype, emtype>::InfrequentEmbedding(const Model<dtype>& model,
                                                        const GPUResource& gpu_resource,
                                                        uint32_t embedding_vec_size)
    : model_(model), gpu_resource(gpu_resource), embedding_vec_size_(embedding_vec_size) {
  auto buf = GeneralBuffer2<CudaAllocator>::create();
  buf->reserve({ceildiv<size_t>(model.num_categories, model.num_instances), embedding_vec_size_},
               &infrequent_embedding_vectors_);
  buf->reserve({model_.num_instances}, &model_indices_sizes_);
  buf->reserve({model_.num_instances}, &model_indices_sizes_ptrs_);
  buf->reserve({model_.num_instances}, &network_indices_sizes_);
  buf->reserve({model_.num_instances}, &network_indices_sizes_ptrs_);

  buf->reserve({model.num_instances, 1}, &interaction_layer_input_pointers_train_);
  buf->reserve({model.num_instances, 1}, &interaction_layer_input_pointers_eval_);
  buf->reserve({model.num_instances, 1}, &gradients_pointers_);
  buf->allocate();

  auto managed_buf = GeneralBuffer2<CudaManagedAllocator>::create();
  managed_buf->reserve({model.num_instances + 1, 1}, &model_indices_offsets_);
  managed_buf->reserve({model.num_instances + 1, 1}, &network_indices_offsets_);
  managed_buf->allocate();
  int current_device;
  HCTR_LIB_THROW(hipGetDevice(&current_device));
  HCTR_LIB_THROW(hipMemAdvise(managed_buf->get_ptr(), managed_buf->get_size_in_bytes(),
                               hipMemAdviseSetReadMostly, current_device));
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::initialize_embedding_vectors(
    const std::vector<size_t>& table_sizes) {
  CudaDeviceContext context(gpu_resource.get_device_id());

  const size_t num_tables = table_sizes.size();
  for (size_t i = 0; i < num_tables; i++) {
    float up_bound = sqrt(1.f / table_sizes[i]);

    const size_t offset = embedding_vec_size_ * model_.h_infrequent_model_table_offsets[i];
    const size_t number_of_vectors =
        model_.h_infrequent_model_table_offsets[i + 1] - model_.h_infrequent_model_table_offsets[i];
    UniformGenerator::fill(
        infrequent_embedding_vectors_.get_ptr() + offset, embedding_vec_size_ * number_of_vectors,
        -up_bound, up_bound, gpu_resource.get_sm_count(),
        gpu_resource.get_replica_variant_curand_generator(), gpu_resource.get_stream());
  }
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::forward_model(emtype* message_buffer,
                                                       hipStream_t stream) {
  auto indices = this->indices_view_;
  auto category_location = model_.category_location.get_ptr();
  auto infrequent_embedding_vectors = infrequent_embedding_vectors_.get_ptr();
  auto embedding_vec_size = embedding_vec_size_;
  auto num_instances = model_.num_instances;

  auto copy_desc = CopyDescriptors::make_OneToOne<float, emtype, 1>(
      embedding_vec_size,
      [=] __device__() { return indices->model_indices_offsets[num_instances]; },
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<float, emtype, 1> {
        uint32_t index = indices->model_indices[i];
        dtype category = indices->samples[index];
        dtype location = category_location[2 * category + 1];

        return {infrequent_embedding_vectors + location * embedding_vec_size,
                {message_buffer + i * embedding_vec_size},
                {true}};
      });

  shuffle(copy_desc, stream, data_->samples.get_num_elements() / model_.num_instances / 8);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::fused_intra_forward_model(emtype** message_buffer,
                                                                   hipStream_t stream) {
  auto indices = this->indices_view_;
  auto category_location = model_.category_location.get_ptr();
  auto infrequent_embedding_vectors = infrequent_embedding_vectors_.get_ptr();
  size_t embedding_vec_size = embedding_vec_size_;
  auto local_instance_id = model_.instance_id;
  auto num_instances = model_.num_instances;
  auto per_node_instances = num_instances / model_.h_num_instances_per_node.size();
  uint32_t local_samples_size =
      ceildiv<uint32_t>(data_->batch_size, num_instances) * data_->table_sizes.size();

  uint32_t local_comm_buff_size =
      ceildiv<uint32_t>(max_num_infrequent_per_batch_, model_.num_instances);

  auto copy_desc = CopyDescriptors::make_OneToOne<float, emtype, 1>(
      embedding_vec_size,
      [=] __device__() { return indices->model_indices_offsets[num_instances]; },
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<float, emtype, 1> {
        uint32_t num_selected = indices->model_indices_offsets[num_instances];
        uint32_t vid =
            (i + indices->model_indices_offsets[(local_instance_id + 1) % per_node_instances]) %
            num_selected;
        uint32_t index = indices->model_indices[vid];
        uint32_t network_id = (index / local_samples_size);
        dtype category = indices->samples[index];
        dtype location = category_location[2 * category + 1];
        uint32_t local_network_id = (network_id % per_node_instances);
        emtype* output_ptr =
            &message_buffer[local_network_id][(network_id - local_network_id + local_instance_id) *
                                              local_comm_buff_size * embedding_vec_size];

        return {
            infrequent_embedding_vectors + location * embedding_vec_size,
            {output_ptr + (vid - indices->model_indices_offsets[network_id]) * embedding_vec_size},
            {true}};
      });

  shuffle(copy_desc, stream, data_->samples.get_num_elements() / model_.num_instances / 8);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::forward_network(const emtype* message_buffer,
                                                         emtype* interaction_layer_input,
                                                         hipStream_t stream) {
  auto indices = this->indices_view_;
  auto embedding_vec_size = embedding_vec_size_;
  auto num_instances = model_.num_instances;

  auto copy_desc = CopyDescriptors::make_OneToOne<emtype, emtype, 1>(
      embedding_vec_size,
      [=] __device__() { return indices->network_indices_offsets[num_instances]; },
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<emtype, emtype, 1> {
        uint32_t index = indices->network_indices[i];
        return {message_buffer + i * embedding_vec_size,
                {interaction_layer_input + index * embedding_vec_size},
                {true}};
      });

  shuffle(copy_desc, stream, data_->samples.get_num_elements() / model_.num_instances / 8);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::hier_forward_network(const emtype* message_buffer,
                                                              emtype* interaction_layer_input,
                                                              hipStream_t stream) {
  auto indices = this->indices_view_;
  auto embedding_vec_size = embedding_vec_size_;
  auto num_instances = model_.num_instances;
  uint32_t local_samples_size =
      ceildiv<uint32_t>(data_->batch_size, model_.num_instances) * data_->table_sizes.size();
  uint32_t local_comm_buff_size =
      ceildiv<uint32_t>(max_num_infrequent_per_batch_, model_.num_instances);

  auto copy_desc = CopyDescriptors::make_OneToOne<emtype, emtype, 1>(
      embedding_vec_size,
      [=] __device__() { return indices->network_indices_offsets[num_instances]; },
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<emtype, emtype, 1> {
        uint32_t index = indices->network_indices[i];
        uint32_t model_id = indices->network_indices_src_model_id[i];
        uint32_t offset = indices->network_indices_offsets[model_id];

        return {
            message_buffer + (model_id * local_comm_buff_size + i - offset) * embedding_vec_size,
            {interaction_layer_input + index * embedding_vec_size},
            {true}};
      });

  shuffle(copy_desc, stream, data_->samples.get_num_elements() / model_.num_instances / 8);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

/** Forward network for single GPU (no communications) */
template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::forward_network_direct(bool is_train,
                                                                hipStream_t stream) {
  const uint32_t num_instances = model_.num_instances;
  const uint32_t model_id = model_.global_instance_id;
  uint32_t local_samples_size =
      ceildiv<uint32_t>(data_->batch_size, num_instances) * data_->table_sizes.size();

  auto interaction_layer_input_pointers = is_train
                                              ? interaction_layer_input_pointers_train_.get_ptr()
                                              : interaction_layer_input_pointers_eval_.get_ptr();
  auto indices = this->indices_view_;
  auto category_location = model_.category_location.get_ptr();
  auto model_table = infrequent_embedding_vectors_.get_ptr();
  auto embedding_vec_size = embedding_vec_size_;

  auto copy_desc = CopyDescriptors::make_OneToOne<float, emtype, 1>(
      embedding_vec_size,
      [=] __device__() { return indices->model_indices_offsets[num_instances]; },
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<float, emtype, 1> {
        const uint32_t offset = indices->model_indices_offsets[model_id + 1];
        const uint32_t num_model_indices = indices->model_indices_offsets[num_instances];
        const uint32_t vid = (i + offset) % num_model_indices;
        const uint32_t index = indices->model_indices[vid];

        const dtype category = indices->samples[index];
        const dtype location = category_location[2 * category + 1];

        const uint32_t network_id = index / local_samples_size;
        const uint32_t local_index = index % local_samples_size;

        emtype* interaction_layer_input = interaction_layer_input_pointers[network_id];

        return {model_table + location * embedding_vec_size,
                {interaction_layer_input + local_index * embedding_vec_size},
                {true}};
      });

  PROFILE_RECORD("inf_forward_network_direct.forward_network_direct.start", stream, false);
  shuffle(copy_desc, stream, local_samples_size / 10);
  HCTR_LIB_THROW(hipPeekAtLastError());
  PROFILE_RECORD("inf_forward_network_direct.forward_network_direct.stop", stream, false);
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::update_network(const emtype* gradients,
                                                        emtype* message_buffer,
                                                        hipStream_t stream) {
  auto indices = this->indices_view_;
  auto embedding_vec_size = embedding_vec_size_;
  auto num_instances = model_.num_instances;

  auto copy_desc = CopyDescriptors::make_OneToOne<emtype, emtype, 1>(
      embedding_vec_size,
      [=] __device__() { return indices->network_indices_offsets[num_instances]; },
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<emtype, emtype, 1> {
        uint32_t index = indices->network_indices[i];

        return {gradients + index * embedding_vec_size,
                {message_buffer + i * embedding_vec_size},
                {true}};
      });

  shuffle(copy_desc, stream, data_->samples.get_num_elements() / model_.num_instances / 8);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::fused_intra_update_network(const emtype* gradients,
                                                                    emtype** message_buffer,
                                                                    hipStream_t stream) {
  auto indices = this->indices_view_;
  size_t embedding_vec_size = embedding_vec_size_;
  auto local_instance_id = model_.instance_id;
  auto num_instances = model_.num_instances;
  auto per_node_instances = num_instances / model_.h_num_instances_per_node.size();
  uint32_t local_comm_buff_size =
      ceildiv<uint32_t>(max_num_infrequent_per_train_batch_, model_.num_instances);

  auto copy_desc = CopyDescriptors::make_OneToOne<emtype, emtype, 1>(
      embedding_vec_size,
      [=] __device__() { return indices->network_indices_offsets[num_instances]; },
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<emtype, emtype, 1> {
        uint32_t num_selected = indices->network_indices_offsets[num_instances];
        uint32_t vid =
            (i + indices->network_indices_offsets[(local_instance_id + 1) % per_node_instances]) %
            num_selected;
        uint32_t index = indices->network_indices[vid];

        uint32_t model_id = indices->network_indices_src_model_id[vid];

        uint32_t local_model_id = (model_id % per_node_instances);
        emtype* output_ptr =
            &message_buffer[local_model_id][(model_id - local_model_id + local_instance_id) *
                                            local_comm_buff_size * embedding_vec_size];

        return {
            gradients + index * embedding_vec_size,
            {output_ptr + (vid - indices->network_indices_offsets[model_id]) * embedding_vec_size},
            {true}};
      });

  shuffle(copy_desc, stream, data_->samples.get_num_elements() / model_.num_instances / 8);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::update_model(const emtype* message_buffer, float* dev_lr,
                                                      float scale, hipStream_t stream) {
  auto indices = this->indices_view_;
  const dtype* __restrict__ category_location = model_.category_location.get_ptr();
  auto num_instances = model_.num_instances;

  uint32_t n_blocks = gpu_resource.get_sm_count();

  sgd_atomic_update(
      message_buffer, infrequent_embedding_vectors_.get_ptr(),
      [indices, num_instances] __device__() {
        return indices->model_indices_offsets[num_instances];
      },
      [indices, category_location] __device__(uint32_t i) {
        uint32_t index = indices->model_indices[i];
        dtype category = indices->samples[index];
        return category_location[2 * category + 1];
      },
      n_blocks, embedding_vec_size_, dev_lr, scale, stream);
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::hier_update_model(const emtype* message_buffer,
                                                           float* dev_lr, float scale,
                                                           hipStream_t stream) {
  const uint32_t& num_instances = model_.num_instances;
  uint32_t local_samples_size =
      ceildiv<uint32_t>(data_->batch_size, num_instances) * data_->table_sizes.size();
  uint32_t local_comm_buff_size =
      ceildiv<uint32_t>(max_num_infrequent_per_train_batch_, model_.num_instances);

  int num_sm = gpu_resource.get_sm_count();
  int n_blocks = 16 * num_sm;  // TODO: better heuristics

  infrequent_embedding_kernels::hier_update_model<<<n_blocks, embedding_vec_size_, 0, stream>>>(
      this->indices_view_, model_.category_location.get_ptr(), message_buffer,
      infrequent_embedding_vectors_.get_ptr(), embedding_vec_size_, model_.num_instances,
      local_samples_size, local_comm_buff_size, dev_lr, scale);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

/** Update model for single GPU (no communications), lr is a device variable */
template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::update_model_direct(float* dev_lr, float scale,
                                                             hipStream_t stream) {
  const uint32_t& num_instances = model_.num_instances;
  uint32_t local_samples_size =
      ceildiv<uint32_t>(data_->batch_size, num_instances) * data_->table_sizes.size();

  int num_sm = gpu_resource.get_sm_count();
  int n_blocks = 16 * num_sm;  // TODO: better heuristics

  /* Each model reads from the gradients of each network */
  PROFILE_RECORD("inf_update_model_direct.infrequent_update_model_direct.start", stream, false);
  infrequent_embedding_kernels::
      infrequent_update_model_direct<<<n_blocks, embedding_vec_size_, 0, stream>>>(
          gradients_pointers_.get_ptr(), infrequent_embedding_vectors_.get_ptr(),
          this->indices_view_, model_.category_location.get_ptr(), model_.num_instances,
          model_.global_instance_id, embedding_vec_size_, local_samples_size, dev_lr, scale);
  HCTR_LIB_THROW(hipPeekAtLastError());
  PROFILE_RECORD("inf_update_model_direct.infrequent_update_model_direct.stop", stream, false);
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::calculate_model_indices_sizes_from_offsets(
    hipStream_t stream) {
  auto indices = this->indices_view_;
  constexpr size_t TPB = 256;
  const size_t n_blocks = ceildiv<size_t>(model_.num_instances, TPB);
  infrequent_embedding_kernels::offsets_to_sizes<<<n_blocks, TPB, 0, stream>>>(
      model_indices_sizes_.get_ptr(), [=] __device__() { return indices->model_indices_offsets; },
      embedding_vec_size_ * sizeof(emtype), model_.num_instances);
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::calculate_network_indices_sizes_from_offsets(
    hipStream_t stream) {
  auto indices = this->indices_view_;
  constexpr size_t TPB = 256;
  const size_t n_blocks = ceildiv<size_t>(model_.num_instances, TPB);
  infrequent_embedding_kernels::offsets_to_sizes<<<n_blocks, TPB, 0, stream>>>(
      network_indices_sizes_.get_ptr(),
      [=] __device__() { return indices->network_indices_offsets; },
      embedding_vec_size_ * sizeof(emtype), model_.num_instances);
}

template class InfrequentEmbeddingBase<uint32_t>;
template class InfrequentEmbeddingBase<long long>;

template class InfrequentEmbedding<uint32_t, __half>;
template class InfrequentEmbedding<uint32_t, float>;
template class InfrequentEmbedding<long long, __half>;
template class InfrequentEmbedding<long long, float>;
}  // namespace hybrid_embedding

}  // namespace HugeCTR
