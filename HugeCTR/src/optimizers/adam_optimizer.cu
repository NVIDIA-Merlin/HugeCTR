#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <general_buffer2.hpp>
#include <optimizers/adam_optimizer.hpp>
#include <utils.cuh>
#include <utils.hpp>

namespace HugeCTR {

namespace {

template <typename T>
__global__ void adam_update_kernel(int len, float* weight, T* m, T* v, const T* wgrad,
                                   float alpha_t, float beta1, float beta2, float epsilon,
                                   float scaler) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    float gi = TypeConvertFunc<float, T>::convert(wgrad[i]) / scaler;
    float mi = beta1 * TypeConvertFunc<float, T>::convert(m[i]) + (1.f - beta1) * gi;
    float vi = beta2 * TypeConvertFunc<float, T>::convert(v[i]) + (1.f - beta2) * gi * gi;
    m[i] = TypeConvertFunc<T, float>::convert(mi);
    v[i] = TypeConvertFunc<T, float>::convert(vi);
    weight[i] -= alpha_t * mi / (sqrt(vi) + epsilon);
  }
}

}  // namespace

AdamOptimizer::AdamOptimizer(const Tensor2<float>& weight_main, const Tensor2<float>& fp32_wgrad,
                             const Tensor2<__half>& fp16_wgrad, bool mixed_precision,
                             const std::shared_ptr<GeneralBuffer2<CudaAllocator>>& buff,
                             const std::shared_ptr<GPUResource>& gpu_resource, float learning_rate,
                             float beta1, float beta2, float epsilon, float scaler)
    : Optimizer(weight_main, fp32_wgrad, fp16_wgrad, mixed_precision, gpu_resource, learning_rate,
                scaler),
      t_(0),
      beta1_(beta1),
      beta2_(beta2),
      epsilon_(epsilon) {
  if (mixed_precision) {
    buff->reserve({weight_main.get_num_elements()}, &fp16_m_);
    buff->reserve({weight_main.get_num_elements()}, &fp16_v_);
  } else {
    buff->reserve({weight_main.get_num_elements()}, &fp32_m_);
    buff->reserve({weight_main.get_num_elements()}, &fp32_v_);
  }
}  // namespace HugeCTR

void AdamOptimizer::initialize() {
  if (mixed_precision_) {
    hipMemsetAsync(fp16_m_.get_ptr(), 0, fp16_m_.get_size_in_bytes(), gpu_resource_->get_stream());
    hipMemsetAsync(fp16_v_.get_ptr(), 0, fp16_v_.get_size_in_bytes(), gpu_resource_->get_stream());
  } else {
    hipMemsetAsync(fp32_m_.get_ptr(), 0, fp32_m_.get_size_in_bytes(), gpu_resource_->get_stream());
    hipMemsetAsync(fp32_v_.get_ptr(), 0, fp32_v_.get_size_in_bytes(), gpu_resource_->get_stream());
  }
}

void AdamOptimizer::update() {
  CudaDeviceContext context(get_device_id());

  const size_t len = weight_main_.get_num_elements();
  constexpr size_t block_dim = 256;
  const size_t grid_dim = (len - 1) / block_dim + 1;

  ++t_;
  const float alpha_t = lr_ * sqrt(1 - pow(beta2_, t_)) / (1 - pow(beta1_, t_));

  float* weight = weight_main_.get_ptr();

  if (mixed_precision_) {
    __half* fp16_m = fp16_m_.get_ptr();
    __half* fp16_v = fp16_v_.get_ptr();
    const __half* fp16_wgrad = fp16_wgrad_.get_ptr();

    adam_update_kernel<<<grid_dim, block_dim, 0, gpu_resource_->get_stream()>>>(
        len, weight, fp16_m, fp16_v, fp16_wgrad, alpha_t, beta1_, beta2_, epsilon_, scaler_);
  } else {
    float* fp32_m = fp32_m_.get_ptr();
    float* fp32_v = fp32_v_.get_ptr();
    const float* fp32_wgrad = fp32_wgrad_.get_ptr();

    adam_update_kernel<<<grid_dim, block_dim, 0, gpu_resource_->get_stream()>>>(
        len, weight, fp32_m, fp32_v, fp32_wgrad, alpha_t, beta1_, beta2_, epsilon_, scaler_);
  }
#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

}  // namespace HugeCTR
