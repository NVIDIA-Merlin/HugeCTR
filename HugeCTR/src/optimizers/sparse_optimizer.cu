#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "HugeCTR/include/embedding.hpp"
#include "HugeCTR/include/utils.cuh"
#include "cub/device/device_radix_sort.cuh"
#include "cub/device/device_run_length_encode.cuh"
#include "cub/device/device_scan.cuh"

namespace HugeCTR {

template <typename TypeHashKey, typename TypeEmbeddingComp>
EmbeddingOptimizer<TypeHashKey, TypeEmbeddingComp>::EmbeddingOptimizer(
    size_t max_vocabulary_size_per_gpu_, SparseEmbeddingHashParams &param,
    const std::shared_ptr<GeneralBuffer2<CudaAllocator>> &buf)
    : param(param) {
  // new optimizer params used by update_params
  switch (param.opt_params.optimizer) {
    case Optimizer_t::Adam:  // adam
    {
      {
        buf->reserve({max_vocabulary_size_per_gpu_, param.embedding_vec_size},
                     &opt_tensors_.opt_m_tensors_);
        buf->reserve({max_vocabulary_size_per_gpu_, param.embedding_vec_size},
                     &opt_tensors_.opt_v_tensors_);
      }
      if (param.opt_params.update_type == Update_t::LazyGlobal) {
        buf->reserve({max_vocabulary_size_per_gpu_, param.embedding_vec_size},
                     &opt_tensors_.opt_prev_time_tensors_);
      }
      break;
    }
    case Optimizer_t::AdaGrad:  // nesterov
    {
      buf->reserve({max_vocabulary_size_per_gpu_, param.embedding_vec_size},
                   &opt_tensors_.opt_accm_tensors_);
      break;
    }
    case Optimizer_t::MomentumSGD:  // momentum_sgd
    {
      buf->reserve({max_vocabulary_size_per_gpu_, param.embedding_vec_size},
                   &opt_tensors_.opt_momentum_tensors_);
      break;
    }

    case Optimizer_t::Nesterov:  // nesterov
    {
      buf->reserve({max_vocabulary_size_per_gpu_, param.embedding_vec_size},
                   &opt_tensors_.opt_accm_tensors_);
      break;
    }

    case Optimizer_t::SGD:
      break;

    default:
      throw std::runtime_error(
          std::string("[HCDEBUG][ERROR] Runtime error: Invalid optimizer type\n"));
  }

  { buf->reserve({1, param.get_batch_size(true) * param.max_feature_num}, &sample_id_tensors_); }
  {
    buf->reserve({1, param.get_batch_size(true) * param.max_feature_num}, &sample_id_sort_tensors_);
  }
  {
    buf->reserve({1, param.get_batch_size(true) * param.max_feature_num},
                 &hash_value_index_sort_tensors_);
  }
  {
    buf->reserve({1, param.get_batch_size(true) * param.max_feature_num + 1},
                 &hash_value_index_count_offset_tensors_);
  }
  {
    buf->reserve({1, param.get_batch_size(true) * param.max_feature_num},
                 &new_hash_value_flag_tensors_);
  }
  {
    buf->reserve({1, param.get_batch_size(true) * param.max_feature_num},
                 &hash_value_flag_sumed_tensors_);
  }
  { buf->reserve({1, 1}, &hash_value_index_count_counter_tensors_); }
  {
    // cal the temp storage bytes for CUB radix sort
    size_t size = 0;
    hipcub::DeviceRadixSort::SortPairs((void *)nullptr, size, (size_t *)nullptr, (size_t *)nullptr,
                                    (TypeHashKey *)nullptr, (TypeHashKey *)nullptr,
                                    param.get_batch_size(true) * param.max_feature_num);

    // new temp storage tensors for CUB radix sort
    buf->reserve({size}, &temp_storage_sort_tensors_);
  }

  {
    size_t size = 0;
    hipcub::DeviceScan::InclusiveSum((void *)nullptr, size, (uint32_t *)nullptr, (uint32_t *)nullptr,
                                  param.get_batch_size(true) * param.max_feature_num);

    buf->reserve({size}, &temp_storage_scan_tensors_);
  }
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void EmbeddingOptimizer<TypeHashKey, TypeEmbeddingComp>::initialize(const GPUResource &local_gpu) {
  switch (param.opt_params.optimizer) {
    case Optimizer_t::Adam:  // adam
      CK_CUDA_THROW_(hipMemsetAsync(opt_tensors_.opt_m_tensors_.get_ptr(), 0,
                                     opt_tensors_.opt_m_tensors_.get_size_in_bytes(),
                                     local_gpu.get_stream()));
      CK_CUDA_THROW_(hipMemsetAsync(opt_tensors_.opt_v_tensors_.get_ptr(), 0,
                                     opt_tensors_.opt_v_tensors_.get_size_in_bytes(),
                                     local_gpu.get_stream()));
      param.opt_params.hyperparams.adam.times = 0;
      if (param.opt_params.update_type == Update_t::LazyGlobal) {
        dim3 grid(local_gpu.get_sm_count() * 4, 1, 1);
        dim3 block(512, 1, 1);
        initialize_array<<<grid, block, 0, local_gpu.get_stream()>>>(
            opt_tensors_.opt_prev_time_tensors_.get_ptr(),
            opt_tensors_.opt_prev_time_tensors_.get_num_elements(), uint64_t(1));
      }
      break;
    case Optimizer_t::AdaGrad:
      CK_CUDA_THROW_(hipMemsetAsync(opt_tensors_.opt_accm_tensors_.get_ptr(),
                                     param.opt_params.hyperparams.adagrad.initial_accu_value,
                                     opt_tensors_.opt_accm_tensors_.get_size_in_bytes(),
                                     local_gpu.get_stream()));
      break;
    case Optimizer_t::MomentumSGD:  // momentum_sgd
      CK_CUDA_THROW_(hipMemsetAsync(opt_tensors_.opt_momentum_tensors_.get_ptr(), 0,
                                     opt_tensors_.opt_momentum_tensors_.get_size_in_bytes(),
                                     local_gpu.get_stream()));
      break;

    case Optimizer_t::Nesterov:  // nesterov
      CK_CUDA_THROW_(hipMemsetAsync(opt_tensors_.opt_accm_tensors_.get_ptr(), 0,
                                     opt_tensors_.opt_accm_tensors_.get_size_in_bytes(),
                                     local_gpu.get_stream()));
      break;

    case Optimizer_t::SGD:
      break;

    default:
      throw std::runtime_error(
          std::string("[HCDEBUG][ERROR] Runtime error: Invalid optimizer type\n"));
  }
}

namespace {

__global__ void value_count_kernel_2(int nnz, const uint32_t *new_hash_value_flag,
                                     const uint32_t *hash_value_flag_sumed,
                                     uint32_t *hash_value_index_index, uint32_t *counter)

{
  for (int gid = blockIdx.x * blockDim.x + threadIdx.x; gid < nnz; gid += blockDim.x * gridDim.x) {
    uint32_t flag = new_hash_value_flag[gid];
    if (flag == 1) {
      hash_value_index_index[hash_value_flag_sumed[gid] - 1] = gid;
    }
  }
  if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
    *counter = hash_value_flag_sumed[nnz - 1];
    hash_value_index_index[*counter] = nnz;
  }
}

// expand sample id by row_offset
template <typename TypeKey>
__global__ void sample_id_expand_kernel(int batch_size, int slot_num, const TypeKey *row_offset,
                                        TypeKey *sample_id) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;

  if (gid < (batch_size * slot_num)) {
    TypeKey offset = row_offset[gid];
    int value_num = row_offset[gid + 1] - offset;
    for (int i = 0; i < value_num; i++) {
      sample_id[offset + i] = gid;
    }
  }
}

__global__ void value_count_kernel_1(int nnz, const size_t *hash_value_index_sort,
                                     uint32_t *new_hash_value_flag) {
  for (int gid = blockIdx.x * blockDim.x + threadIdx.x; gid < nnz; gid += blockDim.x * gridDim.x) {
    size_t cur_value = hash_value_index_sort[gid];
    if (gid > 0) {
      size_t former_value = hash_value_index_sort[gid - 1];
      // decide if this is the start of a group(the elements in this group have the same
      // hash_value_index_sort)
      if (cur_value != former_value) {
        new_hash_value_flag[gid] = 1;
      } else {
        new_hash_value_flag[gid] = 0;
      }
    } else {  // gid == 0
      new_hash_value_flag[gid] = 1;
    }
  }
}

// Helper function to accumulate the weight gradients for a thread's embedding vector
template <typename TypeKey, typename TypeEmbeddingComp>
__device__ __forceinline__ float accumulate_gradients(int embedding_vec_size,
                                                      const TypeKey *sample_id,
                                                      const uint32_t *hash_value_index_count_offset,
                                                      const TypeEmbeddingComp *wgrad, float scaler,
                                                      uint32_t offset, int bid, int tid) {
  uint32_t sample_num = hash_value_index_count_offset[bid + 1] - hash_value_index_count_offset[bid];

  float gi = 0.0f;
  for (int i = 0; i < sample_num; i++) {
    int sample_index = sample_id[offset + i];
    gi += TypeConvertFunc<float, TypeEmbeddingComp>::convert(
        wgrad[sample_index * embedding_vec_size + tid]);
  }
  return gi / scaler;
}

// First step of the global update with the Adam optimizer: compute gradient and add the
// corresponding terms to the moving-average accumulators
template <typename TypeKey, typename TypeEmbeddingComp>
__global__ void opt_adam_kernel_global(uint32_t hash_value_index_count_num, int embedding_vec_size,
                                       const AdamOptHyperParams adam, TypeEmbeddingComp *m_ptr,
                                       TypeEmbeddingComp *v_ptr, const TypeKey *sample_id,
                                       const size_t *hash_value_index_sort,
                                       const uint32_t *hash_value_index_count_offset,
                                       const TypeEmbeddingComp *wgrad, float scaler) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  if (tid < embedding_vec_size && bid < hash_value_index_count_num) {
    uint32_t offset = hash_value_index_count_offset[bid];
    float gi = accumulate_gradients(embedding_vec_size, sample_id, hash_value_index_count_offset,
                                    wgrad, scaler, offset, bid, tid);

    size_t row_index = hash_value_index_sort[offset];
    size_t feature_index = row_index * embedding_vec_size + tid;
    float mi = TypeConvertFunc<float, TypeEmbeddingComp>::convert(m_ptr[feature_index]) +
               (1.0f - adam.beta1) * gi / adam.beta1;
    float vi = TypeConvertFunc<float, TypeEmbeddingComp>::convert(v_ptr[feature_index]) +
               (1.0f - adam.beta2) * gi * gi / adam.beta2;

    m_ptr[feature_index] = TypeConvertFunc<TypeEmbeddingComp, float>::convert(mi);
    v_ptr[feature_index] = TypeConvertFunc<TypeEmbeddingComp, float>::convert(vi);
  }
}

// Second step of the global update with the Adam optimizer: update the moving-average accumulators
// and the weights for all the features
template <typename TypeEmbeddingComp>
__global__ void adam_update_kernel_global(int embedding_vec_size,
                                          size_t table_size,  // vocabulary size / factor
                                          const AdamOptHyperParams adam, TypeEmbeddingComp *m_ptr,
                                          TypeEmbeddingComp *v_ptr, float alpha_t,
                                          float *hash_table_value) {
  const int TILE_SIZE = blockDim.x * gridDim.x;
  for (size_t feature_index = blockIdx.x * blockDim.x + threadIdx.x;
       feature_index < table_size * embedding_vec_size; feature_index += TILE_SIZE) {
    float mi =
        adam.beta1 * TypeConvertFunc<float, TypeEmbeddingComp>::convert(m_ptr[feature_index]);
    float vi =
        adam.beta2 * TypeConvertFunc<float, TypeEmbeddingComp>::convert(v_ptr[feature_index]);

    m_ptr[feature_index] = TypeConvertFunc<TypeEmbeddingComp, float>::convert(mi);
    v_ptr[feature_index] = TypeConvertFunc<TypeEmbeddingComp, float>::convert(vi);

    float weight_diff = -alpha_t * mi / (sqrtf(vi) + adam.epsilon);
    hash_table_value[feature_index] += weight_diff;
  }
}

// First step of the global update with Momentum SGD: compute gradient and add the corresponding
// term to the momentum
template <typename TypeKey, typename TypeEmbeddingComp>
__global__ void opt_momentum_sgd_kernel_global(
    uint32_t hash_value_index_count_num, int embedding_vec_size, float lr,
    const MomentumSGDOptHyperParams momentum, TypeEmbeddingComp *momentum_ptr,
    const TypeKey *sample_id, const size_t *hash_value_index_sort,
    const uint32_t *hash_value_index_count_offset, const TypeEmbeddingComp *wgrad, float scaler) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  if (tid < embedding_vec_size && bid < hash_value_index_count_num) {
    uint32_t offset = hash_value_index_count_offset[bid];
    float gi = accumulate_gradients(embedding_vec_size, sample_id, hash_value_index_count_offset,
                                    wgrad, scaler, offset, bid, tid);

    size_t row_index = hash_value_index_sort[offset];
    size_t feature_index = row_index * embedding_vec_size + tid;
    float mo = TypeConvertFunc<float, TypeEmbeddingComp>::convert(momentum_ptr[feature_index]) -
               lr * gi / momentum.factor;
    momentum_ptr[feature_index] = TypeConvertFunc<TypeEmbeddingComp, float>::convert(mo);
  }
}

// Second step of the global update with Momentum SGD: update the momentum and the weights for all
// the features
template <typename TypeEmbeddingComp>
__global__ void momentum_sgd_update_kernel_global(int embedding_vec_size,
                                                  size_t table_size,  // vocabulary size / factor
                                                  const MomentumSGDOptHyperParams momentum,
                                                  TypeEmbeddingComp *momentum_ptr,
                                                  float *hash_table_value) {
  const int TILE_SIZE = blockDim.x * gridDim.x;
  for (size_t feature_index = blockIdx.x * blockDim.x + threadIdx.x;
       feature_index < table_size * embedding_vec_size; feature_index += TILE_SIZE) {
    float mo = TypeConvertFunc<float, TypeEmbeddingComp>::convert(momentum_ptr[feature_index]);
    mo *= momentum.factor;
    hash_table_value[feature_index] += mo;
    momentum_ptr[feature_index] = TypeConvertFunc<TypeEmbeddingComp, float>::convert(mo);
  }
}

// First step of the global update with Nesterov: update momentum and weights for all the features
template <typename TypeEmbeddingComp>
__global__ void nesterov_global_update_kernel_global(int embedding_vec_size,
                                                     size_t table_size,  // vocabulary size / factor
                                                     const NesterovOptHyperParams nesterov,
                                                     TypeEmbeddingComp *accm_ptr,
                                                     float *hash_table_value) {
  const int TILE_SIZE = blockDim.x * gridDim.x;
  for (size_t feature_index = blockIdx.x * blockDim.x + threadIdx.x;
       feature_index < table_size * embedding_vec_size; feature_index += TILE_SIZE) {
    float accm = TypeConvertFunc<float, TypeEmbeddingComp>::convert(accm_ptr[feature_index]);
    accm *= nesterov.mu;
    accm_ptr[feature_index] = TypeConvertFunc<TypeEmbeddingComp, float>::convert(accm);
    hash_table_value[feature_index] += accm * nesterov.mu;
  }
}

// Second step of the global update with Nesterov: compute gradient, add the corresponding term
// to the momentum and update the weights
template <typename TypeKey, typename TypeEmbeddingComp>
__global__ void nesterov_local_update_kernel_global(
    uint32_t hash_value_index_count_num, int embedding_vec_size, float lr,
    const NesterovOptHyperParams nesterov, TypeEmbeddingComp *accm_ptr, const TypeKey *sample_id,
    const size_t *hash_value_index_sort, const uint32_t *hash_value_index_count_offset,
    const TypeEmbeddingComp *wgrad, float *hash_table_value, float scaler) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  if (tid < embedding_vec_size && bid < hash_value_index_count_num) {
    uint32_t offset = hash_value_index_count_offset[bid];
    float gi = accumulate_gradients(embedding_vec_size, sample_id, hash_value_index_count_offset,
                                    wgrad, scaler, offset, bid, tid);

    size_t row_index = hash_value_index_sort[offset];
    size_t feature_index = row_index * embedding_vec_size + tid;
    float accm = TypeConvertFunc<float, TypeEmbeddingComp>::convert(accm_ptr[feature_index]);
    accm -= lr * gi;
    accm_ptr[feature_index] = TypeConvertFunc<TypeEmbeddingComp, float>::convert(accm);
    hash_table_value[feature_index] -= (1 + nesterov.mu) * (lr * gi);
  }
}

// Local update for the Adam optimizer: compute the gradients and update the accumulators and the
// weights
template <typename TypeKey, typename TypeEmbeddingComp>
__global__ void opt_adam_kernel(uint32_t hash_value_index_count_num, int embedding_vec_size,
                                const AdamOptHyperParams adam, TypeEmbeddingComp *m_ptr,
                                TypeEmbeddingComp *v_ptr, float alpha_t, const TypeKey *sample_id,
                                const size_t *hash_value_index_sort,
                                const uint32_t *hash_value_index_count_offset,
                                const TypeEmbeddingComp *wgrad, float *hash_table_value,
                                float scaler) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  if (tid < embedding_vec_size && bid < hash_value_index_count_num) {
    uint32_t offset = hash_value_index_count_offset[bid];
    float gi = accumulate_gradients(embedding_vec_size, sample_id, hash_value_index_count_offset,
                                    wgrad, scaler, offset, bid, tid);

    size_t row_index = hash_value_index_sort[offset];
    size_t feature_index = row_index * embedding_vec_size + tid;
    float mi =
        adam.beta1 * TypeConvertFunc<float, TypeEmbeddingComp>::convert(m_ptr[feature_index]) +
        (1.0f - adam.beta1) * gi;
    float vi =
        adam.beta2 * TypeConvertFunc<float, TypeEmbeddingComp>::convert(v_ptr[feature_index]) +
        (1.0f - adam.beta2) * gi * gi;
    m_ptr[feature_index] = TypeConvertFunc<TypeEmbeddingComp, float>::convert(mi);
    v_ptr[feature_index] = TypeConvertFunc<TypeEmbeddingComp, float>::convert(vi);
    float weight_diff = -alpha_t * mi / (sqrtf(vi) + adam.epsilon);

    hash_table_value[feature_index] += weight_diff;
  }
}

// Local update for the Adagrad optimizer: compute the gradients and update the accumulators and the
// weights
template <typename TypeKey, typename TypeEmbeddingComp>
__global__ void opt_adagrad_kernel(uint32_t hash_value_index_count_num, int embedding_vec_size,
                                   float lr, const AdaGradParams adagrad,
                                   TypeEmbeddingComp *accum_ptr, const TypeKey *sample_id,
                                   const size_t *hash_value_index_sort,
                                   const uint32_t *hash_value_index_count_offset,
                                   const TypeEmbeddingComp *wgrad, float *hash_table_value,
                                   float scaler) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  if (tid < embedding_vec_size && bid < hash_value_index_count_num) {
    uint32_t offset = hash_value_index_count_offset[bid];

    float gi = accumulate_gradients(embedding_vec_size, sample_id, hash_value_index_count_offset,
                                    wgrad, scaler, offset, bid, tid);

    size_t row_index = hash_value_index_sort[offset];
    size_t feature_index = row_index * embedding_vec_size + tid;
    float accum =
        TypeConvertFunc<float, TypeEmbeddingComp>::convert(accum_ptr[feature_index]) + gi * gi;

    accum_ptr[feature_index] = TypeConvertFunc<TypeEmbeddingComp, float>::convert(accum);
    float weight_diff = -lr * gi / (sqrtf(accum) + adagrad.epsilon);

    hash_table_value[feature_index] += weight_diff;
  }
}

// Local update for Momentum SGD: compute the gradients and update the momentum and the weights
template <typename TypeKey, typename TypeEmbeddingComp>
__global__ void opt_momentum_sgd_kernel(uint32_t hash_value_index_count_num, int embedding_vec_size,
                                        float lr, const MomentumSGDOptHyperParams momentum,
                                        TypeEmbeddingComp *momentum_ptr, const TypeKey *sample_id,
                                        const size_t *hash_value_index_sort,
                                        const uint32_t *hash_value_index_count_offset,
                                        const TypeEmbeddingComp *wgrad, float *hash_table_value,
                                        float scaler) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  if (tid < embedding_vec_size && bid < hash_value_index_count_num) {
    uint32_t offset = hash_value_index_count_offset[bid];
    float gi = accumulate_gradients(embedding_vec_size, sample_id, hash_value_index_count_offset,
                                    wgrad, scaler, offset, bid, tid);

    size_t row_index = hash_value_index_sort[offset];
    size_t feature_index = row_index * embedding_vec_size + tid;
    float mo = momentum.factor *
                   TypeConvertFunc<float, TypeEmbeddingComp>::convert(momentum_ptr[feature_index]) -
               lr * gi;
    momentum_ptr[feature_index] = TypeConvertFunc<TypeEmbeddingComp, float>::convert(mo);

    hash_table_value[feature_index] += mo;
  }
}

// Local update for Nesterov: compute the gradients and update the accumulators and the weights
template <typename TypeKey, typename TypeEmbeddingComp>
__global__ void opt_nesterov_kernel(uint32_t hash_value_index_count_num, int embedding_vec_size,
                                    float lr, const NesterovOptHyperParams nesterov,
                                    TypeEmbeddingComp *accm_ptr, const TypeKey *sample_id,
                                    const size_t *hash_value_index_sort,
                                    const uint32_t *hash_value_index_count_offset,
                                    const TypeEmbeddingComp *wgrad, float *hash_table_value,
                                    float scaler) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  if (tid < embedding_vec_size && bid < hash_value_index_count_num) {
    uint32_t offset = hash_value_index_count_offset[bid];
    float gi = accumulate_gradients(embedding_vec_size, sample_id, hash_value_index_count_offset,
                                    wgrad, scaler, offset, bid, tid);

    size_t row_index = hash_value_index_sort[offset];
    size_t feature_index = row_index * embedding_vec_size + tid;
    float accm_old = TypeConvertFunc<float, TypeEmbeddingComp>::convert(accm_ptr[feature_index]);
    float accm_new = nesterov.mu * accm_old - lr * gi;
    accm_ptr[feature_index] = TypeConvertFunc<TypeEmbeddingComp, float>::convert(accm_new);
    float weight_diff = -nesterov.mu * accm_old + (1.0f + nesterov.mu) * accm_new;

    hash_table_value[feature_index] += weight_diff;
  }
}

// Local update for SGD: compute the gradients and update the weights
template <typename TypeKey, typename TypeEmbeddingComp>
__global__ void opt_sgd_kernel(uint32_t hash_value_index_count_num, int embedding_vec_size,
                               float lr, const TypeKey *sample_id,
                               const size_t *hash_value_index_sort,
                               const uint32_t *hash_value_index_count_offset,
                               const TypeEmbeddingComp *wgrad, float *hash_table_value,
                               float scaler) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  if (tid < embedding_vec_size && bid < hash_value_index_count_num) {
    uint32_t offset = hash_value_index_count_offset[bid];
    float gi = accumulate_gradients(embedding_vec_size, sample_id, hash_value_index_count_offset,
                                    wgrad, scaler, offset, bid, tid);

    size_t row_index = hash_value_index_sort[offset];
    float weight_diff = -lr * gi;

    size_t feature_index = row_index * embedding_vec_size + tid;
    hash_table_value[feature_index] += weight_diff;
  }
}

// Lazy global update for the Adam optimizer: compute the gradients and update the weights and the
// accumulators (local approximation of the global update)
template <typename TypeKey, typename TypeEmbeddingComp>
__global__ void opt_adam_kernel_lazy(uint32_t hash_value_index_count_num, int embedding_vec_size,
                                     const AdamOptHyperParams adam, uint64_t *prev_time_ptr,
                                     TypeEmbeddingComp *m_ptr, TypeEmbeddingComp *v_ptr,
                                     float alpha_t_common, uint64_t times, const TypeKey *sample_id,
                                     const size_t *hash_value_index_sort,
                                     const uint32_t *hash_value_index_count_offset,
                                     const TypeEmbeddingComp *wgrad, float *hash_table_value,
                                     float scaler) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  if (tid < embedding_vec_size && bid < hash_value_index_count_num) {
    uint32_t offset = hash_value_index_count_offset[bid];
    float gi = accumulate_gradients(embedding_vec_size, sample_id, hash_value_index_count_offset,
                                    wgrad, scaler, offset, bid, tid);

    size_t row_index = hash_value_index_sort[offset];
    size_t feature_index = row_index * embedding_vec_size + tid;

    // First update the weights
    uint64_t prev_time = prev_time_ptr[feature_index];
    prev_time_ptr[feature_index] = times;
    uint64_t skipped = times - prev_time;
    float beta1_pow_skipped = powf(adam.beta1, skipped);
    float alpha_t = alpha_t_common * sqrtf(1.0f - powf(adam.beta2, prev_time)) /
                    (1.0f - powf(adam.beta1, prev_time)) * (1.0f - beta1_pow_skipped);
    float mi = TypeConvertFunc<float, TypeEmbeddingComp>::convert(m_ptr[feature_index]);
    float vi = TypeConvertFunc<float, TypeEmbeddingComp>::convert(v_ptr[feature_index]);
    float weight_diff = -alpha_t * mi / (sqrtf(vi) + adam.epsilon);
    hash_table_value[feature_index] += weight_diff;

    // Then update the moving-average accumulators
    mi = beta1_pow_skipped * mi + (1.0f - adam.beta1) * gi;
    vi = powf(adam.beta2, skipped) * vi + (1.0f - adam.beta2) * gi * gi;
    m_ptr[feature_index] = TypeConvertFunc<TypeEmbeddingComp, float>::convert(mi);
    v_ptr[feature_index] = TypeConvertFunc<TypeEmbeddingComp, float>::convert(vi);
  }
}

template <typename TypeKey, typename TypeEmbeddingComp>
__global__ void opt_sgd_atomic_kernel(int nnz, int embedding_vec_size, float lr_scale,
                                      const size_t *hash_value_index, const TypeKey *sample_ids,
                                      const TypeEmbeddingComp *wgrad, float *hash_table_value) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  if (tid < embedding_vec_size && bid < nnz) {
    for (int key_id = bid; key_id < nnz; key_id += gridDim.x) {
      int sample_id = sample_ids[key_id];
      float deltaw = -lr_scale * TypeConvertFunc<float, TypeEmbeddingComp>::convert(
                                     wgrad[sample_id * embedding_vec_size + tid]);

      // atomic update
      size_t value_index = hash_value_index[key_id];
      size_t feature_index = value_index * embedding_vec_size + tid;
      atomicAdd(&hash_table_value[feature_index], deltaw);
    }
  }
}

// only support LocalizedSlotSparseEmbeddingOneHot
template <typename TypeEmbeddingComp>
__global__ void opt_sgd_atomic_kernel(int nnz, int embedding_vec_size, float lr_scale,
                                      const size_t *hash_value_index,
                                      const TypeEmbeddingComp *wgrad, float *hash_table_value) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  if (tid < embedding_vec_size && bid < nnz) {
    for (int key_id = bid; key_id < nnz; key_id += gridDim.x) {
      // for one-hot, the max_feature_per_slot is 1, so sample_id is equal to key_id
      float deltaw = -lr_scale * TypeConvertFunc<float, TypeEmbeddingComp>::convert(
                                     wgrad[key_id * embedding_vec_size + tid]);

      // atomic update
      size_t value_index = hash_value_index[key_id];
      size_t feature_index = value_index * embedding_vec_size + tid;
      atomicAdd(&hash_table_value[feature_index], deltaw);
    }
  }
}

}  // namespace

template <typename TypeHashKey, typename TypeEmbeddingComp>
void EmbeddingOptimizer<TypeHashKey, TypeEmbeddingComp>::update(
    size_t batch_size, size_t slot_num, size_t embedding_vec_size,
    size_t max_vocabulary_size_per_gpu, size_t nnz, const Tensor2<TypeHashKey> &row_offset,
    Tensor2<size_t> &hash_value_index, const Tensor2<TypeEmbeddingComp> &wgrad,
    Tensor2<float> &hash_table_value, size_t sm_count, hipStream_t stream) {
  OptimizerTensor<TypeEmbeddingComp> &opt_tensor = opt_tensors_;
  OptParams &opt_params = param.opt_params;
  Tensor2<TypeHashKey> &sample_id = sample_id_tensors_;
  Tensor2<TypeHashKey> &sample_id_sort = sample_id_sort_tensors_;
  Tensor2<size_t> &hash_value_index_sort = hash_value_index_sort_tensors_;
  Tensor2<uint32_t> &hash_value_index_count_offset = hash_value_index_count_offset_tensors_;
  Tensor2<uint32_t> &new_hash_value_flag = new_hash_value_flag_tensors_;
  Tensor2<uint32_t> &hash_value_flag_sumed = hash_value_flag_sumed_tensors_;
  Tensor2<uint32_t> &hash_value_index_count_counter = hash_value_index_count_counter_tensors_;
  Tensor2<void> &temp_storage_sort = temp_storage_sort_tensors_;
  Tensor2<void> &temp_storage_scan = temp_storage_scan_tensors_;

  if (slot_num == 0) {
    return;
  }

  size_t block_size, grid_size;

  try {
    // step1: expand sample IDs
    block_size = 64;
    grid_size = (batch_size * slot_num - 1) / block_size + 1;
    sample_id_expand_kernel<<<grid_size, block_size, 0, stream>>>(
        batch_size, slot_num, row_offset.get_ptr(), sample_id.get_ptr());

    if (opt_params.optimizer == Optimizer_t::SGD &&
        opt_params.hyperparams.sgd.atomic_update) {  // for SGD, do atomic update
      const size_t block_size = embedding_vec_size;
      const size_t grid_size = min(max(1ul, nnz), sm_count * 32);

      float lr_scale = opt_params.lr / opt_params.scaler;
      opt_sgd_atomic_kernel<<<grid_size, block_size, 0, stream>>>(
          nnz, embedding_vec_size, lr_scale, hash_value_index.get_ptr(), sample_id.get_ptr(),
          wgrad.get_ptr(), hash_table_value.get_ptr());
    } else {
      // step3: sort by hash_value_index
      int end_bit = static_cast<int>(log2(static_cast<float>(max_vocabulary_size_per_gpu))) + 1;
      size_t temp_storage_sort_size = temp_storage_sort.get_size_in_bytes();
      CK_CUDA_THROW_(hipcub::DeviceRadixSort::SortPairs(
          temp_storage_sort.get_ptr(), temp_storage_sort_size, hash_value_index.get_ptr(),
          hash_value_index_sort.get_ptr(), sample_id.get_ptr(), sample_id_sort.get_ptr(), nnz, 0,
          end_bit, stream, false));

      // step4: count the number for each unduplicated hash_value_index
      CK_CUDA_THROW_(
          hipMemsetAsync(hash_value_index_count_counter.get_ptr(), 0, sizeof(uint32_t), stream));

      constexpr size_t max_grid_size = 384;
      block_size = 256;
      grid_size = min(max_grid_size, (nnz - 1) / block_size + 1);

      value_count_kernel_1<<<grid_size, block_size, 0, stream>>>(
          nnz, hash_value_index_sort.get_ptr(), new_hash_value_flag.get_ptr());

      // prefix_sum
      size_t temp_storage_scan_size = temp_storage_scan.get_size_in_bytes();
      CK_CUDA_THROW_(hipcub::DeviceScan::InclusiveSum(
          temp_storage_scan.get_ptr(), temp_storage_scan_size, new_hash_value_flag.get_ptr(),
          hash_value_flag_sumed.get_ptr(), nnz, stream));

      value_count_kernel_2<<<grid_size, block_size, 0, stream>>>(
          nnz, new_hash_value_flag.get_ptr(), hash_value_flag_sumed.get_ptr(),
          hash_value_index_count_offset.get_ptr(), hash_value_index_count_counter.get_ptr());

      uint32_t hash_hash_value_index_count_num = 0;
      // this async memcpy will not perform as a async operation because the host memory is not
      // a pinned memory
      CK_CUDA_THROW_(hipMemcpyAsync(&hash_hash_value_index_count_num,
                                     hash_value_index_count_counter.get_ptr(), sizeof(uint32_t),
                                     hipMemcpyDeviceToHost, stream));

      // step5: use optimizer method to compute deltaw and update the parameters
      block_size = embedding_vec_size;
      grid_size = max(1, hash_hash_value_index_count_num);

      switch (opt_params.update_type) {
        case Update_t::Global: {
          switch (opt_params.optimizer) {
            case Optimizer_t::Adam: {
              float alpha_t =
                  opt_params.lr *
                  sqrt(1 -
                       pow(opt_params.hyperparams.adam.beta2, opt_params.hyperparams.adam.times)) /
                  (1 - pow(opt_params.hyperparams.adam.beta1, opt_params.hyperparams.adam.times));
              // update target mi and vi
              opt_adam_kernel_global<<<grid_size, block_size, 0, stream>>>(
                  hash_hash_value_index_count_num, embedding_vec_size, opt_params.hyperparams.adam,
                  opt_tensor.opt_m_tensors_.get_ptr(), opt_tensor.opt_v_tensors_.get_ptr(),
                  sample_id_sort.get_ptr(), hash_value_index_sort.get_ptr(),
                  hash_value_index_count_offset.get_ptr(), wgrad.get_ptr(), opt_params.scaler);
              // all update according to the mi vi
              adam_update_kernel_global<<<1024, 256, 0, stream>>>(
                  embedding_vec_size, max_vocabulary_size_per_gpu, opt_params.hyperparams.adam,
                  opt_tensor.opt_m_tensors_.get_ptr(), opt_tensor.opt_v_tensors_.get_ptr(), alpha_t,
                  hash_table_value.get_ptr());
              break;
            }
            case Optimizer_t::AdaGrad: {
              opt_adagrad_kernel<<<grid_size, block_size, 0, stream>>>(
                  hash_hash_value_index_count_num, embedding_vec_size, opt_params.lr,
                  opt_params.hyperparams.adagrad, opt_tensor.opt_accm_tensors_.get_ptr(),
                  sample_id_sort.get_ptr(), hash_value_index_sort.get_ptr(),
                  hash_value_index_count_offset.get_ptr(), wgrad.get_ptr(),
                  hash_table_value.get_ptr(), opt_params.scaler);
              break;
            }
            case Optimizer_t::MomentumSGD:
              opt_momentum_sgd_kernel_global<<<grid_size, block_size, 0, stream>>>(
                  hash_hash_value_index_count_num, embedding_vec_size, opt_params.lr,
                  opt_params.hyperparams.momentum, opt_tensor.opt_momentum_tensors_.get_ptr(),
                  sample_id_sort.get_ptr(), hash_value_index_sort.get_ptr(),
                  hash_value_index_count_offset.get_ptr(), wgrad.get_ptr(), opt_params.scaler);
              momentum_sgd_update_kernel_global<<<1024, 256, 0, stream>>>(
                  embedding_vec_size, max_vocabulary_size_per_gpu, opt_params.hyperparams.momentum,
                  opt_tensor.opt_momentum_tensors_.get_ptr(), hash_table_value.get_ptr());
              break;
            case Optimizer_t::Nesterov:
              nesterov_global_update_kernel_global<<<1024, 256, 0, stream>>>(
                  embedding_vec_size, max_vocabulary_size_per_gpu, opt_params.hyperparams.nesterov,
                  opt_tensor.opt_accm_tensors_.get_ptr(), hash_table_value.get_ptr());
              nesterov_local_update_kernel_global<<<grid_size, block_size, 0, stream>>>(
                  hash_hash_value_index_count_num, embedding_vec_size, opt_params.lr,
                  opt_params.hyperparams.nesterov, opt_tensor.opt_accm_tensors_.get_ptr(),
                  sample_id_sort.get_ptr(), hash_value_index_sort.get_ptr(),
                  hash_value_index_count_offset.get_ptr(), wgrad.get_ptr(),
                  hash_table_value.get_ptr(), opt_params.scaler);
              break;
            case Optimizer_t::SGD:
              // Note: this is in fact a local update
              /// TODO: remove duplicate?
              opt_sgd_kernel<<<grid_size, block_size, 0, stream>>>(
                  hash_hash_value_index_count_num, embedding_vec_size, opt_params.lr,
                  sample_id_sort.get_ptr(), hash_value_index_sort.get_ptr(),
                  hash_value_index_count_offset.get_ptr(), wgrad.get_ptr(),
                  hash_table_value.get_ptr(), opt_params.scaler);
              break;
            default:
              CK_THROW_(Error_t::WrongInput, "Error: Invalid opitimizer type");
          }  // switch (optimizer)
          break;
        }
        case Update_t::Local: {
          switch (opt_params.optimizer) {
            case Optimizer_t::Adam: {
              float alpha_t =
                  opt_params.lr *
                  sqrt(1 -
                       pow(opt_params.hyperparams.adam.beta2, opt_params.hyperparams.adam.times)) /
                  (1 - pow(opt_params.hyperparams.adam.beta1, opt_params.hyperparams.adam.times));

              opt_adam_kernel<<<grid_size, block_size, 0, stream>>>(
                  hash_hash_value_index_count_num, embedding_vec_size, opt_params.hyperparams.adam,
                  opt_tensor.opt_m_tensors_.get_ptr(), opt_tensor.opt_v_tensors_.get_ptr(), alpha_t,
                  sample_id_sort.get_ptr(), hash_value_index_sort.get_ptr(),
                  hash_value_index_count_offset.get_ptr(), wgrad.get_ptr(),
                  hash_table_value.get_ptr(), opt_params.scaler);
              break;
            }
            case Optimizer_t::AdaGrad: {
              opt_adagrad_kernel<<<grid_size, block_size, 0, stream>>>(
                  hash_hash_value_index_count_num, embedding_vec_size, opt_params.lr,
                  opt_params.hyperparams.adagrad, opt_tensor.opt_accm_tensors_.get_ptr(),
                  sample_id_sort.get_ptr(), hash_value_index_sort.get_ptr(),
                  hash_value_index_count_offset.get_ptr(), wgrad.get_ptr(),
                  hash_table_value.get_ptr(), opt_params.scaler);
              break;
            }
            case Optimizer_t::MomentumSGD:
              opt_momentum_sgd_kernel<<<grid_size, block_size, 0, stream>>>(
                  hash_hash_value_index_count_num, embedding_vec_size, opt_params.lr,
                  opt_params.hyperparams.momentum, opt_tensor.opt_momentum_tensors_.get_ptr(),
                  sample_id_sort.get_ptr(), hash_value_index_sort.get_ptr(),
                  hash_value_index_count_offset.get_ptr(), wgrad.get_ptr(),
                  hash_table_value.get_ptr(), opt_params.scaler);
              break;
            case Optimizer_t::Nesterov:
              opt_nesterov_kernel<<<grid_size, block_size, 0, stream>>>(
                  hash_hash_value_index_count_num, embedding_vec_size, opt_params.lr,
                  opt_params.hyperparams.nesterov, opt_tensor.opt_accm_tensors_.get_ptr(),
                  sample_id_sort.get_ptr(), hash_value_index_sort.get_ptr(),
                  hash_value_index_count_offset.get_ptr(), wgrad.get_ptr(),
                  hash_table_value.get_ptr(), opt_params.scaler);
              break;
            case Optimizer_t::SGD:
              opt_sgd_kernel<<<grid_size, block_size, 0, stream>>>(
                  hash_hash_value_index_count_num, embedding_vec_size, opt_params.lr,
                  sample_id_sort.get_ptr(), hash_value_index_sort.get_ptr(),
                  hash_value_index_count_offset.get_ptr(), wgrad.get_ptr(),
                  hash_table_value.get_ptr(), opt_params.scaler);
              break;
            default:
              CK_THROW_(Error_t::WrongInput, "Error: Invalid opitimizer type");
          }  // switch (optimizer)
          break;
        }
        case Update_t::LazyGlobal: {
          switch (opt_params.optimizer) {
            case Optimizer_t::Adam: {
              const float alpha_t_common =
                  opt_params.lr / (1.0f - opt_params.hyperparams.adam.beta1);

              opt_adam_kernel_lazy<<<grid_size, block_size, 0, stream>>>(
                  hash_hash_value_index_count_num, embedding_vec_size, opt_params.hyperparams.adam,
                  opt_tensor.opt_prev_time_tensors_.get_ptr(), opt_tensor.opt_m_tensors_.get_ptr(),
                  opt_tensor.opt_v_tensors_.get_ptr(), alpha_t_common,
                  opt_params.hyperparams.adam.times, sample_id_sort.get_ptr(),
                  hash_value_index_sort.get_ptr(), hash_value_index_count_offset.get_ptr(),
                  wgrad.get_ptr(), hash_table_value.get_ptr(), opt_params.scaler);
              break;
            }
            case Optimizer_t::AdaGrad:
            case Optimizer_t::MomentumSGD:
            case Optimizer_t::Nesterov:
            case Optimizer_t::SGD: {
              /// TODO: implement lazy global update for other optimizer types
              CK_THROW_(Error_t::WrongInput,
                        "Error: lazy global update is only implemented for Adam");
              break;
            }
            default:
              CK_THROW_(Error_t::WrongInput, "Error: Invalid opitimizer type");
          }
          break;
        }
        default:
          CK_THROW_(Error_t::WrongInput, "Error: Invalid update type");
      }  // switch (update type)
    }
#ifndef NDEBUG
    hipDeviceSynchronize();
    CK_CUDA_THROW_(hipGetLastError());
#endif
  } catch (const std::runtime_error &rt_err) {
    std::cerr << rt_err.what() << std::endl;
    throw;
  }

  return;
}

template class EmbeddingOptimizer<unsigned int, float>;
template class EmbeddingOptimizer<long long, float>;
template class EmbeddingOptimizer<unsigned int, __half>;
template class EmbeddingOptimizer<long long, __half>;
}  // namespace HugeCTR
