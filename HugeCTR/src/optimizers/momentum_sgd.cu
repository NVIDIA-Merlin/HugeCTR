#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <optimizers/momentum_sgd.hpp>

namespace {

__device__ __forceinline__ void momentumSGD_update_device(
    float* weight_ptr, float* momentum_ptr, float wgrad,
    HugeCTR::MomentumSGDHyperParameters hyper_parameters, float scaler) {
  momentum_ptr[0] =
      hyper_parameters.momentum_factor * momentum_ptr[0] - hyper_parameters.lr * wgrad / scaler;
  weight_ptr[0] += momentum_ptr[0];

  // if(isnan(weight_ptr[0])){
  //   int count = atomicAdd(&debug_counter, 1);
  //   if(count < 10)
  //     printf("%f;", wgrad);
  // }

  return;
}

__global__ void momentumSGD_update_kernel(float* weight_ptr, float* momentum_ptr,
                                          const float* wgrad_ptr, int size,
                                          HugeCTR::MomentumSGDHyperParameters hyper_parameters,
                                          float scaler) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < size) {
    momentumSGD_update_device(weight_ptr + idx, momentum_ptr + idx, wgrad_ptr[idx],
                              hyper_parameters, scaler);
  }
  return;
}

}  // namespace

namespace HugeCTR {

void MomentumSGD::update() {
  CudaDeviceContext context(device_id_);

  constexpr int block_dim = 256;
  int grid_dim = (weight_->get_num_elements() + block_dim - 1) / block_dim;
  float* weight_ptr = weight_->get_ptr_with_offset(0);
  const float* wgrad_ptr = wgrad_->get_ptr_with_offset(0);
  float* momentum_ptr = momentum_->get_ptr_with_offset(0);

  MomentumSGDHyperParameters hyper_parameters = {lr_, momentum_factor_};
  momentumSGD_update_kernel<<<grid_dim, block_dim, 0, stream>>>(
      weight_ptr, momentum_ptr, wgrad_ptr, weight_->get_num_elements(), hyper_parameters, scaler_);
#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

}  // namespace HugeCTR
