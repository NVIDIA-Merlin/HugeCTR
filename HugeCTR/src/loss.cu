#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <loss.hpp>
#include <utils.cuh>
#include <vector>

namespace HugeCTR {

namespace {

template <typename T>
__forceinline__ __device__ void atomic_global_sum_div(T val, T *acc, float div) {
  val = warpReduceSum(val);
  if (threadIdx.x % warpSize == 0) {
    atomicAdd(acc, (T)(val / div));
  }
  return;
}

}  // namespace

template <typename T>
Loss<T>::Loss(const Tensor2<float> &train_label_tensor, const Tensor2<T> &train_input_tensor,
              const Tensor2<float> &evaluate_label_tensor, const Tensor2<T> &evaluate_input_tensor,
              const Tensor2<float> &loss_tensor, const std::shared_ptr<Regularizer<T>> &regularizer,
              const std::shared_ptr<GPUResource> &gpu_resource, int total_gpu_count, float scaler)
    : regularizer_(regularizer),
      gpu_resource_(gpu_resource),
      total_gpu_count_(total_gpu_count),
      scaler_(scaler) {
  train_label_tensors_.push_back(train_label_tensor);
  train_input_tensors_.push_back(train_input_tensor);
  evaluate_label_tensors_.push_back(evaluate_label_tensor);
  evaluate_input_tensors_.push_back(evaluate_input_tensor);
  loss_tensors_.push_back(loss_tensor);
}

template <typename T>
void Loss<T>::compute(bool is_train) {
  Tensor2<T> &input_tensor = get_input_tensors(is_train)[0];
  const auto &input_dim = input_tensor.get_dimensions();
  int batch_size = input_dim[0];
  compute(is_train, batch_size);
}


  //Note: current_batchsize here is the batchsize on this device
template <typename T>
void Loss<T>::compute(bool is_train, long long current_batchsize) {
  CudaDeviceContext context(get_device_id());

  Tensor2<T> &input_tensor = get_input_tensors(is_train)[0];
  const Tensor2<float> &label_tensor = get_label_tensors(is_train)[0];
  Tensor2<float> &loss_tensor = loss_tensors_[0];

  const auto &input_dim = input_tensor.get_dimensions();
  const auto &label_dim = label_tensor.get_dimensions();

  int batch_size = input_dim[0];
  int feature_dim = input_dim[1];

  T *input = input_tensor.get_ptr();
  const float *label = label_tensor.get_ptr();
  float *loss = loss_tensor.get_ptr();

  float rterm = 0.0f;
  if (regularizer_) {
    regularizer_->compute_rterm();
    rterm = regularizer_->get_rterm();
  }

  if (current_batchsize > batch_size && current_batchsize < 0){
    CK_THROW_(Error_t::WrongInput, "current_batchsize > batch_size && current_batchsize < 0");
  }
  
  do_compute(input, label, loss, current_batchsize, feature_dim, scaler_, rterm, is_train,
             get_gpu().get_stream());
  if (is_train) {
    // once current_batchsize < batch_size in train we set the rest dgrad to 0
    if(current_batchsize < batch_size){
      hipMemsetAsync(input+current_batchsize*feature_dim, 0, 
		      (batch_size - current_batchsize)*feature_dim*sizeof(T), get_gpu().get_stream());
    }
  }

  if (is_train && regularizer_) {
    regularizer_->initialize_wgrad();
  }

#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

template <typename T>
CrossEntropyLoss<T>::CrossEntropyLoss(const Tensor2<float> &label_tensor,
                                      const Tensor2<T> &input_tensor,
                                      const Tensor2<float> &loss_tensor,
                                      const std::shared_ptr<Regularizer<T>> &regularizer,
                                      const std::shared_ptr<GPUResource> &gpu_resource,
                                      int total_gpu_count, float scaler)
    : Loss<T>(label_tensor, input_tensor, label_tensor, input_tensor, loss_tensor, regularizer,
              gpu_resource, total_gpu_count, scaler) {
  const auto &input_dim = input_tensor.get_dimensions();
  const auto &label_dim = label_tensor.get_dimensions();
  int feature_dim = input_dim[1];

  if (feature_dim != 2)
    CK_THROW_(Error_t::WrongInput, "The feature dimension of CE loss input should be 2");
  if (input_dim[0] != label_dim[0])
    CK_THROW_(Error_t::WrongInput, "The batch sizes of input tensor and label tensor are not same");
}

// Suppose we use one thread to calculate one sample
template <typename T>
__global__ void CrossEntropy_Kernel(T *input, const float *label, float *cel_loss, int batch_size,
                                    int total_gpu_count, int feature_dim, float scaler, float rterm,
                                    bool is_train) {
  int tid = threadIdx.x;
  extern __shared__ float loss_s[];

  loss_s[tid] = 0.0f;

  float z0_exp, z1_exp, a0, a1;
  int id1, id2;

  for (int i = tid; i < batch_size; i += blockDim.x) {
    id1 = i * feature_dim;
    id2 = i * feature_dim + 1;
    z0_exp = exp((double)input[id1]);
    z1_exp = exp((double)input[id2]);

    a0 = z0_exp / (z0_exp + z1_exp);
    a1 = z1_exp / (z0_exp + z1_exp);

    bool no_click = label[i] < 0.5f;

    if (is_train) {
      // calculate the grad
      input[id1] = (a0 - (no_click ? 1.0f : 0.0f)) / batch_size * scaler / total_gpu_count;
      input[id2] = (a1 - (!no_click ? 1.0f : 0.0f)) / batch_size * scaler / total_gpu_count;
    }

    loss_s[tid] += -1 * log(no_click ? a0 : a1);
  }
  __syncthreads();

  float loss_tmp = 0.0f;

  if (tid == 0) {
    for (int i = 0; i < blockDim.x; ++i) loss_tmp += loss_s[i];
    cel_loss[0] = loss_tmp / batch_size + rterm;
  }
}

template <typename T>
void CrossEntropyLoss<T>::do_compute(T *input, const float *label, float *loss, int batch_size,
                                     int feature_dim, float scaler, float rterm, bool is_train,
                                     hipStream_t stream) {
  int block_size = min(batch_size, 1024);
  size_t smem_size = block_size * sizeof(float);
  CrossEntropy_Kernel<<<1, block_size, smem_size, stream>>>(input, label, loss, batch_size,
                                                            Loss<T>::get_total_gpu_count(),
                                                            feature_dim, scaler, rterm, is_train);
}

template <typename T>
BinaryCrossEntropyLoss<T>::BinaryCrossEntropyLoss(
    const Tensor2<float> &train_label_tensor, const Tensor2<T> &train_input_tensor,
    const Tensor2<float> &evaluate_label_tensor, const Tensor2<T> &evaluate_input_tensor,
    const Tensor2<float> &loss_tensor, const std::shared_ptr<Regularizer<T>> &regularizer,
    const std::shared_ptr<GPUResource> &gpu_resource, int total_gpu_count, float scaler)
    : Loss<T>(train_label_tensor, train_input_tensor, evaluate_label_tensor, evaluate_input_tensor,
              loss_tensor, regularizer, gpu_resource, total_gpu_count, scaler) {
  const auto &input_dim = train_input_tensor.get_dimensions();
  int feature_dim = input_dim[1];
  if (feature_dim != 1)
    CK_THROW_(Error_t::WrongInput, "The feature dimension of BCE loss input should be 1");
}

// Suppose we use one thread to calculate one sample
template <typename T>
__global__ void BinaryCrossEntropy_Kernel(T *input, const float *label, float *bce_loss,
                                          float scaler, int batch_size, int total_gpu_count,
                                          float rterm, bool is_train) {
  int tid = threadIdx.x;
  extern __shared__ float loss_s[];
  loss_s[tid] = 0.0f;

  for (int i = tid; i < batch_size; i += blockDim.x) {
    const float x = input[i];
    const float y = label[i];
    if (x >= 0) {
      float exp_neg_x = exp(-x);
      loss_s[tid] += x * (1 - y) + log(1 + exp_neg_x);
      input[i] = is_train ? ((1 - y) - exp_neg_x / (1 + exp_neg_x)) * scaler / (float)batch_size /
                                total_gpu_count
                          : 1 / (1 + exp_neg_x);
    } else {
      float exp_x = exp(x);
      loss_s[tid] += -x * y + log(1 + exp_x);
      input[i] = is_train
                     ? (-y + exp_x / (1 + exp_x)) * scaler / (float)batch_size / total_gpu_count
                     : exp_x / (exp_x + 1);
    }
  }
  __syncthreads();

  float loss_tmp = 0.0f;
  if (tid == 0) {
    for (int i = 0; i < blockDim.x; ++i) loss_tmp += loss_s[i];
    bce_loss[0] = loss_tmp / batch_size + rterm;
  }
}
template <typename T>
void BinaryCrossEntropyLoss<T>::do_compute(T *input, const float *label, float *loss,
                                           int batch_size, int feature_dim, float scaler,
                                           float rterm, bool is_train, hipStream_t stream) {
  int block_size = min(batch_size, 1024);
  size_t smem_size = block_size * sizeof(float);
  BinaryCrossEntropy_Kernel<<<1, block_size, smem_size, stream>>>(
      input, label, loss, scaler, batch_size, Loss<T>::get_total_gpu_count(), rterm, is_train);
}

__forceinline__ __device__ __host__ float cross_entropy_loss(float x, float y) {
  float loss = 0.f;
  if (x >= 0) {
    float exp_neg_x = exp(-x);
    loss = x * (1 - y) + log(1 + exp_neg_x);
  } else {
    float exp_x = exp(x);
    loss = -x * y + log(1 + exp_x);
  }
  return -loss;
}

__forceinline__ __device__ __host__ float cross_entropy_loss_backward(float x, float y) {
  float grad = 0.f;
  if (x >= 0) {
    float exp_neg_x = exp(-x);
    grad = ((1 - y) - exp_neg_x / (1 + exp_neg_x));
  } else {
    float exp_x = exp(x);
    grad = (-y + exp_x / (1 + exp_x));
  }
  return grad;
}

template <typename T>
__global__ void MultiCrossEntropy_Kernel(T *input, const float *label, const float *target_weight,
                                         float *bce_loss, int batchsize, int total_gpu_count,
                                         int labels_per_sample, float scaler, float rterm,
                                         bool is_train) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int num_threads = blockDim.x * gridDim.x;
  float loss_s = 0.f;
  const int size = batchsize * labels_per_sample;
  for (int i = tid; i < size; i += num_threads) {
    int target_weight_idx = i % labels_per_sample;
    const float x = input[i];
    const float y = label[i];
    float loss =
        (label[i] < -0.5) ? 0.f : (target_weight[target_weight_idx] * cross_entropy_loss(x, y));
    loss_s += loss;
    if (is_train) {
      input[i] = (label[i] < -0.5)
                     ? 0.f
                     : (target_weight[target_weight_idx] * cross_entropy_loss_backward(x, y) /
                        size * scaler / total_gpu_count);
    }
  }

  atomic_global_sum_div(-loss_s, bce_loss, size);
  if (tid == 0) {
    atomicAdd(bce_loss, rterm);
  }
  return;
}

template <typename T>
void MultiCrossEntropyLoss<T>::do_compute(T *input, const float *label, float *loss, int batch_size,
                                          int feature_dim, float scaler, float rterm, bool is_train,
                                          hipStream_t stream) {
  int labels_per_sample = feature_dim;
  hipMemsetAsync(loss, 0, Loss<T>::get_loss_tensors()[0].get_size_in_bytes(), stream);

  const int BLOCK_SIZE = 256;
  const int GRID_SIZE = min(40, (batch_size * labels_per_sample - 1) / BLOCK_SIZE);
  float *target_weight = target_weight_.get_ptr();
  MultiCrossEntropy_Kernel<<<GRID_SIZE, BLOCK_SIZE, 0, stream>>>(
      input, label, target_weight, loss, batch_size, Loss<T>::get_total_gpu_count(),
      labels_per_sample, scaler, rterm, is_train);
}

template <typename T>
MultiCrossEntropyLoss<T>::MultiCrossEntropyLoss(const Tensor2<float> &label_tensor,
                                                const Tensor2<T> &input_tensor,
                                                const Tensor2<float> &loss_tensor,
                                                const std::shared_ptr<Regularizer<T>> &regularizer,
                                                const std::vector<float> &target_weight,
                                                const std::shared_ptr<GPUResource> &gpu_resource,
                                                int total_gpu_count, float scaler)
    : Loss<T>(label_tensor, input_tensor, label_tensor, input_tensor, loss_tensor, regularizer,
              gpu_resource, total_gpu_count, scaler) {
  if (label_tensor.get_dimensions().size() != 2 || input_tensor.get_dimensions().size() != 2 ||
      label_tensor.get_dimensions()[0] != input_tensor.get_dimensions()[0] ||
      label_tensor.get_dimensions()[1] != input_tensor.get_dimensions()[1]) {
    CK_THROW_(Error_t::WrongInput, "Format of input tensor and label tensor don't match");
  }
  // verify the length of target_weight
  if (target_weight.size() != input_tensor.get_dimensions()[1]) {
    CK_THROW_(Error_t::WrongInput, "target_weight.size() != input_tensor.get_dims()[0]");
  }

  // load target_weight to internal Tensor
  std::shared_ptr<GeneralBuffer2<CudaAllocator>> internal_buff =
      GeneralBuffer2<CudaAllocator>::create();
  std::vector<size_t> twdim = {1, label_tensor.get_dimensions()[1]};
  internal_buff->reserve(twdim, &target_weight_);

  CudaDeviceContext context(Loss<T>::get_device_id());
  internal_buff->allocate();
  CK_CUDA_THROW_(hipMemcpy(target_weight_.get_ptr(), target_weight.data(),
                            target_weight_.get_size_in_bytes(), hipMemcpyHostToDevice));

  return;
}

template class Loss<__half>;
template class Loss<float>;
template class MultiCrossEntropyLoss<__half>;
template class MultiCrossEntropyLoss<float>;
template class CrossEntropyLoss<__half>;
template class CrossEntropyLoss<float>;
template class BinaryCrossEntropyLoss<__half>;
template class BinaryCrossEntropyLoss<float>;

}  // namespace HugeCTR
