#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <diagnose.hpp>
#include <fstream>
#include <limits>
#include <utils.cuh>

namespace HugeCTR {

namespace diagnose {

__device__ float atomicMin(float* address, float val) {
  float old = val;
  do {
    val = old;
    old = atomicExch(address, val);
  } while (old < val);
  return old;
}

__device__ float atomicMax(float* address, float val) {
  float old = val;
  do {
    val = old;
    old = atomicExch(address, val);
  } while (old > val);
  return old;
}

template <typename T>
__global__ void histogram_kernel(const T* arr, size_t len, float* range) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < len; i += blockDim.x * gridDim.x) {
    float val = TypeConvertFunc<float, T>::convert(arr[i]);
    if (val <= 0) {
      atomicMin(range + 0, val);
      atomicMax(range + 1, val);
    }
    if (val >= 0) {
      atomicMin(range + 2, val);
      atomicMax(range + 3, val);
    }
  }
}

template <typename T>
__global__ void verify_kernel(const T* arr, size_t len, int* flag);

template <>
__global__ void verify_kernel<float>(const float* arr, size_t len, int* flag) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < len; i += blockDim.x * gridDim.x) {
    if (isnan(arr[i])) atomicAdd(flag, 1);
  }
}

template <>
__global__ void verify_kernel(const __half* arr, size_t len, int* flag) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < len; i += blockDim.x * gridDim.x) {
    if (__hisnan(arr[i])) {
      atomicAdd(flag, 1);
    }
  }
}

template <typename T>
__global__ void sample_kernel(const T* arr, int len, float* arr_sample, int stride,
                              int max_sample_len) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < len; i += blockDim.x * gridDim.x) {
    if (i % stride == 0) {
      int j = i / stride;
      if (j < max_sample_len) {
        arr_sample[j] = TypeConvertFunc<float, T>::convert(arr[i]);
      }
    }
  }
}

template <typename T>
void verify_and_histogram(const char* category, const Tensor2<T>& tensor,
                          const hipStream_t& stream) {
  float h_array[4]{0.0f, -std::numeric_limits<float>::infinity(),
                   std::numeric_limits<float>::infinity(), 0.0f};
  int h_flag;
  float* d_array;
  int* d_flag;
  CK_CUDA_THROW_(hipMalloc(&d_array, sizeof(h_array)));
  CK_CUDA_THROW_(hipMalloc(&d_flag, sizeof(int)));
  CK_CUDA_THROW_(
      hipMemcpyAsync(d_array, h_array, sizeof(h_array), hipMemcpyHostToDevice, stream));
  CK_CUDA_THROW_(hipMemsetAsync(d_flag, 0, sizeof(int), stream));
  histogram_kernel<<<160, 1024, 0, stream>>>(tensor.get_ptr(), tensor.get_num_elements(), d_array);
  verify_kernel<<<160, 1024, 0, stream>>>(tensor.get_ptr(), tensor.get_num_elements(), d_flag);
  CK_CUDA_THROW_(
      hipMemcpyAsync(h_array, d_array, sizeof(h_array), hipMemcpyDeviceToHost, stream));
  CK_CUDA_THROW_(hipMemcpyAsync(&h_flag, d_flag, sizeof(int), hipMemcpyDeviceToHost, stream));
  CK_CUDA_THROW_(hipStreamSynchronize(stream));

  std::stringstream ss;
  ss << "Diagnose for (" << category << "), Histogram [" << h_array[0] << ", " << h_array[1] << "]"
     << ", [" << h_array[2] << ", " << h_array[3] << "]" << std::endl;
  MESSAGE_(ss.str());

  if (h_flag != 0) {
    CK_THROW_(Error_t::DataCheckError, std::string("Nan assert for ") + category + " failed(" +
                                           std::to_string(h_flag) + ").");
  }
  CK_CUDA_THROW_(hipFree(d_array));
  CK_CUDA_THROW_(hipFree(d_flag));
}

template <typename T>
void sample_and_print(const char* category, const Tensor2<T>& tensor, size_t sample_count,
                      const hipStream_t& stream) {
  if (sample_count == 0) return;

  std::unique_ptr<float[]> h_array(new float[sample_count]);

  float* d_array;
  CK_CUDA_THROW_(hipMalloc(&d_array, sample_count * sizeof(float)));
  CK_CUDA_THROW_(hipMemsetAsync(d_array, 0, sample_count * sizeof(float), stream));
  sample_kernel<<<160, 1024, 0, stream>>>(tensor.get_ptr(), tensor.get_num_elements(), d_array,
                                          tensor.get_num_elements() / sample_count, sample_count);
  CK_CUDA_THROW_(hipMemcpyAsync(h_array.get(), d_array, sample_count * sizeof(float),
                                 hipMemcpyDeviceToHost, stream));
  CK_CUDA_THROW_(hipStreamSynchronize(stream));

  std::stringstream ss;
  ss << "Diagnose for (" << category << "), Sampling [";
  for (size_t i = 0; i < min(sample_count, tensor.get_num_elements()); i++) {
    if (i != 0) ss << ",";
    ss << h_array[i];
  }
  ss << "]" << std::endl;
  MESSAGE_(ss.str());

  CK_CUDA_THROW_(hipFree(d_array));
}

template <typename T>
void sample_and_print(const char* category, const Tensor2<T>& tensor, int begin, int end,
                      const hipStream_t& stream) {
  if (begin >= 0 && end <= static_cast<int>(tensor.get_num_elements()) && end > begin) {
  } else if (end < 0 && begin >= -static_cast<int>(tensor.get_num_elements()) && end > begin) {
    begin += tensor.get_num_elements();
    end += tensor.get_num_elements();
  } else {
    return;
  }

  std::unique_ptr<T[]> h_array(new T[end - begin]);
  CK_CUDA_THROW_(hipMemcpyAsync(h_array.get(), tensor.get_ptr() + begin,
                                 (begin - end) * sizeof(float), hipMemcpyDeviceToHost, stream));
  CK_CUDA_THROW_(hipStreamSynchronize(stream));

  std::stringstream ss;
  ss << "Diagnose for (" << category << "), Sampling [";
  for (size_t i = 0; i < end - begin; i++) {
    if (i != 0) ss << ",";
    ss << h_array[i];
  }
  ss << "]" << std::endl;
  MESSAGE_(ss.str());
}

template <typename T>
void dump(const char* filename, const Tensor2<T>& tensor, const hipStream_t& stream) {
  std::unique_ptr<T[]> h_array(new T[tensor.get_num_elements()]);
  CK_CUDA_THROW_(hipMemcpyAsync(h_array.get(), tensor.get_ptr(), tensor.get_size_in_bytes(),
                                 hipMemcpyDeviceToHost, stream));
  CK_CUDA_THROW_(hipStreamSynchronize(stream));

  std::ofstream s(filename, std::ios::out | std::ios::binary);
  s.write(reinterpret_cast<const char*>(h_array.get()), tensor.get_size_in_bytes());
  s.close();
}

template void verify_and_histogram<float>(const char* category, const Tensor2<float>& tensor,
                                          const hipStream_t& stream);

template void dump<unsigned int>(const char* filename, const Tensor2<unsigned int>& tensor,
                                 const hipStream_t& stream);
template void dump<long long>(const char* filename, const Tensor2<long long>& tensor,
                              const hipStream_t& stream);

}  // namespace diagnose

}  // namespace HugeCTR