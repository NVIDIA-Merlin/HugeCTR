#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <HugeCTR/include/utils.hpp>
#include <algorithm>
#include <cstdio>
#include <ctime>
#include <functional>
#include <layers/dropout_cudnn_layer.hpp>
#include <prims/linalg/binary_op.cuh>
#include <utils.cuh>
#include <utils.hpp>

#ifndef NDEBUG
#include <iostream>
#endif

namespace HugeCTR {

template <typename T>
DropoutCudnnLayer<T>::DropoutCudnnLayer(
    const Tensor2<T>& in_tensor, const Tensor2<T>& out_tensor,
    const std::shared_ptr<GeneralBuffer2<CudaAllocator>> blobs_buff, float rate,
    const std::shared_ptr<GPUResource>& gpu_resource)

    : Layer(gpu_resource), rate_(rate), scale_(1.0 / (1.0 - rate)) {
  assert(in_tensor.get_num_elements() == out_tensor.get_num_elements());
  assert(rate_ > 0.f && rate_ < 1.f);

  const auto& in_tensor_dim = in_tensor.get_dimensions();
  in_tensors_.emplace_back(in_tensor);
  out_tensors_.emplace_back(out_tensor);

  CudaDeviceContext context(get_device_id());

  size_t num_feature = in_tensor_dim[1];
  int batch_size = in_tensor_dim[0];
  hipdnnDataType_t data_type = CudnnDataType<T>::getType();
  int n_stride = num_feature;
  int w_stride = 1;
  CK_CUDNN_THROW_(hipdnnCreateTensorDescriptor(&in_out_desc_));
  CK_CUDNN_THROW_(hipdnnSetTensor4dDescriptorEx(in_out_desc_, data_type, batch_size, 1, 1,
                                               num_feature, n_stride, 1, 1, w_stride));

  CK_CUDNN_THROW_(hipdnnCreateDropoutDescriptor(&dropout_descriptor_));

  size_t sizeInBytes = 0;

  CK_CUDNN_THROW_(hipdnnDropoutGetStatesSize(gpu_resource->get_cudnn_handle(), &sizeInBytes));

  assert(sizeInBytes != 0);

  CK_CUDNN_THROW_(cudnnDropoutGetReserveSpaceSize(in_out_desc_, &reserveSpaceSizeInBytes_));

  blobs_buff->reserve({1, reserveSpaceSizeInBytes_}, &mask_);

  hipMalloc(&cudnn_status_, sizeInBytes);

  CK_CUDNN_THROW_(hipdnnSetDropoutDescriptor(dropout_descriptor_, gpu_resource->get_cudnn_handle(),
                                            rate, cudnn_status_, sizeInBytes, 0));
}

template <typename T>
DropoutCudnnLayer<T>::~DropoutCudnnLayer() {
  try {
    CK_CUDNN_THROW_(hipdnnDestroyDropoutDescriptor(dropout_descriptor_));
    hipFree(cudnn_status_);
    CK_CUDNN_THROW_(hipdnnDestroyTensorDescriptor(in_out_desc_));
  } catch (const std::runtime_error& rt_err) {
    std::cerr << rt_err.what() << std::endl;
  }
}

template <typename T>
void DropoutCudnnLayer<T>::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());

  if (is_train) {
    CK_CUDNN_THROW_(cudnnDropoutForward(
        get_gpu().get_cudnn_handle(), dropout_descriptor_, in_out_desc_, in_tensors_[0].get_ptr(),
        in_out_desc_, out_tensors_[0].get_ptr(), mask_.get_ptr(), reserveSpaceSizeInBytes_));
  } else {
    hipMemcpyAsync(out_tensors_[0].get_ptr(), in_tensors_[0].get_ptr(),
                    in_tensors_[0].get_size_in_bytes(), hipMemcpyDeviceToDevice,
                    get_gpu().get_stream());
  }
}

template <typename T>
void DropoutCudnnLayer<T>::bprop() {
  CudaDeviceContext context(get_device_id());
  CK_CUDNN_THROW_(cudnnDropoutBackward(
      get_gpu().get_cudnn_handle(), dropout_descriptor_, in_out_desc_, out_tensors_[0].get_ptr(),
      in_out_desc_, in_tensors_[0].get_ptr(), mask_.get_ptr(), reserveSpaceSizeInBytes_));
}

template class DropoutCudnnLayer<float>;
template class DropoutCudnnLayer<__half>;

}  // namespace HugeCTR
