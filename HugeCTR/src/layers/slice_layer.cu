#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common.hpp>
#include <layers/slice_layer.hpp>
#include <utils.cuh>
#include <utils.hpp>

#ifndef NDEBUG
#include <iostream>
#endif

namespace HugeCTR {

namespace {

template <size_t length, typename T>
__device__ int array_length(T (&arr)[length]) {
  return length;
}

template <typename T, typename... Args>
__global__ void slice_kernel(bool forward, T* in, const int h, const int in_w, const int virt_w,
                             const Args... args) {
  const typename SliceLayer<T>::OutParam out_params[] = {args...};
  const int n_outs = array_length(out_params);

  for (int row = blockIdx.x; row < h; row += gridDim.x) {
    for (int k = 0; k < n_outs; k++) {
      int st = out_params[k].st;
      int ed = out_params[k].ed;
      int out_w = ed - st;
      for (int out_col = threadIdx.x; out_col < out_w; out_col += blockDim.x) {
        int in_col = out_col + st;
        int in_idx = row * in_w + in_col;
        int out_idx = row * out_w + out_col;
        T* out = out_params[k].out;
        if (forward) {
          out[out_idx] = in[in_idx];
        } else {
          in[in_idx] += out[out_idx];
        }
      }
      __syncthreads();
    }
  }
}

}  // anonymous namespace

template <typename T>
SliceLayer<T>::SliceLayer(const Tensor2<T>& train_in_tensor, const Tensor2<T>& evaluate_in_tensor,
                          Tensors2<T>& out_tensors,
                          const std::shared_ptr<GeneralBuffer2<CudaAllocator>>& blobs_buff,
                          std::vector<std::pair<int, int>>& ranges,
                          const std::shared_ptr<GPUResource>& gpu_resource)
    : Layer(gpu_resource), virt_w_(0) {
  try {
    if (ranges.empty()) {
      CK_THROW_(Error_t::WrongInput, "Empty slice ranges is not allowed");
    }

    if (!out_tensors.empty()) {
      CK_THROW_(Error_t::WrongInput, "output tensor vector must be empty");
    }

    auto in_dims = train_in_tensor.get_dimensions();
    if (in_dims.size() != 2) {
      CK_THROW_(Error_t::WrongInput, "Only 2D tensors can be concatenated");
    }

    size_t height = in_dims[0];
    int in_w = in_dims[1];
    int prev_min = -1;
    int prev_max = 0;
    for (auto& range : ranges) {
      int cur_min = range.first;
      int cur_max = range.second;
      if (cur_min >= cur_max) {
        CK_THROW_(Error_t::WrongInput, "Reverse range is not allowed");
      }
      if (cur_min < 0 || cur_max < 0) {
        CK_THROW_(Error_t::WrongInput, "Negative ranges cannot be allowed");
      }
      if (!(prev_min <= cur_min && prev_max <= cur_max)) {
        CK_THROW_(Error_t::WrongInput, "A range cannot be out-order nor included in another");
      }
      if (cur_min >= in_w || cur_max > in_w) {
        CK_THROW_(Error_t::WrongInput, "Ranges cannot be bigger than the input width");
      }
      size_t out_w = cur_max - cur_min;
      std::vector<size_t> out_dims = {height, out_w};
      {
        Tensor2<T> tensor;
        blobs_buff->reserve(out_dims, &tensor);
        out_tensors.push_back(tensor);
      }
      sts_.push_back(cur_min);
      virt_w_ += out_w;

      prev_min = cur_min;
      prev_max = cur_max;
    }

    train_in_tensors_.push_back(train_in_tensor);
    evaluate_in_tensors_.push_back(evaluate_in_tensor);
    for (auto& out_tensor : out_tensors) {
      out_tensors_.push_back(out_tensor);
    }

  } catch (const std::runtime_error& rt_err) {
    std::cerr << rt_err.what() << std::endl;
    throw;
  }
}

template <typename T>
void SliceLayer<T>::fprop(bool is_train) {
  prop_common(true, is_train, get_gpu().get_stream());
}

template <typename T>
void SliceLayer<T>::bprop() {
  prop_common(false, true, get_gpu().get_stream());
}

template <typename T>
void SliceLayer<T>::prop_common(bool forward, bool is_train, hipStream_t stream) {
  CudaDeviceContext context(get_device_id());

  int n_out_tensors = out_tensors_.size();
  if (n_out_tensors == 2) {
    std::vector<OutParam> out_params = set_out_params(2);
    kernel_launch(forward, is_train, stream, out_params[0], out_params[1]);
  } else if (n_out_tensors == 3) {
    std::vector<OutParam> out_params = set_out_params(3);
    kernel_launch(forward, is_train, stream, out_params[0], out_params[1], out_params[2]);
  } else if (n_out_tensors == 4) {
    std::vector<OutParam> out_params = set_out_params(4);
    kernel_launch(forward, is_train, stream, out_params[0], out_params[1], out_params[2],
                  out_params[3]);
  } else if (n_out_tensors == 5) {
    std::vector<OutParam> out_params = set_out_params(5);
    kernel_launch(forward, is_train, stream, out_params[0], out_params[1], out_params[2],
                  out_params[3], out_params[4]);
  } else {
    CK_THROW_(Error_t::UnSupportedFormat, "Slicing into > 5 layers is not supported");
  }

#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

template <typename T>
std::vector<typename SliceLayer<T>::OutParam> SliceLayer<T>::set_out_params(int n) {
  std::vector<OutParam> out_params;
  for (int i = 0; i < n; i++) {
    Tensor2<T>& out_tensor = out_tensors_[i];
    T* out = out_tensor.get_ptr();
    int st = sts_[i];
    int w = out_tensor.get_dimensions()[1];
    out_params.push_back({out, st, st + w});
  }
  return std::move(out_params);
}

template <typename T>
template <typename... Args>
void SliceLayer<T>::kernel_launch(bool forward, bool is_train, hipStream_t stream, Args&... args) {
  int block_size = 512;
  int n_blocks = get_gpu().get_sm_count() * 4;
  Tensor2<T>& in_tensor = get_in_tensors(is_train)[0];
  T* in = in_tensor.get_ptr();
  int h = in_tensor.get_dimensions()[0];
  int in_w = in_tensor.get_dimensions()[1];
  if (!forward) {
    initialize_array<<<n_blocks, block_size, 0, stream>>>(in, h * in_w, T(0));
  }
  slice_kernel<<<n_blocks, block_size, 0, stream>>>(forward, in, h, in_w, virt_w_, args...);
}

template class SliceLayer<float>;
template class SliceLayer<__half>;

}  // namespace HugeCTR
