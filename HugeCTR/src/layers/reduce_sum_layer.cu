#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <layers/reduce_sum_layer.hpp>
#include <utils.cuh>
#include <utils.hpp>

#include <algorithm>
#include <functional>

#ifndef NDEBUG
#include <iostream>
#endif

namespace HugeCTR {

namespace {

template <size_t length, typename T>
__device__ int array_length(T (&arr)[length]) {
  return length;
}

// this kernel can support dims_size=1/2/3
template <typename... Args>
__global__ void reduce_sum_kernel(const float* input, float* output, int axis, Args... args) {
  size_t in_dims[] = {args...};
  int dims_size = array_length(in_dims);
  float local_sum = 0.0f;

  if (axis == 0) {  // block_num = dim1 * dim2, do dim0 number of elements reduction in one block
    if (dims_size == 1) {  // dims_size == 1
      for (int tid = threadIdx.x; tid < in_dims[0]; tid += blockDim.x) {
        local_sum += input[tid];
      }
    } else if (dims_size == 2) {  // dims_size == 2
      for (int tid = threadIdx.x; tid < in_dims[0]; tid += blockDim.x) {
        local_sum += input[tid * in_dims[1] + blockIdx.x];
      }
    } else if (dims_size == 3) {  // dims_size == 3
      for (int tid = threadIdx.x; tid < in_dims[0]; tid += blockDim.x) {
        local_sum += input[tid * (in_dims[1] * in_dims[2]) + blockIdx.x];
      }
    }
  } else if (axis ==
             1) {  // block_num = dim0 * dim2, do dim1 number of elements reduction in one block
    if (dims_size == 2) {  // dims_size == 2
      for (int tid = threadIdx.x; tid < in_dims[1]; tid += blockDim.x) {
        local_sum += input[blockIdx.x * in_dims[1] + tid];
      }
    } else if (dims_size == 3) {  // dims_size == 3
      for (int tid = threadIdx.x; tid < in_dims[1]; tid += blockDim.x) {
        local_sum += input[blockIdx.x / in_dims[2] * (in_dims[1] * in_dims[2]) + tid * in_dims[2] +
                           blockIdx.x % in_dims[2]];
      }
    }
  } else if (axis ==
             2) {  // block_num = dim0 * dim1, do dim2 number of elements reduction in one block
    for (int tid = threadIdx.x; tid < in_dims[2]; tid += blockDim.x) {
      local_sum += input[blockIdx.x * in_dims[2] + tid];
    }
  }

  local_sum = blockReduceSum(local_sum);
  if (threadIdx.x == 0) {
    output[blockIdx.x] = local_sum;
  }
}

template <typename... Args>
__global__ void reduce_sum_dgrad_kernel(const float* top_grad, float* dgrad, int axis,
                                        Args... args) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  size_t in_dims[] = {args...};
  int dims_size = array_length(in_dims);

  if (axis == 0) {
    if (dims_size == 1) {  // dims_size == 1
      if (tid < in_dims[0]) {
        dgrad[tid] = top_grad[0];
      }
    } else if (dims_size == 2) {  // dims_size == 2
      if (tid < (in_dims[0] * in_dims[1])) {
        dgrad[tid] = top_grad[tid % in_dims[1]];
      }
    } else if (dims_size == 3) {  // dims_size == 3
      if (tid < (in_dims[0] * in_dims[1] * in_dims[2])) {
        int dim1_index = tid % (in_dims[1] * in_dims[2]) / in_dims[2];
        int dim2_index = tid % in_dims[2];
        dgrad[tid] = top_grad[dim1_index * in_dims[2] + dim2_index];
      }
    }
  } else if (axis == 1) {
    if (dims_size == 2) {  // dims_size == 2
      if (tid < (in_dims[0] * in_dims[1])) {
        dgrad[tid] = top_grad[tid / in_dims[1]];
      }
    } else if (dims_size == 3) {  // dims_size == 3
      if (tid < (in_dims[0] * in_dims[1] * in_dims[2])) {
        int dim0_index = tid / (in_dims[1] * in_dims[2]);
        int dim2_index = tid % in_dims[2];
        dgrad[tid] = top_grad[dim0_index * in_dims[2] + dim2_index];
      }
    }
  } else if (axis == 2) {
    int dim0_index = tid / (in_dims[1] * in_dims[2]);
    int dim1_index = tid % (in_dims[1] * in_dims[2]) / in_dims[2];
    dgrad[tid] = top_grad[dim0_index * in_dims[1] + dim1_index];
  }
}

}  // end of namespace

ReduceSumLayer::ReduceSumLayer(const Tensor2<float>& in_tensor, Tensor2<float>& out_tensor,
                               const std::shared_ptr<GeneralBuffer2<CudaAllocator>>& blobs_buff,
                               int axis, const std::shared_ptr<GPUResource>& gpu_resource)
    : Layer(gpu_resource), axis_(axis) {
  try {
    // error input checking
    const auto& in_dims = in_tensor.get_dimensions();
    for (auto i : in_dims) {
      if (i == 0) {
        CK_THROW_(Error_t::WrongInput, "The input dims can not be 0");
      }
    }
    if (axis >= (int)(in_dims.size()) || axis < 0) {
      CK_THROW_(Error_t::WrongInput, "The axis is overflow");
    }

    std::vector<size_t> out_dims(in_dims.size());
    for (int i = 0; i < (int)(in_dims.size()); i++) {
      if (i == axis) {
        out_dims[i] = 1;
      } else {
        out_dims[i] = in_dims[i];
      }
    }

    blobs_buff->reserve(out_dims, &out_tensor);
    out_tensors_.push_back(out_tensor);
    in_tensors_.push_back(in_tensor);

  } catch (const std::runtime_error& rt_err) {
    std::cerr << rt_err.what() << std::endl;
    throw;
  }
}

void ReduceSumLayer::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());

  float* input = in_tensors_[0].get_ptr();
  float* output = out_tensors_[0].get_ptr();
  auto in_dims = in_tensors_[0].get_dimensions();
  auto out_dims = out_tensors_[0].get_dimensions();

  int block_num = 1;
  for (auto dim : out_dims) {
    block_num *= dim;
  }

  dim3 blockSize(256, 1, 1);
  dim3 gridSize(block_num, 1, 1);
  if (in_dims.size() == 1) {
    reduce_sum_kernel<<<gridSize, blockSize, 0, get_gpu().get_stream()>>>(input, output, axis_,
                                                                          in_dims[0]);
  } else if (in_dims.size() == 2) {
    reduce_sum_kernel<<<gridSize, blockSize, 0, get_gpu().get_stream()>>>(input, output, axis_,
                                                                          in_dims[0], in_dims[1]);
  } else if (in_dims.size() == 3) {
    reduce_sum_kernel<<<gridSize, blockSize, 0, get_gpu().get_stream()>>>(
        input, output, axis_, in_dims[0], in_dims[1], in_dims[2]);
  }

#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

void ReduceSumLayer::bprop() {
  CudaDeviceContext context(get_device_id());

  float* input = in_tensors_[0].get_ptr();
  float* output = out_tensors_[0].get_ptr();
  auto in_dims = in_tensors_[0].get_dimensions();

  int size = 1;
  for (auto dim : in_dims) {
    size *= dim;
  }

  dim3 blockSize(256, 1, 1);
  dim3 gridSize((size + blockSize.x - 1) / blockSize.x, 1, 1);
  if (in_dims.size() == 1) {
    reduce_sum_dgrad_kernel<<<gridSize, blockSize, 0, get_gpu().get_stream()>>>(output, input,
                                                                                axis_, in_dims[0]);
  } else if (in_dims.size() == 2) {
    reduce_sum_dgrad_kernel<<<gridSize, blockSize, 0, get_gpu().get_stream()>>>(
        output, input, axis_, in_dims[0], in_dims[1]);
  } else if (in_dims.size() == 3) {
    reduce_sum_dgrad_kernel<<<gridSize, blockSize, 0, get_gpu().get_stream()>>>(
        output, input, axis_, in_dims[0], in_dims[1], in_dims[2]);
  }

#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

}  // namespace HugeCTR
