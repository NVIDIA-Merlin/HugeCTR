#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <layers/fused_fully_connected_layer.hpp>
#include <utils.cuh>
#include <utils.hpp>

namespace HugeCTR {

namespace {

__global__ void add_bias_and_re_kernel(__half* top, __half* middle, const __half* bias, int n,
                                       int ldn) {
  const __half2 zero = TypeFunc<__half2>::zero();
  __half2* top2 = reinterpret_cast<__half2*>(top);
  __half2* middle2 = reinterpret_cast<__half2*>(middle);
  const __half2* bias2 = reinterpret_cast<const __half2*>(bias);

  int offset = blockIdx.x * ldn;
  for (int tid = threadIdx.x; tid < n; tid += blockDim.x) {
    __half2 t = __hadd2(middle2[offset + tid], __ldg(bias2 + tid));
    middle2[offset + tid] = t;
    __half2 mask = __hgt2(t, zero);
    top2[offset + tid] = __hmul2(t, mask);
  }
}

template <int BLOCK_WIDTH>
__global__ void reverse_add_bias_and_re_kernel(float* bias, __half* middle, const __half* top,
                                               int ldn) {
  __shared__ __half2 elem[32][BLOCK_WIDTH + 1];
  __shared__ __half2 accu[BLOCK_WIDTH];

  const __half2 zero = TypeFunc<__half2>::zero();

  __half2* middle2 = reinterpret_cast<__half2*>(middle);
  const __half2* top2 = reinterpret_cast<const __half2*>(top);

  int lx, ly, gi;
  int gx_offset = blockIdx.x * BLOCK_WIDTH;
  int gy_offset = blockIdx.y * 32;

  for (int i = 0; i < BLOCK_WIDTH * 32; i += blockDim.x) {
    lx = threadIdx.x % BLOCK_WIDTH;
    ly = (i + threadIdx.x) / BLOCK_WIDTH;
    gi = (ly + gy_offset) * ldn + (lx + gx_offset);

    __half2 t = middle2[gi];
    __half2 mask = __hgt2(t, zero);
    t = __hmul2(__ldg(top2 + gi), mask);

    middle2[gi] = t;
    elem[ly][lx] = t;
  }

  __syncthreads();

  for (int i = 0; i < BLOCK_WIDTH * 32; i += blockDim.x) {
    lx = (i + threadIdx.x) / 32;
    ly = threadIdx.x % 32;

    __half2 val = warpReduceSum(elem[ly][lx]);
    if (ly == 0) {
      accu[lx] = val;
    }
  }

  __syncthreads();

  if (threadIdx.x < BLOCK_WIDTH * 2) {
    __half2 val = accu[threadIdx.x / 2];
    float fval = (threadIdx.x % 2 == 0) ? __low2float(val) : __high2float(val);
    atomicAdd(bias + gx_offset * 2 + threadIdx.x, fval);
  }
}

}  // namespace

FusedFullyConnectedLayer::FusedFullyConnectedLayer(
    const std::shared_ptr<BufferBlock2<float>>& master_weights_buff,
    const std::shared_ptr<BufferBlock2<__half>>& weights_buff,
    const std::shared_ptr<BufferBlock2<__half>>& weights_grad_buff,
    const std::shared_ptr<GeneralBuffer2<CudaAllocator>>& blobs_buff,
    const Tensor2<__half>& train_bottom_tensor, const Tensor2<__half>& evaluate_bottom_tensor,
    const Tensor2<__half>& top_tensor, const std::shared_ptr<GPUResource>& gpu_resource,
    std::vector<Initializer_t> initializer_types)
    : Layer(gpu_resource, initializer_types),
      falgo_k_(CUBLAS_GEMM_DEFAULT_TENSOR_OP),
      balgo_k_(CUBLAS_GEMM_DEFAULT_TENSOR_OP),
      balgo_x_(CUBLAS_GEMM_DEFAULT_TENSOR_OP) {
  const auto& bottom_tensor_dim = train_bottom_tensor.get_dimensions();
  const auto& top_tensor_dim = top_tensor.get_dimensions();

  if (bottom_tensor_dim.size() != 2 || top_tensor_dim.size() != 2) {
    CK_THROW_(Error_t::WrongInput, "input or output tensor doesn't has two dimensions");
  }

  size_t m = bottom_tensor_dim[0];
  size_t n = top_tensor_dim[1];
  size_t k = bottom_tensor_dim[1];

  if (m % 32 != 0 || n % 64 != 0) {
    CK_THROW_(Error_t::WrongInput,
              "The first dimension of bottom tensor must be a multiple of 32, the second dimension "
              "of top tensor must be a multiple of 64.");
  }

  std::vector<size_t> kernel_dim = {k, n};
  std::vector<size_t> bias_dim = {1, n};

  {
    Tensor2<float> tensor;
    master_weights_buff->reserve(kernel_dim, &tensor);
    weights_.push_back(tensor);
  }
  {
    Tensor2<float> tensor;
    master_weights_buff->reserve(bias_dim, &tensor);
    weights_.push_back(tensor);
  }
  {
    Tensor2<__half> tensor;
    weights_buff->reserve(kernel_dim, &tensor);
    weights_half_.push_back(tensor);
  }
  {
    Tensor2<__half> tensor;
    weights_buff->reserve(bias_dim, &tensor);
    weights_half_.push_back(tensor);
  }
  {
    Tensor2<__half> tensor;
    weights_grad_buff->reserve(kernel_dim, &tensor);
    weights_grad_.push_back(tensor);
  }
  {
    Tensor2<__half> tensor;
    weights_grad_buff->reserve(bias_dim, &tensor);
    weights_grad_.push_back(tensor);
  }

  train_bottom_tensor_ = train_bottom_tensor;
  evaluate_bottom_tensor_ = evaluate_bottom_tensor;
  top_tensor_ = top_tensor;
  blobs_buff->reserve(top_tensor_.get_dimensions(), &middle_tensor_);
  blobs_buff->reserve(bias_dim, &bias_grad_tensor_);
}

void FusedFullyConnectedLayer::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());

  const __half* kernel = weights_half_[0].get_ptr();
  const __half* bias = weights_half_[1].get_ptr();
  const __half* bottom = get_bottom_tensor(is_train).get_ptr();
  __half* middle = middle_tensor_.get_ptr();
  __half* top = top_tensor_.get_ptr();

  const auto& bottom_tensor_dim = get_bottom_tensor(is_train).get_dimensions();
  const auto& top_tensor_dim = top_tensor_.get_dimensions();

  size_t m = bottom_tensor_dim[0];
  size_t n = top_tensor_dim[1];
  size_t k = bottom_tensor_dim[1];

  const float alpha = 1.0f;
  const float beta = 0.0f;

  CK_CUBLAS_THROW_(hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                                &alpha, kernel, HIP_R_16F, n, bottom, HIP_R_16F, k, &beta, middle,
                                HIP_R_16F, n, HIP_R_32F, falgo_k_));

  const size_t max_threads = 1024;
  const size_t blocks = m;
  const size_t threads = min(n / 2, max_threads);
  add_bias_and_re_kernel<<<blocks, threads, 0, get_gpu().get_stream()>>>(top, middle, bias, n / 2,
                                                                         n / 2);

#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

void FusedFullyConnectedLayer::bprop() {
  CudaDeviceContext context(get_device_id());

  const __half* kernel = weights_half_[0].get_ptr();
  const __half* top = top_tensor_.get_ptr();
  __half* kernel_grad = weights_grad_[0].get_ptr();
  __half* bias_grad = weights_grad_[1].get_ptr();
  __half* bottom = get_bottom_tensor(true).get_ptr();
  __half* middle = middle_tensor_.get_ptr();
  float* bias_grad_float = bias_grad_tensor_.get_ptr();

  const auto& bottom_tensor_dim = get_bottom_tensor(true).get_dimensions();
  const auto& top_tensor_dim = top_tensor_.get_dimensions();

  int m = bottom_tensor_dim[0];
  int n = top_tensor_dim[1];
  int k = bottom_tensor_dim[1];

  const float alpha = 1.0f;
  const float beta_k = 1.0f;
  const float beta_x = 0.0f;

  initialize_array<<<(n - 1) / 1024 + 1, 1024, 0, get_gpu().get_stream()>>>(bias_grad_float, n,
                                                                            0.0f);

  dim3 blocks(n / 64, m / 32);
  reverse_add_bias_and_re_kernel<32>
      <<<blocks, 512, 0, get_gpu().get_stream()>>>(bias_grad_float, middle, top, n / 2);

  convert_array<<<(n - 1) / 1024 + 1, 1024, 0, get_gpu().get_stream()>>>(bias_grad, bias_grad_float,
                                                                         n);

  CK_CUBLAS_THROW_(hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_T, n, k, m,
                                &alpha, middle, HIP_R_16F, n, bottom, HIP_R_16F, k, &beta_k,
                                kernel_grad, HIP_R_16F, n, HIP_R_32F, balgo_k_));

  CK_CUBLAS_THROW_(hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_N, k, m, n,
                                &alpha, kernel, HIP_R_16F, n, middle, HIP_R_16F, n, &beta_x,
                                bottom, HIP_R_16F, k, HIP_R_32F, balgo_x_));

#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

void FusedFullyConnectedLayer::search_algorithm() {
  // Set to the CUDA device where this layer assigned to
  CudaDeviceContext context(get_device_id());
  const size_t repeat_num = 100;

  // Device Tensors to be used
  __half* bottom = get_bottom_tensor(true).get_ptr();
  __half* top = top_tensor_.get_ptr();
  __half* kernel = weights_half_[0].get_ptr();
  __half* bias = weights_half_[1].get_ptr();
  __half* kernel_grad = weights_grad_[0].get_ptr();
  __half* bias_grad = weights_grad_[1].get_ptr();

  // Tensor dim
  const auto& bottom_tensor_dim = get_bottom_tensor(true).get_dimensions();
  const auto& top_tensor_dim = top_tensor_.get_dimensions();

  size_t m = bottom_tensor_dim[0];
  size_t n = top_tensor_dim[1];
  size_t k = bottom_tensor_dim[1];

  // Record time for each algorithm
  float shortestTime = std::numeric_limits<float>::max();
  float time;
  hipEvent_t start, stop;
  CK_CUDA_THROW_(hipEventCreate(&start));
  CK_CUDA_THROW_(hipEventCreate(&stop));

  // Start, end for search
  const hipblasGemmAlgo_t startAlgo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
  const hipblasGemmAlgo_t endAlgo = CUBLAS_GEMM_ALGO15_TENSOR_OP;

  // Search all the algorithm for falgo_k_
  for (int testAlgo = startAlgo; testAlgo <= endAlgo; testAlgo++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const float alpha = 1.0f;
    const float beta = 1.0f;

    // Record start event
    CK_CUDA_THROW_(hipEventRecord(start, get_gpu().get_stream()));
    for (size_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status = hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                            &alpha, kernel, HIP_R_16F, n, bottom, HIP_R_16F, k, &beta, top,
                            HIP_R_16F, n, HIP_R_32F, static_cast<hipblasGemmAlgo_t>(testAlgo));
    }
    CK_CUDA_THROW_(hipEventRecord(stop, get_gpu().get_stream()));
    CK_CUDA_THROW_(hipEventSynchronize(stop));
    CK_CUDA_THROW_(hipEventElapsedTime(&time, start, stop));
    // Avg Time(ms) for this alorithm for fprop GEMM
    time = time / repeat_num;
    // Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      //      printf("The algorithms %d is not supported for fprop, skipped.\n", testAlgo);
      continue;
    }
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      falgo_k_ = static_cast<hipblasGemmAlgo_t>(testAlgo);
    }
  }

  // Reset shortestTime
  shortestTime = std::numeric_limits<float>::max();

  // Search all the algorithm for balgo_k_
  for (int testAlgo = startAlgo; testAlgo <= endAlgo; testAlgo++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const float alpha = 1.0f;
    const float beta = 1.0f;

    // Record start event
    CK_CUDA_THROW_(hipEventRecord(start, get_gpu().get_stream()));
    for (size_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status = hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_T, n, k, m,
                            &alpha, top, HIP_R_16F, n, bottom, HIP_R_16F, k, &beta, kernel_grad,
                            HIP_R_16F, n, HIP_R_32F, static_cast<hipblasGemmAlgo_t>(testAlgo));
    }
    CK_CUDA_THROW_(hipEventRecord(stop, get_gpu().get_stream()));
    CK_CUDA_THROW_(hipEventSynchronize(stop));
    CK_CUDA_THROW_(hipEventElapsedTime(&time, start, stop));
    // Avg Time(ms) for this alorithm for fprop GEMM
    time = time / repeat_num;
    // Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      //      printf("The algorithms %d is not supported for bprop_W, skipped.\n", testAlgo);
      continue;
    }
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      balgo_k_ = static_cast<hipblasGemmAlgo_t>(testAlgo);
    }
  }

  // Reset shortestTime
  shortestTime = std::numeric_limits<float>::max();

  // Search all the algorithm for balgo_x_
  for (int testAlgo = startAlgo; testAlgo <= endAlgo; testAlgo++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Record start event
    CK_CUDA_THROW_(hipEventRecord(start, get_gpu().get_stream()));
    for (size_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status = hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_N, k, m, n,
                            &alpha, kernel, HIP_R_16F, n, top, HIP_R_16F, n, &beta, bottom,
                            HIP_R_16F, k, HIP_R_32F, static_cast<hipblasGemmAlgo_t>(testAlgo));
    }

    CK_CUDA_THROW_(hipEventRecord(stop, get_gpu().get_stream()));
    CK_CUDA_THROW_(hipEventSynchronize(stop));
    CK_CUDA_THROW_(hipEventElapsedTime(&time, start, stop));
    // Avg Time(ms) for this alorithm for fprop GEMM
    time = time / repeat_num;
    // Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      //      printf("The algorithms %d is not supported for bprop_Xn, skipped.\n", testAlgo);
      continue;
    }
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      balgo_x_ = static_cast<hipblasGemmAlgo_t>(testAlgo);
    }
  }

  // Print selection information
  // printf("The algorithm selection for falgo_k_, balgo_k_, balgo_x_ are: %d, %d and %d.\n",
  //        (int)falgo_k_ - CUBLAS_GEMM_DEFAULT_TENSOR_OP,
  //        (int)balgo_k_ - CUBLAS_GEMM_DEFAULT_TENSOR_OP,
  //        (int)balgo_x_ - CUBLAS_GEMM_DEFAULT_TENSOR_OP);

  // Output msg
  // MESSAGE_("The fully-connected layer has finished choosing the algorithm for cublas Gemm.");
  // Clean-up
  CK_CUDA_THROW_(hipEventDestroy(start));
  CK_CUDA_THROW_(hipEventDestroy(stop));
}  // namespace HugeCTR

std::unique_ptr<DataSimulator> FusedFullyConnectedLayer::get_uniform_initializer(const int index) {
  size_t bottom_dim = get_bottom_tensor(true).get_dimensions()[1];
  size_t top_dim = top_tensor_.get_dimensions()[1];

  float limit = 1.0f / ((0 == index ? bottom_dim : 0) + top_dim);
  return std::make_unique<UniformDataSimulator>(-1 * limit, limit);
}

std::unique_ptr<DataSimulator> FusedFullyConnectedLayer::get_xavier_uniform_initializer(
    const int index) {
  size_t bottom_dim = get_bottom_tensor(true).get_dimensions()[1];
  size_t top_dim = top_tensor_.get_dimensions()[1];

  return std::make_unique<VarianceScalingSimulator>(1.f, data_simu::Mode_t::Fan_avg,
                                                    data_simu::Distribution_t::Uniform,
                                                    0 == index ? bottom_dim : 0, top_dim);
}

std::unique_ptr<DataSimulator> FusedFullyConnectedLayer::get_xavier_norm_initializer(
    const int index) {
  size_t bottom_dim = get_bottom_tensor(true).get_dimensions()[1];
  size_t top_dim = top_tensor_.get_dimensions()[1];

  return std::make_unique<VarianceScalingSimulator>(1.f, data_simu::Mode_t::Fan_avg,
                                                    data_simu::Distribution_t::Norm,
                                                    0 == index ? bottom_dim : 0, top_dim);
}

std::unique_ptr<DataSimulator> FusedFullyConnectedLayer::get_default_initializer(const int index) {
  size_t bottom_dim = get_bottom_tensor(true).get_dimensions()[1];
  size_t top_dim = top_tensor_.get_dimensions()[1];

  std::unique_ptr<DataSimulator> simu(nullptr);
  if (0 == index) {
    simu.reset(new VarianceScalingSimulator(1.f, data_simu::Mode_t::Fan_avg,
                                            data_simu::Distribution_t::Norm, bottom_dim, top_dim));
  } else if (1 == index) {
    float stddev = sqrt(1.f / top_dim);
    simu.reset(new GaussianDataSimulator(0, stddev, -2 * stddev, 2 * stddev));
  } else {
    CK_THROW_(Error_t::OutOfBound, "index != {0, 1}.");
  }

  return simu;
}

}  // namespace HugeCTR
