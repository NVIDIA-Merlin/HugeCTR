#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <layers/relu_layer_half.hpp>
#include <utils.cuh>

namespace HugeCTR {

namespace {

__global__ void forward_half2_relu_kernel(__half* top, const __half* bottom, int size) {
  const __half2 zero = TypeFunc<__half2>::zero();
  __half2* top2 = reinterpret_cast<__half2*>(top);
  const __half2* bottom2 = reinterpret_cast<const __half2*>(bottom);

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
    __half2 t = __ldg(bottom2 + i);
    __half2 mask = __hgt2(t, zero);
    top2[i] = __hmul2(t, mask);
  }
}

__global__ void backward_half2_relu_kernel(__half* bottom, const __half* top, int size) {
  const __half2 zero = TypeFunc<__half2>::zero();
  __half2* bottom2 = reinterpret_cast<__half2*>(bottom);
  const __half2* top2 = reinterpret_cast<const __half2*>(top);

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
    __half2 t = bottom2[i];
    half2 mask = __hgt2(t, zero);
    bottom2[i] = __hmul2(__ldg(top2 + i), mask);
  }
}

}  // namespace

ReluLayerHalf::ReluLayerHalf(const Tensor2<__half>& bottom_tensor,
                             const Tensor2<__half>& top_tensor, int device_id)
    : Layer(device_id) {
  assert(get_size_from_dims(bottom_tensor->get_dims()) ==
         get_size_from_dims(top_tensor->get_dims()));
  assert(get_size_from_dims(bottom_tensor->get_dims()) % 2 == 0);

  bottom_tensor_ = bottom_tensor;
  top_tensor_ = top_tensor;
}

void ReluLayerHalf::fprop(bool is_train, hipStream_t stream) {
  CudaDeviceContext context(get_device_id());

  const size_t BLOCK_DIM = 1024;
  const size_t MAX_GRID_DIM = 1024;

  const size_t size = bottom_tensor_.get_num_elements() / 2;
  const size_t grid_dim = std::min((size - 1) / BLOCK_DIM + 1, MAX_GRID_DIM);
  forward_half2_relu_kernel<<<grid_dim, BLOCK_DIM, 0, stream>>>(top_tensor_->get_ptr(),
                                                                bottom_tensor_->get_ptr(), size);

#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

void ReluLayerHalf::bprop(hipStream_t stream) {
  CudaDeviceContext context(get_device_id());

  const size_t BLOCK_DIM = 1024;
  const size_t MAX_GRID_DIM = 1024;

  const size_t size = bottom_tensor_.get_num_elements() / 2;
  const size_t grid_dim = std::min((size - 1) / BLOCK_DIM + 1, MAX_GRID_DIM);
  backward_half2_relu_kernel<<<grid_dim, BLOCK_DIM, 0, stream>>>(bottom_tensor_->get_ptr(),
                                                                 top_tensor_->get_ptr(), size);

#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

}  // namespace HugeCTR
