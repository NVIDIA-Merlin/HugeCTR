#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <layers/add_layer.hpp>

#include <algorithm>
#include <functional>
#include <utils.cuh>
#include <utils.hpp>

#ifndef NDEBUG
#include <iostream>
#endif

namespace HugeCTR {

namespace {

#define BLOCK_DIM_SIZE 32
template <typename T>
__global__ void add_kernel(T** inputs, T* output, int size, int num) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < size) {
    T tmp = 0;
    for (int i = 0; i < num; i++) {
      tmp += inputs[i][tid];
    }
    output[tid] = tmp;
  }
}

template <typename T>
__global__ void add_dgrad_kernel(const T* top_grad, T** dgrads, int size, int num) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < size) {
    for (int i = 0; i < num; i++) {
      dgrads[i][tid] = top_grad[tid];
    }
  }
}

}  // end of namespace

template <typename T>
AddLayer<T>::AddLayer(const Tensors2<T>& in_tensors, const Tensor2<T>& out_tensor,
                      const std::shared_ptr<GeneralBuffer2<CudaAllocator>>& blobs_buff,
                      const std::shared_ptr<GPUResource>& gpu_resource)
    : Layer(gpu_resource) {
  try {
    size_ = in_tensors[0].get_num_elements();
    num_ = in_tensors.size();

    // error input checking
    auto dims = in_tensors[0].get_dimensions();
    if (num_ < 2) {
      CK_THROW_(Error_t::WrongInput, "AddLayer needs at least 2 input tensors");
    }
    for (size_t i = 1; i < num_; i++) {
      if (in_tensors[i].get_dimensions().size() != dims.size()) {
        CK_THROW_(Error_t::WrongInput, "All the input tensors must have the same num of dims");
      }
      for (unsigned int j = 0; j < dims.size(); j++) {
        if (in_tensors[i].get_dimensions()[j] != dims[j]) {
          CK_THROW_(Error_t::WrongInput, "All the input tensors must have the same dims");
        }
      }
    }

    for (size_t i = 0; i < num_; i++) {
      in_tensors_.push_back(in_tensors[i]);
    }
    out_tensors_.push_back(out_tensor);

    blobs_buff->reserve({num_}, &d_inputs_);

  } catch (const std::runtime_error& rt_err) {
    std::cerr << rt_err.what() << std::endl;
    throw;
  }
}

template <typename T>
void AddLayer<T>::initialize() {
  std::shared_ptr<GeneralBuffer2<CudaHostAllocator>> pinned_host_buf =
      GeneralBuffer2<CudaHostAllocator>::create();
  pinned_host_buf->reserve({num_}, &h_inputs_);
  pinned_host_buf->allocate();

  for (size_t i = 0; i < num_; i++) {
    h_inputs_.get_ptr()[i] = in_tensors_[i].get_ptr();
  }

  CK_CUDA_THROW_(hipMemcpyAsync((void*)d_inputs_.get_ptr(), (void*)h_inputs_.get_ptr(),
                                 num_ * sizeof(T*), hipMemcpyHostToDevice,
                                 get_gpu().get_stream()));
}

template <typename T>
void AddLayer<T>::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());

  T* output = out_tensors_[0].get_ptr();

  dim3 block_size(256, 1, 1);
  dim3 grid_size((size_ + block_size.x - 1) / block_size.x, 1, 1);
  add_kernel<<<grid_size, block_size, 0, get_gpu().get_stream()>>>(d_inputs_.get_ptr(), output,
                                                                   size_, num_);
}

template <typename T>
void AddLayer<T>::bprop() {
  CudaDeviceContext context(get_device_id());

  T* output = out_tensors_[0].get_ptr();

  dim3 blockSize(256, 1, 1);
  dim3 gridSize((size_ + blockSize.x - 1) / blockSize.x, 1, 1);
  add_dgrad_kernel<<<gridSize, blockSize, 0, get_gpu().get_stream()>>>(output, d_inputs_.get_ptr(),
                                                                       size_, num_);
}

template class AddLayer<float>;
template class AddLayer<__half>;

}  // namespace HugeCTR
