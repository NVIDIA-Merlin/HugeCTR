#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utils.cuh>

namespace HugeCTR {

void conv_weight_gpu(size_t grid, size_t block, __half* dst, float* src, int elems,
                     hipStream_t stream) {
  convert_array<<<grid, block, 0, stream>>>(dst, src, elems);
}

}  // end namespace HugeCTR
