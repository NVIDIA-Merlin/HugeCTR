/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <data_simulator.hpp>
#include <diagnose.hpp>
#include <random>
#include <utils.cuh>

namespace HugeCTR {

template <>
void UniformGenerator::fill<float>(Tensor2<float>& tensor, float a, float b,
                                   const GPUResource& gpu) {
  if (a >= b) {
    CK_THROW_(Error_t::WrongInput, "a must be smaller than b");
  }

  CK_CURAND_THROW_(hiprandGenerateUniform(gpu.get_curand_generator(), tensor.get_ptr(),
                                         tensor.get_num_elements()));

  auto op = [a, b] __device__(float val) { return val * (b - a) + a; };
  transform_array<<<gpu.get_sm_count() * 2, 1024, 0, gpu.get_stream()>>>(
      tensor.get_ptr(), tensor.get_ptr(), tensor.get_num_elements(), op);
}

template <>
void HostUniformGenerator::fill<float>(Tensor2<float>& tensor, float a, float b,
                                       const hiprandGenerator_t& gen) {
  if (a >= b) {
    CK_THROW_(Error_t::WrongInput, "a must be smaller than b");
  }
  CK_CURAND_THROW_(hiprandGenerateUniform(gen, tensor.get_ptr(),
                                         tensor.get_num_elements() % 2 != 0
                                             ? tensor.get_num_elements() + 1
                                             : tensor.get_num_elements()));
  float* p = tensor.get_ptr();
  for (size_t i = 0; i < tensor.get_num_elements(); i++) {
    p[i] = p[i] * (b - a) + a;
  }
}

template <>
void NormalGenerator::fill<float>(Tensor2<float>& tensor, float mean, float stddev,
                                  const GPUResource& gpu) {
  CK_CURAND_THROW_(hiprandGenerateNormal(gpu.get_curand_generator(), tensor.get_ptr(),
                                        tensor.get_num_elements(), mean, stddev));
}

template <>
void HostNormalGenerator::fill<float>(Tensor2<float>& tensor, float mean, float stddev,
                                      const hiprandGenerator_t& gen) {
  CK_CURAND_THROW_(hiprandGenerateNormal(gen, tensor.get_ptr(),
                                        tensor.get_num_elements() % 2 != 0
                                            ? tensor.get_num_elements() + 1
                                            : tensor.get_num_elements(),
                                        mean, stddev));
}

void ConstantDataSimulator::fill(Tensor2<float>& tensor, const hiprandGenerator_t& gen) {
  float* p = tensor.get_ptr();
  for (size_t i = 0; i < tensor.get_num_elements(); i++) {
    p[i] = value_;
  }
}

void UniformDataSimulator::fill(Tensor2<float>& tensor, const hiprandGenerator_t& gen) {
  HostUniformGenerator::fill(tensor, min_, max_, gen);
}

void GaussianDataSimulator::fill(Tensor2<float>& tensor, const hiprandGenerator_t& gen) {
  HostNormalGenerator::fill(tensor, mu_, sigma_, gen);
}
}  // namespace HugeCTR