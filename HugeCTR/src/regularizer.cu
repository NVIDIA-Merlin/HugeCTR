#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <regularizer.hpp>
#include <utility>

#ifndef NDEBUG
#include <iostream>
#endif

namespace HugeCTR {

template <typename T>
Regularizer<T>::Regularizer(const Tensor2<float>& weight_buff, const Tensor2<T>& wgrad_buff,
                            const int batch_size, const std::shared_ptr<GPUResource>& gpu_resource)
    : weight_buff_(weight_buff),
      wgrad_buff_(wgrad_buff),
      batch_size_(batch_size),
      gpu_resource_(gpu_resource) {}

template <typename T>
void Regularizer<T>::compute_rterm() {
  CudaDeviceContext context(get_device_id());

  const float* weight = weight_buff_.get_ptr();
  do_compute_rterm(weight, &h_rterm_, weight_buff_.get_num_elements());

#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

template <typename T>
void Regularizer<T>::initialize_wgrad() {
  CudaDeviceContext context(get_device_id());

  const float* weight = weight_buff_.get_ptr();
  T* wgrad = wgrad_buff_.get_ptr();
  do_initialize_wgrad(weight, wgrad, weight_buff_.get_num_elements());

#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

template class Regularizer<float>;
template class Regularizer<__half>;
}  // namespace HugeCTR
