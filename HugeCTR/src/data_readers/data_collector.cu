#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <nvtx3/nvToolsExt.h>

#include <common.hpp>
#include <core23/tensor_operations.hpp>
#include <data_readers/data_collector.hpp>
namespace HugeCTR {

template <typename TypeComp>
__global__ void split_kernel__(int batchsize, float* label_ptr, int label_dim, TypeComp* dense_ptr,
                               int dense_dim, const float* label_dense, int label_dense_dim) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < batchsize * label_dense_dim) {
    const int in_col = idx % label_dense_dim;
    const int in_row = idx / label_dense_dim;
    const int out_row = in_row;
    if (in_col < label_dim) {
      const int out_col = in_col;
      label_ptr[out_row * label_dim + out_col] = label_dense[idx];
    } else {
      const int out_col = in_col - label_dim;
      dense_ptr[out_row * dense_dim + out_col] = label_dense[idx];
    }
  }
  return;
}

template <typename TypeComp>
void split(core23::Tensor& label_tensor, core23::Tensor& dense_tensor,
           const core23::Tensor& label_dense_buffer, const int label_dense_dim,
           hipStream_t stream) {
  const int batchsize = label_tensor.shape()[0];
  const int label_dim = label_tensor.shape()[1];
  const int dense_dim = dense_tensor.shape()[1];

  const int BLOCK_DIM = 256;
  const int GRID_DIM = (label_dense_buffer.num_elements() - 1) / BLOCK_DIM + 1;
  assert(dense_dim >= 0 || "dense_dim should be >= 0");

  if (dense_dim > 0) {
    split_kernel__<<<GRID_DIM, BLOCK_DIM, 0, stream>>>(
        batchsize, label_tensor.data<float>(), label_dim, dense_tensor.data<TypeComp>(), dense_dim,
        label_dense_buffer.data<float>(), label_dense_dim);
  } else if (dense_dim == 0) {
    split_kernel__<<<GRID_DIM, BLOCK_DIM, 0, stream>>>(
        batchsize, label_tensor.data<float>(), label_dim, (TypeComp*)0, 0,
        label_dense_buffer.data<float>(), label_dense_dim);

  } else {
    HCTR_OWN_THROW(Error_t::WrongInput, "dense_dim < 0");
  }

  return;
};
// broadcast, called by bg thread
// threadbuffer broadcast to broadbuffers
template <typename T>
void broadcast(const std::shared_ptr<ThreadBuffer23>& thread_buffer,
               std::shared_ptr<BroadcastBuffer23>& broadcast_buffer,
               std::vector<size_t>& last_batch_nnz_,
               const std::shared_ptr<ResourceManager>& resource_manager) {
  nvtxRangePushA("collector_broadcast");
  int param_num = thread_buffer->param_num;
  int dense_dim = thread_buffer->dense_dim;
  int label_dim = thread_buffer->label_dim;
  int batch_size = thread_buffer->batch_size;
  int batch_size_per_gpu = batch_size / resource_manager->get_global_gpu_count();
  int local_gpu_count = resource_manager->get_local_gpu_count();

#pragma omp parallel for num_threads(local_gpu_count)
  for (int i = 0; i < local_gpu_count; ++i) {
    auto local_gpu = resource_manager->get_local_gpu(i);
    auto gpu_id = local_gpu->get_device_id();
    CudaDeviceContext ctx(gpu_id);

    for (int param_id = 0; param_id < param_num; ++param_id) {
      auto src_sparse_tensor = thread_buffer->device_sparse_buffers[param_id];
      auto dst_sparse_tensor = broadcast_buffer->sparse_buffers[i * param_num + param_id];
      *dst_sparse_tensor.get_nnz_ptr() = src_sparse_tensor.nnz();

      if (thread_buffer->is_fixed_length[param_id] &&
          last_batch_nnz_[i * param_num + param_id] ==
              static_cast<size_t>(src_sparse_tensor.nnz())) {
        HCTR_LIB_THROW(hipMemcpyAsync(dst_sparse_tensor.get_value_ptr(),
                                       src_sparse_tensor.get_value_ptr(),
                                       src_sparse_tensor.nnz() * sizeof(T),
                                       hipMemcpyDeviceToDevice, local_gpu->get_p2p_stream()));
      } else {
        HCTR_LIB_THROW(hipMemcpyAsync(dst_sparse_tensor.get_value_ptr(),
                                       src_sparse_tensor.get_value_ptr(),
                                       src_sparse_tensor.nnz() * sizeof(T),
                                       hipMemcpyDeviceToDevice, local_gpu->get_p2p_stream()));
        auto src_tensor23 = src_sparse_tensor.get_rowoffset_tensor();
        // TODO remove this conversion after changing output of data reader.
        HCTR_LIB_THROW(hipMemcpyAsync(dst_sparse_tensor.get_rowoffset_ptr(),
                                       src_sparse_tensor.get_rowoffset_ptr(),
                                       src_sparse_tensor.get_rowoffset_tensor().num_bytes(),
                                       hipMemcpyDeviceToDevice, local_gpu->get_p2p_stream()));
        last_batch_nnz_[i * param_num + param_id] = src_sparse_tensor.nnz();
      }
    }
    auto dst_dense_tensor = broadcast_buffer->dense_tensors[i];
    auto src_dense_tensor = thread_buffer->device_dense_buffers;
    HCTR_LIB_THROW(hipMemcpyAsync(
        dst_dense_tensor.data<float>(),
        src_dense_tensor.data<float>() + i * batch_size_per_gpu * (label_dim + dense_dim),
        batch_size_per_gpu * (label_dim + dense_dim) * sizeof(float), hipMemcpyDeviceToDevice,
        local_gpu->get_p2p_stream()));
    HCTR_LIB_THROW(hipStreamSynchronize(local_gpu->get_p2p_stream()));
  }
  nvtxRangePop();
}

template void broadcast<unsigned int>(const std::shared_ptr<ThreadBuffer23>& thread_buffer,
                                      std::shared_ptr<BroadcastBuffer23>& broadcast_buffer,
                                      std::vector<size_t>& last_batch_nnz_,
                                      const std::shared_ptr<ResourceManager>& resource_manager);
template void broadcast<long long>(const std::shared_ptr<ThreadBuffer23>& thread_buffer,
                                   std::shared_ptr<BroadcastBuffer23>& broadcast_buffer,
                                   std::vector<size_t>& last_batch_nnz_,
                                   const std::shared_ptr<ResourceManager>& resource_manager);

template void split<float>(core23::Tensor& label_tensor, core23::Tensor& dense_tensor,
                           const core23::Tensor& label_dense_buffer, const int label_dense_dim,
                           hipStream_t stream);

template void split<__half>(core23::Tensor& label_tensor, core23::Tensor& dense_tensor,
                            const core23::Tensor& label_dense_buffer, const int label_dense_dim,
                            hipStream_t stream);

}  // namespace HugeCTR
