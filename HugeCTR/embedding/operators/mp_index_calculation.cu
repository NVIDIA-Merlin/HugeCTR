#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <hipcub/hipcub.hpp>

#include "HugeCTR/include/utils.cuh"
#include "generic_lookup.cuh"
#include "mp_index_calculation.hpp"

namespace embedding {

namespace {

template <typename key_t, typename offset_t>
__global__ void index_calculation_kernel(const key_t* key, const offset_t* bucket_range,
                                         const int* local_embedding_list, int shard_id,
                                         int shards_count, int batch_size, int num_local_embedding,
                                         uint32_t* model_idx_offsets, char* flag) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < batch_size * num_local_embedding) {
    int batch_id = tid % batch_size;
    int embedding_id = local_embedding_list[tid / batch_size];

    uint32_t bucket_start =
        static_cast<uint32_t>(bucket_range[batch_size * embedding_id + batch_id]);
    uint32_t bucket_end =
        static_cast<uint32_t>(bucket_range[batch_size * embedding_id + batch_id + 1]);
    uint32_t flag_cnt = 0;
    for (uint32_t idx = 0; idx < (bucket_end - bucket_start); ++idx) {
      key_t k = key[idx + bucket_start];
      if (k % shards_count == shard_id) {
        flag[idx + bucket_start] = 1;
        flag_cnt += 1;
      }
    }
    model_idx_offsets[1 + tid] = flag_cnt;
    if (tid == 0) {
      model_idx_offsets[0] = 0;
    }
  }
}

__global__ void expand_bucket_id_kernel(const uint32_t* model_offset, uint32_t* bucket_idx,
                                        int batch_size, int num_local_embedding,
                                        int batch_size_per_gpu) {
  for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < batch_size * num_local_embedding;
       idx += blockDim.x * gridDim.x) {
    int batch_id = idx % batch_size;
    int local_embedding_id = idx / batch_size;
    int gpu_id = batch_id / batch_size_per_gpu;
    int local_batch_id = batch_id % batch_size_per_gpu;

    uint32_t start = model_offset[idx];
    uint32_t end = model_offset[idx + 1];
    for (int i = start; i < end; ++i) {
      bucket_idx[i] = idx;
    }
  }
}

template <typename key_t>
constexpr key_t empty_key = std::numeric_limits<key_t>::max();

template <typename key_t>
class Hash {
 public:
  __forceinline__ __device__ Hash() {}
  __forceinline__ __device__ uint32_t operator()(key_t key) { return static_cast<uint32_t>(key); }
};

template <typename key_t>
__global__ void initialize_hash_key(key_t* hash_key, int num_hash_key) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < num_hash_key) {
    hash_key[idx] = empty_key<key_t>;
  }
}

__forceinline__ __device__ int32_t _atomicCAS(int32_t* address, int32_t compare, int32_t val) {
  return (int32_t)atomicCAS((int*)address, (int)compare, (int)val);
}

__forceinline__ __device__ uint32_t _atomicCAS(uint32_t* address, uint32_t compare, uint32_t val) {
  return (uint32_t)atomicCAS((unsigned int*)address, (unsigned int)compare, (unsigned int)val);
}

__forceinline__ __device__ int64_t _atomicCAS(int64_t* address, int64_t compare, int64_t val) {
  return (int64_t)atomicCAS((unsigned long long*)address, (unsigned long long)compare,
                            (unsigned long long)val);
}

__forceinline__ __device__ uint64_t _atomicCAS(uint64_t* address, uint64_t compare, uint64_t val) {
  return (uint64_t)atomicCAS((unsigned long long*)address, (unsigned long long)compare,
                             (unsigned long long)val);
}

template <typename key_t, typename hasher_t = Hash<key_t>>
__global__ void get_unique_index_kernel(const key_t* key_list, size_t num_key,
                                        const uint32_t* id_space_offset, const int* id_space_list,
                                        size_t num_id_space, const int* unique_id_space_list,
                                        size_t num_unique_id_space, const uint32_t* hash_offset,
                                        key_t* hash_key_list, uint32_t* local_index) {
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;

  hasher_t hasher;

  if (idx < num_key) {
    uint32_t idx_id_space = binary_search_index_lower_bound(id_space_offset, num_id_space + 1, idx);
    int id_space = id_space_list[idx_id_space];
    int idx_unique_id_space =
        binary_search_index_lower_bound(unique_id_space_list, num_unique_id_space, id_space);

    uint32_t start = hash_offset[idx_unique_id_space];
    uint32_t end = hash_offset[idx_unique_id_space + 1];
    key_t target_key = key_list[idx];

    uint32_t capacity = end - start;
    uint32_t hash_index = hasher(target_key) % capacity;
    while (true) {
      const key_t old_key =
          _atomicCAS(hash_key_list + start + hash_index, (key_t)empty_key<key_t>, target_key);

      if ((empty_key<key_t> == old_key) || (target_key == old_key)) {
        local_index[idx] = start + hash_index;
        break;
      }
      hash_index = (hash_index + 1) % capacity;
    }
  }
}

template <typename key_t>
__global__ void extract_unique_key_and_dst_offset_kernel(
    const key_t* hash_keys, const uint32_t* hash_offset, const uint32_t* unique_local_index,
    const size_t* num_unique_key, const int* unique_id_space_list, int num_unique_id_space_list,
    const int* id_space_list, int num_id_space_list, const int* unique_id_space_ev_size_list,
    key_t* unique_key, uint32_t* unique_dst_idx) {
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < *num_unique_key) {
    uint32_t local_index = unique_local_index[idx];
    unique_key[idx] = hash_keys[local_index];

    uint32_t idx_id_space =
        binary_search_index_lower_bound(hash_offset, num_unique_id_space_list + 1, local_index);

    unique_dst_idx[1 + idx] = unique_id_space_ev_size_list[idx_id_space];
  }

  if (idx == 0) {
    unique_dst_idx[0] = 0;
  }
}

template <typename key_t, int kWarpPerBlock = 1, int kWarpSize = 32>
__global__ void count_unique_key_kernel(const key_t *hash_keys, const uint32_t *hash_offset, 
                                        int num_unique_id_space, uint32_t *unique_key_count) {
  int warp_id = 0;
  int lane_id = threadIdx.x;
  int block_id = blockIdx.x;

  int count = 0;
  if (block_id < num_unique_id_space) {
    int start = hash_offset[block_id];
    int end = hash_offset[block_id + 1];
    for (int i = 0; i * kWarpSize + lane_id < (end - start); ++i) {
      count += (hash_keys[start + i * kWarpSize + lane_id] == empty_key<key_t>) ? 0 : 1;
    }
  }

  typedef hipcub::WarpReduce<int> WarpReduce;
  __shared__ typename WarpReduce::TempStorage temp_storage[kWarpPerBlock];
  int aggregate = WarpReduce(temp_storage[warp_id]).Sum(count);

  if (lane_id == 0) {
    unique_key_count[block_id + 1] = aggregate;
    if (block_id == 0) {
      unique_key_count[0] = 0;
    }
  }
}

template <typename key_t, int kWarpPerBlock, int kWarpSize = 32>
__global__ void scan_id_space_offset(const key_t* hash_keys, const uint32_t* hash_offset,
                                     int num_unique_id_space, uint32_t* unique_id_space_offset,
                                     uint32_t* temp_id_space_value) {
  int warp_id = threadIdx.y;
  int lane_id = threadIdx.x;

  int count = 0;
  if (warp_id < num_unique_id_space) {
    int start = hash_offset[warp_id];
    int end = hash_offset[warp_id + 1];
    for (int i = 0; i * kWarpSize + lane_id < (end - start); ++i) {
      count += (hash_keys[start + i * kWarpSize + lane_id] == empty_key<key_t>) ? 0 : 1;
    }
  }

  typedef hipcub::WarpReduce<int> WarpReduce;
  __shared__ typename WarpReduce::TempStorage temp_storage[kWarpPerBlock];
  int aggregate = WarpReduce(temp_storage[warp_id]).Sum(count);

  __shared__ int s_id_space_offset[kWarpPerBlock];
  if (lane_id == 0) {
    s_id_space_offset[warp_id] = aggregate;
  }
  __syncthreads();

  if (threadIdx.x + threadIdx.y * blockDim.x == 0) {
    uint32_t prefix_sum = 0;
    for (int i = 0; i < num_unique_id_space + 1; ++i) {
      unique_id_space_offset[i] = prefix_sum;

      prefix_sum += static_cast<uint32_t>(s_id_space_offset[i]);
    }
  }
}
}  // namespace

ModelIndexCalculation::ModelIndexCalculation(std::shared_ptr<CoreResourceManager> core,
                                             int num_local_embedding,
                                             const std::vector<int>& local_hotness_list,
                                             const std::vector<int>& hotness_list,
                                             int universal_batch_size, DataType key_type)
    : core_(core),
      num_local_embedding_(num_local_embedding),
      local_hotness_sum_(0),
      hotness_list_sum_(0),
      universal_batch_size_(universal_batch_size) {
  for (size_t idx = 0; idx < local_hotness_list.size(); ++idx) {
    local_hotness_sum_ += local_hotness_list[idx];
  }
  for (size_t idx = 0; idx < hotness_list.size(); ++idx) {
    hotness_list_sum_ += hotness_list[idx];
  }
  CudaDeviceContext ctx(core_->get_device_id());
  Device device{DeviceType::GPU, core->get_device_id()};

  auto buffer_ptr_ = GetBuffer(core);
  model_key_ = buffer_ptr_->reserve({universal_batch_size_ * local_hotness_sum_}, device, key_type);
  model_idx_offsets_ = buffer_ptr_->reserve({universal_batch_size_ * num_local_embedding_ + 1},
                                            device, TensorScalarType::UInt32);
  num_key_in_bucket_for_combiner_ = buffer_ptr_->reserve(
      {universal_batch_size_ * num_local_embedding_}, device, TensorScalarType::UInt32);
  num_model_key_ = buffer_ptr_->reserve({1}, DeviceType::CPU, TensorScalarType::Size_t);
  flag_ = buffer_ptr_->reserve({universal_batch_size_ * hotness_list_sum_}, device,
                               TensorScalarType::Char);
  {
    size_t temp_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(nullptr, temp_bytes, (uint32_t*)nullptr, (uint32_t*)nullptr,
                                  universal_batch_size_ * num_local_embedding_ + 1);
    d_temp_scan_storage_ = buffer_ptr_->reserve({temp_bytes}, device, TensorScalarType::Void);
  }
  {
    size_t temp_bytes = 0;
    DISPATCH_INTEGRAL_FUNCTION(key_type.type(), key_t, [&] {
      hipcub::DeviceSelect::Flagged(nullptr, temp_bytes, (key_t*)nullptr, (char*)nullptr,
                                 (key_t*)nullptr, (size_t*)nullptr,
                                 universal_batch_size * hotness_list_sum_);
    });
    d_temp_select_storage_ = buffer_ptr_->reserve({temp_bytes}, device, TensorScalarType::Void);
  }
  buffer_ptr_->allocate();
}

void ModelIndexCalculation::compute(const Tensor& key, const Tensor& bucket_range, size_t num_key,
                                    const Tensor& d_local_embedding_list, int shard_id,
                                    int shards_count, int batch_size, Tensor* model_key,
                                    Tensor* model_idx_offsets, size_t* num_model_key) {
  CudaDeviceContext ctx(core_->get_device_id());

  if (num_local_embedding_ > 0) {
    DISPATCH_INTEGRAL_FUNCTION(key.dtype().type(), key_t, [&] {
      DISPATCH_INTEGRAL_FUNCTION(bucket_range.dtype().type(), offset_t, [&] {
        auto stream = core_->get_local_gpu()->get_stream();

        HCTR_LIB_THROW(hipMemsetAsync(model_key_.get(), 0, model_key_.nbytes(), stream));
        HCTR_LIB_THROW(
            hipMemsetAsync(model_idx_offsets_.get(), 0, model_idx_offsets_.nbytes(), stream));
        HCTR_LIB_THROW(hipMemsetAsync(flag_.get(), 0, flag_.nbytes(), stream));

        key_t* model_key_ptr = model_key_.get<key_t>();
        uint32_t* model_idx_offsets_ptr = model_idx_offsets_.get<uint32_t>();
        size_t* num_model_key_ptr = num_model_key_.get<size_t>();
        char* flag_ptr = flag_.get<char>();
        const key_t* key_ptr = key.get<key_t>();
        const offset_t* bucket_range_ptr = bucket_range.get<offset_t>();
        const int* local_embedding_list_ptr = d_local_embedding_list.get<int>();

        // in cub implementation, the flag must be 0 or 1. See
        // https://github.com/NVIDIA/cub/issues/235 we can fuse thie memset with next kernel
        int thread_cnt = 128;
        int block_cnt = (batch_size * num_local_embedding_ - 1) / thread_cnt + 1;
        index_calculation_kernel<<<block_cnt, thread_cnt, 0, stream>>>(
            key_ptr, bucket_range_ptr, local_embedding_list_ptr, shard_id, shards_count,
            batch_size, num_local_embedding_, model_idx_offsets_ptr, flag_ptr);

        size_t d_temp_scan_storage_nbytes = d_temp_scan_storage_.nbytes();
        hipcub::DeviceScan::InclusiveSum(d_temp_scan_storage_.get(), d_temp_scan_storage_nbytes,
                                      model_idx_offsets_ptr, model_idx_offsets_ptr,
                                      batch_size * num_local_embedding_ + 1, stream);

        size_t d_temp_select_storage_nbytes = d_temp_select_storage_.nbytes();
        hipcub::DeviceSelect::Flagged(d_temp_select_storage_.get(), d_temp_select_storage_nbytes,
                                   key_ptr, flag_ptr, model_key_ptr, num_model_key_ptr, num_key,
                                   stream);
        HCTR_LIB_THROW(hipStreamSynchronize(stream));
      });
    });
  }
  *model_key = model_key_;
  *model_idx_offsets = model_idx_offsets_;
  *num_model_key = *(num_model_key_.get<size_t>());
}

ModelBackwardIndexCalculation::ModelBackwardIndexCalculation(
    std::shared_ptr<CoreResourceManager> core, int num_gpus, int num_local_embedding,
    const std::vector<int>& h_local_hotness_list, const std::vector<int>& h_local_id_space_list,
    const std::vector<int>& h_local_ev_size_list, int universal_batch_size, DataType key_type)
    : core_(core), num_gpus_(num_gpus), num_local_embedding_(num_local_embedding) {
  CudaDeviceContext ctx(core_->get_device_id());
  Device device{DeviceType::GPU};

  int local_hotness_sum =
      std::accumulate(h_local_hotness_list.begin(), h_local_hotness_list.end(), 0);
  sort_end_bit_ =
      static_cast<int>(log2(static_cast<float>(universal_batch_size * local_hotness_sum))) + 1;

  std::vector<int> h_unique_id_space_list;
  std::vector<int> h_unique_id_space_ev_size_list;
  for (size_t i = 0; i < h_local_id_space_list.size(); ++i) {
    if (h_unique_id_space_list.size() == 0) {
      h_unique_id_space_list.push_back(h_local_id_space_list[i]);
      h_unique_id_space_ev_size_list.push_back(h_local_ev_size_list[i]);
      continue;
    }
    if (h_local_id_space_list[i] > h_unique_id_space_list.back()) {
      h_unique_id_space_list.push_back(h_local_id_space_list[i]);
      h_unique_id_space_ev_size_list.push_back(h_local_ev_size_list[i]);
    }
  }

  auto buffer_ptr = GetBuffer(core);
  bucket_id_list_ = buffer_ptr->reserve(universal_batch_size * local_hotness_sum, DeviceType::GPU,
                                        TensorScalarType::UInt32);
  hash_keys_ = buffer_ptr->reserve({universal_batch_size, local_hotness_sum}, device, key_type);
  hash_offset_ =
      buffer_ptr->reserve({1 + h_unique_id_space_list.size()}, device, TensorScalarType::UInt32);
  local_index_ = buffer_ptr->reserve({universal_batch_size, local_hotness_sum}, device,
                                     TensorScalarType::UInt32);
  sorted_local_index_ = buffer_ptr->reserve({universal_batch_size, local_hotness_sum}, device,
                                            TensorScalarType::UInt32);
  unique_local_index_ = buffer_ptr->reserve({universal_batch_size, local_hotness_sum}, device,
                                            TensorScalarType::UInt32);

  unique_key_ = buffer_ptr->reserve({universal_batch_size, local_hotness_sum}, device, key_type);
  num_unique_key_ = buffer_ptr->reserve({1}, DeviceType::CPU, TensorScalarType::Size_t);
  unique_dst_idx_ = buffer_ptr->reserve({1 + universal_batch_size * local_hotness_sum}, device,
                                        TensorScalarType::UInt32);
  sorted_bucket_id_list_ = buffer_ptr->reserve({universal_batch_size, local_hotness_sum}, device,
                                               TensorScalarType::UInt32);
  sorted_bucket_id_offset_ = buffer_ptr->reserve({1 + universal_batch_size * local_hotness_sum},
                                                 device, TensorScalarType::UInt32);
  unique_id_space_offset_ =
      buffer_ptr->reserve({1 + h_unique_id_space_list.size()}, device, TensorScalarType::UInt32);
  unique_id_space_list_ = buffer_ptr->reserve({h_unique_id_space_list.size()}, DeviceType::GPU,
                                              TensorScalarType::Int32);
  unique_id_space_ev_size_list_ = buffer_ptr->reserve({h_unique_id_space_ev_size_list.size()},
                                                      DeviceType::GPU, TensorScalarType::Int32);
  {
    size_t temp_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs(nullptr, temp_bytes, (uint32_t*)nullptr, (uint32_t*)nullptr,
                                    (uint32_t*)nullptr, (uint32_t*)nullptr,
                                    universal_batch_size * local_hotness_sum, 0, sort_end_bit_);
    d_temp_sort_storage_ = buffer_ptr->reserve({temp_bytes}, device, TensorScalarType::Void);
  }
  {
    size_t temp_bytes = 0;
    hipcub::DeviceRunLengthEncode::Encode(nullptr, temp_bytes, (uint32_t*)nullptr, (uint32_t*)nullptr,
                                       (uint32_t*)nullptr, (size_t*)nullptr,
                                       universal_batch_size * local_hotness_sum);
    d_temp_run_length_encode_storage_ =
        buffer_ptr->reserve({temp_bytes}, device, TensorScalarType::Void);
  }
  {
    size_t temp_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(nullptr, temp_bytes, (uint32_t*)nullptr, (uint32_t*)nullptr,
                                  std::max(static_cast<int64_t>(universal_batch_size * local_hotness_sum), unique_id_space_offset_.get_num_elements()));
    d_temp_scan_encode_storage_ = buffer_ptr->reserve({temp_bytes}, device, TensorScalarType::Void);
  }
  buffer_ptr->allocate();
  unique_id_space_list_.copy_from(h_unique_id_space_list);
  unique_id_space_ev_size_list_.copy_from(h_unique_id_space_ev_size_list);

  std::vector<uint32_t> h_hash_offset(1 + h_unique_id_space_list.size(), 0);
  for (int i = 0; i < num_local_embedding; ++i) {
    int id_space = h_local_id_space_list[i];
    auto iter = find(h_unique_id_space_list.begin(), h_unique_id_space_list.end(), id_space);
    HCTR_CHECK_HINT(iter != h_unique_id_space_list.end(),
                    "can not find id space in unique id space");
    int idx = std::distance(h_unique_id_space_list.begin(), iter);
    h_hash_offset[1 + idx] += universal_batch_size * h_local_hotness_list[i];
  }
  std::partial_sum(h_hash_offset.begin(), h_hash_offset.end(), h_hash_offset.begin());
  hash_offset_.copy_from(h_hash_offset);
}

void ModelBackwardIndexCalculation::compute(
    const Tensor& model_key, size_t num_model_key, const Tensor& model_offset,
    const Tensor& id_space_offset, const Tensor& id_space_list, int batch_size, Tensor* unique_key,
    size_t* num_unique_key, Tensor* unique_dst_idx, Tensor* sorted_bucket_id_list,
    Tensor* sorted_bucket_id_offset, Tensor* unique_id_space_list, Tensor* unique_id_space_offset) {
  CudaDeviceContext ctx(core_->get_device_id());
  int batch_size_per_gpu = batch_size / num_gpus_;

  DISPATCH_INTEGRAL_FUNCTION(model_key.dtype().type(), key_t, [&] {
    auto stream = core_->get_local_gpu()->get_stream();

    HCTR_LIB_THROW(
        hipMemsetAsync(bucket_id_list_.get<uint32_t>(), 0, bucket_id_list_.nbytes(), stream));
    HCTR_LIB_THROW(hipMemsetAsync(sorted_local_index_.get<uint32_t>(), 0,
                                   sorted_local_index_.nbytes(), stream));
    HCTR_LIB_THROW(hipMemsetAsync(unique_local_index_.get<uint32_t>(), 0,
                                   unique_local_index_.nbytes(), stream));
    HCTR_LIB_THROW(hipMemsetAsync(unique_key_.get<key_t>(), 0, unique_key_.nbytes(), stream));
    HCTR_LIB_THROW(
        hipMemsetAsync(unique_dst_idx_.get<uint32_t>(), 0, unique_dst_idx_.nbytes(), stream));
    HCTR_LIB_THROW(hipMemsetAsync(sorted_bucket_id_list_.get<uint32_t>(), 0,
                                   sorted_bucket_id_list_.nbytes(), stream));
    HCTR_LIB_THROW(hipMemsetAsync(sorted_bucket_id_offset_.get<uint32_t>(), 0,
                                   sorted_bucket_id_offset_.nbytes(), stream));
    if (num_local_embedding_ > 0 && num_model_key > 0ul) {
      {
        // this can be fused with sort pair in 4th code
        int block_size = 256;
        int grid_size = (batch_size * num_local_embedding_ - 1) / block_size + 1;
        expand_bucket_id_kernel<<<grid_size, block_size, 0, stream>>>(
            model_offset.get<uint32_t>(), bucket_id_list_.get<uint32_t>(), batch_size,
            num_local_embedding_, batch_size_per_gpu);
      }
      {
        int num_hash_key = hash_keys_.get_num_elements();
        constexpr int block_size = 256;
        int grid_size = (num_hash_key - 1) / block_size + 1;
        initialize_hash_key<<<grid_size, block_size, 0, stream>>>(hash_keys_.get<key_t>(),
                                                                  num_hash_key);
      }
      {
        constexpr int block_size = 256;
        int grid_size = (num_model_key - 1) / block_size + 1;
        get_unique_index_kernel<<<grid_size, block_size, 0, stream>>>(
            model_key.get<key_t>(), num_model_key, id_space_offset.get<uint32_t>(),
            id_space_list.get<int>(), num_local_embedding_, unique_id_space_list_.get<int>(),
            unique_id_space_list_.get_num_elements(), hash_offset_.get<uint32_t>(),
            hash_keys_.get<key_t>(), local_index_.get<uint32_t>());
      }

      {
        size_t nbytes = d_temp_sort_storage_.nbytes();
        hipcub::DeviceRadixSort::SortPairs(
            d_temp_sort_storage_.get(), nbytes, local_index_.get<uint32_t>(),
            sorted_local_index_.get<uint32_t>(), bucket_id_list_.get<uint32_t>(),
            sorted_bucket_id_list_.get<uint32_t>(), num_model_key, 0, sort_end_bit_, stream);
      }
      {
        size_t nbytes = d_temp_run_length_encode_storage_.nbytes();
        hipcub::DeviceRunLengthEncode::Encode(
            d_temp_run_length_encode_storage_.get(), nbytes, sorted_local_index_.get<uint32_t>(),
            unique_local_index_.get<uint32_t>(), sorted_bucket_id_offset_.get<uint32_t>() + 1,
            num_unique_key_.get<size_t>(), num_model_key, stream);
      }
      {
        constexpr int block_size = 256;
        int grid_size = (hash_keys_.get_num_elements() - 1) / block_size + 1;
        extract_unique_key_and_dst_offset_kernel<<<grid_size, block_size, 0, stream>>>(
            hash_keys_.get<key_t>(), hash_offset_.get<uint32_t>(),
            unique_local_index_.get<uint32_t>(), num_unique_key_.get<size_t>(),
            unique_id_space_list_.get<int>(), unique_id_space_list_.get_num_elements(),
            id_space_list.get<int>(), id_space_list.get_num_elements(),
            unique_id_space_ev_size_list_.get<int>(), unique_key_.get<key_t>(),
            unique_dst_idx_.get<uint32_t>());
      }
      {
        int num_unique_id_space = static_cast<int>(unique_id_space_list_.get_num_elements());
        count_unique_key_kernel<<<num_unique_id_space, 32, 0, stream>>>(
            hash_keys_.get<key_t>(), hash_offset_.get<uint32_t>(), num_unique_id_space,
            unique_id_space_offset_.get<uint32_t>());

        HCTR_LIB_THROW(hipPeekAtLastError());
      }
      {
        size_t nbytes = d_temp_scan_encode_storage_.nbytes();
        hipcub::DeviceScan::InclusiveSum(
            d_temp_scan_encode_storage_.get(), nbytes, unique_id_space_offset_.get<uint32_t>(),
            unique_id_space_offset_.get<uint32_t>(), unique_id_space_offset_.get_num_elements(), stream);
        hipcub::DeviceScan::InclusiveSum(
            d_temp_scan_encode_storage_.get(), nbytes, unique_dst_idx_.get<uint32_t>(),
            unique_dst_idx_.get<uint32_t>(), unique_dst_idx_.get_num_elements(), stream);
        hipcub::DeviceScan::InclusiveSum(d_temp_scan_encode_storage_.get(), nbytes,
                                      sorted_bucket_id_offset_.get<uint32_t>(),
                                      sorted_bucket_id_offset_.get<uint32_t>(),
                                      sorted_bucket_id_offset_.get_num_elements(), stream);
      }
      HCTR_LIB_THROW(hipStreamSynchronize(stream));
    }
  });
  *unique_key = unique_key_;
  *num_unique_key = *num_unique_key_.get<size_t>();
  *unique_dst_idx = unique_dst_idx_;
  *sorted_bucket_id_list = sorted_bucket_id_list_;
  *sorted_bucket_id_offset = sorted_bucket_id_offset_;
  *unique_id_space_list = unique_id_space_list_;
  *unique_id_space_offset = unique_id_space_offset_;
}
}  // namespace embedding
